#include "hip/hip_runtime.h"
// -*- c++ -*-

#include <Eigen/Dense>

#include "CUDADataFormats/HcalRecHitSoA/interface/RecHitCollection.h"
#include "DataFormats/HcalDetId/interface/HcalSubdetector.h"
#include "DataFormats/ParticleFlowReco/interface/PFLayer.h"
#include "DataFormats/DetId/interface/DetId.h"
#include "DataFormats/HcalDetId/interface/HcalDetId.h"
#include "HeterogeneousCore/CUDAUtilities/interface/copyAsync.h"
#include "DeclsForKernels.h"
#include "SimplePFGPUAlgos.h"
//#include "RecoParticleFlow/PFClusterProducer/interface/PFHBHERecHitParamsGPU.h"

// Uncomment for debug mode
//#define DEBUG_ENABLE

namespace PFRecHit {
  namespace HCAL {

    __constant__ Constants constantsGPU_d;
    // member methods:
    //  initializeCudaConstants [called from producer]
    //  initializeArrays
    //  buildDetIdMapPerBlockMulti (not used)
    //  buildDetIdMapPerBlock
    //  testDetIdMap (can be used for tesing maps)
    //  applyQTests (apply a single threshold)
    //  applyDepthThresholdQTests
    //  applyMaskSerial (simplier version)
    //  applyMask
    //  convert_rechits_to_PFRechits
    //  entryPoint [called from producer] utilizes:
    //   initializeArrays
    //   buildDetIdMapPerBlock
    //   applyDepthThresholdQTests
    //   applyMask
    //   convert_rechits_to_PFRechits

    void initializeCudaConstants(const PFRecHit::HCAL::Constants& cudaConstants, const hipStream_t cudaStream) {
      cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(constantsGPU_d), &cudaConstants, sizeof(cudaConstants),
					0, hipMemcpyHostToDevice, cudaStream));
    }

    // Initialize arrays used to store temporary values for each event
    __global__ void initializeArrays(uint32_t nTopoArraySize,
				     uint32_t nRHIn,          // Number of input rechits
                                     int* rh_mask,            // Mask for input rechit index
                                     int* rh_inputToFullIdx,  // Mapping of input rechit index -> reference table index
                                     int* rh_fullToInputIdx,  // Mapping of reference table index -> input rechit index
                                     int* pfrhToInputIdx,     // Mapping of output PFRecHit index -> input rechit index
                                     int* inputToPFRHIdx) {   // Mapping of input rechit index -> output PFRecHit index

      if (blockIdx.x==0 && threadIdx.x==0){
	printf("bb3 %8d %8d\n",
	       (int)constantsGPU_d.nDenseIdsInRange,
	       nTopoArraySize);
      }

      // Reset mappings of reference table index. Total length = number of all valid HCAL detIds
      //for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (constantsGPU_d.nValidBarrelIds + constantsGPU_d.nValidEndcapIds); i += blockDim.x * gridDim.x) {
      for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (constantsGPU_d.nDenseIdsInRange); i += blockDim.x * gridDim.x) {
        rh_fullToInputIdx[i] = -1;
      }

      // Reset mappings of input,output indices and rechit mask
      for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < nRHIn; i += blockDim.x * gridDim.x) {
        pfrhToInputIdx[i] = -1;
        inputToPFRHIdx[i] = -1;
        rh_inputToFullIdx[i] = -1;
        rh_mask[i] = -2;
      }
    }

    __global__ void buildDetIdMapPerBlockMulti(
        uint32_t size,
        uint32_t const* rh_detIdRef,    // Reference table index -> detId
        uint32_t* rh_detIdMap,          // Map for input rechit detId -> reference table index
        uint32_t const* recHits_did) {  // Input rechit detIds

      __shared__ uint32_t detId, subdet, minval, maxval, notDone;

      for (uint32_t i = blockIdx.x; i < size; i += gridDim.x) {
        if (threadIdx.x == 0) {
          notDone = 1;
          detId = recHits_did[i];

          // Get subdetector encoded in detId
          // cmssdt.cern.ch/lxr/source/DataFormats/DetId/interface/DetId.h#0048
          subdet = (detId >> DetId::kSubdetOffset) & DetId::kSubdetMask;
          if (subdet == HcalBarrel) {
            minval = 0;
            maxval = constantsGPU_d.nValidBarrelIds;
          } else if (subdet == HcalEndcap) {
            minval = constantsGPU_d.nValidEndcapIds;
            maxval = (constantsGPU_d.nValidBarrelIds + constantsGPU_d.nValidEndcapIds);
          } else {
            printf("Rechit %u detId %u has invalid subdetector %u!\n", blockIdx.x, detId, subdet);
            return;
          }
        }
        __syncthreads();

        for (uint32_t j = threadIdx.x + minval; j < maxval && notDone; j += blockDim.x) {
          if (detId == rh_detIdRef[j]) {
            // Found it
            rh_detIdMap[i] = j;
            notDone = 0;
            //atomicAdd(&notDone, -1);
            //    break;
          }
          __syncthreads();
        }
      }
    }



    __global__ void buildDetIdMapHackathon(
        uint32_t size,
        uint32_t const* rh_detIdRef,    // Reference table index -> detId
        int* rh_inputToFullIdx,     // Map for input rechit detId -> reference table index
        int* rh_fullToInputIdx,     // Map for reference table index -> input rechit index
        uint32_t const* recHits_did)    // Input rechit detIds
        {

          int first = blockIdx.x*blockDim.x + threadIdx.x;
          for (int i = first; i < size; i += gridDim.x * blockDim.x) {
            auto detId = rh_detIdRef[i];
            for(int j = 0; j< size; ++j)
            {
              if(recHits_did[j] == detId)
              {
                rh_inputToFullIdx[j] = i;
                rh_fullToInputIdx[i] = j;
                return;
              }
            }
          }
        }

    __global__ void buildDetIdMapKH(
        uint32_t size,
        uint32_t const* rh_detIdRef,    // Reference table index -> detId
        int* rh_inputToFullIdx,     // Map for input rechit detId -> reference table index
        int* rh_fullToInputIdx,     // Map for reference table index -> input rechit index
        uint32_t const* recHits_did)    // Input rechit detIds
        {

          int first = blockIdx.x*blockDim.x + threadIdx.x;
          for (int i = first; i < size; i += gridDim.x * blockDim.x) {
            auto detId = recHits_did[i];

            // Get subdetector encoded in detId to narrow the range of reference table values to search
            // cmssdt.cern.ch/lxr/source/DataFormats/DetId/interface/DetId.h#0048
            uint32_t subdet = (detId >> DetId::kSubdetOffset) & DetId::kSubdetMask;
            uint32_t minval, maxval;
            if (subdet == HcalBarrel) {
              minval = 0;
              maxval = constantsGPU_d.nValidBarrelIds;
            } else if (subdet == HcalEndcap) {
              minval = constantsGPU_d.nValidEndcapIds;
              maxval = (constantsGPU_d.nValidBarrelIds + constantsGPU_d.nValidEndcapIds);
            } else {
              printf("Rechit %u detId %u has invalid subdetector %u!\n", blockIdx.x, detId, subdet);
              return;
            }

            // Search all valid rechits for matching detId
            for (uint32_t j = minval; j < maxval; j += 1) {
              if (detId == rh_detIdRef[j]) {
                // Found it
                rh_inputToFullIdx[i] = j;  // Input rechit index -> reference table index
                rh_fullToInputIdx[j] = i;  // Reference table index -> input rechit index
                return;
              }
            }
          }

        }

    // Get subdetector encoded in detId to narrow the range of reference table values to search
    // cmssdt.cern.ch/lxr/source/DataFormats/DetId/interface/DetId.h#0048
    __device__ uint32_t getSubdet(uint32_t detId) {return ((detId >> DetId::kSubdetOffset) & DetId::kSubdetMask);}

    //https://cmssdt.cern.ch/lxr/source/DataFormats/HcalDetId/interface/HcalDetId.h#0163
    __device__ uint32_t getDepth(uint32_t detId) {return ((detId >> HcalDetId::kHcalDepthOffset2) & HcalDetId::kHcalDepthMask2);}

    //https://cmssdt.cern.ch/lxr/source/DataFormats/HcalDetId/interface/HcalDetId.h#0148
    __device__ uint32_t getIetaAbs(uint32_t detId) {return ((detId >> HcalDetId::kHcalEtaOffset2) & HcalDetId::kHcalEtaMask2);}

    //https://cmssdt.cern.ch/lxr/source/DataFormats/HcalDetId/interface/HcalDetId.h#0157
    __device__ uint32_t getIphi(uint32_t detId) {return (detId & HcalDetId::kHcalPhiMask2);}

    //https://cmssdt.cern.ch/lxr/source/DataFormats/HcalDetId/interface/HcalDetId.h#0141
    __device__ int getZside(uint32_t detId) {return ((detId & HcalDetId::kHcalZsideMask2) ? (1) : (-1));}

    //https://cmssdt.cern.ch/lxr/source/Geometry/CaloTopology/src/HcalTopology.cc#1170
    __device__ uint32_t detId2denseIdHB(uint32_t detId) {
      const int maxDepthHB_ = 4;
      //const int maxDepthHE_ = 7;
      const int firstHBRing_ = 1;
      const int lastHBRing_ = 16;
      const int nEtaHB_ = (lastHBRing_ - firstHBRing_ + 1);
      const int IPHI_MAX = 72;
      const int ip = getIphi(detId);
      const int ie = getIetaAbs(detId);
      const int dp = getDepth(detId);
      const int zn = getZside(detId);
      unsigned int retval = 0xFFFFFFFFu;
      retval = (dp - 1) + maxDepthHB_ * (ip - 1);
      if (zn > 0)
	retval += maxDepthHB_ * IPHI_MAX * (ie*zn - firstHBRing_);
      else
	retval += maxDepthHB_ * IPHI_MAX * (ie*zn + lastHBRing_ + nEtaHB_);

      return retval;
    }

    //https://cmssdt.cern.ch/lxr/source/Geometry/CaloTopology/src/HcalTopology.cc#1189
    __device__ uint32_t detId2denseIdHE(uint32_t detId) {
      const int maxDepthHB_ = 4;
      const int maxDepthHE_ = 7;
      const int firstHERing_ = 16;
      const int lastHERing_ = 29;
      const int nEtaHE_ = (lastHERing_ - firstHERing_ + 1);
      const int maxPhiHE_ = 72;
      const int IPHI_MAX = 72;
      const int ip = getIphi(detId);
      const int ie = getIetaAbs(detId);
      const int dp = getDepth(detId);
      const int zn = getZside(detId);
      unsigned int retval = 0xFFFFFFFFu;
      const int HBSize_ = maxDepthHB_ * 16 * IPHI_MAX * 2;
      retval = (dp - 1) + maxDepthHE_ * (ip - 1) + HBSize_;
      if (zn > 0)
	retval += maxDepthHE_ * maxPhiHE_ * (ie*zn - firstHERing_);
      else
	retval += maxDepthHE_ * maxPhiHE_ * (ie*zn + lastHERing_ + nEtaHE_);

      return retval;
    }

    __device__ uint32_t detId2denseId(uint32_t detId) {
      if (getSubdet(detId)==HcalBarrel) return detId2denseIdHB(detId);
      else if (getSubdet(detId)==HcalEndcap) return detId2denseIdHE(detId);
      else printf("invalid detId\n");
    }

    __global__ void checkPersistentDataInputs(
	uint32_t denseIdHcalMin,        // min denseIdHcal
	uint32_t nDenseIdsInRange,      // denseIdHcal ranges (# of elements) i.e. max-min+1
        uint32_t const* rh_detIdRef,    // Reference table index -> detId
        float3 const* rh_posRef,      // Reference table index -> position
        int const* rh_neighboursRef)    // Reference table index -> neighbours
    {
      for (int i = 0; i < nDenseIdsInRange; i++ ){
        float3 pos = rh_posRef[i];  // position vector of this rechit
	int denseid = -1;
	if (rh_detIdRef[i]>0) denseid = detId2denseId(rh_detIdRef[i]);
 	printf("check dense, hid, dense: %d %d %d  %8.3f %8.3f %8.3f  %d %d %d %d %d %d %d %d\n",
	       i,rh_detIdRef[i],
	       denseid,
	       pos.x,pos.y,pos.z,
	       rh_neighboursRef[8*i],
	       rh_neighboursRef[8*i+1],
	       rh_neighboursRef[8*i+2],
	       rh_neighboursRef[8*i+3],
	       rh_neighboursRef[8*i+4],
	       rh_neighboursRef[8*i+5],
	       rh_neighboursRef[8*i+6],
	       rh_neighboursRef[8*i+7]
	       );
      }
    }

    __global__ void buildDetIdMapKH2(
        uint32_t size,
	uint32_t denseIdHcalMin,        // min denseIdHcal
        uint32_t const* rh_detIdRef,    // Reference table index -> detId
	uint32_t const* detId,
        int* rh_inputToFullIdx,     // Map for input rechit detId -> reference table index
        int* rh_fullToInputIdx,     // Map for reference table index -> input rechit index
        uint32_t const* recHits_did)    // Input rechit detIds
        {

	  //printf("Access detId via ES on device %8d\n",detId[1]);

          int first = blockIdx.x*blockDim.x + threadIdx.x;
          for (int i = first; i < size; i += gridDim.x * blockDim.x) {
	    // i: index for input rechits
            auto detId = recHits_did[i];
	    auto denseId = detId2denseId(detId);
	    auto fullIdx = denseId - denseIdHcalMin;
	    rh_inputToFullIdx[i] = fullIdx;  // Input rechit index -> reference table index
	    rh_fullToInputIdx[fullIdx] = i;  // Reference table index -> input rechit index
	  }
        }

    // Build detId map with 1 block per input rechit
    // Searches by detId for the matching index in reference table
    __global__ void buildDetIdMapPerBlock(
        uint32_t size,                  // Number of input rechits
        uint32_t const* rh_detIdRef,    // Reference table index -> detId
        int* rh_inputToFullIdx,         // Map for input rechit index -> reference table index
        int* rh_fullToInputIdx,         // Map for reference table index -> input rechit index
        uint32_t const* recHits_did) {  // Input rechit detIds

      __shared__ uint32_t detId, subdet, minval, maxval;

      if (threadIdx.x == 0) {
        detId = recHits_did[blockIdx.x];

        // Get subdetector encoded in detId to narrow the range of reference table values to search
        // cmssdt.cern.ch/lxr/source/DataFormats/DetId/interface/DetId.h#0048
        subdet = (detId >> DetId::kSubdetOffset) & DetId::kSubdetMask;
        if (subdet == HcalBarrel) {
          minval = 0;
          maxval = constantsGPU_d.nValidBarrelIds;
        } else if (subdet == HcalEndcap) {
          minval = constantsGPU_d.nValidEndcapIds;
          maxval = (constantsGPU_d.nValidBarrelIds + constantsGPU_d.nValidEndcapIds);
        } else {
          printf("Rechit %u detId %u has invalid subdetector %u!\n", blockIdx.x, detId, subdet);
          return;
        }
      }
      __syncthreads();

      // Search all valid rechits for matching detId
      for (uint32_t j = threadIdx.x + minval; j < maxval; j += blockDim.x) {
        if (detId == rh_detIdRef[j]) {
          // Found it
          rh_inputToFullIdx[blockIdx.x] = j;  // Input rechit index -> reference table index
          rh_fullToInputIdx[j] = blockIdx.x;  // Reference table index -> input rechit index
          return;
        }
      }
    }

    // Debugging function used to check the mapping of input index <-> reference table index
    __global__ void testDetIdMap(uint32_t size,                  // Number of input rechits
                                 const uint32_t* rh_detIdRef,    // Reference table index -> detId
                                 const int* rh_inputToFullIdx,   //  Map for input rh index -> reference table index
                                 const int* rh_fullToInputIdx,   //  Map for reference table index -> input rh index
                                 const uint32_t* recHits_did) {  //  Rechit detIds

      uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
      if (i >= size)
        return;

      uint32_t detId = recHits_did[i];
      int index = rh_inputToFullIdx[i];
      int fullToInputIdx = index > -1 ? rh_fullToInputIdx[index] : -1;
      if (fullToInputIdx != i) {
        printf("Rechit %d detId %u doesn't match index from rh_fullToInputIdx %d!\n", i, detId, fullToInputIdx);
      }
      if (index >= (constantsGPU_d.nValidBarrelIds + constantsGPU_d.nValidEndcapIds) || detId != rh_detIdRef[index])
        printf(
            "Rechit %u detId %u MISMATCH with reference table index %u detId %u\n", i, detId, index, rh_detIdRef[index]);
    }

    // Phase 0 threshold test corresponding to PFRecHitQTestThreshold
    __global__ void applyQTests(const uint32_t nRHIn,
                                int* rh_mask,  // Mask for rechits by input index
                                const uint32_t* recHits_did,
                                const float* recHits_energy) {
      for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < nRHIn; i += gridDim.x * blockDim.x) {
        rh_mask[i] = (recHits_energy[i] > constantsGPU_d.qTestThresh);
      }
    }

    // Phase I threshold test corresponding to PFRecHitQTestHCALThresholdVsDepth
    __global__ void applyDepthThresholdQTests(const uint32_t nRHIn,           // Number of input rechits
					      //const PFHBHERecHitParamsGPU::Product recHitParamsProduct,
					      int const* depthHB,
					      int const* depthHE,
					      float const* thresholdE_HB,
					      float const* thresholdE_HE,
                                              int* rh_mask,                   // Mask for rechit index
                                              const uint32_t* recHits_did,    // Input rechit detIds
                                              const float* recHits_energy) {  // Input rechit energy

      for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < nRHIn; i += gridDim.x * blockDim.x) {
        uint32_t detid = recHits_did[i];
        uint32_t subdet = (detid >> DetId::kSubdetOffset) & DetId::kSubdetMask;
        uint32_t depth = (detid >> HcalDetId::kHcalDepthOffset2) & HcalDetId::kHcalDepthMask2;
        float threshold = 9999.;
        if (subdet == HcalBarrel) {
	  bool found = false;
	  for (uint32_t j=0; j<4; j++){
	    if (depth == depthHB[j]){
	      /*
	      printf("aa %6d %8.2f %8d\n",
		     depthHB[j],
		     thresholdE_HB[j],
		     depthHB[j]);
	      */
	      threshold = thresholdE_HB[j];
	      found = true; // found depth and threshold
	    }
	  }
	  if (!found)
            printf("i = %u\tInvalid depth %u for barrel rechit %u!\n", i, depth, detid);
        } else if (subdet == HcalEndcap) {
	  bool found = false;
	  for (uint32_t j=0; j<7; j++){
	    if (depth == depthHE[j]){
	      threshold = thresholdE_HE[j];
	      found = true; // found depth and threshold
	    }
	  }
	  if (!found)
            printf("i = %u\tInvalid depth %u for endcap rechit %u!\n", i, depth, detid);
        } else {
          printf("Rechit %u detId %u has invalid subdetector %u!\n", blockIdx.x, detid, subdet);
          return;
        }
        // If this PFRecHit:
        //  Passes threshold cuts, set mask to 1
        //  Fails cuts and discarded, set mask to 0
        //  Should be cleaned (only applicable to HF), mask = -1 (default value)
        rh_mask[i] = (recHits_energy[i] >= threshold);
        if (rh_mask[i] < 0)
          printf("WARNING: Found input rechit %d has rh_mask = %d\n", i, rh_mask[i]);
      }
    }

    __global__ void applyMaskSerial(uint32_t nRHIn,
                                    uint32_t* nPFRHOut,
                                    //const bool* rh_mask,
                                    const int* rh_mask,
                                    int* pfrhToInputIdx,
                                    int* inputToPFRHIdx) {
      extern __shared__ uint16_t serial_cleanedList[];
      __shared__ uint16_t cleanedTotal, pos;

      pos = cleanedTotal = 0;
      for (uint16_t i = 0; i < nRHIn; i++) {
        if (rh_mask[i] == 1) {
          pfrhToInputIdx[pos] = i;
          inputToPFRHIdx[i] = pos;
          pos++;
        } else if (rh_mask[i] == -1) {
          serial_cleanedList[cleanedTotal] = i;
          cleanedTotal++;
        }
      }
      for (uint16_t i = 0; i < cleanedTotal; i++) {
        pfrhToInputIdx[pos + i] = serial_cleanedList[i];
        inputToPFRHIdx[serial_cleanedList[i]] = pos + i;
      }
      *nPFRHOut = pos;  // Total number of PFRecHits passing cuts
    }

    // Apply rechit mask and determine output PFRecHit ordering
    __global__ void applyMask(uint32_t nRHIn,          // Number of input rechits
                              uint32_t* nPFRHOut,      // Number of passing output PFRecHits
                              uint32_t* nPFRHCleaned,  // Number of cleaned output PFRecHits
                              const int* rh_mask,      // Rechit mask
                              int* pfrhToInputIdx,     // Mapping of output PFRecHit index -> input rechit index
                              int* inputToPFRHIdx) {   // Mapping of input rechit index -> output PFRecHit index

      extern __shared__ uint32_t cleanedList[];
      __shared__ uint32_t cleanedTotal, pos;

      if (threadIdx.x == 0) {
        pos = cleanedTotal = 0;
      }
      __syncthreads();

      for (uint32_t i = threadIdx.x; i < nRHIn; i += blockDim.x) {
        if (rh_mask[i] == 1) {  // Passing
          int k = atomicAdd(&pos, 1);
          pfrhToInputIdx[k] = i;
          inputToPFRHIdx[i] = k;
        } else if (rh_mask[i] == -1) {  // Cleaned
          int k = atomicAdd(&cleanedTotal, 1);
          cleanedList[k] = i;
        }
      }
      __syncthreads();

      // Loop over cleaned PFRecHits and append to the end of the output array
      for (uint32_t i = threadIdx.x; i < cleanedTotal; i += blockDim.x) {
        pfrhToInputIdx[pos + i] = cleanedList[i];
        inputToPFRHIdx[cleanedList[i]] = pos + i;
      }
      __syncthreads();
      if (threadIdx.x == 0) {
        *nPFRHOut = pos;               // Total number of PFRecHits passing cuts
        *nPFRHCleaned = cleanedTotal;  // Total number of cleaned PFRecHits
      }
    }

    // Fill output PFRecHit arrays
    __global__ void convert_rechits_to_PFRechits(const uint32_t nRHIn,
                                                 const uint32_t* nPFRHOut,
                                                 const uint32_t* nPFRHCleaned,
                                                 const int* rh_mask,
                                                 const int* pfrhToInputIdx,
                                                 const int* inputToPFRHIdx,
                                                 const float3* rh_pos,
                                                 const int* rh_neighbours,
						 const float3* position,
						 const int* neighbours,
                                                 const int* rh_inputToFullIdx,
                                                 const int* rh_fullToInputIdx,
                                                 const float* recHits_energy,
                                                 const float* recHits_chi2,
                                                 const float* recHits_energyM0,
                                                 const float* recHits_timeM0,
                                                 const uint32_t* recHits_did,
                                                 int* pfrechits_depth,
                                                 int* pfrechits_layer,
                                                 int* pfrechits_detId,
                                                 float* pfrechits_time,
                                                 float* pfrechits_energy,
                                                 float* pfrechits_x,
                                                 float* pfrechits_y,
                                                 float* pfrechits_z,
                                                 int* pfrechits_neighbours,
                                                 short* pfrechits_neighbourInfos) {
      for (uint32_t pfIdx = blockIdx.x * blockDim.x + threadIdx.x; pfIdx < (*nPFRHOut + *nPFRHCleaned);
           pfIdx += blockDim.x * gridDim.x) {

        int i = pfrhToInputIdx[pfIdx];  // Get input rechit index corresponding to output PFRecHit index pfIdx
        if (i < 0)
          printf("convert kernel with pfIdx = %u has input index i = %u\n", pfIdx, i);
        pfrechits_time[pfIdx] = recHits_timeM0[i];
        float energy = recHits_energy[i];
        pfrechits_energy[pfIdx] = energy;

        uint32_t detid = recHits_did[i];
        pfrechits_detId[pfIdx] = detid;

        //bool debug = (detid == 1158706177) ? true : false;
        bool debug = false;
        // cmssdt.cern.ch/lxr/source/DataFormats/HcalDetId/interface/HcalDetId.h#0168
        pfrechits_depth[pfIdx] = (detid >> HcalDetId::kHcalDepthOffset2) & HcalDetId::kHcalDepthMask2;

        // cmssdt.cern.ch/lxr/source/DataFormats/DetId/interface/DetId.h#0050
        int subdet = (detid >> DetId::kSubdetOffset) & DetId::kSubdetMask;
        int layer = 0;
        if (subdet == HcalBarrel)
          layer = PFLayer::HCAL_BARREL1;
        else if (subdet == HcalEndcap)
          layer = PFLayer::HCAL_ENDCAP;
        else
          printf("Invalid subdetector (%d) for detId %d: pfIdx = %d\tinputIdx = %d\tfullIdx = %d\n",
                 subdet,
                 detid,
                 pfIdx,
                 i,
                 rh_inputToFullIdx[i]);

        pfrechits_layer[pfIdx] = layer;

        int index = rh_inputToFullIdx[i];  // Determine reference table index corresponding to this input index
        if (index < 0)
          printf("convert kernel with pfIdx = %u has full index = %u\n", pfIdx, index);
	float3 pos2 = position[index];
        float3 pos = rh_pos[index];  // position vector of this rechit
	if (pos.x!=pos2.x || pos.y!=pos2.y || pos.z!=pos2.z)
	  printf("DDD pos check %8.2f %8.2f %8.2f vs %8.2f %8.2f %8.2f\n",
		 pos.x,pos.y,pos.z,
		 position[index].x,position[index].y,position[index].z);
        pfrechits_x[pfIdx] = pos.x;
        pfrechits_y[pfIdx] = pos.y;
        pfrechits_z[pfIdx] = pos.z;

	if (rh_neighbours[index * 8]    !=neighbours[index * 8])     printf("neigh  %8d %8d\n",rh_neighbours[index * 8],     neighbours[index * 8]);
	if (rh_neighbours[index * 8 + 1]!=neighbours[index * 8 + 1]) printf("neigh1 %8d %8d\n",rh_neighbours[index * 8 + 1], neighbours[index * 8 + 1]);
	if (rh_neighbours[index * 8 + 2]!=neighbours[index * 8 + 2]) printf("neigh2 %8d %8d\n",rh_neighbours[index * 8 + 2], neighbours[index * 8 + 2]);
	if (rh_neighbours[index * 8 + 3]!=neighbours[index * 8 + 3]) printf("neigh3 %8d %8d\n",rh_neighbours[index * 8 + 3], neighbours[index * 8 + 3]);
	if (rh_neighbours[index * 8 + 4]!=neighbours[index * 8 + 4]) printf("neigh4 %8d %8d\n",rh_neighbours[index * 8 + 4], neighbours[index * 8 + 4]);
	if (rh_neighbours[index * 8 + 5]!=neighbours[index * 8 + 5]) printf("neigh5 %8d %8d\n",rh_neighbours[index * 8 + 5], neighbours[index * 8 + 5]);
	if (rh_neighbours[index * 8 + 6]!=neighbours[index * 8 + 6]) printf("neigh6 %8d %8d\n",rh_neighbours[index * 8 + 6], neighbours[index * 8 + 6]);
	if (rh_neighbours[index * 8 + 7]!=neighbours[index * 8 + 7]) printf("neigh7 %8d %8d\n",rh_neighbours[index * 8 + 7], neighbours[index * 8 + 7]);

	/*
          printf("\trh_neighbours = [%d, %d, %d, %d, %d, %d, %d, %d] [%d, %d, %d, %d, %d, %d, %d, %d]\n",
                 rh_neighbours[index * 8],
                 rh_neighbours[index * 8 + 1],
                 rh_neighbours[index * 8 + 2],
                 rh_neighbours[index * 8 + 3],
                 rh_neighbours[index * 8 + 4],
                 rh_neighbours[index * 8 + 5],
                 rh_neighbours[index * 8 + 6],
                 rh_neighbours[index * 8 + 7],
                 neighbours[index * 8],
                 neighbours[index * 8 + 1],
                 neighbours[index * 8 + 2],
                 neighbours[index * 8 + 3],
                 neighbours[index * 8 + 4],
                 neighbours[index * 8 + 5],
                 neighbours[index * 8 + 6],
                 neighbours[index * 8 + 7]);
	*/

        if (debug) {
          printf("Now debugging rechit %d\tpfIdx %u\ti = %d\tindex = %d\tpos = (%f, %f, %f)\n",
                 detid,
                 pfIdx,
                 i,
                 index,
                 pos.x,
                 pos.y,
                 pos.z);
          printf("\trh_neighbours = [%d, %d, %d, %d, %d, %d, %d, %d]\n\n",
                 rh_neighbours[index * 8],
                 rh_neighbours[index * 8 + 1],
                 rh_neighbours[index * 8 + 2],
                 rh_neighbours[index * 8 + 3],
                 rh_neighbours[index * 8 + 4],
                 rh_neighbours[index * 8 + 5],
                 rh_neighbours[index * 8 + 6],
                 rh_neighbours[index * 8 + 7]);
        }

        // Lambda function for filling PFRecHit neighbour arrays
        // pos: Order in PFRecHit neighbour array. First four values correspond to 4-neighbours: N,S,E,W
        // refPos: Order of rechit neighbors given in neighboursHcal_ array from PFHCALDenseIdNavigator
        // eta: ieta for this direction relative to center
        // phi: iphi for this direction relative to center
        // depth: idepth for this direction relative to center (always 0 for layer clusters)
        auto associateNeighbour = [&] __device__(uint32_t pos, uint32_t refPos, int eta, int phi, int depth) {
          int fullIdx = rh_neighbours[index * 8 + refPos];                // Reference table index for this neighbour
          int inputIdx = fullIdx > -1 ? rh_fullToInputIdx[fullIdx] : -1;  // Input rechit index for this neighbour
          int pfrhIdx = inputIdx > -1 ? inputToPFRHIdx[inputIdx] : -1;    // Output PFRecHit index for this neighbour
          short infos = pfrhIdx > -1 ? 0 : -1;
          if (debug)
            printf(
                "associateNeighbour for rechit %d pos %d refPos %d: fullIdx = %d%sinputIdx = %d\tpfrhIdx = "
                "%d\trecHits_did[inputIdx] = %d\n",
                detid,
                pos,
                refPos,
                fullIdx,
                (fullIdx == 0) ? "\t\t" : "\t",
                inputIdx,
                pfrhIdx,
                recHits_did[inputIdx]);
          if (pfrhIdx < 0 ||
              pfrhIdx >= *nPFRHOut) {  // Only include valid PFRecHit indices. Don't include cleaned rechits
            pfrechits_neighbours[pfIdx * 8 + pos] = -1;
            pfrechits_neighbourInfos[pfIdx * 8 + pos] = -1;
            if (debug)
              printf("\tNeigh %u has invalid pfrhIdx %d!\n", pos, pfrhIdx);
          } else {
            // Valid neighbour found. Compute neighbour infos
            if (eta > 0)
              infos |= 1;
            infos |= (abs(eta) << 1);

            if (phi > 0)
              infos |= (1 << 4);
            infos |= (abs(phi) << 5);

            if (depth > 0)
              infos |= (1 << 8);
            infos |= (abs(depth) << 9);

            // Set PFRecHit index and infos for this neighbour
            pfrechits_neighbours[pfIdx * 8 + pos] = pfrhIdx;
            pfrechits_neighbourInfos[pfIdx * 8 + pos] = infos;
            if (debug)
              printf("\tNeigh %u has pfrhIdx %d and infos %d\n", pos, pfrhIdx, infos);
          }
        };

        // Now fill neighbours and neighbourInfos
        // Reference neighbor array order from navigator:
        // SOUTH,SOUTHEAST,SOUTHWEST,EAST,WEST,NORTHEAST,NORTHWEST,NORTH
        associateNeighbour(0, 7, 0, 1, 0);    // N
        associateNeighbour(1, 0, 0, -1, 0);   // $
        associateNeighbour(2, 3, 1, 0, 0);    // E
        associateNeighbour(3, 4, -1, 0, 0);   // W
        associateNeighbour(4, 5, 1, 1, 0);    // NE
        associateNeighbour(5, 2, -1, -1, 0);  // SW
        associateNeighbour(6, 1, 1, -1, 0);   // SE
        associateNeighbour(7, 6, -1, 1, 0);   // NW
      }
    }

    void entryPoint(::hcal::RecHitCollection<::calo::common::DevStoragePolicy> const& HBHERecHits_asInput,
		    const PFRecHit::HCAL::Constants& cudaConstants,
		    const ConstantProducts& constantProducts,
		    //const PFRecHit::HCAL::ConstantProducts& constantProducts,
                    OutputPFRecHitDataGPU& HBHEPFRecHits_asOutput,
                    PersistentDataGPU& persistentDataGPU,
                    ScratchDataGPU& scratchDataGPU,
                    hipStream_t cudaStream,
                    std::array<float, 5>& timer) {

      //printf("bb %8d\n",recHitParametersProduct.valuesdepthHB[1]);
      //printf("bb %8d\n",constantProducts.depthHB[1]);
      std::cout << constantProducts.denseId.size() << std::endl;
      std::cout << constantProducts.detId.size() << std::endl;
      std::cout << constantProducts.position.size() << std::endl;
      std::cout << constantProducts.neighbours.size() << std::endl;
      // printf("bb2 %8d %8d %8d %8d %8d %8d\n",
      // 	     cudaConstants.nDenseIdsInRange,
      // 	     constantProducts.denseId[0],
      // 	     constantProducts.denseId[13325],
      // 	     *(&(constantProducts.denseId) + 1) - constantProducts.denseId;
      // 	     (int)sizeof(constantProducts.denseId),
      // 	     (int)sizeof(constantProducts.denseId[0]));

      uint32_t nRHIn = HBHERecHits_asInput.size;  // Number of input rechits
      if (nRHIn == 0) {
        HBHEPFRecHits_asOutput.PFRecHits.size = 0;
        HBHEPFRecHits_asOutput.PFRecHits.sizeCleaned = 0;
        return;
      }

      // uint32_t *h_nPFRHOut, *d_nPFRHOut;          // Number of output PFRecHits (total passing cuts)
      // uint32_t *h_nPFRHCleaned, *d_nPFRHCleaned;  // Number of cleaned PFRecHits
      // h_nPFRHOut = new uint32_t(0);
      // h_nPFRHCleaned = new uint32_t(0);
      // cudaCheck(hipMallocAsync(&d_nPFRHOut, sizeof(int), cudaStream));
      // cudaCheck(hipMallocAsync(&d_nPFRHCleaned, sizeof(int), cudaStream));

      cms::cuda::device::unique_ptr<uint32_t[]> d_nPFRHOut; // Number of output PFRecHits (total passing cuts)
      cms::cuda::device::unique_ptr<uint32_t[]> d_nPFRHCleaned; // Number of cleaned PFRecHits
      cms::cuda::host::unique_ptr<uint32_t[]> h_nPFRHOut;
      cms::cuda::host::unique_ptr<uint32_t[]> h_nPFRHCleaned;

      d_nPFRHOut = cms::cuda::make_device_unique<uint32_t[]>(sizeof(uint32_t) , cudaStream);
      d_nPFRHCleaned = cms::cuda::make_device_unique<uint32_t[]>(sizeof(uint32_t) , cudaStream);

      h_nPFRHOut = cms::cuda::make_host_unique<uint32_t[]>(sizeof(uint32_t) , cudaStream);
      h_nPFRHCleaned = cms::cuda::make_host_unique<uint32_t[]>(sizeof(uint32_t) , cudaStream);

#ifdef DEBUG_ENABLE
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipDeviceSynchronize();
      hipEventRecord(start, cudaStream);
#endif
      int threadsPerBlock = 256;
      // Initialize scratch arrays
      initializeArrays<<<(max(scratchDataGPU.maxSize,cudaConstants.nDenseIdsInRange) + threadsPerBlock-1) / threadsPerBlock,
	threadsPerBlock, 0, cudaStream>>>(
	  constantProducts.detId.size(),
          nRHIn,
          scratchDataGPU.rh_mask.get(),
          scratchDataGPU.rh_inputToFullIdx.get(),
          scratchDataGPU.rh_fullToInputIdx.get(),
          scratchDataGPU.pfrhToInputIdx.get(),
          scratchDataGPU.inputToPFRHIdx.get());
      cudaCheck(hipGetLastError());

#ifdef DEBUG_ENABLE
      hipEventRecord(stop, cudaStream);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&timer[0], start, stop);
      printf("\ninitializeArrays took %f ms\n", timer[0]);
      hipEventRecord(start, cudaStream);
#endif

      // checkPersistentDataInputs<<<1,1,0, cudaStream>>>(cudaConstants.denseIdHcalMin,
      // 					 cudaConstants.nDenseIdsInRange,
      // 					 persistentDataGPU.rh_detId.get(),
      // 					 persistentDataGPU.rh_pos.get(),
      // 					 persistentDataGPU.rh_neighbours.get()
      // 						       );

      // // First build the mapping for input rechits to reference table indices
      // buildDetIdMapPerBlock<<<nRHIn, 256, 0, cudaStream>>>(nRHIn,
      //                                                      persistentDataGPU.rh_detId.get(),
      //                                                      scratchDataGPU.rh_inputToFullIdx.get(),
      //                                                      scratchDataGPU.rh_fullToInputIdx.get(),
      //                                                      HBHERecHits_asInput.did.get());
      // cudaCheck(hipGetLastError());

      // First build the mapping for input rechits to reference table indices
      // buildDetIdMapHackathon<<<(nRHIn + threadsPerBlock - 1)/threadsPerBlock, threadsPerBlock, 0, cudaStream>>>(nRHIn,
      // buildDetIdMapKH<<<(nRHIn + threadsPerBlock - 1)/threadsPerBlock, threadsPerBlock, 0, cudaStream>>>(nRHIn,
      buildDetIdMapKH2<<<(nRHIn + threadsPerBlock - 1)/threadsPerBlock, threadsPerBlock, 0, cudaStream>>>(nRHIn,
							   cudaConstants.denseIdHcalMin,
                                                           persistentDataGPU.rh_detId.get(),
							   constantProducts.topoDataProduct.detId,
                                                           scratchDataGPU.rh_inputToFullIdx.get(),
                                                           scratchDataGPU.rh_fullToInputIdx.get(),
                                                           HBHERecHits_asInput.did.get());
      cudaCheck(hipGetLastError());


    // Debugging function used to check the mapping of input index <-> reference table index
    // testDetIdMap<<<(nRHIn + threadsPerBlock - 1)/threadsPerBlock, threadsPerBlock, 0, cudaStream>>>(nRHIn,
    //                                                        persistentDataGPU.rh_detId.get(),
    //                                                        scratchDataGPU.rh_inputToFullIdx.get(),
    //                                                        scratchDataGPU.rh_fullToInputIdx.get(),
    //                                                        HBHERecHits_asInput.did.get());
     cudaCheck(hipGetLastError());
#ifdef DEBUG_ENABLE
      hipEventRecord(stop, cudaStream);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&timer[1], start, stop);
      printf("\nbuildDetIdMapPerBlock took %f ms\n", timer[1]);

      hipEventRecord(start, cudaStream);
#endif

      // Apply PFRecHit threshold & quality tests

      //applyQTests<<<(nRHIn+127)/128, 256, 0, cudaStream>>>(nRHIn, scratchDataGPU.rh_mask.get(), HBHERecHits_asInput.did.get(), HBHERecHits_asInput.energy.get());

      applyDepthThresholdQTests<<<(nRHIn + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock, 0, cudaStream>>>(
          nRHIn,
	  //constantProducts.recHitParametersProduct,
	  constantProducts.recHitParametersProduct.depthHB,
	  constantProducts.recHitParametersProduct.depthHE,
	  constantProducts.recHitParametersProduct.thresholdE_HB,
	  constantProducts.recHitParametersProduct.thresholdE_HE,
	  scratchDataGPU.rh_mask.get(), HBHERecHits_asInput.did.get(), HBHERecHits_asInput.energy.get());
      cudaCheck(hipGetLastError());

#ifdef DEBUG_ENABLE
      hipEventRecord(stop, cudaStream);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&timer[2], start, stop);
      printf("\napplyQTests took %f ms\n", timer[2]);
      hipEventRecord(start, cudaStream);
#endif

      // Apply rechit mask and determine output PFRecHit order
      applyMask<<<1, threadsPerBlock, 0, cudaStream>>>(nRHIn,
                                                             d_nPFRHOut.get(),
                                                             d_nPFRHCleaned.get(),
                                                             scratchDataGPU.rh_mask.get(),
                                                             scratchDataGPU.pfrhToInputIdx.get(),
                                                             scratchDataGPU.inputToPFRHIdx.get());
      cudaCheck(hipGetLastError());

#ifdef DEBUG_ENABLE
      hipEventRecord(stop, cudaStream);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&timer[3], start, stop);
      printf("\napplyMask took %f ms\n\n", timer[3]);
#endif

      cms::cuda::copyAsync(h_nPFRHOut, d_nPFRHOut, sizeof(uint32_t), cudaStream);
      cms::cuda::copyAsync(h_nPFRHCleaned, d_nPFRHCleaned, sizeof(uint32_t), cudaStream);

#ifdef DEBUG_ENABLE
      hipDeviceSynchronize();
      hipEventRecord(start);
#endif

      // Fill output PFRecHit arrays
      convert_rechits_to_PFRechits<<<(nRHIn + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock, 0, cudaStream>>>(
      //convert_rechits_to_PFRechits<<<1, 1, 0, cudaStream>>>(
          nRHIn,
          d_nPFRHOut.get(),
          d_nPFRHCleaned.get(),
          scratchDataGPU.rh_mask.get(),
          scratchDataGPU.pfrhToInputIdx.get(),
          scratchDataGPU.inputToPFRHIdx.get(),
          persistentDataGPU.rh_pos.get(),
          persistentDataGPU.rh_neighbours.get(),
	  constantProducts.topoDataProduct.position,
	  constantProducts.topoDataProduct.neighbours,
          scratchDataGPU.rh_inputToFullIdx.get(),
          scratchDataGPU.rh_fullToInputIdx.get(),
          HBHERecHits_asInput.energy.get(),
          HBHERecHits_asInput.chi2.get(),
          HBHERecHits_asInput.energyM0.get(),
          HBHERecHits_asInput.timeM0.get(),
          HBHERecHits_asInput.did.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_depth.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_layer.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_detId.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_time.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_energy.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_x.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_y.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_z.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_neighbours.get(),
          HBHEPFRecHits_asOutput.PFRecHits.pfrh_neighbourInfos.get());

      cudaCheck(hipGetLastError());

      // Make sure output size has finished copying before freeing memory
      //if (hipEventQuery(sizeCopyEvt) != hipSuccess) hipEventSynchronize(sizeCopyEvt);
      if (hipStreamQuery(cudaStream) != hipSuccess)
        cudaCheck(hipStreamSynchronize(cudaStream));

#ifdef DEBUG_ENABLE
      hipEventRecord(stop, cudaStream);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&timer[4], start, stop);
      printf("\nconvert_rechits_to_PFRechits took %f ms\n\n", timer[4]);
#endif

      HBHEPFRecHits_asOutput.PFRecHits.size = *(h_nPFRHOut.get());
      HBHEPFRecHits_asOutput.PFRecHits.sizeCleaned = *(h_nPFRHCleaned.get());

      // cudaCheck(hipFree(d_nPFRHOut));
      // cudaCheck(hipFree(d_nPFRHCleaned));
      // delete h_nPFRHOut;
      // delete h_nPFRHCleaned;
    }
  }  // namespace HCAL
}  //  namespace PFRecHit
