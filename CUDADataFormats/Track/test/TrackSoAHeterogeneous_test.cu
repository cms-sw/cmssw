#include "hip/hip_runtime.h"
#include "CUDADataFormats/Track/interface/PixelTrackUtilities.h"
#include "HeterogeneousCore/CUDAUtilities/interface/OneToManyAssoc.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"

namespace testTrackSoA {

  // Kernel which fills the TrackSoAView with data
  // to test writing to it
  template <typename TrackerTraits>
  __global__ void fill(TrackSoAView<TrackerTraits> tracks_view) {
    int i = threadIdx.x;
    if (i == 0) {
      tracks_view.nTracks() = 420;
    }

    for (int j = i; j < tracks_view.metadata().size(); j += blockDim.x) {
      tracks_view[j].pt() = (float)j;
      tracks_view[j].eta() = (float)j;
      tracks_view[j].chi2() = (float)j;
      tracks_view[j].quality() = (pixelTrack::Quality)(j % 256);
      tracks_view[j].nLayers() = j % 128;
      tracks_view.hitIndices().off[j] = j;
    }
  }

  // Kernel which reads from the TrackSoAView to verify
  // that it was written correctly from the fill kernel
  template <typename TrackerTraits>
  __global__ void verify(TrackSoAConstView<TrackerTraits> tracks_view) {
    int i = threadIdx.x;

    if (i == 0) {
      printf("SoA size: % d, block dims: % d\n", tracks_view.metadata().size(), blockDim.x);
      assert(tracks_view.nTracks() == 420);
    }
    for (int j = i; j < tracks_view.metadata().size(); j += blockDim.x) {
      assert(abs(tracks_view[j].pt() - (float)j) < .0001);
      assert(abs(tracks_view[j].eta() - (float)j) < .0001);
      assert(abs(tracks_view[j].chi2() - (float)j) < .0001);
      assert(tracks_view[j].quality() == (pixelTrack::Quality)(j % 256));
      assert(tracks_view[j].nLayers() == j % 128);
      assert(tracks_view.hitIndices().off[j] == j);
    }
  }

  // Host function which invokes the two kernels above
  template <typename TrackerTraits>
  void runKernels(TrackSoAView<TrackerTraits>& tracks_view, hipStream_t stream) {
    fill<TrackerTraits><<<1, 1024, 0, stream>>>(tracks_view);
    cudaCheck(hipGetLastError());
    cudaCheck(hipDeviceSynchronize());

    verify<TrackerTraits><<<1, 1024, 0, stream>>>(tracks_view);
    cudaCheck(hipGetLastError());
    cudaCheck(hipDeviceSynchronize());
  }

  template void runKernels<pixelTopology::Phase1>(TrackSoAView<pixelTopology::Phase1>& tracks_view,
                                                  hipStream_t stream);
  template void runKernels<pixelTopology::Phase2>(TrackSoAView<pixelTopology::Phase2>& tracks_view,
                                                  hipStream_t stream);

}  // namespace testTrackSoA
