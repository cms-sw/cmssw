#include "BrokenLineFitOnGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"

template <typename TrackerTraits>
void HelixFitOnGPU<TrackerTraits>::launchBrokenLineKernels(const TrackingRecHitSoAConstView<TrackerTraits>& hv,
                                                           uint32_t hitsInFit,
                                                           uint32_t maxNumberOfTuples,
                                                           hipStream_t stream) {
  assert(tuples_);

  auto blockSize = 64;
  auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

  //  Fit internals
  auto tkidGPU =
      cms::cuda::make_device_unique<typename TrackerTraits::tindex_type[]>(maxNumberOfConcurrentFits_, stream);
  auto hitsGPU = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Matrix3xNd<6>) / sizeof(double), stream);
  auto hits_geGPU = cms::cuda::make_device_unique<float[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Matrix6xNf<6>) / sizeof(float), stream);
  auto fast_fit_resultsGPU = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Vector4d) / sizeof(double), stream);

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // fit triplets

    kernel_BLFastFit<3, TrackerTraits><<<numberOfBlocks, blockSize, 0, stream>>>(tuples_,
                                                                                 tupleMultiplicity_,
                                                                                 hv,
                                                                                 tkidGPU.get(),
                                                                                 hitsGPU.get(),
                                                                                 hits_geGPU.get(),
                                                                                 fast_fit_resultsGPU.get(),
                                                                                 3,
                                                                                 3,
                                                                                 offset);
    cudaCheck(hipGetLastError());

    kernel_BLFit<3, TrackerTraits><<<numberOfBlocks, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                             bField_,
                                                                             outputSoa_,
                                                                             tkidGPU.get(),
                                                                             hitsGPU.get(),
                                                                             hits_geGPU.get(),
                                                                             fast_fit_resultsGPU.get());
    cudaCheck(hipGetLastError());

    if (fitNas4_) {
      // fit all as 4
      riemannFit::rolling_fits<4, TrackerTraits::maxHitsOnTrack, 1>([this,
                                                                     &hv,
                                                                     &tkidGPU,
                                                                     &hitsGPU,
                                                                     &hits_geGPU,
                                                                     &fast_fit_resultsGPU,
                                                                     &offset,
                                                                     &numberOfBlocks,
                                                                     &blockSize,
                                                                     &stream](auto i) {
        kernel_BLFastFit<4, TrackerTraits><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tuples_,
                                                                                         tupleMultiplicity_,
                                                                                         hv,
                                                                                         tkidGPU.get(),
                                                                                         hitsGPU.get(),
                                                                                         hits_geGPU.get(),
                                                                                         fast_fit_resultsGPU.get(),
                                                                                         4,
                                                                                         4,
                                                                                         offset);

        cudaCheck(hipGetLastError());

        kernel_BLFit<4, TrackerTraits><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                                     bField_,
                                                                                     outputSoa_,
                                                                                     tkidGPU.get(),
                                                                                     hitsGPU.get(),
                                                                                     hits_geGPU.get(),
                                                                                     fast_fit_resultsGPU.get());

        cudaCheck(hipGetLastError());
      });

    } else {
      riemannFit::rolling_fits<4, TrackerTraits::maxHitsOnTrackForFullFit, 1>([this,
                                                                               &hv,
                                                                               &tkidGPU,
                                                                               &hitsGPU,
                                                                               &hits_geGPU,
                                                                               &fast_fit_resultsGPU,
                                                                               &offset,
                                                                               &numberOfBlocks,
                                                                               &blockSize,
                                                                               &stream](auto i) {
        kernel_BLFastFit<i, TrackerTraits><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tuples_,
                                                                                         tupleMultiplicity_,
                                                                                         hv,
                                                                                         tkidGPU.get(),
                                                                                         hitsGPU.get(),
                                                                                         hits_geGPU.get(),
                                                                                         fast_fit_resultsGPU.get(),
                                                                                         i,
                                                                                         i,
                                                                                         offset);

        kernel_BLFit<i, TrackerTraits><<<8, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                    bField_,
                                                                    outputSoa_,
                                                                    tkidGPU.get(),
                                                                    hitsGPU.get(),
                                                                    hits_geGPU.get(),
                                                                    fast_fit_resultsGPU.get());
      });

      static_assert(TrackerTraits::maxHitsOnTrackForFullFit < TrackerTraits::maxHitsOnTrack);

      //Fit all the rest using the maximum from previous call
      kernel_BLFastFit<TrackerTraits::maxHitsOnTrackForFullFit, TrackerTraits>
          <<<numberOfBlocks / 4, blockSize, 0, stream>>>(tuples_,
                                                         tupleMultiplicity_,
                                                         hv,
                                                         tkidGPU.get(),
                                                         hitsGPU.get(),
                                                         hits_geGPU.get(),
                                                         fast_fit_resultsGPU.get(),
                                                         TrackerTraits::maxHitsOnTrackForFullFit,
                                                         TrackerTraits::maxHitsOnTrack - 1,
                                                         offset);

      kernel_BLFit<TrackerTraits::maxHitsOnTrackForFullFit, TrackerTraits>
          <<<8, blockSize, 0, stream>>>(tupleMultiplicity_,
                                        bField_,
                                        outputSoa_,
                                        tkidGPU.get(),
                                        hitsGPU.get(),
                                        hits_geGPU.get(),
                                        fast_fit_resultsGPU.get());
    }

  }  // loop on concurrent fits
}

template class HelixFitOnGPU<pixelTopology::Phase1>;
template class HelixFitOnGPU<pixelTopology::Phase2>;
template class HelixFitOnGPU<pixelTopology::HIonPhase1>;
