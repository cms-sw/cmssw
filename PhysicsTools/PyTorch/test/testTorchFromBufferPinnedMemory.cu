#include "hip/hip_runtime.h"
// from https://stackoverflow.com/questions/77390607/how-to-convert-a-cudaarray-to-a-torch-tensor

#include <hip/hip_runtime.h>
#include <torch/torch.h>
#include <iostream>
#include <exception>
#include <memory>
#include <math.h>
#include <cppunit/extensions/HelperMacros.h>

#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

namespace torchtest {

  __global__ void vector_add_kernel(int* a, int* b, int* c, int N) {
    int t_id = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (t_id < N) {
      c[t_id] = a[t_id] + b[t_id];
    }
  }

  void vector_add(int* a, int* b, int* c, int N, int cuda_grid_size, int cuda_block_size) {
    vector_add_kernel<<<cuda_grid_size, cuda_block_size>>>(a, b, c, N);
    hipGetLastError();
  }

}  // namespace torchtest

int main(int argc, const char* argv[]) {
  // temporary workaround to disable test on non-CUDA devices
  if (not cms::cudatest::testDevices())
    return 0;

  // Setup array, here 2^16 = 65536 items
  const int N = 1 << 16;
  size_t bytes = N * sizeof(int);

  // Declare pinned memory pointers
  int *a_cpu, *b_cpu, *c_cpu;

  // Allocate pinned memory for the pointers
  hipHostMalloc(&a_cpu, bytes);
  hipHostMalloc(&b_cpu, bytes);
  hipHostMalloc(&c_cpu, bytes);

  // Init vectors
  for (int i = 0; i < N; ++i) {
    a_cpu[i] = rand() % 100;
    b_cpu[i] = rand() % 100;
  }

  // Declare GPU memory pointers
  int *a_gpu, *b_gpu, *c_gpu;

  // Allocate memory on the device
  hipMalloc(&a_gpu, bytes);
  hipMalloc(&b_gpu, bytes);
  hipMalloc(&c_gpu, bytes);

  // Copy data from the host to the device (CPU -> GPU)
  hipMemcpy(a_gpu, a_cpu, bytes, hipMemcpyHostToDevice);
  hipMemcpy(b_gpu, b_cpu, bytes, hipMemcpyHostToDevice);

  int NUM_THREADS = 1 << 10;
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
  torchtest::vector_add(a_gpu, b_gpu, c_gpu, N, NUM_BLOCKS, NUM_THREADS);

  try {
    // Convert pinned memory on GPU to Torch tensor on GPU
    auto options = torch::TensorOptions().dtype(torch::kInt).device(torch::kCUDA, 0).pinned_memory(true);
    torch::Tensor a_gpu_tensor = torch::from_blob(a_gpu, {N}, options);
    torch::Tensor b_gpu_tensor = torch::from_blob(b_gpu, {N}, options);
    torch::Tensor c_gpu_tensor = torch::from_blob(c_gpu, {N}, options);
  } catch (std::exception& e) {
    std::cout << e.what() << std::endl;

    hipHostFree(a_cpu);
    hipHostFree(b_cpu);
    hipHostFree(c_cpu);

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);

    return 1;
  }

  hipMemcpy(c_cpu, c_gpu, bytes, hipMemcpyDeviceToHost);
  for (int i = 0; i < N; ++i) {
    assert(c_cpu[i] == a_cpu[i] + b_cpu[i]);
  }

  hipHostFree(a_cpu);
  hipHostFree(b_cpu);
  hipHostFree(c_cpu);

  hipFree(a_gpu);
  hipFree(b_gpu);
  hipFree(c_gpu);

  return 0;
}
