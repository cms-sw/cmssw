#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth
 * for pageable and pinned memory, and device to host copy bandwidth for pageable
 * and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization

#include <hip/hip_runtime.h>

#include <memory>
#include <iostream>
#include <cassert>

static const char *sSDKsample = "CUDA Bandwidth Test";

// defines, project
#define MEMCOPY_ITERATIONS  100
#define DEFAULT_SIZE        ( 32 * ( 1 << 20 ) )    //32 M
#define DEFAULT_INCREMENT   (1 << 22)               //4 M
#define CACHE_CLEAR_SIZE    (1 << 24)               //16 M

//shmoo mode defines
#define SHMOO_MEMSIZE_MAX     (1 << 26)         //64 M
#define SHMOO_MEMSIZE_START   (1 << 10)         //1 KB
#define SHMOO_INCREMENT_1KB   (1 << 10)         //1 KB
#define SHMOO_INCREMENT_2KB   (1 << 11)         //2 KB
#define SHMOO_INCREMENT_10KB  (10 * (1 << 10))  //10KB
#define SHMOO_INCREMENT_100KB (100 * (1 << 10)) //100 KB
#define SHMOO_INCREMENT_1MB   (1 << 20)         //1 MB
#define SHMOO_INCREMENT_2MB   (1 << 21)         //2 MB
#define SHMOO_INCREMENT_4MB   (1 << 22)         //4 MB
#define SHMOO_LIMIT_20KB      (20 * (1 << 10))  //20 KB
#define SHMOO_LIMIT_50KB      (50 * (1 << 10))  //50 KB
#define SHMOO_LIMIT_100KB     (100 * (1 << 10)) //100 KB
#define SHMOO_LIMIT_1MB       (1 << 20)         //1 MB
#define SHMOO_LIMIT_16MB      (1 << 24)         //16 MB
#define SHMOO_LIMIT_32MB      (1 << 25)         //32 MB

//enums, project
enum testMode   { QUICK_MODE, RANGE_MODE, SHMOO_MODE };
enum memcpyKind { DEVICE_TO_HOST, HOST_TO_DEVICE, DEVICE_TO_DEVICE };
enum printMode  { USER_READABLE, CSV };
enum memoryMode { PINNED, PAGEABLE };

const char *sMemoryCopyKind[] =
{
    "Device to Host",
    "Host to Device",
    "Device to Device",
    NULL
};

const char *sMemoryMode[] =
{
    "PINNED",
    "PAGEABLE",
    NULL
};

// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;

int *pArgc = NULL;
char **pArgv = NULL;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest(const int argc, const char **argv);
void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthRange(unsigned int start, unsigned int end, unsigned int increment,
                        memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthShmoo(memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode, bool wc);
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc);
float testDeviceToDeviceTransfer(unsigned int memSize);
void printResultsReadable(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc);
void printResultsCSV(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc);
void printHelp(void);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    pArgc = &argc;
    pArgv = argv;

    // set logfile name and start logs
    printf("[%s] - Starting...\n", sSDKsample);

    int iRetVal = runTest(argc, (const char **)argv);

    if (iRetVal < 0)
    {
        checkCudaErrors(hipSetDevice(0));
    }

    // finish
    printf("%s\n", (iRetVal==0) ? "Result = PASS" : "Result = FAIL");

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    exit((iRetVal==0) ? EXIT_SUCCESS : EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////
//Parse args, run the appropriate tests
///////////////////////////////////////////////////////////////////////////////
int runTest(const int argc, const char **argv)
{
    int start = DEFAULT_SIZE;
    int end = DEFAULT_SIZE;
    int startDevice = 0;
    int endDevice = 0;
    int increment = DEFAULT_INCREMENT;
    testMode mode = QUICK_MODE;
    bool htod = false;
    bool dtoh = false;
    bool dtod = false;
    bool wc = false;
    char *modeStr;
    char *device = NULL;
    printMode printmode = USER_READABLE;
    char *memModeStr = NULL;
    memoryMode memMode = PINNED;

    //process command line args
    if (checkCmdLineFlag(argc, argv, "help"))
    {
        printHelp();
        return 0;
    }

    if (checkCmdLineFlag(argc, argv, "csv"))
    {
        printmode = CSV;
    }

    if (getCmdLineArgumentString(argc, argv, "memory", &memModeStr))
    {
        if (strcmp(memModeStr, "pageable") == 0)
        {
            memMode = PAGEABLE;
        }
        else if (strcmp(memModeStr, "pinned") == 0)
        {
            memMode = PINNED;
        }
        else
        {
            printf("Invalid memory mode - valid modes are pageable or pinned\n");
            printf("See --help for more information\n");
            return -1000;
        }
    }
    else
    {
        //default - pinned memory
        memMode = PINNED;
    }

    if (getCmdLineArgumentString(argc, argv, "device", &device))
    {
        int deviceCount;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);

        if (error_id != hipSuccess)
        {
            printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
            exit(EXIT_FAILURE);
        }

        if (deviceCount == 0)
        {
            printf("!!!!!No devices found!!!!!\n");
            return -2000;
        }

        if (strcmp(device, "all") == 0)
        {
            printf("\n!!!!!Cumulative Bandwidth to be computed from all the devices !!!!!!\n\n");
            startDevice = 0;
            endDevice = deviceCount-1;
        }
        else
        {
            startDevice = endDevice = atoi(device);

            if (startDevice >= deviceCount || startDevice < 0)
            {
                printf("\n!!!!!Invalid GPU number %d given hence default gpu %d will be used !!!!!\n", startDevice,0);
                startDevice = endDevice = 0;
            }
        }
    }

    printf("Running on...\n\n");

    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipDeviceProp_t deviceProp;
        hipError_t error_id = hipGetDeviceProperties(&deviceProp, currentDevice);

        if (error_id == hipSuccess)
        {
            printf(" Device %d: %s\n", currentDevice, deviceProp.name);

            if (deviceProp.computeMode == hipComputeModeProhibited)
            {
                fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
                checkCudaErrors(hipSetDevice(currentDevice));

                exit(EXIT_FAILURE);
            }
        }
        else
        {
            printf("hipGetDeviceProperties returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
            checkCudaErrors(hipSetDevice(currentDevice));

            exit(EXIT_FAILURE);
        }
    }

    if (getCmdLineArgumentString(argc, argv, "mode", &modeStr))
    {
        //figure out the mode
        if (strcmp(modeStr, "quick") == 0)
        {
            printf(" Quick Mode\n\n");
            mode = QUICK_MODE;
        }
        else if (strcmp(modeStr, "shmoo") == 0)
        {
            printf(" Shmoo Mode\n\n");
            mode = SHMOO_MODE;
        }
        else if (strcmp(modeStr, "range") == 0)
        {
            printf(" Range Mode\n\n");
            mode = RANGE_MODE;
        }
        else
        {
            printf("Invalid mode - valid modes are quick, range, or shmoo\n");
            printf("See --help for more information\n");
            return -3000;
        }
    }
    else
    {
        //default mode - quick
        printf(" Quick Mode\n\n");
        mode = QUICK_MODE;
    }

    if (checkCmdLineFlag(argc, argv, "htod"))
    {
        htod = true;
    }

    if (checkCmdLineFlag(argc, argv, "dtoh"))
    {
        dtoh = true;
    }

    if (checkCmdLineFlag(argc, argv, "dtod"))
    {
        dtod = true;
    }

#if CUDART_VERSION >= 2020

    if (checkCmdLineFlag(argc, argv, "wc"))
    {
        wc = true;
    }

#endif

    if (checkCmdLineFlag(argc, argv, "cputiming"))
    {
        bDontUseGPUTiming = true;
    }

    if (!htod && !dtoh && !dtod)
    {
        //default:  All
        htod = true;
        dtoh = true;
        dtod = true;
    }

    if (RANGE_MODE == mode)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "start"))
        {
            start = getCmdLineArgumentInt(argc, argv, "start");

            if (start <= 0)
            {
                printf("Illegal argument - start must be greater than zero\n");
                return -4000;
            }
        }
        else
        {
            printf("Must specify a starting size in range mode\n");
            printf("See --help for more information\n");
            return -5000;
        }

        if (checkCmdLineFlag(argc, (const char **)argv, "end"))
        {
            end = getCmdLineArgumentInt(argc, argv, "end");

            if (end <= 0)
            {
                printf("Illegal argument - end must be greater than zero\n");
                return -6000;
            }

            if (start > end)
            {
                printf("Illegal argument - start is greater than end\n");
                return -7000;
            }
        }
        else
        {
            printf("Must specify an end size in range mode.\n");
            printf("See --help for more information\n");
            return -8000;
        }


        if (checkCmdLineFlag(argc, argv, "increment"))
        {
            increment = getCmdLineArgumentInt(argc, argv, "increment");

            if (increment <= 0)
            {
                printf("Illegal argument - increment must be greater than zero\n");
                return -9000;
            }
        }
        else
        {
            printf("Must specify an increment in user mode\n");
            printf("See --help for more information\n");
            return -10000;
        }
    }

    if (htod)
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                      mode, HOST_TO_DEVICE, printmode, memMode, startDevice, endDevice, wc);
    }

    if (dtoh)
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                      mode, DEVICE_TO_HOST, printmode, memMode, startDevice, endDevice, wc);
    }

    if (dtod)
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                      mode, DEVICE_TO_DEVICE, printmode, memMode, startDevice, endDevice, wc);
    }

    // Ensure that we reset all CUDA Devices in question
    for (int nDevice = startDevice; nDevice <= endDevice; nDevice++)
    {
        hipSetDevice(nDevice);
    }

    return 0;
}

///////////////////////////////////////////////////////////////////////////////
//  Run a bandwidth test
///////////////////////////////////////////////////////////////////////////////
void
testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
              testMode mode, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    switch (mode)
    {
        case QUICK_MODE:
            testBandwidthQuick(DEFAULT_SIZE, kind, printmode, memMode, startDevice, endDevice, wc);
            break;

        case RANGE_MODE:
            testBandwidthRange(start, end, increment, kind, printmode, memMode, startDevice, endDevice, wc);
            break;

        case SHMOO_MODE:
            testBandwidthShmoo(kind, printmode, memMode, startDevice, endDevice, wc);
            break;

        default:
            break;
    }
}

//////////////////////////////////////////////////////////////////////
//  Run a quick mode bandwidth test
//////////////////////////////////////////////////////////////////////
void
testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    testBandwidthRange(size, size, DEFAULT_INCREMENT, kind, printmode, memMode, startDevice, endDevice, wc);
}

///////////////////////////////////////////////////////////////////////
//  Run a range mode bandwidth test
//////////////////////////////////////////////////////////////////////
void
testBandwidthRange(unsigned int start, unsigned int end, unsigned int increment,
                   memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    //count the number of copies we're going to run
    unsigned int count = 1 + ((end - start) / increment);

    unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
    double *bandwidths = (double *) malloc(count * sizeof(double));

    // Before calculating the cumulative bandwidth, initialize bandwidths array to NULL
    for (unsigned int i = 0; i < count; i++)
    {
        bandwidths[i] = 0.0;
    }

    // Use the device asked by the user
    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipSetDevice(currentDevice);

        //run each of the copies
        for (unsigned int i = 0; i < count; i++)
        {

            memSizes[i] = start + i * increment;

            switch (kind)
            {
                case DEVICE_TO_HOST:
                    bandwidths[i] += testDeviceToHostTransfer(memSizes[i], memMode, wc);
                    break;

                case HOST_TO_DEVICE:
                    bandwidths[i] += testHostToDeviceTransfer(memSizes[i], memMode, wc);
                    break;

                case DEVICE_TO_DEVICE:
                    bandwidths[i] += testDeviceToDeviceTransfer(memSizes[i]);
                    break;
            }
        }
    } // Complete the bandwidth computation on all the devices

    //print results
    if (printmode == CSV)
    {
        printResultsCSV(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }
    else
    {
        printResultsReadable(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }

    //clean up
    free(memSizes);
    free(bandwidths);
}

//////////////////////////////////////////////////////////////////////////////
// Intense shmoo mode - covers a large range of values with varying increments
//////////////////////////////////////////////////////////////////////////////
void
testBandwidthShmoo(memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    //count the number of copies to make
    unsigned int count = 1 + (SHMOO_LIMIT_20KB  / SHMOO_INCREMENT_1KB)
                         + ((SHMOO_LIMIT_50KB - SHMOO_LIMIT_20KB) / SHMOO_INCREMENT_2KB)
                         + ((SHMOO_LIMIT_100KB - SHMOO_LIMIT_50KB) / SHMOO_INCREMENT_10KB)
                         + ((SHMOO_LIMIT_1MB - SHMOO_LIMIT_100KB) / SHMOO_INCREMENT_100KB)
                         + ((SHMOO_LIMIT_16MB - SHMOO_LIMIT_1MB) / SHMOO_INCREMENT_1MB)
                         + ((SHMOO_LIMIT_32MB - SHMOO_LIMIT_16MB) / SHMOO_INCREMENT_2MB)
                         + ((SHMOO_MEMSIZE_MAX - SHMOO_LIMIT_32MB) / SHMOO_INCREMENT_4MB);

    unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
    double *bandwidths = (double *) malloc(count * sizeof(double));

    // Before calculating the cumulative bandwidth, initialize bandwidths array to NULL
    for (unsigned int i = 0; i < count; i++)
    {
        bandwidths[i] = 0.0;
    }

    // Use the device asked by the user
    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipSetDevice(currentDevice);
        //Run the shmoo
        int iteration = 0;
        unsigned int memSize = 0;

        while (memSize <= SHMOO_MEMSIZE_MAX)
        {
            if (memSize < SHMOO_LIMIT_20KB)
            {
                memSize += SHMOO_INCREMENT_1KB;
            }
            else if (memSize < SHMOO_LIMIT_50KB)
            {
                memSize += SHMOO_INCREMENT_2KB;
            }
            else if (memSize < SHMOO_LIMIT_100KB)
            {
                memSize += SHMOO_INCREMENT_10KB;
            }
            else if (memSize < SHMOO_LIMIT_1MB)
            {
                memSize += SHMOO_INCREMENT_100KB;
            }
            else if (memSize < SHMOO_LIMIT_16MB)
            {
                memSize += SHMOO_INCREMENT_1MB;
            }
            else if (memSize < SHMOO_LIMIT_32MB)
            {
                memSize += SHMOO_INCREMENT_2MB;
            }
            else
            {
                memSize += SHMOO_INCREMENT_4MB;
            }

            memSizes[iteration] = memSize;

            switch (kind)
            {
                case DEVICE_TO_HOST:
                    bandwidths[iteration] += testDeviceToHostTransfer(memSizes[iteration], memMode, wc);
                    break;

                case HOST_TO_DEVICE:
                    bandwidths[iteration] += testHostToDeviceTransfer(memSizes[iteration], memMode, wc);
                    break;

                case DEVICE_TO_DEVICE:
                    bandwidths[iteration] += testDeviceToDeviceTransfer(memSizes[iteration]);
                    break;
            }

            iteration++;
            printf(".");
        }
    } // Complete the bandwidth computation on all the devices

    //print results
    printf("\n");

    if (CSV == printmode)
    {
        printResultsCSV(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }
    else
    {
        printResultsReadable(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }

    //clean up
    free(memSizes);
    free(bandwidths);
}

///////////////////////////////////////////////////////////////////////////////
//  test the bandwidth of a device to host memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode, bool wc)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    unsigned char *h_idata = NULL;
    unsigned char *h_odata = NULL;
    hipEvent_t start, stop;

    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    //allocate host memory
    if (PINNED == memMode)
    {
        //pinned memory mode - use special function to get OS-pinned memory
#if CUDART_VERSION >= 2020
        checkCudaErrors(hipHostAlloc((void **)&h_idata, memSize, (wc) ? hipHostMallocWriteCombined : 0));
        checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize, (wc) ? hipHostMallocWriteCombined : 0));
#else
        checkCudaErrors(hipHostMalloc((void **)&h_idata, memSize));
        checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
    }
    else
    {
        //pageable memory mode - use malloc
        h_idata = (unsigned char *)malloc(memSize);
        h_odata = (unsigned char *)malloc(memSize);

        if (h_idata == 0 || h_odata == 0)
        {
            fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
            exit(EXIT_FAILURE);
        }
    }

    //initialize the memory
    for (unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_idata[i] = (unsigned char)(i & 0xff);
    }

    // allocate device memory
    unsigned char *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, memSize));

    //initialize the device memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, memSize,
                               hipMemcpyHostToDevice));

    //copy data from GPU to Host
    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    if (PINNED == memMode)
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpyAsync(h_odata, d_idata, memSize,
                                            hipMemcpyDeviceToHost, 0));
        }
    }
    else
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpy(h_odata, d_idata, memSize,
                                       hipMemcpyDeviceToHost));
        }
    }

    checkCudaErrors(hipEventRecord(stop, 0));

    // make sure GPU has finished copying
    checkCudaErrors(hipDeviceSynchronize());
    //get the total elapsed time in ms
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    if (PINNED != memMode || bDontUseGPUTiming)
    {
        elapsedTimeInMs = sdkGetTimerValue(&timer);
    }

    //calculate bandwidth in MB/s
    bandwidthInMBs = ((float)(1<<10) * memSize * (float)MEMCOPY_ITERATIONS) /
                     (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    sdkDeleteTimer(&timer);

    if (PINNED == memMode)
    {
        checkCudaErrors(hipHostFree(h_idata));
        checkCudaErrors(hipHostFree(h_odata));
    }
    else
    {
        free(h_idata);
        free(h_odata);
    }

    checkCudaErrors(hipFree(d_idata));

    return bandwidthInMBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;
    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    //allocate host memory
    unsigned char *h_odata = NULL;

    if (PINNED == memMode)
    {
#if CUDART_VERSION >= 2020
        //pinned memory mode - use special function to get OS-pinned memory
        checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize, (wc) ? hipHostMallocWriteCombined : 0));
#else
        //pinned memory mode - use special function to get OS-pinned memory
        checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
    }
    else
    {
        //pageable memory mode - use malloc
        h_odata = (unsigned char *)malloc(memSize);

        if (h_odata == 0)
        {
            fprintf(stderr, "Not enough memory available on host to run test!\n");
            exit(EXIT_FAILURE);
        }
    }

    unsigned char *h_cacheClear1 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);
    unsigned char *h_cacheClear2 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);

    if (h_cacheClear1 == 0 || h_cacheClear2 == 0)
    {
        fprintf(stderr, "Not enough memory available on host to run test!\n");
        exit(EXIT_FAILURE);
    }

    //initialize the memory
    for (unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_odata[i] = (unsigned char)(i & 0xff);
    }

    for (unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(unsigned char); i++)
    {
        h_cacheClear1[i] = (unsigned char)(i & 0xff);
        h_cacheClear2[i] = (unsigned char)(0xff - (i & 0xff));
    }

    //allocate device memory
    unsigned char *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, memSize));

    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    //copy host memory to device memory
    if (PINNED == memMode)
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpyAsync(d_idata, h_odata, memSize,
                                            hipMemcpyHostToDevice, 0));
        }
    }
    else
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpy(d_idata, h_odata, memSize,
                                       hipMemcpyHostToDevice));
        }
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    //total elapsed time in ms
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    if (PINNED != memMode || bDontUseGPUTiming)
    {
        elapsedTimeInMs = sdkGetTimerValue(&timer);
    }

    sdkResetTimer(&timer);

    //calculate bandwidth in MB/s
    bandwidthInMBs = ((float)(1<<10) * memSize * (float)MEMCOPY_ITERATIONS) /
                     (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    sdkDeleteTimer(&timer);

    if (PINNED == memMode)
    {
        checkCudaErrors(hipHostFree(h_odata));
    }
    else
    {
        free(h_odata);
    }

    free(h_cacheClear1);
    free(h_cacheClear2);
    checkCudaErrors(hipFree(d_idata));

    return bandwidthInMBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testDeviceToDeviceTransfer(unsigned int memSize)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;

    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    //allocate host memory
    unsigned char *h_idata = (unsigned char *)malloc(memSize);

    if (h_idata == 0)
    {
        fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
        exit(EXIT_FAILURE);
    }

    //initialize the host memory
    for (unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_idata[i] = (unsigned char)(i & 0xff);
    }

    //allocate device memory
    unsigned char *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, memSize));
    unsigned char *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, memSize));

    //initialize memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, memSize,
                               hipMemcpyHostToDevice));

    //run the memcopy
    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
        checkCudaErrors(hipMemcpy(d_odata, d_idata, memSize,
                                   hipMemcpyDeviceToDevice));
    }

    checkCudaErrors(hipEventRecord(stop, 0));

    //Since device to device memory copies are non-blocking,
    //hipDeviceSynchronize() is required in order to get
    //proper timing.
    checkCudaErrors(hipDeviceSynchronize());

    //get the total elapsed time in ms
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    if (bDontUseGPUTiming)
    {
        elapsedTimeInMs = sdkGetTimerValue(&timer);
    }

    //calculate bandwidth in MB/s
    bandwidthInMBs = 2.0f * ((float)(1<<10) * memSize * (float)MEMCOPY_ITERATIONS) /
                     (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    sdkDeleteTimer(&timer);
    free(h_idata);
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return bandwidthInMBs;
}

/////////////////////////////////////////////////////////
//print results in an easily read format
////////////////////////////////////////////////////////
void printResultsReadable(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc)
{
    printf(" %s Bandwidth, %i Device(s)\n", sMemoryCopyKind[kind], iNumDevs);
    printf(" %s Memory Transfers\n", sMemoryMode[memMode]);

    if (wc)
    {
        printf(" Write-Combined Memory Writes are Enabled");
    }

    printf("   Transfer Size (Bytes)\tBandwidth(MB/s)\n");
    unsigned int i;

    for (i = 0; i < (count - 1); i++)
    {
        printf("   %u\t\t\t%s%.1f\n", memSizes[i], (memSizes[i] < 10000)? "\t" : "", bandwidths[i]);
    }

    printf("   %u\t\t\t%s%.1f\n\n", memSizes[i], (memSizes[i] < 10000)? "\t" : "", bandwidths[i]);
}

///////////////////////////////////////////////////////////////////////////
//print results in a database format
///////////////////////////////////////////////////////////////////////////
void printResultsCSV(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc)
{
    std::string sConfig;

    // log config information
    if (kind == DEVICE_TO_DEVICE)
    {
        sConfig += "D2D";
    }
    else
    {
        if (kind == DEVICE_TO_HOST)
        {
            sConfig += "D2H";
        }
        else if (kind == HOST_TO_DEVICE)
        {
            sConfig += "H2D";
        }

        if (memMode == PAGEABLE)
        {
            sConfig += "-Paged";
        }
        else if (memMode == PINNED)
        {
            sConfig += "-Pinned";

            if (wc)
            {
                sConfig += "-WriteCombined";
            }
        }
    }

    unsigned int i;
    double dSeconds = 0.0;

    for (i = 0; i < count; i++)
    {
        dSeconds = (double)memSizes[i] / (bandwidths[i] * (double)(1<<20));
        printf("bandwidthTest-%s, Bandwidth = %.1f MB/s, Time = %.5f s, Size = %u bytes, NumDevsUsed = %d\n",
               sConfig.c_str(), bandwidths[i], dSeconds, memSizes[i], iNumDevs);
    }
}

///////////////////////////////////////////////////////////////////////////
//Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void)
{
    printf("Usage:  bandwidthTest [OPTION]...\n");
    printf("Test the bandwidth for device to host, host to device, and device to device transfers\n");
    printf("\n");
    printf("Example:  measure the bandwidth of device to host pinned memory copies in the range 1024 Bytes to 102400 Bytes in 1024 Byte increments\n");
    printf("./bandwidthTest --memory=pinned --mode=range --start=1024 --end=102400 --increment=1024 --dtoh\n");

    printf("\n");
    printf("Options:\n");
    printf("--help\tDisplay this help menu\n");
    printf("--csv\tPrint results as a CSV\n");
    printf("--device=[deviceno]\tSpecify the device device to be used\n");
    printf("  all - compute cumulative bandwidth on all the devices\n");
    printf("  0,1,2,...,n - Specify any particular device to be used\n");
    printf("--memory=[MEMMODE]\tSpecify which memory mode to use\n");
    printf("  pageable - pageable memory\n");
    printf("  pinned   - non-pageable system memory\n");
    printf("--mode=[MODE]\tSpecify the mode to use\n");
    printf("  quick - performs a quick measurement\n");
    printf("  range - measures a user-specified range of values\n");
    printf("  shmoo - performs an intense shmoo of a large range of values\n");

    printf("--htod\tMeasure host to device transfers\n");
    printf("--dtoh\tMeasure device to host transfers\n");
    printf("--dtod\tMeasure device to device transfers\n");
#if CUDART_VERSION >= 2020
    printf("--wc\tAllocate pinned memory as write-combined\n");
#endif
    printf("--cputiming\tForce CPU-based timing always\n");

    printf("Range mode options\n");
    printf("--start=[SIZE]\tStarting transfer size in bytes\n");
    printf("--end=[SIZE]\tEnding transfer size in bytes\n");
    printf("--increment=[SIZE]\tIncrement size in bytes\n");
}
