#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <cstdio>
#include <vector>

#include <hip/hip_runtime_api.h>

using namespace std;

const char *sSampleName = "P2P (Peer-to-Peer) GPU Bandwidth Latency Test";

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }
__global__ void delay(int * null) {
  float j=threadIdx.x;
  for(int i=1;i<10000;i++)
      j=(j+1)/j;

  if(threadIdx.x == j) null[0] = j;
}

void checkP2Paccess(int numGPUs)
{
    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access,i,j);
                printf("Device=%d %s Access Peer Device=%d\n", i, access ? "CAN" : "CANNOT", j);
            }
        }
    }
    printf("\n***NOTE: In case a device doesn't have P2P access to other one, it falls back to normal memcopy procedure.\nSo you can see lesser Bandwidth (GB/s) in those cases.\n\n");
}

void outputBandwidthMatrix(int numGPUs, bool p2p)
{
    int numElems=10000000;
    int repeat=5;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],numElems*sizeof(int));
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> bandwidthMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            if(p2p) {
                hipDeviceCanAccessPeer(&access,i,j);
                if (access)
                {
                    hipDeviceEnablePeerAccess(j,0 );
                    cudaCheckError();
                }
            }

            hipDeviceSynchronize();
            cudaCheckError();
            delay<<<1,1>>>((int *)NULL);
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,sizeof(int)*numElems);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);
            double time_s=time_ms/1e3;

            double gb=numElems*sizeof(int)*repeat/(double)1e9;
            if(i==j) gb*=2;  //must count both the read and the write here
            bandwidthMatrix[i*numGPUs+j]=gb/time_s;
            if (p2p && access)
            {
                hipDeviceDisablePeerAccess(j);
                cudaCheckError();
            }
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", bandwidthMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
}

void outputBidirectionalBandwidthMatrix(int numGPUs, bool p2p)
{
    int numElems=10000000;
    int repeat=5;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);
    vector<hipStream_t> stream0(numGPUs);
    vector<hipStream_t> stream1(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],numElems*sizeof(int));
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
        hipStreamCreate(&stream0[d]);
        cudaCheckError();
        hipStreamCreate(&stream1[d]);
        cudaCheckError();
    }

    vector<double> bandwidthMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            if(p2p) {
                hipDeviceCanAccessPeer(&access,i,j);
                if (access)
                {
                    hipSetDevice(i);
                    hipDeviceEnablePeerAccess(j,0);
                    cudaCheckError();
                    hipSetDevice(j);
                    hipDeviceEnablePeerAccess(i,0);
                    cudaCheckError();
                }
            }

            hipSetDevice(i);
            hipDeviceSynchronize();
            cudaCheckError();
            delay<<<1,1>>>((int *)NULL);
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,sizeof(int)*numElems,stream0[i]);
                hipMemcpyPeerAsync(buffers[j],j,buffers[i],i,sizeof(int)*numElems,stream1[i]);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);
            double time_s=time_ms/1e3;

            double gb=2.0*numElems*sizeof(int)*repeat/(double)1e9;
            if(i==j) gb*=2;  //must count both the read and the write here
            bandwidthMatrix[i*numGPUs+j]=gb/time_s;
            if(p2p && access)
            {
                hipSetDevice(i);
                hipDeviceDisablePeerAccess(j);
                hipSetDevice(j);
                hipDeviceDisablePeerAccess(i);
            }
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", bandwidthMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
        hipStreamDestroy(stream0[d]);
        cudaCheckError();
        hipStreamDestroy(stream1[d]);
        cudaCheckError();
    }
}

void outputLatencyMatrix(int numGPUs, bool p2p)
{
    int repeat=10000;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],1);
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> latencyMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            if(p2p) {
                hipDeviceCanAccessPeer(&access,i,j);
                if (access)
                {
                    hipDeviceEnablePeerAccess(j,0);
                    cudaCheckError();
                }
            }
            hipDeviceSynchronize();
            cudaCheckError();
            delay<<<1,1>>>((int *)NULL);
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,1);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);

            latencyMatrix[i*numGPUs+j]=time_ms*1e3/repeat;
            if(p2p && access)
            {
                hipDeviceDisablePeerAccess(j);
            }
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", latencyMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
}

int main(int argc, char **argv)
{

    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    printf("[%s]\n", sSampleName);

    //output devices
    for (int i=0; i<numGPUs; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);
        printf("Device: %d, %s, pciBusID: %x, pciDeviceID: %x, pciDomainID:%x\n",i,prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);
    }

    checkP2Paccess(numGPUs);

    //Check peer-to-peer connectivity
    printf("P2P Connectivity Matrix\n");
    printf("     D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d", j);
    }
    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d\t", i);
        for (int j=0; j<numGPUs; j++)
        {
            if (i!=j)
            {
               int access;
               hipDeviceCanAccessPeer(&access,i,j);
               printf("%6d", (access) ? 1 : 0);
            }
            else
            {
                printf("%6d", 1);
            }
        }
        printf("\n");
    }

    printf("Unidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
    outputBandwidthMatrix(numGPUs, false);
    printf("Unidirectional P2P=Enabled Bandwidth Matrix (GB/s)\n");
    outputBandwidthMatrix(numGPUs, true);
    printf("Bidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
    outputBidirectionalBandwidthMatrix(numGPUs, false);
    printf("Bidirectional P2P=Enabled Bandwidth Matrix (GB/s)\n");
    outputBidirectionalBandwidthMatrix(numGPUs, true);


    printf("P2P=Disabled Latency Matrix (us)\n");
    outputLatencyMatrix(numGPUs, false);
    printf("P2P=Enabled Latency Matrix (us)\n");
    outputLatencyMatrix(numGPUs, true);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    exit(EXIT_SUCCESS);
}
