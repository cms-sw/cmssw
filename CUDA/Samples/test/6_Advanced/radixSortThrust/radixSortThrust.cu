#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

#include <hip/hip_runtime_api.h>

#include <algorithm>
#include <time.h>
#include <limits.h>

template <typename T, bool floatKeys>
bool testSort(int argc, char **argv)
{
    int cmdVal;
    int keybits = 32;

    unsigned int numElements = 1048576;
    bool keysOnly = checkCmdLineFlag(argc, (const char **)argv, "keysonly");
    bool quiet    = checkCmdLineFlag(argc, (const char **)argv, "quiet");

    if (checkCmdLineFlag(argc, (const char **)argv, "n"))
    {
        cmdVal = getCmdLineArgumentInt(argc, (const char **)argv, "n");
        numElements = cmdVal;

        if (cmdVal < 0)
        {
            printf("Error: elements must be > 0, elements=%d is invalid\n", cmdVal);
            exit(EXIT_SUCCESS);
        }
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "keybits"))
    {
        cmdVal = getCmdLineArgumentInt(argc, (const char **)argv, "keybits");
        keybits = cmdVal;

        if (keybits <= 0)
        {
            printf("Error: keybits must be > 0, keybits=%d is invalid\n", keybits);
            exit(EXIT_SUCCESS);
        }
    }

    unsigned int numIterations = (numElements >= 16777216) ? 10 : 100;

    if (checkCmdLineFlag(argc, (const char **)argv, "iterations"))
    {
        cmdVal = getCmdLineArgumentInt(argc, (const char **)argv, "iterations");
        numIterations = cmdVal;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "help"))
    {
        printf("Command line:\nradixSortThrust [-option]\n");
        printf("Valid options:\n");
        printf("-n=<N>        : number of elements to sort\n");
        printf("-keybits=bits : keybits must be > 0\n");
        printf("-keysonly     : only sort an array of keys (default sorts key-value pairs)\n");
        printf("-float        : use 32-bit float keys (default is 32-bit unsigned int)\n");
        printf("-quiet        : Output only the number of elements and the time to sort\n");
        printf("-help         : Output a help message\n");
        exit(EXIT_SUCCESS);
    }

    if (!quiet)
        printf("\nSorting %d %d-bit %s keys %s\n\n", numElements, keybits, floatKeys ? "float" : "unsigned int", keysOnly ? "(only)" : "and values");

    int deviceID = -1;

    if (hipSuccess == hipGetDevice(&deviceID))
    {
        hipDeviceProp_t devprop;
        hipGetDeviceProperties(&devprop, deviceID);
        unsigned int totalMem = (keysOnly ? 2 : 4) * numElements * sizeof(T);

        if (devprop.totalGlobalMem < totalMem)
        {
            printf("Error: insufficient amount of memory to sort %d elements.\n", numElements);
            printf("%d bytes needed, %d bytes available\n", (int) totalMem, (int) devprop.totalGlobalMem);
            exit(EXIT_SUCCESS);
        }
    }

    thrust::host_vector<T> h_keys(numElements);
    thrust::host_vector<T> h_keysSorted(numElements);
    thrust::host_vector<unsigned int> h_values;

    if (!keysOnly)
        h_values = thrust::host_vector<unsigned int>(numElements);

    // Fill up with some random data
    thrust::default_random_engine rng(clock());

    if (floatKeys)
    {
        thrust::uniform_real_distribution<float> u01(0, 1);

        for (int i = 0; i < (int)numElements; i++)
            h_keys[i] = u01(rng);
    }
    else
    {
        thrust::uniform_int_distribution<unsigned int> u(0, UINT_MAX);

        for (int i = 0; i < (int)numElements; i++)
            h_keys[i] = u(rng);
    }

    if (!keysOnly)
        thrust::sequence(h_values.begin(), h_values.end());

    // Copy data onto the GPU
    thrust::device_vector<T> d_keys;
    thrust::device_vector<unsigned int> d_values;

    // run multiple iterations to compute an average sort time
    hipEvent_t start_event, stop_event;
    checkCudaErrors(hipEventCreate(&start_event));
    checkCudaErrors(hipEventCreate(&stop_event));

    float totalTime = 0;

    for (unsigned int i = 0; i < numIterations; i++)
    {
        // reset data before sort
        d_keys= h_keys;

        if (!keysOnly)
            d_values = h_values;

        checkCudaErrors(hipEventRecord(start_event, 0));

        if (keysOnly)
            thrust::sort(d_keys.begin(), d_keys.end());
        else
            thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_values.begin());

        checkCudaErrors(hipEventRecord(stop_event, 0));
        checkCudaErrors(hipEventSynchronize(stop_event));

        float time = 0;
        checkCudaErrors(hipEventElapsedTime(&time, start_event, stop_event));
        totalTime += time;
    }

    totalTime /= (1.0e3f * numIterations);
    printf("radixSortThrust, Throughput = %.4f MElements/s, Time = %.5f s, Size = %u elements\n",
           1.0e-6f * numElements / totalTime, totalTime, numElements);

    getLastCudaError("after radixsort");

    // Get results back to host for correctness checking
    thrust::copy(d_keys.begin(), d_keys.end(), h_keysSorted.begin());

    if (!keysOnly)
        thrust::copy(d_values.begin(), d_values.end(), h_values.begin());

    getLastCudaError("copying results to host memory");

    // Check results
    bool bTestResult = thrust::is_sorted(h_keysSorted.begin(), h_keysSorted.end());

    checkCudaErrors(hipEventDestroy(start_event));
    checkCudaErrors(hipEventDestroy(stop_event));

    if (!bTestResult  && !quiet)
    {
        return false;
    }

    return bTestResult;
}

int main(int argc, char **argv)
{
    // Start logs
    printf("%s Starting...\n\n", argv[0]);

    findCudaDevice(argc, (const char **)argv);

    bool bTestResult = false;

    if (checkCmdLineFlag(argc, (const char **)argv, "float"))
        bTestResult = testSort<float, true>(argc, argv);
    else
        bTestResult = testSort<unsigned int, false>(argc, argv);

    printf(bTestResult ? "Test passed\n" : "Test failed!\n");
}

