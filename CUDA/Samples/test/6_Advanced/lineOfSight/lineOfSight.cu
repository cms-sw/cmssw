#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This sample is an implementation of a simple line-of-sight algorithm:
// Given a height map and a ray originating at some observation point,
// it computes all the points along the ray that are visible from the
// observation point.
// It is based on the description made in "Guy E. Blelloch.  Vector models
// for data-parallel computing. MIT Press, 1990" and uses open source CUDA
// Thrust Library

#ifdef _WIN32
#  define NOMINMAX
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

// includes, library
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/copy.h>

////////////////////////////////////////////////////////////////////////////////
// declaration, types

// Boolean
typedef unsigned char Bool;
enum
{
    False = 0,
    True = 1
};

// 2D height field
struct HeightField
{
    int     width;
    float  *height;
};

// Ray
struct Ray
{
    float3 origin;
    float2 dir;
    int    length;
    float  oneOverLength;
};

////////////////////////////////////////////////////////////////////////////////
// declaration, variables

// Height field texture reference
texture<float, 2, hipReadModeElementType> g_HeightFieldTex;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest(int argc, char **argv);
__global__ void computeAngles_kernel(const Ray, float *);
__global__ void computeVisibilities_kernel(const float *, const float *, int, Bool *);
void lineOfSight_gold(const HeightField, const Ray, Bool *);
__device__ __host__ float2 getLocation(const Ray, int);
__device__ __host__ float getAngle(const Ray, float2, float);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    int res = runTest(argc, argv);

    if (res != 1)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    exit(EXIT_SUCCESS);

}

////////////////////////////////////////////////////////////////////////////////
//! Run a line-of-sight test for CUDA
////////////////////////////////////////////////////////////////////////////////
int runTest(int argc, char **argv)
{
    ////////////////////////////////////////////////////////////////////////////
    // Device initialization

    printf("[%s] - Starting...\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    ////////////////////////////////////////////////////////////////////////////
    // Timer

    // Create
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    // Number of iterations to get accurate timing
    uint numIterations = 100;

    ////////////////////////////////////////////////////////////////////////////
    // Height field

    HeightField heightField;

    // Allocate in host memory
    int2 dim = make_int2(10000, 100);
    heightField.width = dim.x;
    thrust::host_vector<float> height(dim.x * dim.y);
    heightField.height = (float *)&height[0];

    //
    // Fill in with an arbitrary sine surface
    for (int x = 0; x < dim.x; ++x)
        for (int y = 0; y < dim.y; ++y)
        {
            float amp = 0.1f * (x + y);
            float period = 2.0f + amp;
            *(heightField.height + dim.x * y + x) =
                amp * (sinf(sqrtf((float)(x * x + y * y)) * 2.0f * 3.1416f / period) + 1.0f);
        }

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *heightFieldArray;
    checkCudaErrors(hipMallocArray(&heightFieldArray, &channelDesc, dim.x, dim.y));

    // Initialize device memory
    checkCudaErrors(hipMemcpyToArray(heightFieldArray, 0, 0, heightField.height,
                                      dim.x * dim.y * sizeof(float), hipMemcpyHostToDevice));

    // Set texture parameters
    g_HeightFieldTex.addressMode[0] = hipAddressModeClamp;
    g_HeightFieldTex.addressMode[1] = hipAddressModeClamp;
    g_HeightFieldTex.filterMode = hipFilterModePoint;
    g_HeightFieldTex.normalized = 0;

    // Bind CUDA array to texture reference
    checkCudaErrors(hipBindTextureToArray(g_HeightFieldTex, heightFieldArray,
                                           channelDesc));

    ////////////////////////////////////////////////////////////////////////////
    // Ray (starts at origin and traverses the height field diagonally)

    Ray ray;
    ray.origin = make_float3(0, 0, 2.0f);
    int2 dir = make_int2(dim.x - 1, dim.y - 1);
    ray.dir = make_float2((float)dir.x, (float)dir.y);
    ray.length = max(abs(dir.x), abs(dir.y));
    ray.oneOverLength = 1.0f / ray.length;

    ////////////////////////////////////////////////////////////////////////////
    // View angles

    // Allocate view angles for each point along the ray
    thrust::device_vector<float> d_angles(ray.length);

    // Allocate result of max-scan operation on the array of view angles
    thrust::device_vector<float> d_scannedAngles(ray.length);

    ////////////////////////////////////////////////////////////////////////////
    // Visibility results

    // Allocate visibility results for each point along the ray
    thrust::device_vector<Bool> d_visibilities(ray.length);
    thrust::host_vector<Bool> h_visibilities(ray.length);
    thrust::host_vector<Bool> h_visibilitiesRef(ray.length);

    ////////////////////////////////////////////////////////////////////////////
    // Reference solution
    lineOfSight_gold(heightField, ray, (Bool *)&h_visibilitiesRef[0]);

    ////////////////////////////////////////////////////////////////////////////
    // Device solution

    // Execution configuration
    dim3 block(256);
    dim3 grid((uint)ceil(ray.length / (double)block.x));

    // Compute device solution
    printf("Line of sight\n");
    sdkStartTimer(&timer);

    for (uint i = 0; i < numIterations; ++i)
    {

        // Compute view angle for each point along the ray
        computeAngles_kernel<<<grid, block>>>(ray, thrust::raw_pointer_cast(&d_angles[0]));
        getLastCudaError("Kernel execution failed");

        // Perform a max-scan operation on the array of view angles
        thrust::inclusive_scan(d_angles.begin(), d_angles.end(), d_scannedAngles.begin(), thrust::maximum<float>());
        getLastCudaError("Kernel execution failed");

        // Compute visibility results based on the array of view angles
        // and its scanned version
        computeVisibilities_kernel<<<grid, block>>>(thrust::raw_pointer_cast(&d_angles[0]),
                                                    thrust::raw_pointer_cast(&d_scannedAngles[0]),
                                                    ray.length,
                                                    thrust::raw_pointer_cast(&d_visibilities[0]));
        getLastCudaError("Kernel execution failed");
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    getLastCudaError("Kernel execution failed");

    // Copy visibility results back to the host
    thrust::copy(d_visibilities.begin(), d_visibilities.end(), h_visibilities.begin());

    // Compare device visibility results against reference results
    bool res = compareData(thrust::raw_pointer_cast(&h_visibilitiesRef[0]),
                           thrust::raw_pointer_cast(&h_visibilities[0]), ray.length, 0.0f, 0.0f);
    printf("Average time: %f ms\n\n", sdkGetTimerValue(&timer) / numIterations);
    sdkResetTimer(&timer);

    // Cleanup memory
    checkCudaErrors(hipFreeArray(heightFieldArray));
    return res;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute view angles for each point along the ray
//! @param ray         ray
//! @param angles      view angles
////////////////////////////////////////////////////////////////////////////////
__global__ void computeAngles_kernel(const Ray ray, float *angles)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < ray.length)
    {
        float2 location = getLocation(ray, i + 1);
        float height = tex2D(g_HeightFieldTex, location.x, location.y);
        float angle = getAngle(ray, location, height);
        angles[i] = angle;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute visibility for each point along the ray
//! @param angles          view angles
//! @param scannedAngles   max-scanned view angles
//! @param numAngles       number of view angles
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
__global__ void computeVisibilities_kernel(const float *angles,
                                           const float *scannedAngles,
                                           int numAngles,
                                           Bool *visibilities)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numAngles)
    {
        visibilities[i] = scannedAngles[i] <= angles[i];
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! @param heightField     height field
//! @param ray             ray
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
void lineOfSight_gold(const HeightField heightField, const Ray ray,
                      Bool *visibilities)
{
    float angleMax = asinf(-1.0f);

    for (int i = 0; i < ray.length; ++i)
    {
        float2 location = getLocation(ray, i + 1);
        float height = *(heightField.height
                         + heightField.width * (int)floorf(location.y)
                         + (int)floorf(location.x));
        float angle = getAngle(ray, location, height);

        if (angle > angleMax)
        {
            angleMax = angle;
            visibilities[i] = True;
        }
        else
        {
            visibilities[i] = False;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the 2D coordinates of the point located at i steps from the origin
//! of the ray
//! @param ray      ray
//! @param i        integer offset along the ray
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float2 getLocation(const Ray ray, int i)
{
    float step = i * ray.oneOverLength;
    return make_float2(ray.origin.x, ray.origin.y) + ray.dir * step;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the angle of view between a 3D point and the origin of the ray
//! @param ray        ray
//! @param location   2D coordinates of the input point
//! @param height     height of the input point
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float getAngle(const Ray ray, float2 location, float height)
{
    float2 dir = location - make_float2(ray.origin.x, ray.origin.y);
    return atanf((height - ray.origin.z) / length(dir));
}
