/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Computation of eigenvalues of a small symmetric, tridiagonal matrix */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include "config.h"
#include "structs.h"
#include "matlab.h"

// includes, kernels
#include "bisect_kernel_small.cuh"

// includes, file
#include "bisect_small.cuh"

////////////////////////////////////////////////////////////////////////////////
//! Determine eigenvalues for matrices smaller than MAX_SMALL_MATRIX
//! @param TimingIterations  number of iterations for timing
//! @param  input  handles to input data of kernel
//! @param  result handles to result of kernel
//! @param  mat_size  matrix size
//! @param  lg  lower limit of Gerschgorin interval
//! @param  ug  upper limit of Gerschgorin interval
//! @param  precision  desired precision of eigenvalues
//! @param  iterations  number of iterations for timing
////////////////////////////////////////////////////////////////////////////////
void
computeEigenvaluesSmallMatrix(const InputData &input, ResultDataSmall &result,
                              const unsigned int mat_size,
                              const float lg, const float ug,
                              const float precision,
                              const unsigned int iterations)
{
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (unsigned int i = 0; i < iterations; ++i)
    {

        dim3  blocks(1, 1, 1);
        dim3  threads(MAX_THREADS_BLOCK_SMALL_MATRIX, 1, 1);

        bisectKernel<<< blocks, threads >>>(input.g_a, input.g_b, mat_size,
                                            result.g_left, result.g_right,
                                            result.g_left_count,
                                            result.g_right_count,
                                            lg, ug, 0, mat_size,
                                            precision
                                           );
    }

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    getLastCudaError("Kernel launch failed");
    printf("Average time: %f ms (%i iterations)\n",
           sdkGetTimerValue(&timer) / (float) iterations, iterations);

    sdkDeleteTimer(&timer);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize variables and memory for the result for small matrices
//! @param result  handles to the necessary memory
//! @param  mat_size  matrix_size
////////////////////////////////////////////////////////////////////////////////
void
initResultSmallMatrix(ResultDataSmall &result, const unsigned int mat_size)
{

    result.mat_size_f = sizeof(float) * mat_size;
    result.mat_size_ui = sizeof(unsigned int) * mat_size;

    result.eigenvalues = (float *) malloc(result.mat_size_f);

    // helper variables
    result.zero_f = (float *) malloc(result.mat_size_f);
    result.zero_ui = (unsigned int *) malloc(result.mat_size_ui);

    for (unsigned int i = 0; i < mat_size; ++i)
    {

        result.zero_f[i] = 0.0f;
        result.zero_ui[i] = 0;

        result.eigenvalues[i] = 0.0f;
    }

    checkCudaErrors(hipMalloc((void **) &result.g_left, result.mat_size_f));
    checkCudaErrors(hipMalloc((void **) &result.g_right, result.mat_size_f));

    checkCudaErrors(hipMalloc((void **) &result.g_left_count,
                               result.mat_size_ui));
    checkCudaErrors(hipMalloc((void **) &result.g_right_count,
                               result.mat_size_ui));

    // initialize result memory
    checkCudaErrors(hipMemcpy(result.g_left, result.zero_f, result.mat_size_f,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(result.g_right, result.zero_f, result.mat_size_f,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(result.g_right_count, result.zero_ui,
                               result.mat_size_ui,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(result.g_left_count, result.zero_ui,
                               result.mat_size_ui,
                               hipMemcpyHostToDevice));
}

////////////////////////////////////////////////////////////////////////////////
//! Cleanup memory and variables for result for small matrices
//! @param  result  handle to variables
////////////////////////////////////////////////////////////////////////////////
void
cleanupResultSmallMatrix(ResultDataSmall &result)
{

    freePtr(result.eigenvalues);
    freePtr(result.zero_f);
    freePtr(result.zero_ui);

    checkCudaErrors(hipFree(result.g_left));
    checkCudaErrors(hipFree(result.g_right));
    checkCudaErrors(hipFree(result.g_left_count));
    checkCudaErrors(hipFree(result.g_right_count));
}

////////////////////////////////////////////////////////////////////////////////
//! Process the result obtained on the device, that is transfer to host and
//! perform basic sanity checking
//! @param  input  handles to input data
//! @param  result  handles to result data
//! @param  mat_size   matrix size
//! @param  filename  output filename
////////////////////////////////////////////////////////////////////////////////
void
processResultSmallMatrix(const InputData &input, const ResultDataSmall &result,
                         const unsigned int mat_size,
                         const char *filename)
{

    const unsigned int mat_size_f = sizeof(float) * mat_size;
    const unsigned int mat_size_ui = sizeof(unsigned int) * mat_size;

    // copy data back to host
    float *left = (float *) malloc(mat_size_f);
    unsigned int *left_count = (unsigned int *) malloc(mat_size_ui);

    checkCudaErrors(hipMemcpy(left, result.g_left, mat_size_f,
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(left_count, result.g_left_count, mat_size_ui,
                               hipMemcpyDeviceToHost));

    float *eigenvalues = (float *) malloc(mat_size_f);

    for (unsigned int i = 0; i < mat_size; ++i)
    {
        eigenvalues[left_count[i]] = left[i];
    }

    // save result in matlab format
    writeTridiagSymMatlab(filename, input.a, input.b+1, eigenvalues, mat_size);

    freePtr(left);
    freePtr(left_count);
    freePtr(eigenvalues);
}
