#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



//Based on http://www.iti.fh-flensburg.de/lang/algorithmen/sortieren/networks/oemen.htm



#include <assert.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <hip/hip_runtime_api.h>
#include "sortingNetworks_common.h"
#include "sortingNetworks_common.cuh"



////////////////////////////////////////////////////////////////////////////////
// Monolithic Bacther's sort kernel for short arrays fitting into shared memory
////////////////////////////////////////////////////////////////////////////////
__global__ void oddEvenMergeSortShared(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength,
    uint dir
)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    //Shared memory storage for one or more small vectors
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    //Offset to the beginning of subbatch and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    for (uint size = 2; size <= arrayLength; size <<= 1)
    {
        uint stride = size / 2;
        uint offset = threadIdx.x & (stride - 1);

        {
            cg::sync(cta);
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                dir
            );
            stride >>= 1;
        }

        for (; stride > 0; stride >>= 1)
        {
            cg::sync(cta);
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));

            if (offset >= stride)
                Comparator(
                    s_key[pos - stride], s_val[pos - stride],
                    s_key[pos +      0], s_val[pos +      0],
                    dir
                );
        }
    }

    cg::sync(cta);
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}



////////////////////////////////////////////////////////////////////////////////
// Odd-even merge sort iteration kernel
// for large arrays (not fitting into shared memory)
////////////////////////////////////////////////////////////////////////////////
__global__ void oddEvenMergeGlobal(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength,
    uint size,
    uint stride,
    uint dir
)
{
    uint global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;

    //Odd-even merge
    uint pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

    if (stride < size / 2)
    {
        uint offset = global_comparatorI & ((size / 2) - 1);

        if (offset >= stride)
        {
            uint keyA = d_SrcKey[pos - stride];
            uint valA = d_SrcVal[pos - stride];
            uint keyB = d_SrcKey[pos +      0];
            uint valB = d_SrcVal[pos +      0];

            Comparator(
                keyA, valA,
                keyB, valB,
                dir
            );

            d_DstKey[pos - stride] = keyA;
            d_DstVal[pos - stride] = valA;
            d_DstKey[pos +      0] = keyB;
            d_DstVal[pos +      0] = valB;
        }
    }
    else
    {
        uint keyA = d_SrcKey[pos +      0];
        uint valA = d_SrcVal[pos +      0];
        uint keyB = d_SrcKey[pos + stride];
        uint valB = d_SrcVal[pos + stride];

        Comparator(
            keyA, valA,
            keyB, valB,
            dir
        );

        d_DstKey[pos +      0] = keyA;
        d_DstVal[pos +      0] = valA;
        d_DstKey[pos + stride] = keyB;
        d_DstVal[pos + stride] = valB;
    }
}



////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
//Helper function
extern "C" uint factorRadix2(uint *log2L, uint L);

extern "C" void oddEvenMergeSort(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint batchSize,
    uint arrayLength,
    uint dir
)
{
    //Nothing to sort
    if (arrayLength < 2)
        return;

    //Only power-of-two array lengths are supported by this implementation
    uint log2L;
    uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
    assert(factorizationRemainder == 1);

    dir = (dir != 0);

    uint  blockCount = (batchSize * arrayLength) / SHARED_SIZE_LIMIT;
    uint threadCount = SHARED_SIZE_LIMIT / 2;

    if (arrayLength <= SHARED_SIZE_LIMIT)
    {
        assert(SHARED_SIZE_LIMIT % arrayLength == 0);
        oddEvenMergeSortShared<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength, dir);
    }
    else
    {
        oddEvenMergeSortShared<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, SHARED_SIZE_LIMIT, dir);

        for (uint size = 2 * SHARED_SIZE_LIMIT; size <= arrayLength; size <<= 1)
            for (unsigned stride = size / 2; stride > 0; stride >>= 1)
            {
                //Unlike with bitonic sort, combining bitonic merge steps with
                //stride = [SHARED_SIZE_LIMIT / 2 .. 1] seems to be impossible as there are
                //dependencies between data elements crossing the SHARED_SIZE_LIMIT borders
                oddEvenMergeGlobal<<<(batchSize * arrayLength) / 512, 256>>>(d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, stride, dir);
            }
    }
}
