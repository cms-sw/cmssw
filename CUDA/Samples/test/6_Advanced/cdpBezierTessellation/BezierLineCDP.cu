#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <string.h>

__forceinline__ __device__ float2 operator+(float2 a, float2 b)
{
    float2 c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

__forceinline__ __device__ float2 operator-(float2 a, float2 b)
{
    float2 c;
    c.x = a.x - b.x;
    c.y = a.y - b.y;
    return c;
}

__forceinline__ __device__ float2 operator*(float a, float2 b)
{
    float2 c;
    c.x = a * b.x;
    c.y = a * b.y;
    return c;
}

__forceinline__ __device__ float length(float2 a)
{
    return sqrtf(a.x*a.x + a.y*a.y);
}

#define MAX_TESSELLATION 32
struct BezierLine
{
    float2 CP[3];
    float2 *vertexPos;
    int nVertices;
};

__global__ void computeBezierLinePositions(int lidx, BezierLine *bLines, int nTessPoints)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;

    if (idx < nTessPoints)
    {
        float u = (float)idx/(float)(nTessPoints-1);
        float omu = 1.0f - u;

        float B3u[3];

        B3u[0] = omu*omu;
        B3u[1] = 2.0f*u*omu;
        B3u[2] = u*u;

        float2 position = {0,0};

        for (int i = 0; i < 3; i++)
        {
            position = position + B3u[i] * bLines[lidx].CP[i];
        }

        bLines[lidx].vertexPos[idx] = position;
    }
}

__global__ void computeBezierLinesCDP(BezierLine *bLines, int nLines)
{
    int lidx = threadIdx.x + blockDim.x*blockIdx.x;

    if (lidx < nLines)
    {
        float curvature = length(bLines[lidx].CP[1] - 0.5f*(bLines[lidx].CP[0] + bLines[lidx].CP[2]))/length(bLines[lidx].CP[2] - bLines[lidx].CP[0]);
        int nTessPoints = min(max((int)(curvature*16.0f),4),MAX_TESSELLATION);

        if (bLines[lidx].vertexPos == NULL)
        {
            bLines[lidx].nVertices = nTessPoints;
            hipMalloc((void **)&bLines[lidx].vertexPos, nTessPoints*sizeof(float2));
        }

        computeBezierLinePositions<<<ceil((float)bLines[lidx].nVertices/32.0f), 32>>>(lidx, bLines, bLines[lidx].nVertices);
    }
}

__global__ void freeVertexMem(BezierLine *bLines, int nLines)
{
    int lidx = threadIdx.x + blockDim.x*blockIdx.x;

    if (lidx < nLines)
        hipFree(bLines[lidx].vertexPos);
}

unsigned int checkCapableSM35Device(int argc, char** argv)
{
    // Get device properties
    hipDeviceProp_t properties;
    int device_count = 0, device = -1;
    
    if(checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        device = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        
        hipDeviceProp_t properties;
        checkCudaErrors(hipGetDeviceProperties(&properties, device));
        
        if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5))
        {
            printf("Running on GPU  %d (%s)\n", device , properties.name);
        }
        else
        {
            printf("cdpBezierTessellation requires GPU devices with compute SM 3.5 or higher.");
            printf("Current GPU device has compute SM %d.%d. Exiting...\n",properties.major, properties.minor);
            return EXIT_FAILURE;
        }

    }
    else
    {
    
        checkCudaErrors(hipGetDeviceCount(&device_count));

        for (int i=0; i < device_count; ++i)
        {
            checkCudaErrors(hipGetDeviceProperties(&properties, i));

            if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5))
            {
                device = i;
                printf("Running on GPU %d (%s)\n", i, properties.name);
                break;
            }

            printf("GPU %d %s does not support CUDA Dynamic Parallelism\n", i, properties.name);
        }
    }
    if (device == -1)
    {
        fprintf(stderr, "cdpBezierTessellation requires GPU devices with compute SM 3.5 or higher.  Exiting...\n");
        return EXIT_WAIVED;
    }

    return EXIT_SUCCESS;
}


#define N_LINES 256
#define BLOCK_DIM 64
int main(int argc, char **argv)
{
    BezierLine *bLines_h = new BezierLine[N_LINES];

    float2 last = {0,0};

    for (int i = 0; i < N_LINES; i++)
    {
        bLines_h[i].CP[0] = last;

        for (int j = 1; j < 3; j++)
        {
            bLines_h[i].CP[j].x = (float)rand()/(float)RAND_MAX;
            bLines_h[i].CP[j].y = (float)rand()/(float)RAND_MAX;
        }

        last = bLines_h[i].CP[2];
        bLines_h[i].vertexPos = NULL;
        bLines_h[i].nVertices = 0;
    }

    unsigned int sm35Ret = checkCapableSM35Device(argc, argv);
    if (sm35Ret != EXIT_SUCCESS)
    {
        exit(sm35Ret);
    }

    BezierLine *bLines_d;
    checkCudaErrors(hipMalloc((void **)&bLines_d, N_LINES*sizeof(BezierLine)));
    checkCudaErrors(hipMemcpy(bLines_d, bLines_h, N_LINES*sizeof(BezierLine), hipMemcpyHostToDevice));
    printf("Computing Bezier Lines (CUDA Dynamic Parallelism Version) ... ");
    computeBezierLinesCDP<<< (unsigned int)ceil((float)N_LINES/(float)BLOCK_DIM), BLOCK_DIM >>>(bLines_d, N_LINES);
    printf("Done!\n");

    //Do something to draw the lines here

    freeVertexMem<<< (unsigned int)ceil((float)N_LINES/(float)BLOCK_DIM), BLOCK_DIM >>>(bLines_d, N_LINES);
    checkCudaErrors(hipFree(bLines_d));
    delete[] bLines_h;

    exit(EXIT_SUCCESS);
}
