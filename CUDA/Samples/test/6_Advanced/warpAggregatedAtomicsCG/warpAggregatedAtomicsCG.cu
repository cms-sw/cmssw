#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
// includes, project
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#define NUM_ELEMS 10000000
#define NUM_THREADS_PER_BLOCK 512

// warp-aggregated atomic increment
__device__ int atomicAggInc(int *counter)
{
    cg::coalesced_group active = cg::coalesced_threads();

    int mask = active.ballot(1);
    // select the leader
    int leader = __ffs(mask) - 1;

    // leader does the update
    int res = 0;
    if (active.thread_rank() == leader)
    {
        res = atomicAdd(counter, __popc(mask));
    }

    // broadcast result
    res = active.shfl(res, leader);

    // each thread computes its own value
    return res + __popc(mask & ((1 << active.thread_rank()) - 1));
}

__global__ void filter_arr(int *dst, int *nres, const int *src, int n)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = id; i < n; i += gridDim.x*blockDim.x)
    {
        if(src[i] > 0)
            dst[atomicAggInc(nres)] = src[i];
    }
}


int main(int argc, char **argv)
{
    int *data_to_filter, *filtered_data, nres=0;
    int *d_data_to_filter, *d_filtered_data, *d_nres;

    data_to_filter = (int *) malloc(sizeof(int)*NUM_ELEMS);

    // Generate input data.
    for (int i=0; i < NUM_ELEMS; i++)
    {
        data_to_filter[i] = rand() % 20;
    }

    findCudaDevice(argc, (const char **)argv);

    checkCudaErrors(hipMalloc(&d_data_to_filter, sizeof(int)*NUM_ELEMS));
    checkCudaErrors(hipMalloc(&d_filtered_data, sizeof(int)*NUM_ELEMS));
    checkCudaErrors(hipMalloc(&d_nres, sizeof(int)));

    checkCudaErrors(hipMemcpy(d_data_to_filter, data_to_filter, sizeof(int)*NUM_ELEMS, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_nres, 0, sizeof(int)));

    dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1, 1);
    dim3 dimGrid((NUM_ELEMS/NUM_THREADS_PER_BLOCK) + 1, 1, 1);

    filter_arr<<<dimGrid, dimBlock>>>(d_filtered_data, d_nres, d_data_to_filter, NUM_ELEMS);

    checkCudaErrors(hipMemcpy(&nres, d_nres, sizeof(int), hipMemcpyDeviceToHost));

    filtered_data = (int *) malloc(sizeof(int)*nres);

    checkCudaErrors(hipMemcpy(filtered_data, d_filtered_data, sizeof(int)*nres, hipMemcpyDeviceToHost));

    int *host_filtered_data = (int *) malloc(sizeof(int)*NUM_ELEMS);

    // Generate host output with host filtering code.
    int host_flt_count=0;
    for (int i=0; i < NUM_ELEMS; i++)
    {
        if (data_to_filter[i] > 0)
        {
            host_filtered_data[host_flt_count++] = data_to_filter[i];
        }
    }

    printf("\nWarp Aggregated Atomics %s \n",  host_flt_count == nres ? "PASSED" : "FAILED");

    checkCudaErrors(hipFree(d_data_to_filter));
    checkCudaErrors(hipFree(d_filtered_data));
    checkCudaErrors(hipFree(d_nres));
    free(data_to_filter); 
    free(filtered_data);
    free(host_filtered_data);
}

