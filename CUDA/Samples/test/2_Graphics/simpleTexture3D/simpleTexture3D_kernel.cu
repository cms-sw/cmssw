#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _SIMPLETEXTURE3D_KERNEL_CU_
#define _SIMPLETEXTURE3D_KERNEL_CU_


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

texture<uchar, 3, hipReadModeNormalizedFloat> tex;  // 3D texture

hipArray *d_volumeArray = 0;

__global__ void
d_render(uint *d_output, uint imageW, uint imageH, float w)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    float u = x / (float) imageW;
    float v = y / (float) imageH;
    // read from 3D texture
    float voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH))
    {
        // write output color
        uint i = __umul24(y, imageW) + x;
        d_output[i] = voxel*255;
    }
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}


extern "C"
void initCuda(const uchar *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void *)h_volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeWrap;
    tex.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH, float w)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, w);
}

#endif // #ifndef _SIMPLETEXTURE3D_KERNEL_CU_
