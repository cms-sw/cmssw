#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "volumeRender.h"

// random ray offset texture, didn't pay off much in this demo
//#define VOLUMERENDER_RANDSIZE         256

#define VOLUMERENDER_TFS              2
#define VOLUMERENDER_TF_PREINTSIZE    1024
#define VOLUMERENDER_TF_PREINTSTEPS   1024
#define VOLUMERENDER_TF_PREINTRAY     4

enum TFMode
{
    TF_SINGLE_1D = 0,         // single 1D TF for everything
    TF_LAYERED_2D_PREINT = 1, // layered 2D TF uses pre-integration
    TF_LAYERED_2D = 2,        // layered 2D TF without pre-integration behavior
};

typedef unsigned int  uint;
typedef unsigned char uchar;

static bool usePreInt = true;
static hipArray *d_transferIntegrate = 0;
static hipArray *d_transferFunc = 0;
static hipArray *d_transferArray = 0;

#ifdef VOLUMERENDER_RANDSIZE
// 2D ray offsets
static hipArray *d_rayArray = 0;
texture<uchar, 2, hipReadModeNormalizedFloat>        rayTex;
#endif

// 3D texture
texture<VolumeType, 3, VolumeTypeInfo<VolumeType>::readMode>  volumeTex;
// 1D transfer function texture
texture<float4, 1, hipReadModeElementType>           transferTex;
// 1D transfer integration texture
texture<float4, 1, hipReadModeElementType>           transferIntegrateTex;
surface<void, 1>                                      transferIntegrateSurf;
// 2D layered preintegrated transfer function texture
texture<float4, hipTextureType2DLayered,hipReadModeElementType>   transferLayerPreintTex;
surface<void, cudaSurfaceType2DLayered>                             transferLayerPreintSurf;


typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;    // origin
    float3 d;    // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin  = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}


template <int TFMODE >
__device__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale, float transferWeight = 0.0f)
{
    const float rayscale =  float(TFMODE != TF_SINGLE_1D ? VOLUMERENDER_TF_PREINTRAY : 1);
    const int maxSteps = 512;
    const float tstep = 0.01f * rayscale;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    density *= rayscale;

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;
#ifdef VOLUMERENDER_RANDSIZE
    float  offset = (tex2D(rayTex,u,v));
    pos += step * offset;
#endif
    float lastsample = 0;

    //lastsample = (lastsample-transferOffset)*transferScale;
    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        float3 coord = make_float3(pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        float sample = tex3D(volumeTex, coord.x, coord.y, coord.z);
        //sample = (sample-transferOffset)*transferScale;
        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        float4 col;
        int tfid = (pos.x < 0);

        if (TFMODE != TF_SINGLE_1D)
        {
            col = tex2DLayered(transferLayerPreintTex, sample, TFMODE==TF_LAYERED_2D ? sample : lastsample, tfid);
            col.w *= density;
            lastsample = sample;
        }
        else
        {
            col = tex1D(transferTex, sample);
            col.w *= 0;
        }

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);


        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_render_regular(uint *d_output, uint imageW, uint imageH,
                 float density, float brightness,
                 float transferOffset, float transferScale, float transferWeight = 0.0f)
{
    d_render<TF_SINGLE_1D>(d_output,imageW,imageH,density,brightness,transferOffset,transferScale,transferWeight);
}

__global__ void
d_render_preint(uint *d_output, uint imageW, uint imageH,
                float density, float brightness,
                float transferOffset, float transferScale, float transferWeight = 0.0f)
{
    d_render<TF_LAYERED_2D_PREINT>(d_output,imageW,imageH,density,brightness,transferOffset,transferScale,transferWeight);
}

__global__ void
d_render_preint_off(uint *d_output, uint imageW, uint imageH,
                    float density, float brightness,
                    float transferOffset, float transferScale, float transferWeight = 0.0f)
{
    d_render<TF_LAYERED_2D>(d_output,imageW,imageH,density,brightness,transferOffset,transferScale,transferWeight);
}

//////////////////////////////////////////////////////////////////////////

__global__ void
d_integrate_trapezoidal(hipExtent extent)
{
    uint x = blockIdx.x*blockDim.x + threadIdx.x;

    // for higher speed could use hierarchical approach for sum
    if (x >= extent.width)
    {
        return;
    }

    float stepsize = 1.0/float(extent.width-1);
    float to = float(x) * stepsize;

    float4 outclr = make_float4(0,0,0,0);
    float incr = stepsize;

    float4 lastval = tex1D(transferTex,0);

    float cur = incr;

    while (cur < to + incr * 0.5)
    {
        float4 val = tex1D(transferTex,cur);
        float4 trapezoid = (lastval+val)/2.0f;
        lastval = val;

        outclr += trapezoid;
        cur += incr;
    }

    // surface writes need byte offsets for x!
    surf1Dwrite(outclr,transferIntegrateSurf,x * sizeof(float4));
}

__global__ void
d_preintegrate(int layer, float steps, hipExtent extent)
{
    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= extent.width || y >= extent.height)
    {
        return;
    }

    float sx = float(x)/float(extent.width);
    float sy = float(y)/float(extent.height);

    float smax = max(sx,sy);
    float smin = min(sx,sy);

    float4 iv;

    if (x != y)
    {
        // assumes square textures!
        float fracc = smax - smin;
        fracc = 1.0 /(fracc*steps);

        float4 intmax = tex1D(transferIntegrateTex,smax);
        float4 intmin = tex1D(transferIntegrateTex,smin);
        iv.x = (intmax.x - intmin.x)*fracc;
        iv.y = (intmax.y - intmin.y)*fracc;
        iv.z = (intmax.z - intmin.z)*fracc;
        //iv.w = (intmax.w - intmin.w)*fracc;
        iv.w   = (1.0 - exp(-(intmax.w - intmin.w) * fracc));
    }
    else
    {
        float4 sample = tex1D(transferTex,smin);
        iv.x = sample.x;
        iv.y = sample.y;
        iv.z = sample.z;
        //iv.w = sample.w;
        iv.w   = (1.0 - exp(-sample.w));
    }

    iv.x =  __saturatef(iv.x);
    iv.y =  __saturatef(iv.y);
    iv.z =  __saturatef(iv.z);
    iv.w =  __saturatef(iv.w);

    // surface writes need byte offsets for x!
    surf2DLayeredwrite(iv,transferLayerPreintSurf, x * sizeof(float4), y, layer);
}

//////////////////////////////////////////////////////////////////////////


void VolumeRender_setTextureFilterMode(bool bLinearFilter)
{
    volumeTex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

void VolumeRender_setVolume(const Volume *vol)
{
    checkCudaErrors(hipBindTextureToArray(volumeTex, vol->content, vol->channelDesc));
}

static unsigned int iDivUp(size_t a, size_t b)
{
    size_t val = (a % b != 0) ? (a / b + 1) : (a / b);
    if (val > UINT_MAX)
    {
        fprintf(stderr, "\nUINT_MAX limit exceeded in iDivUp() exiting.....\n");
        exit(EXIT_FAILURE);    // val exceeds limit
    }

    return static_cast<unsigned int>(val);
}

void VolumeRender_updateTF(int tfIdx, int numColors, float4 *colors)
{

    if (d_transferFunc)
    {
        checkCudaErrors(hipFreeArray(d_transferFunc));
        d_transferFunc = 0;
    }

    hipChannelFormatDesc channelFloat4 = hipCreateChannelDesc<float4>();
    checkCudaErrors(hipMallocArray(&d_transferFunc, &channelFloat4, numColors, 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFunc, 0, 0, colors, sizeof(float4)*numColors, hipMemcpyHostToDevice));
    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFunc, channelFloat4));

    if (tfIdx < 0 || tfIdx >= VOLUMERENDER_TFS)
    {
        return;
    }

    {
        hipExtent extent = {VOLUMERENDER_TF_PREINTSTEPS, 0,0};
        dim3 blockSize(32,1,1);
        dim3 gridSize(iDivUp(extent.width,blockSize.x),1,1);
        d_integrate_trapezoidal<<<gridSize, blockSize>>>(extent);
    }

    {
        hipExtent extent = {VOLUMERENDER_TF_PREINTSIZE, VOLUMERENDER_TF_PREINTSIZE,VOLUMERENDER_TFS};
        dim3 blockSize(16,16,1);
        dim3 gridSize(iDivUp(extent.width,blockSize.x),iDivUp(extent.height,blockSize.y),1);
        d_preintegrate<<<gridSize, blockSize>>>(tfIdx, float(VOLUMERENDER_TF_PREINTSTEPS), extent);
    }

}

void VolumeRender_init()
{
#ifdef VOLUMERENDER_RANDSIZE
    // random ray offsets to fight aliasing
    srand(128123);
    uchar *randoms = new uchar[VOLUMERENDER_RANDSIZE*VOLUMERENDER_RANDSIZE];

    for (int i = 0; i < VOLUMERENDER_RANDSIZE*VOLUMERENDER_RANDSIZE; i++)
    {
        randoms[i] = rand();
    }

    hipChannelFormatDesc channelUchar = hipCreateChannelDesc<uchar>();
    checkCudaErrors(hipMallocArray(&d_rayArray, &channelUchar, VOLUMERENDER_RANDSIZE, VOLUMERENDER_RANDSIZE));
    checkCudaErrors(hipMemcpyToArray(d_rayArray, 0, 0, randoms, sizeof(uchar)*VOLUMERENDER_RANDSIZE*VOLUMERENDER_RANDSIZE, hipMemcpyHostToDevice));
    delete [] randoms;

    rayTex.normalized = 1;
    rayTex.filterMode = hipFilterModeLinear;      // linear interpolation
    rayTex.addressMode[0] = hipAddressModeWrap;
    rayTex.addressMode[1] = hipAddressModeWrap;

    checkCudaErrors(hipBindTextureToArray(rayTex,  d_rayArray, channelUchar));
#endif
    // set texture parameters
    volumeTex.normalized = true;                      // access with normalized texture coordinates
    volumeTex.filterMode = hipFilterModeLinear;      // linear interpolation
    volumeTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    volumeTex.addressMode[1] = hipAddressModeClamp;
    volumeTex.addressMode[2] = hipAddressModeClamp;

    transferTex.normalized = true;
    transferTex.filterMode = hipFilterModeLinear;
    transferTex.addressMode[0] = hipAddressModeClamp;

    transferLayerPreintTex.normalized = true;
    transferLayerPreintTex.filterMode = hipFilterModeLinear;
    transferLayerPreintTex.addressMode[0] = hipAddressModeClamp;
    transferLayerPreintTex.addressMode[1] = hipAddressModeClamp;

    hipChannelFormatDesc channelFloat4 = hipCreateChannelDesc<float4>();
    hipExtent extent = {VOLUMERENDER_TF_PREINTSIZE, VOLUMERENDER_TF_PREINTSIZE,VOLUMERENDER_TFS};
    checkCudaErrors(hipMalloc3DArray(&d_transferArray, &channelFloat4, extent, hipArrayLayered | hipArraySurfaceLoadStore));
    checkCudaErrors(hipBindTextureToArray(transferLayerPreintTex,  d_transferArray, channelFloat4));
    checkCudaErrors(cudaBindSurfaceToArray(transferLayerPreintSurf, d_transferArray, channelFloat4));

    transferIntegrateTex.normalized = true;
    transferIntegrateTex.filterMode = hipFilterModeLinear;
    transferIntegrateTex.addressMode[0] = hipAddressModeClamp;
    transferIntegrateTex.addressMode[1] = hipAddressModeClamp;
    transferIntegrateTex.addressMode[2] = hipAddressModeClamp;
    checkCudaErrors(hipMallocArray(&d_transferIntegrate, &channelFloat4, VOLUMERENDER_TF_PREINTSTEPS,0,hipArraySurfaceLoadStore));
    checkCudaErrors(hipBindTextureToArray(transferIntegrateTex,  d_transferIntegrate, channelFloat4));
    checkCudaErrors(cudaBindSurfaceToArray(transferIntegrateSurf, d_transferIntegrate, channelFloat4));

    // create transfer function texture
    float4 transferFunc0[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    float4 transferFunc1[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  0.0, 1.0, 0.0, 0.125, },
        {  0.0, 0.5, 1.0, 0.125, },
        {  0.0, 1.0, 1.0, 0.125, },
        {  0.0, 1.0, 0.0, 0.125, },
        {  0.25, 0.75, 0.0, 1.0, },
        {  0.75, 0.25, 0.0, 0.125, },
        {  1.0, 0.75, 0.0, 0.125, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    VolumeRender_updateTF(1,sizeof(transferFunc1)/sizeof(float4),transferFunc1);
    VolumeRender_updateTF(0,sizeof(transferFunc0)/sizeof(float4),transferFunc0);
}

void VolumeRender_deinit()
{
    checkCudaErrors(hipFreeArray(d_transferFunc));
    checkCudaErrors(hipFreeArray(d_transferArray));
    checkCudaErrors(hipFreeArray(d_transferIntegrate));
    d_transferArray = 0;
    d_transferFunc = 0;
    d_transferIntegrate = 0;

#ifdef VOLUMERENDER_RANDSIZE
    checkCudaErrors(hipFreeArray(d_rayArray));
    d_rayArray = 0;
#endif

}



void VolumeRender_setPreIntegrated(int state)
{
    usePreInt = !!state;
}

void VolumeRender_render(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                         float density, float brightness, float transferOffset, float transferScale)
{
    if (usePreInt)
    {
        d_render_preint<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                                 brightness, transferOffset, transferScale);
    }
    else
    {
        d_render_preint_off<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                                     brightness, transferOffset, transferScale);
    }

}

void VolumeRender_copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
