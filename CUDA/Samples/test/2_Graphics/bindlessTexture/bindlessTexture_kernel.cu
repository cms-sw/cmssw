#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
    This sample has two kernels, one doing the rendering every frame, and another
    one used to generate the mip map levels at startup.

    For rendering we use a "virtual" texturing approach, where one 2d texture
    stores pointers to the actual textures used. This can be achieved by the
    new cudaTextureObject introduced in CUDA 5.0 and requiring sm3+ hardware.

    The mipmap generation kernel uses cudaSurfaceObject and cudaTextureObject
    passed as kernel arguments to compute the higher mip map level based on
    the lower.

*/

#ifndef _BINDLESSTEXTURE_KERNEL_CU_
#define _BINDLESSTEXTURE_KERNEL_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <vector>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "bindlessTexture.h"

// set this to just see the mipmap chain of first image
//#define SHOW_MIPMAPS


// local references to resources

Image               atlasImage;
std::vector<Image>  contentImages;
float               highestLod = 1.0f;

#ifndef MAX
#define MAX(a,b) ((a > b) ? a : b)
#endif

//////////////////////////////////////////////////////////////////////////

__host__ __device__ __inline__ uint2 encodeTextureObject(hipTextureObject_t obj)
{
    return make_uint2((uint)(obj & 0xFFFFFFFF), (uint)(obj >> 32));
}

__host__ __device__ __inline__ hipTextureObject_t decodeTextureObject(uint2 obj)
{
    return (((hipTextureObject_t)obj.x) | ((hipTextureObject_t)obj.y) << 32);
}

__device__ __inline__ float4 to_float4(uchar4 vec)
{
    return make_float4(vec.x, vec.y, vec.z, vec.w);
}

__device__ __inline__ uchar4 to_uchar4(float4 vec)
{
    return make_uchar4((uchar)vec.x, (uchar)vec.y, (uchar)vec.z, (uchar)vec.w);
}

//////////////////////////////////////////////////////////////////////////
// Rendering

texture<uint2, 2, hipReadModeElementType> atlasTexture;
// the atlas texture stores the 64 bit cudaTextureObjects
// we use it for "virtual" texturing

__global__ void
d_render(uchar4 *d_output, uint imageW, uint imageH, float lod)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float) imageW;
    float v = y / (float) imageH;

    if ((x < imageW) && (y < imageH))
    {
        // read from 2D atlas texture and decode texture object
        uint2 texCoded = tex2D(atlasTexture, u, v);
        hipTextureObject_t tex = decodeTextureObject(texCoded);

        // read from cuda texture object, use template to specify what data will be
        // returned. tex2DLod allows us to pass the lod (mip map level) directly.
        // There is other functions with CUDA 5, e.g. tex2DGrad,    that allow you
        // to pass derivatives to perform automatic mipmap/anisotropic filtering.
        float4 color = tex2DLod<float4>(tex, u, 1-v, lod);
        // In our sample tex is always valid, but for something like your own
        // sparse texturing you would need to make sure to handle the zero case.

        // write output color
        uint i = y * imageW + x;
        d_output[i] = to_uchar4(color * 255.0);
    }
}

extern "C"
void renderAtlasImage(dim3 gridSize, dim3 blockSize, uchar4 *d_output, uint imageW, uint imageH, float lod)
{
    // psuedo animate lod
    lod = fmodf(lod,highestLod*2);
    lod = highestLod-fabs(lod-highestLod);

#ifdef SHOW_MIPMAPS
    lod = 0.0f;
#endif

    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, lod);

    checkCudaErrors(hipGetLastError());
}

//////////////////////////////////////////////////////////////////////////
// MipMap Generation


//  A key benefit of using the new surface objects is that we don't need any global
//  binding points anymore. We can directly pass them as function arguments.

__global__ void
d_mipmap(hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput, uint imageW, uint imageH)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    float px = 1.0/float(imageW);
    float py = 1.0/float(imageH);


    if ((x < imageW) && (y < imageH))
    {
        // take the average of 4 samples

        // we are using the normalized access to make sure non-power-of-two textures
        // behave well when downsized.
        float4 color =
            (tex2D<float4>(mipInput,(x + 0) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput,(x + 1) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput,(x + 1) * px, (y + 1) * py)) +
            (tex2D<float4>(mipInput,(x + 0) * px, (y + 1) * py));


        color /= 4.0;
        color *= 255.0;
        color = fminf(color,make_float4(255.0));

        surf2Dwrite(to_uchar4(color),mipOutput,x * sizeof(uchar4),y);
    }
}

void generateMipMaps(hipMipmappedArray_t mipmapArray, hipExtent size)
{
    size_t width    = size.width;
    size_t height   = size.height;

#ifdef SHOW_MIPMAPS
    hipArray_t levelFirst;
    checkCudaErrors(hipGetMipmappedArrayLevel(&levelFirst, mipmapArray, 0));
#endif

    uint level = 0;

    while (width != 1 || height != 1)
    {
        width     /= 2;
        width      = MAX((size_t)1,width);
        height    /= 2;
        height     = MAX((size_t)1,height);

        hipArray_t levelFrom;
        checkCudaErrors(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
        hipArray_t levelTo;
        checkCudaErrors(hipGetMipmappedArrayLevel(&levelTo,   mipmapArray, level + 1));

        hipExtent  levelToSize;
        checkCudaErrors(hipArrayGetInfo(NULL,&levelToSize,NULL,levelTo));
        checkHost(levelToSize.width  == width);
        checkHost(levelToSize.height == height);
        checkHost(levelToSize.depth  == 0);

        // generate texture object for reading
        hipTextureObject_t         texInput;
        hipResourceDesc            texRes;
        memset(&texRes,0,sizeof(hipResourceDesc));

        texRes.resType            = hipResourceTypeArray;
        texRes.res.array.array    = levelFrom;

        hipTextureDesc             texDescr;
        memset(&texDescr,0,sizeof(hipTextureDesc));

        texDescr.normalizedCoords = 1;
        texDescr.filterMode       = hipFilterModeLinear;

        texDescr.addressMode[0] = hipAddressModeClamp;
        texDescr.addressMode[1] = hipAddressModeClamp;
        texDescr.addressMode[2] = hipAddressModeClamp;

        texDescr.readMode = hipReadModeNormalizedFloat;

        checkCudaErrors(hipCreateTextureObject(&texInput, &texRes, &texDescr, NULL));

        // generate surface object for writing

        hipSurfaceObject_t surfOutput;
        hipResourceDesc    surfRes;
        memset(&surfRes,0,sizeof(hipResourceDesc));
        surfRes.resType = hipResourceTypeArray;
        surfRes.res.array.array = levelTo;

        checkCudaErrors(hipCreateSurfaceObject(&surfOutput,&surfRes));

        // run mipmap kernel
        dim3 blockSize(16,16,1);
        dim3 gridSize(((uint)width+blockSize.x-1)/blockSize.x,((uint)height+blockSize.y-1)/blockSize.y,1);

        d_mipmap<<<gridSize, blockSize>>>(surfOutput, texInput, (uint)width, (uint)height);

        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipGetLastError());

        checkCudaErrors(hipDestroySurfaceObject(surfOutput));

        checkCudaErrors(hipDestroyTextureObject(texInput));

#ifdef SHOW_MIPMAPS
        // we blit the current mipmap back into first level
        hipMemcpy3DParms copyParams = {0};
        copyParams.dstArray     = levelFirst;
        copyParams.srcArray     = levelTo;
        copyParams.extent       = make_hipExtent(width,height,1);
        copyParams.kind         = hipMemcpyDeviceToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));
#endif

        level++;
    }
}

uint getMipMapLevels(hipExtent size)
{
    size_t sz = MAX(MAX(size.width,size.height),size.depth);

    uint levels = 0;

    while (sz)
    {
        sz /= 2;
        levels++;
    }

    return levels;
}


//////////////////////////////////////////////////////////////////////////
// Initalization

extern "C"
void randomizeAtlas()
{
    uint2 *h_data = (uint2 *) atlasImage.h_data;

    // assign random texture object handles to our atlas image tiles
    for (size_t i = 0; i < atlasImage.size.width * atlasImage.size.height; i++)
    {
#ifdef SHOW_MIPMAPS
        h_data[i] = encodeTextureObject(contentImages[ 0 ].textureObject);
#else
        h_data[i] = encodeTextureObject(contentImages[ rand() % contentImages.size() ].textureObject);
#endif
    }

    // copy data to atlas array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr       = make_hipPitchedPtr(atlasImage.h_data, atlasImage.size.width * sizeof(uint2), atlasImage.size.width, atlasImage.size.height);
    copyParams.dstArray     = atlasImage.dataArray;
    copyParams.extent       = atlasImage.size;
    copyParams.extent.depth = 1;
    copyParams.kind         = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

};

extern "C"
void deinitAtlasAndImages()
{
    for (size_t i = 0; i < contentImages.size(); i++)
    {
        Image &image = contentImages[i];

        if (image.h_data)
        {
            free(image.h_data);
        }

        if (image.textureObject)
        {
            checkCudaErrors(hipDestroyTextureObject(image.textureObject));
        }

        if (image.mipmapArray)
        {
            checkCudaErrors(hipFreeMipmappedArray(image.mipmapArray));
        }
    }

    checkCudaErrors(hipUnbindTexture(atlasTexture));

    if (atlasImage.h_data)
    {
        free(atlasImage.h_data);
    }

    if (atlasImage.dataArray)
    {
        checkCudaErrors(hipFreeArray(atlasImage.dataArray));
    }
}

extern "C"
void initAtlasAndImages(const Image *images, size_t numImages, hipExtent atlasSize)
{
    // create individual textures
    contentImages.resize(numImages);

    for (size_t i = 0; i < numImages; i++)
    {
        Image &image = contentImages[i];
        image.size = images[i].size;
        image.size.depth = 0;
        image.type = hipResourceTypeMipmappedArray;

        // how many mipmaps we need
        uint levels = getMipMapLevels(image.size);
        highestLod  = MAX(highestLod, (float) levels-1);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        checkCudaErrors(hipMallocMipmappedArray(&image.mipmapArray, &desc, image.size, levels));

        // upload level 0
        hipArray_t level0;
        checkCudaErrors(hipGetMipmappedArrayLevel(&level0,image.mipmapArray,0));

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr       = make_hipPitchedPtr(images[i].h_data, image.size.width * sizeof(uchar4), image.size.width, image.size.height);
        copyParams.dstArray     = level0;
        copyParams.extent       = image.size;
        copyParams.extent.depth = 1;
        copyParams.kind         = hipMemcpyHostToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));

        // compute rest of mipmaps based on level 0
        generateMipMaps(image.mipmapArray, image.size);

        // generate bindless texture object

        hipResourceDesc            resDescr;
        memset(&resDescr,0,sizeof(hipResourceDesc));

        resDescr.resType            = hipResourceTypeMipmappedArray;
        resDescr.res.mipmap.mipmap  = image.mipmapArray;

        hipTextureDesc             texDescr;
        memset(&texDescr,0,sizeof(hipTextureDesc));

        texDescr.normalizedCoords = 1;
        texDescr.filterMode       = hipFilterModeLinear;
        texDescr.mipmapFilterMode = hipFilterModeLinear;

        texDescr.addressMode[0] = hipAddressModeClamp;
        texDescr.addressMode[1] = hipAddressModeClamp;
        texDescr.addressMode[2] = hipAddressModeClamp;

        texDescr.maxMipmapLevelClamp = float(levels - 1);

        texDescr.readMode = hipReadModeNormalizedFloat;

        checkCudaErrors(hipCreateTextureObject(&image.textureObject, &resDescr, &texDescr, NULL));
    }

    // create atlas array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint2>();
    checkCudaErrors(hipMallocArray(&atlasImage.dataArray, &channelDesc, atlasSize.width, atlasSize.height));
    atlasImage.h_data              = malloc(atlasSize.width * atlasSize.height * sizeof(uint2));
    atlasImage.type                = hipResourceTypeArray;
    atlasImage.size                = atlasSize;

    // set texture parameters
    atlasTexture.normalized     = 1;                    // access with normalized texture coordinates
    atlasTexture.filterMode     = hipFilterModePoint;  // and without any filtering (we want raw 2x32bit values)
    atlasTexture.addressMode[0] = hipAddressModeClamp;
    atlasTexture.addressMode[1] = hipAddressModeClamp;
    atlasTexture.addressMode[2] = hipAddressModeClamp;

    randomizeAtlas();

    // Bind array to atlas texture using the classic binding approach.
    // An alternative approach,would be to create a cudaTextureObject and pass it as kernel argument
    checkCudaErrors(hipBindTextureToArray(atlasTexture, atlasImage.dataArray, channelDesc));
}


#endif // #ifndef _SIMPLETEXTURE3D_KERNEL_CU_
