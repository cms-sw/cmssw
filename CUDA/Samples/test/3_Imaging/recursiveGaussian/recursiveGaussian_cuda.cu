#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Recursive Gaussian filter
    sgreen 8/1/08

    This code sample implements a Gaussian blur using Deriche's recursive method:
    http://citeseer.ist.psu.edu/deriche93recursively.html

    This is similar to the box filter sample in the SDK, but it uses the previous
    outputs of the filter as well as the previous inputs. This is also known as an
    IIR (infinite impulse response) filter, since its response to an input impulse
    can last forever.

    The main advantage of this method is that the execution time is independent of
    the filter width.

    The GPU processes columns of the image in parallel. To avoid uncoalesced reads
    for the row pass we transpose the image and then transpose it back again
    afterwards.

    The implementation is based on code from the CImg library:
    http://cimg.sourceforge.net/
    Thanks to David Tschumperl� and all the CImg contributors!
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "recursiveGaussian_kernel.cuh"

#define USE_SIMPLE_FILTER 0

//Round a / b to nearest higher integer value
int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

/*
    Transpose a 2D array (see SDK transpose example)
*/
extern "C"
void transpose(uint *d_src, uint *d_dest, uint width, int height)
{
    dim3 grid(iDivUp(width, BLOCK_DIM), iDivUp(height, BLOCK_DIM), 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    d_transpose<<< grid, threads >>>(d_dest, d_src, width, height);
    getLastCudaError("Kernel execution failed");
}

/*
    Perform Gaussian filter on a 2D image using CUDA

    Parameters:
    d_src  - pointer to input image in device memory
    d_dest - pointer to destination image in device memory
    d_temp - pointer to temporary storage in device memory
    width  - image width
    height - image height
    sigma  - sigma of Gaussian
    order  - filter order (0, 1 or 2)
*/

// 8-bit RGBA version
extern "C"
void gaussianFilterRGBA(uint *d_src, uint *d_dest, uint *d_temp, int width, int height, float sigma, int order, int nthreads)
{
    // compute filter coefficients
    const float
    nsigma = sigma < 0.1f ? 0.1f : sigma,
    alpha = 1.695f / nsigma,
    ema = (float)std::exp(-alpha),
    ema2 = (float)std::exp(-2*alpha),
    b1 = -2*ema,
    b2 = ema2;

    float a0 = 0, a1 = 0, a2 = 0, a3 = 0, coefp = 0, coefn = 0;

    switch (order)
    {
        case 0:
            {
                const float k = (1-ema)*(1-ema)/(1+2*alpha*ema-ema2);
                a0 = k;
                a1 = k*(alpha-1)*ema;
                a2 = k*(alpha+1)*ema;
                a3 = -k*ema2;
            }
            break;

        case 1:
            {
                const float k = (1-ema)*(1-ema)/ema;
                a0 = k*ema;
                a1 = a3 = 0;
                a2 = -a0;
            }
            break;

        case 2:
            {
                const float
                ea = (float)std::exp(-alpha),
                k = -(ema2-1)/(2*alpha*ema),
                kn = (-2*(-1+3*ea-3*ea*ea+ea*ea*ea)/(3*ea+1+3*ea*ea+ea*ea*ea));
                a0 = kn;
                a1 = -kn*(1+k*alpha)*ema;
                a2 = kn*(1-k*alpha)*ema;
                a3 = -kn*ema2;
            }
            break;

        default:
            fprintf(stderr, "gaussianFilter: invalid order parameter!\n");
            return;
    }

    coefp = (a0+a1)/(1+b1+b2);
    coefn = (a2+a3)/(1+b1+b2);

    // process columns
#if USE_SIMPLE_FILTER
    d_simpleRecursive_rgba<<< iDivUp(width, nthreads), nthreads >>>(d_src, d_temp, width, height, ema);
#else
    d_recursiveGaussian_rgba<<< iDivUp(width, nthreads), nthreads >>>(d_src, d_temp, width, height, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
    getLastCudaError("Kernel execution failed");

    transpose(d_temp, d_dest, width, height);
    getLastCudaError("transpose: Kernel execution failed");

    // process rows
#if USE_SIMPLE_FILTER
    d_simpleRecursive_rgba<<< iDivUp(height, nthreads), nthreads >>>(d_dest, d_temp, height, width, ema);
#else
    d_recursiveGaussian_rgba<<< iDivUp(height, nthreads), nthreads >>>(d_dest, d_temp, height, width, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
    getLastCudaError("Kernel execution failed");

    transpose(d_temp, d_dest, height, width);
}
