#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    NV12ToARGB color space conversion CUDA kernel

    This sample uses CUDA to perform a simple NV12 (YUV 4:2:0 planar)
    source and converts to output in ARGB format
*/


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "cudaProcessFrame.h"

__constant__ uint32 constAlpha;

#define MUL(x,y)    (x*y)
__constant__ float  constHueColorSpaceMat[9];


__device__ void YUV2RGB(uint32 *yuvi, float *red, float *green, float *blue)
{
    float luma, chromaCb, chromaCr;

    // Prepare for hue adjustment
    luma     = (float)yuvi[0];
    chromaCb = (float)((int32)yuvi[1] - 512.0f);
    chromaCr = (float)((int32)yuvi[2] - 512.0f);

    // Convert YUV To RGB with hue adjustment
    *red  = MUL(luma,     constHueColorSpaceMat[0]) +
            MUL(chromaCb, constHueColorSpaceMat[1]) +
            MUL(chromaCr, constHueColorSpaceMat[2]);
    *green= MUL(luma,     constHueColorSpaceMat[3]) +
            MUL(chromaCb, constHueColorSpaceMat[4]) +
            MUL(chromaCr, constHueColorSpaceMat[5]);
    *blue = MUL(luma,     constHueColorSpaceMat[6]) +
            MUL(chromaCb, constHueColorSpaceMat[7]) +
            MUL(chromaCr, constHueColorSpaceMat[8]);
}


__device__ uint32 RGBAPACK_8bit(float red, float green, float blue, uint32 alpha)
{
    uint32 ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 255.0f);
    green = min(max(green, 0.0f), 255.0f);
    blue  = min(max(blue,  0.0f), 255.0f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = (((uint32)blue) |
                 (((uint32)green) << 8)  |
                 (((uint32)red) << 16) | (uint32)alpha);

    return  ARGBpixel;
}

__device__ uint32 RGBAPACK_10bit(float red, float green, float blue, uint32 alpha)
{
    uint32 ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 1023.f);
    green = min(max(green, 0.0f), 1023.f);
    blue  = min(max(blue,  0.0f), 1023.f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = (((uint32)blue  >> 2) |
                 (((uint32)green >> 2) << 8)  |
                 (((uint32)red   >> 2) << 16) | (uint32)alpha);

    return  ARGBpixel;
}


// CUDA kernel for outputing the final ARGB output from NV12;
extern "C"
__global__ void Passthru_drvapi(uint32 *srcImage,   size_t nSourcePitch,
                                uint32 *dstImage,   size_t nDestPitch,
                                uint32 width,       uint32 height)
{
    int32 x, y;
    uint32 yuv101010Pel[2];
    uint32 processingPitch = ((width) + 63) & ~63;
    uint32 dstImagePitch   = nDestPitch >> 2;
    uint8 *srcImageU8     = (uint8 *)srcImage;

    processingPitch = nSourcePitch;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]);
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]);

    // this steps performs the color conversion
    float luma[2];

    luma[0]   = (yuv101010Pel[0]        & 0x00FF);
    luma[1]   = (yuv101010Pel[1]        & 0x00FF);

    // Clamp the results to RGBA
    dstImage[y * dstImagePitch + x     ] = RGBAPACK_8bit(luma[0], luma[0], luma[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_8bit(luma[1], luma[1], luma[1], constAlpha);
}


// CUDA kernel for outputing the final ARGB output from NV12;
extern "C"
__global__ void NV12ToARGB_drvapi(uint32 *srcImage,     size_t nSourcePitch,
                                  uint32 *dstImage,     size_t nDestPitch,
                                  uint32 width,         uint32 height)
{
    int32 x, y;
    uint32 yuv101010Pel[2];
    uint32 processingPitch = ((width) + 63) & ~63;
    uint32 dstImagePitch   = nDestPitch >> 2;
    uint8 *srcImageU8     = (uint8 *)srcImage;

    processingPitch = nSourcePitch;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32 chromaOffset    = processingPitch * height;
    int32 y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        uint32 chromaCb;
        uint32 chromaCr;

        chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
        yuv101010Pel[0] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    // this steps performs the color conversion
    uint32 yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] &   COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] &   COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA
    dstImage[y * dstImagePitch + x     ] = RGBAPACK_10bit(red[0], green[0], blue[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_10bit(red[1], green[1], blue[1], constAlpha);
}

