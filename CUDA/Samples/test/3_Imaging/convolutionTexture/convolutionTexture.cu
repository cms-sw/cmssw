#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>

#include "convolutionTexture_common.h"

////////////////////////////////////////////////////////////////////////////////
// GPU-specific defines
////////////////////////////////////////////////////////////////////////////////
//Maps to a single instruction on G8x / G9x / G10x
#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )

//Use unrolled innermost convolution loop
#define UNROLL_INNER 1

//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
inline int iAlignUp(int a, int b)
{
    return (a % b != 0) ? (a - a % b + b) : a;
}



////////////////////////////////////////////////////////////////////////////////
// Convolution kernel and input array storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[KERNEL_LENGTH];

extern "C" void setConvolutionKernel(float *h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}

texture<float, 2, hipReadModeElementType> texSrc;

extern "C" void setInputArray(hipArray *a_Src)
{
}

extern "C" void detachInputArray(void)
{
}



////////////////////////////////////////////////////////////////////////////////
// Loop unrolling templates, needed for best performance
////////////////////////////////////////////////////////////////////////////////
template<int i> __device__ float convolutionRow(float x, float y)
{
    return
        tex2D(texSrc, x + (float)(KERNEL_RADIUS - i), y) * c_Kernel[i]
        + convolutionRow<i - 1>(x, y);
}

template<> __device__ float convolutionRow<-1>(float x, float y)
{
    return 0;
}

template<int i> __device__ float convolutionColumn(float x, float y)
{
    return
        tex2D(texSrc, x, y + (float)(KERNEL_RADIUS - i)) * c_Kernel[i]
        + convolutionColumn<i - 1>(x, y);
}

template<> __device__ float convolutionColumn<-1>(float x, float y)
{
    return 0;
}



////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowsKernel(
    float *d_Dst,
    int imageW,
    int imageH
)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if (ix >= imageW || iy >= imageH)
    {
        return;
    }

    float sum = 0;

#if(UNROLL_INNER)
    sum = convolutionRow<2 *KERNEL_RADIUS>(x, y);
#else

    for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
    {
        sum += tex2D(texSrc, x + (float)k, y) * c_Kernel[KERNEL_RADIUS - k];
    }

#endif

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}


extern "C" void convolutionRowsGPU(
    float *d_Dst,
    hipArray *a_Src,
    int imageW,
    int imageH
)
{
    dim3 threads(16, 12);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    checkCudaErrors(hipBindTextureToArray(texSrc, a_Src));
    convolutionRowsKernel<<<blocks, threads>>>(
        d_Dst,
        imageW,
        imageH
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");

    checkCudaErrors(hipUnbindTexture(texSrc));
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnsKernel(
    float *d_Dst,
    int imageW,
    int imageH
)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if (ix >= imageW || iy >= imageH)
    {
        return;
    }

    float sum = 0;

#if(UNROLL_INNER)
    sum = convolutionColumn<2 *KERNEL_RADIUS>(x, y);
#else

    for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
    {
        sum += tex2D(texSrc, x, y + (float)k) * c_Kernel[KERNEL_RADIUS - k];
    }

#endif

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}

extern "C" void convolutionColumnsGPU(
    float *d_Dst,
    hipArray *a_Src,
    int imageW,
    int imageH
)
{
    dim3 threads(16, 12);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    checkCudaErrors(hipBindTextureToArray(texSrc, a_Src));
    convolutionColumnsKernel<<<blocks, threads>>>(
        d_Dst,
        imageW,
        imageH
    );
    getLastCudaError("convolutionColumnsKernel() execution failed\n");

    checkCudaErrors(hipUnbindTexture(texSrc));
}
