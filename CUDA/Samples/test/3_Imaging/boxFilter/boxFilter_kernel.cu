#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

#ifndef _BOXFILTER_KERNEL_CH_
#define _BOXFILTER_KERNEL_CH_

#include <hip/hip_vector_types.h>
#include <helper_functions.h>

texture<float, 2> tex;
texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
hipArray *d_array, *d_tempArray;

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*
    Perform a fast box filter using the sliding window method.

    As the kernel moves from left to right, we add in the contribution of the new
    sample on the right, and subtract the value of the exiting sample on the left.
    This only requires 2 adds and a mul per output value, independent of the filter radius.
    The box filter is separable, so to perform a 2D box filter we perform the filter in
    the x direction, followed by the same filter in the y direction.
    Applying multiple iterations of the box filter converges towards a Gaussian blur.
    Using CUDA, rows or columns of the image are processed in parallel.
    This version duplicates edge pixels.

    Note that the x (row) pass suffers from uncoalesced global memory reads,
    since each thread is reading from a different row. For this reason it is
    better to use texture lookups for the x pass.
    The y (column) pass is perfectly coalesced.

    Parameters
    id - pointer to input data in global memory
    od - pointer to output data in global memory
    w  - image width
    h  - image height
    r  - filter radius

    e.g. for r = 2, w = 8:

    0 1 2 3 4 5 6 7
    x - -
    - x - -
    - - x - -
      - - x - -
        - - x - -
          - - x - -
            - - x -
              - - x
*/

// process row
__device__ void
d_boxfilter_x(float *id, float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);

    float t;
    // do left edge
    t = id[0] * r;

    for (int x = 0; x < (r + 1); x++)
    {
        t += id[x];
    }

    od[0] = t * scale;

    for (int x = 1; x < (r + 1); x++)
    {
        t += id[x + r];
        t -= id[0];
        od[x] = t * scale;
    }

    // main loop
    for (int x = (r + 1); x < w - r; x++)
    {
        t += id[x + r];
        t -= id[x - r - 1];
        od[x] = t * scale;
    }

    // do right edge
    for (int x = w - r; x < w; x++)
    {
        t += id[w - 1];
        t -= id[x - r - 1];
        od[x] = t * scale;
    }
}

// process column
__device__ void
d_boxfilter_y(float *id, float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);

    float t;
    // do left edge
    t = id[0] * r;

    for (int y = 0; y < (r + 1); y++)
    {
        t += id[y * w];
    }

    od[0] = t * scale;

    for (int y = 1; y < (r + 1); y++)
    {
        t += id[(y + r) * w];
        t -= id[0];
        od[y * w] = t * scale;
    }

    // main loop
    for (int y = (r + 1); y < (h - r); y++)
    {
        t += id[(y + r) * w];
        t -= id[((y - r) * w) - w];
        od[y * w] = t * scale;
    }

    // do right edge
    for (int y = h - r; y < h; y++)
    {
        t += id[(h-1) * w];
        t -= id[((y - r) * w) - w];
        od[y * w] = t * scale;
    }
}

__global__ void
d_boxfilter_x_global(float *id, float *od, int w, int h, int r)
{
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
    d_boxfilter_x(&id[y * w], &od[y * w], w, h, r);
}

__global__ void
d_boxfilter_y_global(float *id, float *od, int w, int h, int r)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    d_boxfilter_y(&id[x], &od[x], w, h, r);
}

// texture version
// texture fetches automatically clamp to edge of image
__global__ void
d_boxfilter_x_tex(float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;

    for (int x =- r; x <= r; x++)
    {
        t += tex2D(tex, x, y);
    }

    od[y * w] = t * scale;

    for (int x = 1; x < w; x++)
    {
        t += tex2D(tex, x + r, y);
        t -= tex2D(tex, x - r - 1, y);
        od[y * w + x] = t * scale;
    }
}

__global__ void
d_boxfilter_y_tex(float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;

    for (int y = -r; y <= r; y++)
    {
        t += tex2D(tex, x, y);
    }

    od[x] = t * scale;

    for (int y = 1; y < h; y++)
    {
        t += tex2D(tex, x, y + r);
        t -= tex2D(tex, x, y - r - 1);
        od[y * w + x] = t * scale;
    }
}

// RGBA version
// reads from 32-bit unsigned int array holding 8-bit RGBA

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
           ((unsigned int)(rgba.z * 255.0f) << 16) |
           ((unsigned int)(rgba.y * 255.0f) <<  8) |
           ((unsigned int)(rgba.x * 255.0f));
}

__device__ float4 rgbaIntToFloat(unsigned int c)
{
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
    rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
    rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
    rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
    return rgba;
}

// row pass using texture lookups
__global__ void
d_boxfilter_rgba_x(unsigned int *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

    // as long as address is always less than height, we do work
    if (y < h)
    {
        float4 t = make_float4(0.0f);

        for (int x = -r; x <= r; x++)
        {
            t += tex2D(rgbaTex, x, y);
        }

        od[y * w] = rgbaFloatToInt(t * scale);

        for (int x = 1; x < w; x++)
        {
            t += tex2D(rgbaTex, x + r, y);
            t -= tex2D(rgbaTex, x - r - 1, y);
            od[y * w + x] = rgbaFloatToInt(t * scale);
        }
    }
}

// column pass using coalesced global memory reads
__global__ void
d_boxfilter_rgba_y(unsigned int *id, unsigned int *od, int w, int h, int r)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    id = &id[x];
    od = &od[x];

    float scale = 1.0f / (float)((r << 1) + 1);

    float4 t;
    // do left edge
    t = rgbaIntToFloat(id[0]) * r;

    for (int y = 0; y < (r + 1); y++)
    {
        t += rgbaIntToFloat(id[y*w]);
    }

    od[0] = rgbaFloatToInt(t * scale);

    for (int y = 1; y < (r + 1); y++)
    {
        t += rgbaIntToFloat(id[(y + r) * w]);
        t -= rgbaIntToFloat(id[0]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }

    // main loop
    for (int y = (r + 1); y < (h - r); y++)
    {
        t += rgbaIntToFloat(id[(y + r) * w]);
        t -= rgbaIntToFloat(id[((y - r) * w) - w]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }

    // do right edge
    for (int y = h - r; y < h; y++)
    {
        t += rgbaIntToFloat(id[(h - 1) * w]);
        t -= rgbaIntToFloat(id[((y - r) * w) - w]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }
}

extern "C"
void initTexture(int width, int height, void *pImage, bool useRGBA)
{
    int size = width * height * (useRGBA ? sizeof(uchar4) : sizeof(float));

    // copy image data to array
    hipChannelFormatDesc channelDesc;
    if (useRGBA)
    {
        channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    }
    else
    {
        channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    }
    checkCudaErrors(hipMallocArray(&d_array, &channelDesc, width, height));
    checkCudaErrors(hipMemcpyToArray(d_array, 0, 0, pImage, size, hipMemcpyHostToDevice));

    checkCudaErrors(hipMallocArray(&d_tempArray,   &channelDesc, width, height));

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = true;

    // Bind the array to the texture
    if (useRGBA)
    {
        checkCudaErrors(hipBindTextureToArray(rgbaTex, d_array, channelDesc));
    }
    else
    {
        checkCudaErrors(hipBindTextureToArray(tex, d_array, channelDesc));
    }
}

extern "C"
void freeTextures()
{
    checkCudaErrors(hipFreeArray(d_array));
    checkCudaErrors(hipFreeArray(d_tempArray));
}


/*
    Perform 2D box filter on image using CUDA

    Parameters:
    d_src  - pointer to input image in device memory
    d_temp - pointer to temporary storage in device memory
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    radius - filter radius
    iterations - number of iterations

*/
extern "C"
double boxFilter(float *d_src, float *d_temp, float *d_dest, int width, int height,
                 int radius, int iterations, int nthreads, StopWatchInterface *timer)
{
    // var for kernel timing
    double dKernelTime = 0.0;

    // sync host and start computation timer_kernel
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipBindTextureToArray(tex, d_array));

    for (int i=0; i<iterations; i++)
    {
        sdkResetTimer(&timer);
        // use texture for horizontal pass
        d_boxfilter_x_tex<<< height / nthreads, nthreads, 0 >>>(d_temp, width, height, radius);
        d_boxfilter_y_global<<< width / nthreads, nthreads, 0 >>>(d_temp, d_dest, width, height, radius);

        // sync host and stop computation timer_kernel
        checkCudaErrors(hipDeviceSynchronize());
        dKernelTime += sdkGetTimerValue(&timer);

        if (iterations > 1)
        {
            // copy result back from global memory to array
            checkCudaErrors(hipMemcpyToArray(d_tempArray, 0, 0, d_dest, width * height * sizeof(float), hipMemcpyDeviceToDevice));
            checkCudaErrors(hipBindTextureToArray(tex, d_tempArray));
        }
    }

    return ((dKernelTime/1000.)/(double)iterations);
}

// RGBA version
extern "C"
double boxFilterRGBA(unsigned int *d_src, unsigned int *d_temp, unsigned int *d_dest, int width, int height,
                     int radius, int iterations, int nthreads, StopWatchInterface *timer)
{
    checkCudaErrors(hipBindTextureToArray(rgbaTex, d_array));

    // var for kernel computation timing
    double dKernelTime;

    for (int i=0; i<iterations; i++)
    {
        // sync host and start kernel computation timer_kernel
        dKernelTime = 0.0;
        checkCudaErrors(hipDeviceSynchronize());
        sdkResetTimer(&timer);

        // use texture for horizontal pass
        d_boxfilter_rgba_x<<< height / nthreads, nthreads, 0 >>>(d_temp, width, height, radius);
        d_boxfilter_rgba_y<<< width / nthreads, nthreads, 0 >>>(d_temp, d_dest, width, height, radius);

        // sync host and stop computation timer_kernel
        checkCudaErrors(hipDeviceSynchronize());
        dKernelTime += sdkGetTimerValue(&timer);

        if (iterations > 1)
        {
            // copy result back from global memory to array
            checkCudaErrors(hipMemcpyToArray(d_tempArray, 0, 0, d_dest, width * height * sizeof(float), hipMemcpyDeviceToDevice));
            checkCudaErrors(hipBindTextureToArray(rgbaTex, d_tempArray));
        }
    }

    return ((dKernelTime/1000.)/(double)iterations);
}

#endif // #ifndef _BOXFILTER_KERNEL_H_
