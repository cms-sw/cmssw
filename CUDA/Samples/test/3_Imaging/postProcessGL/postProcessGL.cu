#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Utilities and system includes

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <hip/hip_runtime_api.h>

#ifndef USE_TEXTURE_RGBA8UI
texture<float4, 2, hipReadModeElementType> inTex;
#else
texture<uchar4, 2, hipReadModeElementType> inTex;
#endif

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

// get pixel from 2D image, with clamping to border
__device__ uchar4 getPixel(int x, int y)
{
#ifndef USE_TEXTURE_RGBA8UI
    float4 res = tex2D(inTex, x, y);
    uchar4 ucres = make_uchar4(res.x*255.0f, res.y*255.0f, res.z*255.0f, res.w*255.0f);
#else
    uchar4 ucres = tex2D(inTex, x, y);
#endif
    return ucres;
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew+(X)]

/*
    2D convolution using shared memory
    - operates on 8-bit RGB data stored in 32-bit int
    - assumes kernel radius is less than or equal to block size
    - not optimized for performance
     _____________
    |   :     :   |
    |_ _:_____:_ _|
    |   |     |   |
    |   |     |   |
    |_ _|_____|_ _|
  r |   :     :   |
    |___:_____:___|
      r    bw   r
    <----tilew---->
*/

__global__ void
cudaProcess(unsigned int *g_odata, int imgw, int imgh,
            int tilew, int r, float threshold, float highlight)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

#if 0
    uchar4 c4 = getPixel(x, y);
    g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
#else
    // copy tile to shared memory
    // center region
    SMEM(r + tx, r + ty) = getPixel(x, y);

    // borders
    if (threadIdx.x < r)
    {
        // left
        SMEM(tx, r + ty) = getPixel(x - r, y);
        // right
        SMEM(r + bw + tx, r + ty) = getPixel(x + bw, y);
    }

    if (threadIdx.y < r)
    {
        // top
        SMEM(r + tx, ty) = getPixel(x, y - r);
        // bottom
        SMEM(r + tx, r + bh + ty) = getPixel(x, y + bh);
    }

    // load corners
    if ((threadIdx.x < r) && (threadIdx.y < r))
    {
        // tl
        SMEM(tx, ty) = getPixel(x - r, y - r);
        // bl
        SMEM(tx, r + bh + ty) = getPixel(x - r, y + bh);
        // tr
        SMEM(r + bw + tx, ty) = getPixel(x + bh, y - r);
        // br
        SMEM(r + bw + tx, r + bh + ty) = getPixel(x + bw, y + bh);
    }

    // wait for loads to complete
    cg::sync(cta);

    // perform convolution
    float rsum = 0.0f;
    float gsum = 0.0f;
    float bsum = 0.0f;
    float samples = 0.0f;

    for (int dy=-r; dy<=r; dy++)
    {
        for (int dx=-r; dx<=r; dx++)
        {
#if 0
            // try this to see the benefit of using shared memory
            uchar4 pixel = getPixel(x+dx, y+dy);
#else
            uchar4 pixel = SMEM(r+tx+dx, r+ty+dy);
#endif

            // only sum pixels within disc-shaped kernel
            float l = dx*dx + dy*dy;

            if (l <= r*r)
            {
                float r = float(pixel.x);
                float g = float(pixel.y);
                float b = float(pixel.z);
#if 1
                // brighten highlights
                float lum = (r + g + b) / (255*3);

                if (lum > threshold)
                {
                    r *= highlight;
                    g *= highlight;
                    b *= highlight;
                }

#endif
                rsum += r;
                gsum += g;
                bsum += b;
                samples += 1.0f;
            }
        }
    }

    rsum /= samples;
    gsum /= samples;
    bsum /= samples;
    // ABGR
    g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
    //g_odata[y*imgw+x] = rgbToInt(x,y,0);
#endif
}

extern "C" void
launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                   hipArray *g_data_array, unsigned int *g_odata,
                   int imgw, int imgh, int tilew,
                   int radius, float threshold, float highlight)
{
    checkCudaErrors(hipBindTextureToArray(inTex, g_data_array));

    struct hipChannelFormatDesc desc;
    checkCudaErrors(hipGetChannelDesc(&desc, g_data_array));

#if 0
    printf("CUDA Array channel descriptor, bits per component:\n");
    printf("X %d Y %d Z %d W %d, kind %d\n",
           desc.x,desc.y,desc.z,desc.w,desc.f);

    printf("Possible values for channel format kind: i %d, u%d, f%d:\n",
           hipChannelFormatKindSigned, hipChannelFormatKindUnsigned,
           hipChannelFormatKindFloat);
#endif

    //printf("\n");
#ifdef GPU_PROFILING
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    int nIter = 30;

    for (int i = -1; i < nIter; ++i)
    {
        if (i == 0)
        {
            sdkStartTimer(&timer);
        }

#endif

        cudaProcess<<< grid, block, sbytes >>>(g_odata, imgw, imgh,
                                               block.x+(2*radius), radius, 0.8f, 4.0f);

#ifdef GPU_PROFILING
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    double dSeconds = sdkGetTimerValue(&timer)/((double)nIter * 1000.0);
    double dNumTexels = (double)imgw * (double)imgh;
    double mtexps = 1.0e-6 * dNumTexels/dSeconds;

    if (radius == 4)
    {
        printf("\n");
        printf("postprocessGL, Throughput = %.4f MTexels/s, Time = %.5f s, Size = %.0f Texels, NumDevsUsed = %d, Workgroup = %u\n",
               mtexps, dSeconds, dNumTexels, 1, block.x * block.y);
    }

#endif
}
