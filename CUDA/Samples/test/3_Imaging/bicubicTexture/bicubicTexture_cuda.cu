#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

// includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include "bicubicTexture_kernel.cuh"

hipArray *d_imageArray = 0;

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight));
    uint size = imageWidth * imageHeight * sizeof(uchar);
    checkCudaErrors(hipMemcpyToArray(d_imageArray, 0, 0, h_data, size, hipMemcpyHostToDevice));
    free(h_data);

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = false;    // access with integer texture coordinates

    getLastCudaError("initTexture");

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, d_imageArray));

    // bind same array to 2nd texture reference with point sampling
    tex2.addressMode[0] = hipAddressModeClamp;
    tex2.addressMode[1] = hipAddressModeClamp;
    tex2.filterMode = hipFilterModePoint;
    tex2.normalized = false;    // access with integer texture coordinates

    checkCudaErrors(hipBindTextureToArray(tex2, d_imageArray));
}

extern "C"
void freeTexture()
{
    checkCudaErrors(hipFreeArray(d_imageArray));
}


// render image using CUDA
extern "C"
void render(int width, int height, float tx, float ty, float scale, float cx, float cy,
            dim3 blockSize, dim3 gridSize, int filter_mode, uchar4 *output)
{
    // call CUDA kernel, writing results to PBO memory
    switch (filter_mode)
    {
        case MODE_NEAREST:
            tex.filterMode = hipFilterModePoint;
            d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_BILINEAR:
            tex.filterMode = hipFilterModeLinear;
            d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_BICUBIC:
            tex.filterMode = hipFilterModePoint;
            d_renderBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_FAST_BICUBIC:
            tex.filterMode = hipFilterModeLinear;
            d_renderFastBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_CATROM:
            tex.filterMode = hipFilterModePoint;
            d_renderCatRom<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;
    }

    getLastCudaError("kernel failed");
}

#endif
