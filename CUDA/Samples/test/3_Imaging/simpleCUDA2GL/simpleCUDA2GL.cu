#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Utilities and system includes

#include <hip/hip_runtime_api.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

__global__ void
cudaProcess(unsigned int *g_odata, int imgw)
{
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    uchar4 c4 = make_uchar4((x & 0x20)?100:0,0,(y & 0x20)?100:0,0);
    g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void
launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                   unsigned int *g_odata,
                   int imgw)
{
    cudaProcess<<< grid, block, sbytes >>>(g_odata, imgw);

}
