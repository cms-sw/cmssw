#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



/*
 * This sample demonstrates two adaptive image denoising techniques:
 * KNN and NLM, based on computation of both geometric and color distance
 * between texels. While both techniques are already implemented in the
 * DirectX SDK using shaders, massively speeded up variation
 * of the latter technique, taking advantage of shared memory, is implemented
 * in addition to DirectX counterparts.
 * See supplied whitepaper for more explanations.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "imageDenoising.h"


////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
float Max(float x, float y)
{
    return (x > y) ? x : y;
}

float Min(float x, float y)
{
    return (x < y) ? x : y;
}

int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__ float lerpf(float a, float b, float c)
{
    return a + (b - a) * c;
}

__device__ float vecLen(float4 a, float4 b)
{
    return (
               (b.x - a.x) * (b.x - a.x) +
               (b.y - a.y) * (b.y - a.y) +
               (b.z - a.z) * (b.z - a.z)
           );
}

__device__ TColor make_color(float r, float g, float b, float a)
{
    return
        ((int)(a * 255.0f) << 24) |
        ((int)(b * 255.0f) << 16) |
        ((int)(g * 255.0f) <<  8) |
        ((int)(r * 255.0f) <<  0);
}



////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
//Texture reference and channel descriptor for image texture
texture<uchar4, 2, hipReadModeNormalizedFloat> texImage;
hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

//CUDA array descriptor
hipArray *a_Src;

////////////////////////////////////////////////////////////////////////////////
// Filtering kernels
////////////////////////////////////////////////////////////////////////////////
#include "imageDenoising_copy_kernel.cuh"
#include "imageDenoising_knn_kernel.cuh"
#include "imageDenoising_nlm_kernel.cuh"
#include "imageDenoising_nlm2_kernel.cuh"

extern "C"
hipError_t CUDA_Bind2TextureArray()
{
    return hipBindTextureToArray(texImage, a_Src);
}

extern "C"
hipError_t CUDA_UnbindTexture()
{
    return hipUnbindTexture(texImage);
}

extern "C"
hipError_t CUDA_MallocArray(uchar4 **h_Src, int imageW, int imageH)
{
    hipError_t error;

    error = hipMallocArray(&a_Src, &uchar4tex, imageW, imageH);
    error = hipMemcpyToArray(a_Src, 0, 0,
                              *h_Src, imageW * imageH * sizeof(uchar4),
                              hipMemcpyHostToDevice
                             );

    return error;
}


extern "C"
hipError_t CUDA_FreeArray()
{
    return hipFreeArray(a_Src);
}

