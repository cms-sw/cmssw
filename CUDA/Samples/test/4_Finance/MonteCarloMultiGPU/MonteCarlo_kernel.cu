#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

////////////////////////////////////////////////////////////////////////////////
// Global types
////////////////////////////////////////////////////////////////////////////////
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo_common.h"

////////////////////////////////////////////////////////////////////////////////
// Helper reduction template
// Please see the "reduction" CUDA Sample for more information
////////////////////////////////////////////////////////////////////////////////
#include "MonteCarlo_reduction.cuh"

////////////////////////////////////////////////////////////////////////////////
// Internal GPU-side data structures
////////////////////////////////////////////////////////////////////////////////
#define MAX_OPTIONS (1024*1024)

//Preprocessed input option data
typedef struct
{
    real S;
    real X;
    real MuByT;
    real VBySqrtT;
} __TOptionData;

////////////////////////////////////////////////////////////////////////////////
// Overloaded shortcut payoff functions for different precision modes
////////////////////////////////////////////////////////////////////////////////
__device__ inline float endCallValue(float S, float X, float r, float MuByT, float VBySqrtT)
{
    float callValue = S * __expf(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0F) ? callValue : 0.0F;
}

__device__ inline double endCallValue(double S, double X, double r, double MuByT, double VBySqrtT)
{
    double callValue = S * exp(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0) ? callValue : 0.0;
}

#define THREAD_N 256

////////////////////////////////////////////////////////////////////////////////
// This kernel computes the integral over all paths using a single thread block
// per option. It is fastest when the number of thread blocks times the work per
// block is high enough to keep the GPU busy.
////////////////////////////////////////////////////////////////////////////////
static __global__ void MonteCarloOneBlockPerOption(
    hiprandState * __restrict rngStates,
    const __TOptionData * __restrict d_OptionData,
    __TOptionValue * __restrict d_CallValue,
    int pathN,
    int optionN)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    const int SUM_N = THREAD_N;
    __shared__ real s_SumCall[SUM_N];
    __shared__ real s_Sum2Call[SUM_N];

    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Copy random number state to local memory for efficiency
    hiprandState localState = rngStates[tid];
    for(int optionIndex = blockIdx.x; optionIndex < optionN; optionIndex += gridDim.x)
    {
        const real        S = d_OptionData[optionIndex].S;
        const real        X = d_OptionData[optionIndex].X;
        const real    MuByT = d_OptionData[optionIndex].MuByT;
        const real VBySqrtT = d_OptionData[optionIndex].VBySqrtT;

        //Cycle through the entire samples array:
        //derive end stock price for each path
        //accumulate partial integrals into intermediate shared memory buffer
        for (int iSum = threadIdx.x; iSum < SUM_N; iSum += blockDim.x)
        {
            __TOptionValue sumCall = {0, 0};

            #pragma unroll 8
            for (int i = iSum; i < pathN; i += SUM_N)
            {
                real              r = hiprand_normal(&localState);
                real      callValue = endCallValue(S, X, r, MuByT, VBySqrtT);
                sumCall.Expected   += callValue;
                sumCall.Confidence += callValue * callValue;
            }

            s_SumCall[iSum]  = sumCall.Expected;
            s_Sum2Call[iSum] = sumCall.Confidence;
        }

        //Reduce shared memory accumulators
        //and write final result to global memory
        cg::sync(cta);
        sumReduce<real, SUM_N, THREAD_N>(s_SumCall, s_Sum2Call, cta, tile32, &d_CallValue[optionIndex]);
    }
}

static __global__ void rngSetupStates(
    hiprandState *rngState,
    int device_id)
{
    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each threadblock gets different seed,
    // Threads within a threadblock get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x * device_id, threadIdx.x, 0, &rngState[tid]);
}



////////////////////////////////////////////////////////////////////////////////
// Host-side interface to GPU Monte Carlo
////////////////////////////////////////////////////////////////////////////////

extern "C" void initMonteCarloGPU(TOptionPlan *plan)
{
    checkCudaErrors(hipMalloc(&plan->d_OptionData, sizeof(__TOptionData)*(plan->optionCount)));
    checkCudaErrors(hipMalloc(&plan->d_CallValue, sizeof(__TOptionValue)*(plan->optionCount)));
    checkCudaErrors(hipHostMalloc(&plan->h_OptionData, sizeof(__TOptionData)*(plan->optionCount)));
    //Allocate internal device memory
    checkCudaErrors(hipHostMalloc(&plan->h_CallValue, sizeof(__TOptionValue)*(plan->optionCount)));
    //Allocate states for pseudo random number generators
    checkCudaErrors(hipMalloc((void **) &plan->rngStates,
                               plan->gridSize * THREAD_N * sizeof(hiprandState)));
    checkCudaErrors(hipMemset(plan->rngStates, 0, plan->gridSize * THREAD_N * sizeof(hiprandState)));

    // place each device pathN random numbers apart on the random number sequence
    rngSetupStates<<<plan->gridSize, THREAD_N>>>(plan->rngStates, plan->device);
    getLastCudaError("rngSetupStates kernel failed.\n");
}

//Compute statistics and deallocate internal device memory
extern "C" void closeMonteCarloGPU(TOptionPlan *plan)
{
    for (int i = 0; i < plan->optionCount; i++)
    {
        const double    RT = plan->optionData[i].R * plan->optionData[i].T;
        const double   sum = plan->h_CallValue[i].Expected;
        const double  sum2 = plan->h_CallValue[i].Confidence;
        const double pathN = plan->pathN;
        //Derive average from the total sum and discount by riskfree rate
        plan->callValue[i].Expected = (float)(exp(-RT) * sum / pathN);
        //Standard deviation
        double stdDev = sqrt((pathN * sum2 - sum * sum)/ (pathN * (pathN - 1)));
        //Confidence width; in 95% of all cases theoretical value lies within these borders
        plan->callValue[i].Confidence = (float)(exp(-RT) * 1.96 * stdDev / sqrt(pathN));
    }

    checkCudaErrors(hipFree(plan->rngStates));
    checkCudaErrors(hipHostFree(plan->h_CallValue));
    checkCudaErrors(hipHostFree(plan->h_OptionData));
    checkCudaErrors(hipFree(plan->d_CallValue));
    checkCudaErrors(hipFree(plan->d_OptionData));
}

//Main computations
extern "C" void MonteCarloGPU(TOptionPlan *plan, hipStream_t stream)
{
    __TOptionValue *h_CallValue = plan->h_CallValue;

    if (plan->optionCount <= 0 || plan->optionCount > MAX_OPTIONS)
    {
        printf("MonteCarloGPU(): bad option count.\n");
        return;
    }

    __TOptionData * h_OptionData = (__TOptionData *)plan->h_OptionData;

    for (int i = 0; i < plan->optionCount; i++)
    {
        const double           T = plan->optionData[i].T;
        const double           R = plan->optionData[i].R;
        const double           V = plan->optionData[i].V;
        const double       MuByT = (R - 0.5 * V * V) * T;
        const double    VBySqrtT = V * sqrt(T);
        h_OptionData[i].S        = (real)plan->optionData[i].S;
        h_OptionData[i].X        = (real)plan->optionData[i].X;
        h_OptionData[i].MuByT    = (real)MuByT;
        h_OptionData[i].VBySqrtT = (real)VBySqrtT;
    }

    checkCudaErrors(hipMemcpyAsync(
                        plan->d_OptionData,
                        h_OptionData,
                        plan->optionCount * sizeof(__TOptionData),
                        hipMemcpyHostToDevice, stream
                    ));

    MonteCarloOneBlockPerOption<<<plan->gridSize, THREAD_N, 0, stream>>>(
        plan->rngStates,
        (__TOptionData *)(plan->d_OptionData),
        (__TOptionValue *)(plan->d_CallValue),
        plan->pathN,
        plan->optionCount
    );
    getLastCudaError("MonteCarloOneBlockPerOption() execution failed\n");


    checkCudaErrors(hipMemcpyAsync(
                        h_CallValue,
                        plan->d_CallValue,
                        plan->optionCount * sizeof(__TOptionValue), hipMemcpyDeviceToHost, stream
                    ));

    //hipDeviceSynchronize();
}

