#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_runtime_api.h>
#include <math.h>

//#include <GL/glew.h>
//#include <GL/freeglut.h>

// CUDA standard includes
#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>

#include "bodysystem.h"

__constant__ float softeningSquared;
__constant__ double softeningSquared_fp64;

hipError_t setSofteningSquared(float softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared),
                              &softeningSq,
                              sizeof(float), 0,
                              hipMemcpyHostToDevice);
}

hipError_t setSofteningSquared(double softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared_fp64),
                              &softeningSq,
                              sizeof(double), 0,
                              hipMemcpyHostToDevice);
}

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<typename T>
__device__ T rsqrt_T(T x)
{
    return rsqrt(x);
}

template<>
__device__ float rsqrt_T<float>(float x)
{
    return rsqrtf(x);
}

template<>
__device__ double rsqrt_T<double>(double x)
{
    return rsqrt(x);
}


// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i+blockDim.x*threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i,j) sharedPos[i+blockDim.x*j]

template <typename T>
__device__ T getSofteningSquared()
{
    return softeningSquared;
}
template <>
__device__ double getSofteningSquared<double>()
{
    return softeningSquared_fp64;
}

template <typename T>
struct DeviceData
{
    T *dPos[2]; // mapped host pointers
    T *dVel;
    hipEvent_t  event;
    unsigned int offset;
    unsigned int numBodies;
};


template <typename T>
__device__ typename vec3<T>::Type
bodyBodyInteraction(typename vec3<T>::Type ai,
                    typename vec4<T>::Type bi,
                    typename vec4<T>::Type bj)
{
    typename vec3<T>::Type r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    T distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
    distSqr += getSofteningSquared<T>();

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    T invDist = rsqrt_T(distSqr);
    T invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    T s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}

template <typename T>
__device__ typename vec3<T>::Type
computeBodyAccel(typename vec4<T>::Type bodyPos,
                 typename vec4<T>::Type *positions,
                 int numTiles)
{
    typename vec4<T>::Type *sharedPos = SharedMemory<typename vec4<T>::Type>();

    typename vec3<T>::Type acc = {0.0f, 0.0f, 0.0f};

    for (int tile = 0; tile < numTiles; tile++)
    {
        sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

        __syncthreads();

        // This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128

        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
            acc = bodyBodyInteraction<T>(acc, bodyPos, sharedPos[counter]);
        }

        __syncthreads();
    }

    return acc;
}

template<typename T>
__global__ void
integrateBodies(typename vec4<T>::Type *__restrict__ newPos,
                typename vec4<T>::Type *__restrict__ oldPos,
                typename vec4<T>::Type *vel,
                unsigned int deviceOffset, unsigned int deviceNumBodies,
                float deltaTime, float damping, int numTiles)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= deviceNumBodies)
    {
        return;
    }

    typename vec4<T>::Type position = oldPos[deviceOffset + index];

    typename vec3<T>::Type accel = computeBodyAccel<T>(position,
                                                       oldPos,
                                                       numTiles);

    // acceleration = force / mass;
    // new velocity = old velocity + acceleration * deltaTime
    // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
    // (because they cancel out).  Thus here force == acceleration
    typename vec4<T>::Type velocity = vel[deviceOffset + index];

    velocity.x += accel.x * deltaTime;
    velocity.y += accel.y * deltaTime;
    velocity.z += accel.z * deltaTime;

    velocity.x *= damping;
    velocity.y *= damping;
    velocity.z *= damping;

    // new position = old position + velocity * deltaTime
    position.x += velocity.x * deltaTime;
    position.y += velocity.y * deltaTime;
    position.z += velocity.z * deltaTime;

    // store new position and velocity
    newPos[deviceOffset + index] = position;
    vel[deviceOffset + index]    = velocity;
}

template <typename T>
void integrateNbodySystem(DeviceData<T> *deviceData,
                          hipGraphicsResource **pgres,
                          unsigned int currentRead,
                          float deltaTime,
                          float damping,
                          unsigned int numBodies,
                          unsigned int numDevices,
                          int blockSize,
                          bool bUsePBO)
{
    if (bUsePBO)
    {
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[currentRead], cudaGraphicsMapFlagsReadOnly));
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[1-currentRead], cudaGraphicsMapFlagsWriteDiscard));
        checkCudaErrors(hipGraphicsMapResources(2, pgres, 0));
        size_t bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[currentRead]), &bytes, pgres[currentRead]));
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[1-currentRead]), &bytes, pgres[1-currentRead]));
    }

    for (unsigned int dev = 0; dev != numDevices; dev++)
    {
        if (numDevices > 1)
        {
            hipSetDevice(dev);
        }

        int numBlocks = (deviceData[dev].numBodies + blockSize-1) / blockSize;
        int numTiles = (numBodies + blockSize - 1) / blockSize;
        int sharedMemSize = blockSize * 4 * sizeof(T); // 4 floats for pos

        integrateBodies<T><<< numBlocks, blockSize, sharedMemSize >>>
        ((typename vec4<T>::Type *)deviceData[dev].dPos[1-currentRead],
         (typename vec4<T>::Type *)deviceData[dev].dPos[currentRead],
         (typename vec4<T>::Type *)deviceData[dev].dVel,
         deviceData[dev].offset, deviceData[dev].numBodies,
         deltaTime, damping, numTiles);

        if (numDevices > 1)
        {
            checkCudaErrors(hipEventRecord(deviceData[dev].event));
            // MJH: Hack on older driver versions to force kernel launches to flush!
            hipStreamQuery(0);
        }

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    if (numDevices > 1)
    {
        for (unsigned int dev = 0; dev < numDevices; dev++)
        {
            checkCudaErrors(hipEventSynchronize(deviceData[dev].event));
        }
    }

    if (bUsePBO)
    {
        checkCudaErrors(hipGraphicsUnmapResources(2, pgres, 0));
    }
}


// Explicit specializations needed to generate code
template void integrateNbodySystem<float>(DeviceData<float> *deviceData,
                                          hipGraphicsResource **pgres,
                                          unsigned int currentRead,
                                          float deltaTime,
                                          float damping,
                                          unsigned int numBodies,
                                          unsigned int numDevices,
                                          int blockSize,
                                          bool bUsePBO);

template void integrateNbodySystem<double>(DeviceData<double> *deviceData,
                                           hipGraphicsResource **pgres,
                                           unsigned int currentRead,
                                           float deltaTime,
                                           float damping,
                                           unsigned int numBodies,
                                           unsigned int numDevices,
                                           int blockSize,
                                           bool bUsePBO);
