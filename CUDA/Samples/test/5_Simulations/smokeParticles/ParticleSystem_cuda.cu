#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
This file contains simple wrapper functions that call the CUDA kernels
*/
#define HELPERGL_EXTERN_GL_FUNC_IMPLEMENTATION
#include <helper_gl.h>
#include <hip/hip_runtime_api.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_device.cuh"
#include "ParticleSystem.cuh"

extern "C"
{

    hipArray *noiseArray;

    void initCuda(bool bUseGL)
    {
        if (bUseGL)
        {
            cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
        }
        else
        {
            hipSetDevice(gpuGetMaxGflopsDeviceId());
        }
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    int iDivUp(int a, int b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    inline float frand()
    {
        return rand() / (float) RAND_MAX;
    }

    // create 3D texture containing random values
    void createNoiseTexture(int w, int h, int d)
    {
        hipExtent size = make_hipExtent(w, h, d);
        size_t elements = size.width*size.height*size.depth;

        float *volumeData = (float *)malloc(elements*4*sizeof(float));
        float *ptr = volumeData;

        for (size_t i=0; i<elements; i++)
        {
            *ptr++ = frand()*2.0f-1.0f;
            *ptr++ = frand()*2.0f-1.0f;
            *ptr++ = frand()*2.0f-1.0f;
            *ptr++ = frand()*2.0f-1.0f;
        }


        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
        checkCudaErrors(hipMalloc3DArray(&noiseArray, &channelDesc, size));

        hipMemcpy3DParms copyParams = { 0 };
        copyParams.srcPtr   = make_hipPitchedPtr((void *)volumeData, size.width*sizeof(float4), size.width, size.height);
        copyParams.dstArray = noiseArray;
        copyParams.extent   = size;
        copyParams.kind     = hipMemcpyHostToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));

        free(volumeData);

        // set texture parameters
        noiseTex.normalized = true;                      // access with normalized texture coordinates
        noiseTex.filterMode = hipFilterModeLinear;      // linear interpolation
        noiseTex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
        noiseTex.addressMode[1] = hipAddressModeWrap;
        noiseTex.addressMode[2] = hipAddressModeWrap;

        // bind array to 3D texture
        checkCudaErrors(hipBindTextureToArray(noiseTex, noiseArray, channelDesc));
    }

    void
    integrateSystem(float4 *oldPos, float4 *newPos,
                    float4 *oldVel, float4 *newVel,
                    float deltaTime,
                    int numParticles)
    {
        thrust::device_ptr<float4> d_newPos(newPos);
        thrust::device_ptr<float4> d_newVel(newVel);
        thrust::device_ptr<float4> d_oldPos(oldPos);
        thrust::device_ptr<float4> d_oldVel(oldVel);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_newPos, d_newVel, d_oldPos, d_oldVel)),
            thrust::make_zip_iterator(thrust::make_tuple(d_newPos+numParticles, d_newVel+numParticles, d_oldPos+numParticles, d_oldVel+numParticles)),
            integrate_functor(deltaTime));
    }

    void
    calcDepth(float4  *pos,
              float   *keys,        // output
              uint    *indices,     // output
              float3   sortVector,
              int      numParticles)
    {
        thrust::device_ptr<float4> d_pos(pos);
        thrust::device_ptr<float> d_keys(keys);
        thrust::device_ptr<uint> d_indices(indices);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos, d_keys)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos+numParticles, d_keys+numParticles)),
            calcDepth_functor(sortVector));

        thrust::sequence(d_indices, d_indices + numParticles);
    }

    void sortParticles(float *sortKeys, uint *indices, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<float>(sortKeys),
                            thrust::device_ptr<float>(sortKeys + numParticles),
                            thrust::device_ptr<uint>(indices));
    }

}   // extern "C"
