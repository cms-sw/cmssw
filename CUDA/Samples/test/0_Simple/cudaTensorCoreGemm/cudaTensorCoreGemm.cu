#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// CUDA sample demonstrating a GEMM computation using the Warp Matrix Multiply
// and Accumulate API introduced in CUDA 9.

// In this program, the compute_gemm kernel computes the result of a matrix multiplication
// and addition: D = alpha * A * B + beta * C. The dimensions of both C and D matrices
// are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x K_GLOBAL (row-major), the B matrix
// is K_GLOBAL x N_GLOBAL (column-major).
// In that kernel, each CTA computes one 128 x 128 tile of the resulting matrix
// per iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes eight
// 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array.
// Warps compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and accumulating
// the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments from
//   shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the A and B
//   data from shared memory, thus reducing the number of data copies from global memory.
// - The portions of the A and B matrices are stored in shared memory with an additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_HALF macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory contents to
//   global memory, again avoiding redundant random global memory accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

// GEMM configuration.

#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#define CHUNK_K 8

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B matrix
// in shared memory to minimize possible bank conflicts.
// Before performing the nvcuda::wmma::mma_sync operation, the warp must load the matrix
// data using the nvcuda::wmma::load_matrix_sync operation. Although the memory access pattern
// is not specified for that function, each lane in the warp can read one or multiple matrix
// elements from different matrix rows or columns.
// For shared memory, such access can result in bank conflicts if different rows / columns
// of the matrix map to the same bank. By shifting each row and column by a few bytes, we
// make sure that they map to different banks, thus reducing the number of possible bank
// conflicts.
// The number of 8 two-byte "half" elements is chosen as the minimum possible shift because
// we must keep each row and column 128-bit aligned, as required by nvcuda::wmma::load_matrix_sync.
#define SKEW_HALF 8

#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)

using namespace nvcuda;

__host__ void init_host_matrices(float *a, float *b, float *c)
{
    for (int i = 0; i < M_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            a[i*K_GLOBAL+j] = (float)(rand() % 3);
        }
    }

    for (int i = 0; i < N_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            b[i*K_GLOBAL+j] = (float)(rand() % 3);
        }
    }

    for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
        c[t] = (float)(rand() % 3);
    }
}

__global__ void init_device_matrices(const float *A_h, const float *B_h, const float *C_h, half *A, half *B, float *C, float *D)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < M_GLOBAL * K_GLOBAL; i += gridDim.x * blockDim.x)
        A[i] = __float2half(A_h[i]);

    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < N_GLOBAL * K_GLOBAL; i += gridDim.x * blockDim.x)
        B[i] = __float2half(B_h[i]);

    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < M_GLOBAL * N_GLOBAL; i += gridDim.x * blockDim.x)
        C[i] = C_h[i];

    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < M_GLOBAL * N_GLOBAL; i += gridDim.x * blockDim.x)
        D[i] = 0;
}

__global__ void compute_gemm(const half *A, const half *B, const float *C, float *D, float alpha, float beta)
{
    extern __shared__ half shmem[][CHUNK_K * K + SKEW_HALF];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    float *shmem_warp_tile_ptr = (float*)&shmem[0][0] + (warpId/2) * SHMEM_STRIDE * K * 2 + (warpId%2) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    float *shmem_warp_stream_ptr = (float*)&shmem[0][0] + warpId * SHMEM_STRIDE * K;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    // Each CTA slides along the 128 x 128 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_COL_TILES) / N_TILES) * (BLOCK_ROW_WARPS * WARP_ROW_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < K; i++) {
            typedef int4 copy_t;

            *((copy_t *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) = 
                *((copy_t *)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId);
        }

        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Scale the C matrix.
#pragma unroll
       for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const half *warp_ptr = (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % 4) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % 4) * 2);

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            size_t shmem_idx = warpId < (WARPS_PER_BLOCK/2) ? (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) : 
                                                              (N * (warpId % (WARPS_PER_BLOCK/2)) * 2 + shmem_idx_b_off);

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            int4 *lane_ptr = (int4*)(warp_ptr + tile_k * K + (laneId / (WARP_SIZE/2)) * K_GLOBAL) + (laneId % (WARP_SIZE/2));

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += laneId / (WARP_SIZE/2);

#pragma unroll
            for(int i = 0; i < (WARP_SIZE/2); i++) {
                // Copy 16 bytes at once in each lane.
                *((int4*)&shmem[shmem_idx][0] + (laneId % (WARP_SIZE/2))) = *lane_ptr;

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = (int4*)((half*)lane_ptr + K_GLOBAL * 2);
                shmem_idx += 2;
            }

            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId/2) * M * 2 + (i * M);
                    const half *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_HALF);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const half *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_HALF);
                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }

            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < K; i++) {
            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
}

int main(int argc, char **argv)
{
    printf("Initializing...\n");

    int dev = findCudaDevice(argc, (const char **)argv);

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    // Tensor cores require a GPU of Volta (SM7X) architecture or higher.
    if (deviceProp.major < 7) {
        printf("cudaTensorCoreGemm requires requires SM 7.0 or higher to use Tensor Cores.  Exiting...\n");
        exit(EXIT_WAIVED);
    }

    printf("M: %d (%d x %d)\n", M_GLOBAL, M, M_TILES);
    printf("N: %d (%d x %d)\n", N_GLOBAL, N, N_TILES);
    printf("K: %d (%d x %d)\n", K_GLOBAL, K, K_TILES);

    float *A_h = NULL;
    float *B_h = NULL;
    float *C_h = NULL;

    checkCudaErrors(hipMallocManaged((void**)&A_h, sizeof(float) * M_GLOBAL * K_GLOBAL));
    checkCudaErrors(hipMallocManaged((void**)&B_h, sizeof(float) * K_GLOBAL * N_GLOBAL));
    checkCudaErrors(hipMallocManaged((void**)&C_h, sizeof(float) * M_GLOBAL * N_GLOBAL));

    half *A = NULL;
    half *B = NULL;
    float *C = NULL;
    float *D = NULL;

    checkCudaErrors(hipMalloc((void**)&A, sizeof(half) * M_GLOBAL * K_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&B, sizeof(half) * N_GLOBAL * K_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&C, sizeof(float) * M_GLOBAL * N_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&D, sizeof(float) * M_GLOBAL * N_GLOBAL));

    assert(((unsigned long long)A) % 128 == 0);
    assert(((unsigned long long)B) % 128 == 0);
    assert(((unsigned long long)C) % 128 == 0);
    assert(((unsigned long long)D) % 128 == 0);

    init_host_matrices(A_h, B_h, C_h);

    printf("Preparing data for GPU...\n");

    checkKernelErrors((init_device_matrices<<<deviceProp.multiProcessorCount, THREADS_PER_BLOCK>>>(A_h, B_h, C_h, A, B, C, D)));

    checkCudaErrors(hipDeviceSynchronize());

    enum { SHMEM_SZ = sizeof(half) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_HALF) * 2 };

    printf("Required shared memory size: %lu Kb\n", SHMEM_SZ / 1024UL);

    checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(compute_gemm), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));

    printf("Computing...\n");

    hipEvent_t start, stop;

    checkCudaErrors(hipEventCreate(&start));    
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));

    const float alpha = 1.1f;
    const float beta = 1.2f;

    checkKernelErrors((compute_gemm<<<deviceProp.multiProcessorCount, THREADS_PER_BLOCK, SHMEM_SZ>>>(A, B, C, D, alpha, beta)));

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    
    float milliseconds = 0;

    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    printf("Time: %f ms\n", milliseconds);
    printf("TFLOPS: %.2f\n", (((double)M_GLOBAL * N_GLOBAL * K_GLOBAL * 2)/(milliseconds/1000.)) / 1e12);

    checkCudaErrors(hipFree((void*)A_h));
    checkCudaErrors(hipFree((void*)B_h));
    checkCudaErrors(hipFree((void*)C_h));
    checkCudaErrors(hipFree((void*)A));
    checkCudaErrors(hipFree((void*)B));
    checkCudaErrors(hipFree((void*)C));
    checkCudaErrors(hipFree((void*)D));

    return 0;
}
