#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 *
 * This sample is a simple code that illustrates basic usage of
 * cooperative groups within the thread block. The code launches a single
 * thread block, creates a cooperative group of all threads in the block,
 * and a set of tiled partition cooperative groups. For each, it uses a
 * generic reduction function to calculate the sum of all the ranks in
 * that group. In each case the result is printed, together with the
 * expected answer (which is calculated using the analytical formula
 * (n-1)*n)/2, noting that the ranks start at zero).
 *
 */

#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;


/**
 * CUDA device function
 * 
 * calculates the sum of val across the group g. The workspace array, x, 
 * must be large enough to contain g.size() integers.
 */
__device__ int sumReduction(thread_group g, int *x, int val) 
{ 
    // rank of this thread in the group 
    int lane = g.thread_rank(); 

    // for each iteration of this loop, the number of threads active in the
    // reduction, i, is halved, and each active thread (with index [lane])
    // performs a single summation of it's own value with that
    // of a "partner" (with index [lane+i]). 
    for (int i = g.size()/2; i > 0; i /= 2) 
        { 

            // store value for this thread in temporary array
            x[lane] = val;

            // synchronize all threads in group
            g.sync();


            if(lane<i)
                // active threads perform summation of their value with
                // their partner's value
                val += x[lane + i];
      
            // synchronize all threads in group
            g.sync();
      
        }

    // master thread in group returns result, and others return -1.
    if (g.thread_rank()==0)
        return val; 
    else
        return -1;
}


/**
 * CUDA kernel device code
 * 
 * Creates cooperative groups and performs reductions
 */
__global__ void cgkernel(){


    // threadBlockGroup includes all threads in the block
    thread_block threadBlockGroup = this_thread_block();
    int threadBlockGroupSize=threadBlockGroup.size();

    // workspace array in shared memory required for reduction
    extern __shared__ int workspace[];

    int input, output, expectedOutput;

    // input to reduction, for each thread, is its' rank in the group
    input=threadBlockGroup.thread_rank();

    // expected output from analytical formula (n-1)(n)/2
    // (noting that indexing starts at 0 rather than 1)
    expectedOutput=(threadBlockGroupSize-1)*threadBlockGroupSize/2;

    // perform reduction
    output=sumReduction(threadBlockGroup, workspace, input);

    // master thread in group prints out result
    if(threadBlockGroup.thread_rank()==0){
    
        printf(" Sum of all ranks 0..%d in threadBlockGroup is %d (expected %d)\n\n",
               threadBlockGroup.size()-1,output,
               expectedOutput);
    
        printf(" Now creating %d groups, each of size 16 threads:\n\n",
               threadBlockGroup.size()/16);
    
    }

    threadBlockGroup.sync();
  
    // each tiledPartition16 group includes 16 threads
    thread_block_tile<16> tiledPartition16 = tiled_partition<16>(threadBlockGroup);

    // This offset allows each group to have its own unique area in the workspace array
    int workspaceOffset=threadBlockGroup.thread_rank()-tiledPartition16.thread_rank();

    // input to reduction, for each thread, is its' rank in the group
    input=tiledPartition16.thread_rank();

    // expected output from analytical formula (n-1)(n)/2
    // (noting that indexing starts at 0 rather than 1)
    expectedOutput=15*16/2;
  
    // Perform reduction
    output=sumReduction(tiledPartition16, workspace+workspaceOffset, input);

    // each master thread prints out result
    if(tiledPartition16.thread_rank()==0)
        printf("   Sum of all ranks 0..15 in this tiledPartition16 group is %d (expected %d)\n",output,expectedOutput);

    return;
  
}


/**
 * Host main routine
 */
int main(){

    // Error code to check return values for CUDA calls
    hipError_t err;

    //Launch the kernel

    int blocksPerGrid=1;
    int threadsPerBlock=64;

    printf("\nLaunching a single block with %d threads...\n\n",threadsPerBlock);

    // we use the optional third argument to specify the size
    // of shared memory required in the kernel
    cgkernel <<<blocksPerGrid,threadsPerBlock,threadsPerBlock*sizeof(int)>>> (); 
    err = hipDeviceSynchronize();

    if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

    printf("\n...Done.\n\n");

    return 0;
}
