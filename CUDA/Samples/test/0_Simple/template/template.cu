#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size,
                               hipMemcpyHostToDevice));

    // allocate device memory for result
    float *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,
                               hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // compute reference solution
    float *reference = (float *) malloc(mem_size);
    computeGold(reference, h_idata, num_threads);

    // check result
    if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
    {
        // write file for regression test
        sdkWriteFile("./data/regression.dat", h_odata, num_threads, 0.0f, false);
    }
    else
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected solution
        bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);
    }

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
