#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2016 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* A program demonstrating trivial use of system-wide atomics on migratable memory.
 */
 
#include <math.h>
#include <cstdio>
#include <ctime>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define min(a,b) (a) < (b) ? (a) : (b)
#define max(a,b) (a) > (b) ? (a) : (b)

#define LOOP_NUM 50

 __global__ void atomicKernel(int *atom_arr)
 {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i=0; i < LOOP_NUM; i++)
    {
        // Atomic addition
        atomicAdd_system(&atom_arr[0], 10);

        // Atomic exchange
        atomicExch_system(&atom_arr[1], tid);

        // Atomic maximum
        atomicMax_system(&atom_arr[2], tid);

        // Atomic minimum
        atomicMin_system(&atom_arr[3], tid);

        // Atomic increment (modulo 17+1)
        atomicInc_system((unsigned int *)&atom_arr[4], 17);

        // Atomic decrement
        atomicDec_system((unsigned int *)&atom_arr[5], 137);

        // Atomic compare-and-swap
        atomicCAS_system(&atom_arr[6], tid-1, tid);

        // Bitwise atomic instructions

        // Atomic AND
        atomicAnd_system(&atom_arr[7], 2*tid+7);

        // Atomic OR
        atomicOr_system(&atom_arr[8], 1 << tid);

        // Atomic XOR
        atomicXor_system(&atom_arr[9], tid);
    }
 }

void atomicKernel_CPU(int *atom_arr, int no_of_threads)
{

    for (int i=no_of_threads; i<2*no_of_threads; i++)
    {

        for (int j=0; j < LOOP_NUM; j++)
        {
            // Atomic addition
            __sync_fetch_and_add(&atom_arr[0],10);

            // Atomic exchange
             __sync_lock_test_and_set(&atom_arr[1], i);

            // Atomic maximum
            int old, expected;
            do {
                expected = atom_arr[2];
                old = __sync_val_compare_and_swap(&atom_arr[2], expected, max(expected, i));
            } while (old != expected);

            // Atomic minimum
            do {
                expected = atom_arr[3];
                old = __sync_val_compare_and_swap(&atom_arr[3], expected, min(expected, i));
            } while (old != expected);

            // Atomic increment (modulo 17+1)
            int limit = 17;
            do {
                expected = atom_arr[4];
                old = __sync_val_compare_and_swap(&atom_arr[4], expected, (expected >= limit) ? 0 : expected+1);
            } while (old != expected);

            // Atomic decrement
            limit = 137;
            do {
                expected = atom_arr[5];
                old = __sync_val_compare_and_swap(&atom_arr[5], expected, ((expected == 0) || (expected > limit)) ? limit : expected-1);
            } while (old != expected);

            // Atomic compare-and-swap
           __sync_val_compare_and_swap(&atom_arr[6], i-1, i);


            // Bitwise atomic instructions

            // Atomic AND
             __sync_fetch_and_and(&atom_arr[7] , 2 * i + 7);

            // Atomic OR
             __sync_fetch_and_or(&atom_arr[8], 1 << i);

            // Atomic XOR
             // 11th element should be 0xff
            __sync_fetch_and_xor (&atom_arr[9] ,i);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! Each element is multiplied with the number of threads / array length
//! @param reference  reference data, computed but preallocated
//! @param idata      input data as provided to device
//! @param len        number of elements in reference / idata
////////////////////////////////////////////////////////////////////////////////
int verify(int *testData, const int len)
{
    int val = 0;

    for (int i = 0; i < len*LOOP_NUM; ++i)
    {
        val += 10;
    }

    if (val != testData[0])
    {
        printf("atomicAdd failed val = %d testData = %d\n", val, testData[0]);
        return false;
    }

    val = 0;

    bool found = false;

    for (int i = 0; i < len; ++i)
    {
        // second element should be a member of [0, len)
        if (i == testData[1])
        {
            found = true;
            break;
        }
    }

    if (!found)
    {
        printf("atomicExch failed\n");
        return false;
    }

    val = -(1 << 8);

    for (int i = 0; i < len; ++i)
    {
        // third element should be len-1
        val = max(val, i);
    }

    if (val != testData[2])
    {
        printf("atomicMax failed\n");
        return false;
    }

    val = 1 << 8;

    for (int i = 0; i < len; ++i)
    {
        val = min(val, i);
    }

    if (val != testData[3])
    {
        printf("atomicMin failed\n");
        return false;
    }

    int limit = 17;
    val = 0;

    for (int i = 0; i < len*LOOP_NUM; ++i)
    {
        val = (val >= limit) ? 0 : val+1;
    }

    if (val != testData[4])
    {
        printf("atomicInc failed\n");
        return false;
    }

    limit = 137;
    val = 0;

    for (int i = 0; i < len*LOOP_NUM; ++i)
    {
        val = ((val == 0) || (val > limit)) ? limit : val-1;
    }

    if (val != testData[5])
    {
        printf("atomicDec failed\n");
        return false;
    }

    found = false;

    for (int i = 0; i < len; ++i)
    {
        // seventh element should be a member of [0, len)
        if (i == testData[6])
        {
            found = true;
            break;
        }
    }

    if (!found)
    {
        printf("atomicCAS failed\n");
        return false;
    }

    val = 0xff;

    for (int i = 0; i < len; ++i)
    {
        // 8th element should be 1
        val &= (2 * i + 7);
    }

    if (val != testData[7])
    {
        printf("atomicAnd failed\n");
        return false;
    }

    val = 0;

    for (int i = 0; i < len; ++i)
    {
        // 9th element should be 0xff
        val |= (1 << i);
    }

    if (val != testData[8])
    {
        printf("atomicOr failed\n");
        return false;
    }

    val = 0xff;

    for (int i = 0; i < len; ++i)
    {
        // 11th element should be 0xff
        val ^= i;
    }

    if (val != testData[9])
    {
        printf("atomicXor failed\n");
        return false;
    }

    return true;
}


int main(int argc, char **argv)
{
    // set device
    hipDeviceProp_t device_prop;
    int dev_id = findCudaDevice(argc, (const char **) argv);
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));

    if (!device_prop.managedMemory) {
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");
        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");
        exit(EXIT_WAIVED);
    }

    if (device_prop.major < 6)
    {
        printf("%s: requires a minimum CUDA compute 6.0 capability, waiving testing.\n", argv[0]);
        exit(EXIT_WAIVED);
    }

    unsigned int numThreads = 256;
    unsigned int numBlocks = 64;
    unsigned int numData = 10;

    int *atom_arr;

    if (device_prop.pageableMemoryAccess)
    {
        printf("CAN access pageable memory\n");
        atom_arr = (int *) malloc(sizeof(int)*numData);
    }
    else
    {
        printf("CANNOT access pageable memory\n");
        checkCudaErrors(hipMallocManaged(&atom_arr, sizeof(int)*numData));
    }

    for (unsigned int i = 0; i < numData; i++)
        atom_arr[i] = 0;

    //To make the AND and XOR tests generate something other than 0...
    atom_arr[7] = atom_arr[9] = 0xff;

    atomicKernel<<<numBlocks, numThreads>>>(atom_arr);
    atomicKernel_CPU(atom_arr, numBlocks*numThreads);

    checkCudaErrors(hipDeviceSynchronize());

    // Compute & verify reference solution
    int testResult = verify(atom_arr, 2*numThreads*numBlocks);

    if (device_prop.pageableMemoryAccess)
    {
        free(atom_arr);
    }
    else
    {
        hipFree(atom_arr);
    }

    printf("systemWideAtomics completed, returned %s \n", testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
