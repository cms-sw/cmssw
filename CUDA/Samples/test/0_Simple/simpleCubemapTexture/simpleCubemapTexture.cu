#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* This sample demonstrates how to use texture fetches from layered 2D textures in CUDA C
*
* This sample first generates a 3D input data array for the layered texture
* and the expected output. Then it starts CUDA C kernels, one for each layer,
* which fetch their layer's texture data (using normalized texture coordinates)
* transform it to the expected output, and write it to a 3D output data array.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

static const char *sSDKname = "simpleCubemapTexture";

// includes, kernels
// declare texture reference for layered 2D float texture
// Note: The "dim" field in the texture reference template is now deprecated.
// Instead, please use a texture type macro such as hipTextureType1D, etc.

texture<float, hipTextureTypeCubemap> tex;

////////////////////////////////////////////////////////////////////////////////
//! Transform a cubemap face of a linear buffe using cubemap texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
transformKernel(float *g_odata, int width)
{
    // calculate this thread's data point
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // 0.5f offset and division are necessary to access the original data points
    // in the texture (such that bilinear interpolation will not be activated).
    // For details, see also CUDA Programming Guide, Appendix D

    float u = ((x+0.5f) / (float) width) * 2.f - 1.f;
    float v = ((y+0.5f) / (float) width) * 2.f - 1.f;

    float cx, cy, cz;

    for (unsigned int face = 0; face < 6; face ++)
    {
        //Layer 0 is positive X face
        if (face == 0)
        {
            cx = 1;
            cy = -v;
            cz = -u;
        }
        //Layer 1 is negative X face
        else if (face == 1)
        {
            cx = -1;
            cy = -v;
            cz = u;
        }
        //Layer 2 is positive Y face
        else if (face == 2)
        {
            cx = u;
            cy = 1;
            cz = v;
        }
        //Layer 3 is negative Y face
        else if (face == 3)
        {
            cx = u;
            cy = -1;
            cz = -v;
        }
        //Layer 4 is positive Z face
        else if (face == 4)
        {
            cx = u;
            cy = -v;
            cz = 1;
        }
        //Layer 4 is negative Z face
        else if (face == 5)
        {
            cx = -u;
            cy = -v;
            cz = -1;
        }

        // read from texture, do expected transformation and write to global memory
        g_odata[face*width*width + y*width + x] = -texCubemap(tex, cx, cy, cz);
    }
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    bool bResult = true;

    // get number of SMs on this GPU
    hipDeviceProp_t deviceProps;

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors ", deviceProps.name, deviceProps.multiProcessorCount);
    printf("SM %d.%d\n", deviceProps.major, deviceProps.minor);

    if (deviceProps.major < 2)
    {
        printf("%s requires SM 2.0 or higher for support of Texture Arrays.  Test will exit... \n", sSDKname);

        exit(EXIT_WAIVED);
    }

    // generate input data for layered texture
    unsigned int width=64, num_faces = 6, num_layers = 1;
    unsigned int cubemap_size = width * width * num_faces;
    unsigned int size = cubemap_size * num_layers * sizeof(float);
    float *h_data = (float *) malloc(size);

    for (int i = 0; i < (int)(cubemap_size * num_layers); i++)
    {
        h_data[i] = (float)i;
    }



    // this is the expected transformation of the input data (the expected output)
    float *h_data_ref = (float *) malloc(size);

    for (unsigned int layer = 0; layer < num_layers; layer++)
    {
        for (int i = 0; i < (int)(cubemap_size); i++)
        {
            h_data_ref[layer*cubemap_size + i] = -h_data[layer*cubemap_size + i] + layer;
        }
    }

    // allocate device memory for result
    float *d_data = NULL;
    checkCudaErrors(hipMalloc((void **) &d_data, size));

    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cu_3darray;
    //    checkCudaErrors(hipMalloc3DArray( &cu_3darray, &channelDesc, make_hipExtent(width, height, num_layers), hipArrayLayered ));
    checkCudaErrors(hipMalloc3DArray(&cu_3darray, &channelDesc, make_hipExtent(width, width, num_faces), hipArrayCubemap));
    hipMemcpy3DParms myparms = {0};
    myparms.srcPos = make_hipPos(0,0,0);
    myparms.dstPos = make_hipPos(0,0,0);
    myparms.srcPtr = make_hipPitchedPtr(h_data, width * sizeof(float), width, width);
    myparms.dstArray = cu_3darray;
    myparms.extent = make_hipExtent(width, width, num_faces);
    myparms.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&myparms));

    // set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;  // access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, cu_3darray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, width / dimBlock.y, 1);

    printf("Covering Cubemap data array of %d~3 x %d: Grid size is %d x %d, each block has 8 x 8 threads\n",
           width, num_layers, dimGrid.x, dimGrid.y);

    transformKernel<<< dimGrid, dimBlock >>>(d_data, width);  // warmup (for better timing)

    // check if kernel execution generated an error
    getLastCudaError("warmup Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // execute the kernel
    transformKernel<<< dimGrid, dimBlock, 0 >>>(d_data, width);

    // check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %.3f msec\n", sdkGetTimerValue(&timer));
    printf("%.2f Mtexlookups/sec\n", (cubemap_size / (sdkGetTimerValue(&timer) / 1000.0f) / 1e6));
    sdkDeleteTimer(&timer);

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_data, size, hipMemcpyDeviceToHost));

    // write regression file if necessary
    if (checkCmdLineFlag(argc, (const char **)argv, "regression"))
    {
        // write file for regression test
        sdkWriteFile<float>("./data/regression.dat", h_odata, width*width, 0.0f, false);
    }
    else
    {
        printf("Comparing kernel output to expected data\n");

#define MIN_EPSILON_ERROR 5e-3f
        bResult = compareData(h_odata, h_data_ref, cubemap_size, MIN_EPSILON_ERROR, 0.0f);
    }

    // cleanup memory
    free(h_data);
    free(h_data_ref);
    free(h_odata);

    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFreeArray(cu_3darray));

    exit(bResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
