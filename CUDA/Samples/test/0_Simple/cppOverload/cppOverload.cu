#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#define THREAD_N 256
#define N 1024
#define DIV_UP(a, b) (((a) + (b) - 1) / (b))

// Includes, system
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <hip/hip_vector_types.h>
#include "cppOverload_kernel.cuh"

const char *sampleName = "C++ Function Overloading";

#define OUTPUT_ATTR(attr)  \
    printf("Shared Size:   %d\n", (int)attr.sharedSizeBytes);   \
    printf("Constant Size: %d\n", (int)attr.constSizeBytes);                 \
    printf("Local Size:    %d\n", (int)attr.localSizeBytes);                 \
    printf("Max Threads Per Block: %d\n", attr.maxThreadsPerBlock);          \
    printf("Number of Registers: %d\n", attr.numRegs);                       \
    printf("PTX Version: %d\n", attr.ptxVersion);                            \
    printf("Binary Version: %d\n", attr.binaryVersion);                      \
     

bool check_func1(int *hInput, int *hOutput, int a)
{
    for (int i = 0; i < N; ++i)
    {
        int cpuRes = hInput[i]*a + i;

        if (hOutput[i] != cpuRes)
        {
            return false;
        }
    }

    return true;
}

bool check_func2(int2 *hInput, int *hOutput, int a)
{
    for (int i = 0; i < N; i++)
    {
        int cpuRes = (hInput[i].x + hInput[i].y)*a + i;

        if (hOutput[i] != cpuRes)
        {
            return false;
        }
    }

    return true;
}

bool check_func3(int *hInput1, int *hInput2, int *hOutput, int a)
{
    for (int i = 0; i < N; i++)
    {
        if (hOutput[i] != (hInput1[i] + hInput2[i])*a + i)
        {
            return false;
        }
    }

    return true;
}

int main(int argc, const char *argv[])
{
    int *hInput  = NULL;
    int *hOutput = NULL;
    int *dInput  = NULL;
    int *dOutput = NULL;

    printf("%s starting...\n", sampleName);

    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));
    printf("DevicecheckCudaErrors Count: %d\n", deviceCount);

    int deviceID = findCudaDevice(argc, argv);
	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, deviceID));
	if (prop.major < 2)    
    {
        printf("ERROR: cppOverload requires GPU devices with compute SM 2.0 or higher.\n");
        printf("Current GPU device has compute SM%d.%d, Exiting...", prop.major, prop.minor);
        exit(EXIT_WAIVED);
    }
	
    checkCudaErrors(hipSetDevice(deviceID));

    // Allocate device memory
    checkCudaErrors(hipMalloc(&dInput , sizeof(int)*N*2));
    checkCudaErrors(hipMalloc(&dOutput, sizeof(int)*N));

    // Allocate host memory
    checkCudaErrors(hipHostMalloc(&hInput , sizeof(int)*N*2));
    checkCudaErrors(hipHostMalloc(&hOutput, sizeof(int)*N));

    for (int i = 0; i < N*2; i++)
    {
        hInput[i] = i;
    }

    // Copy data from host to device
    checkCudaErrors(hipMemcpy(dInput, hInput, sizeof(int)*N*2, hipMemcpyHostToDevice));

    // Test C++ overloading
    bool testResult = true;
    bool funcResult = true;
    int a = 1;

    void (*func1)(const int *, int *, int);
    void (*func2)(const int2 *, int *, int);
    void (*func3)(const int *, const int *, int *, int);
    struct hipFuncAttributes attr;

    // overload function 1
    func1 = simple_kernel;
    memset(&attr, 0, sizeof(attr));
    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*func1), hipFuncCachePreferShared));
    checkCudaErrors(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(*func1)));
    OUTPUT_ATTR(attr);
    (*func1)<<<DIV_UP(N, THREAD_N), THREAD_N>>>(dInput, dOutput, a);
    checkCudaErrors(hipMemcpy(hOutput, dOutput, sizeof(int)*N, hipMemcpyDeviceToHost));
    funcResult = check_func1(hInput, hOutput, a);
    printf("simple_kernel(const int *pIn, int *pOut, int a) %s\n\n", funcResult ? "PASSED" : "FAILED");
    testResult &= funcResult;

    // overload function 2
    func2 = simple_kernel;
    memset(&attr, 0, sizeof(attr));
    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*func2), hipFuncCachePreferShared));
    checkCudaErrors(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(*func2)));
    OUTPUT_ATTR(attr);
    (*func2)<<<DIV_UP(N, THREAD_N), THREAD_N>>>((int2 *)dInput, dOutput, a);
    checkCudaErrors(hipMemcpy(hOutput, dOutput, sizeof(int)*N, hipMemcpyDeviceToHost));
    funcResult = check_func2(reinterpret_cast<int2 *>(hInput), hOutput, a);
    printf("simple_kernel(const int2 *pIn, int *pOut, int a) %s\n\n", funcResult ? "PASSED" : "FAILED");
    testResult &= funcResult;

    // overload function 3
    func3 = simple_kernel;
    memset(&attr, 0, sizeof(attr));
    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*func3), hipFuncCachePreferShared));
    checkCudaErrors(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(*func3)));
    OUTPUT_ATTR(attr);
    (*func3)<<<DIV_UP(N, THREAD_N), THREAD_N>>>(dInput, dInput+N, dOutput, a);
    checkCudaErrors(hipMemcpy(hOutput, dOutput, sizeof(int)*N, hipMemcpyDeviceToHost));
    funcResult = check_func3(&hInput[0], &hInput[N], hOutput, a);
    printf("simple_kernel(const int *pIn1, const int *pIn2, int *pOut, int a) %s\n\n", funcResult ? "PASSED" : "FAILED");
    testResult &= funcResult;

    checkCudaErrors(hipFree(dInput));
    checkCudaErrors(hipFree(dOutput));
    checkCudaErrors(hipHostFree(hOutput));
    checkCudaErrors(hipHostFree(hInput));

    checkCudaErrors(hipDeviceSynchronize());

    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
