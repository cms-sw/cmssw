#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

////////////////////////////////////////////////////////////////////////////////
// Variable on the GPU used to generate unique identifiers of blocks.
////////////////////////////////////////////////////////////////////////////////
__device__ int g_uids = 0;

////////////////////////////////////////////////////////////////////////////////
// Print a simple message to signal the block which is currently executing.
////////////////////////////////////////////////////////////////////////////////
__device__ void print_info(int depth, int thread, int uid, int parent_uid)
{
    if (threadIdx.x == 0)
    {
        if (depth == 0)
            printf("BLOCK %d launched by the host\n", uid);
        else
        {
            char buffer[32];

            for (int i = 0 ; i < depth ; ++i)
            {
                buffer[3*i+0] = '|';
                buffer[3*i+1] = ' ';
                buffer[3*i+2] = ' ';
            }

            buffer[3*depth] = '\0';
            printf("%sBLOCK %d launched by thread %d of block %d\n", buffer, uid, thread, parent_uid);
        }
    }

    __syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
// The kernel using CUDA dynamic parallelism.
//
// It generates a unique identifier for each block. Prints the information
// about that block. Finally, if the 'max_depth' has not been reached, the
// block launches new blocks directly from the GPU.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_kernel(int max_depth, int depth, int thread, int parent_uid)
{
    // We create a unique ID per block. Thread 0 does that and shares the value with the other threads.
    __shared__ int s_uid;

    if (threadIdx.x == 0)
    {
        s_uid = atomicAdd(&g_uids, 1);
    }

    __syncthreads();

    // We print the ID of the block and information about its parent.
    print_info(depth, thread, s_uid, parent_uid);

    // We launch new blocks if we haven't reached the max_depth yet.
    if (++depth >= max_depth)
    {
        return;
    }

    cdp_kernel<<<gridDim.x, blockDim.x>>>(max_depth, depth, threadIdx.x, s_uid);
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("starting Simple Print (CUDA Dynamic Parallelism)\n");

    // Parse a few command-line arguments.
    int max_depth = 2;

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "h"))
    {
        printf("Usage: %s depth=<max_depth>\t(where max_depth is a value between 1 and 8).\n", argv[0]);
        exit(EXIT_SUCCESS);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "depth"))
    {
        max_depth = getCmdLineArgumentInt(argc, (const char **)argv, "depth");

        if (max_depth < 1 || max_depth > 8)
        {
            printf("depth parameter has to be between 1 and 8\n");
            exit(EXIT_FAILURE);
        }
    }

    // Find/set the device.
    int device_count = 0, device = -1;
    
    if(checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        device = getCmdLineArgumentInt(argc, (const char **)argv, "device");

        hipDeviceProp_t properties;
        checkCudaErrors(hipGetDeviceProperties(&properties, device));
        
        if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5))
        {
            std::cout << "Running on GPU " << device << " (" << properties.name << ")" << std::endl;
        }
        else
        {
            std::cout << "ERROR: cdpsimplePrint requires GPU devices with compute SM 3.5 or higher."<< std::endl;
            std::cout << "Current GPU device has compute SM" << properties.major <<"."<< properties.minor <<". Exiting..." << std::endl;
            exit(EXIT_FAILURE);
        }

    }
    else
    {
        checkCudaErrors(hipGetDeviceCount(&device_count));
        for (int i = 0 ; i < device_count ; ++i)
        {
            hipDeviceProp_t properties;
            checkCudaErrors(hipGetDeviceProperties(&properties, i));
            if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5))
            {
                device = i;
                std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
                break;
            }
            std::cout << "GPU " << i << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;
        }
    }
    if (device == -1)
    {
              std::cerr << "cdpSimplePrint requires GPU devices with compute SM 3.5 or higher.  Exiting..." << std::endl;
              exit(EXIT_WAIVED);
     }
    hipSetDevice(device);

    // Print a message describing what the sample does.
    printf("***************************************************************************\n");
    printf("The CPU launches 2 blocks of 2 threads each. On the device each thread will\n");
    printf("launch 2 blocks of 2 threads each. The GPU we will do that recursively\n");
    printf("until it reaches max_depth=%d\n\n", max_depth);
    printf("In total 2");
    int num_blocks = 2, sum = 2;

    for (int i = 1 ; i < max_depth ; ++i)
    {
        num_blocks *= 4;
        printf("+%d", num_blocks);
        sum += num_blocks;
    }

    printf("=%d blocks are launched!!! (%d from the GPU)\n", sum, sum-2);
    printf("***************************************************************************\n\n");

    // We set the recursion limit for CDP to max_depth.
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, max_depth);

    // Launch the kernel from the CPU.
    printf("Launching cdp_kernel() with CUDA Dynamic Parallelism:\n\n");
    cdp_kernel<<<2, 2>>>(max_depth, 0, 0, -1);
    checkCudaErrors(hipGetLastError());

    // Finalize.
    checkCudaErrors(hipDeviceSynchronize());

    exit(EXIT_SUCCESS);
}
