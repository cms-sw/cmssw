#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// System includes.
#include <stdio.h>
#include <iostream>

// STL.
#include <vector>

// CUDA runtime.
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA.
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Device library includes.
#include "simpleDeviceLibrary.cuh"

using std::cout;
using std::endl;

using std::vector;

#define EPS 1e-5

typedef unsigned int uint;
typedef float(*deviceFunc)(float);

const char *sampleName = "simpleSeparateCompilation";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Static device pointers to __device__ functions.
__device__ deviceFunc dMultiplyByTwoPtr = multiplyByTwo;
__device__ deviceFunc dDivideByTwoPtr = divideByTwo;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Transforms vector.
//! Applies the __device__ function "f" to each element of the vector "v".
////////////////////////////////////////////////////////////////////////////////
__global__ void transformVector(float *v, deviceFunc f, uint size)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        v[tid] = (*f)(v[tid]);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, const char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    cout << sampleName << " starting..." << endl;

    runTest(argc, (const char **)argv);

    cout << sampleName << " completed, returned "
         << (testResult ? "OK" : "ERROR") << endl;

    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}


void runTest(int argc, const char **argv)
{
    try
    {
        int devID;

        //hipError_t error;

        // This will pick the best possible CUDA capable device.
        devID = findCudaDevice(argc, (const char **) argv);

        // Create host vector.
        const uint kVectorSize = 1000;

        vector<float> hVector(kVectorSize);

        for (uint i = 0; i < kVectorSize; ++i)
        {
            hVector[i] = rand() / static_cast<float>(RAND_MAX);
        }

        // Create and populate device vector.
        float *dVector;
        checkCudaErrors(hipMalloc(&dVector, kVectorSize * sizeof(float)));

        checkCudaErrors(hipMemcpy(dVector,
                                   &hVector[0],
                                   kVectorSize * sizeof(float),
                                   hipMemcpyHostToDevice));

        // Kernel configuration, where a one-dimensional
        // grid and one-dimensional blocks are configured.
        const int nThreads = 1024;
        const int nBlocks = 1;

        dim3 dimGrid(nBlocks);
        dim3 dimBlock(nThreads);

        // Test library functions.
        deviceFunc hFunctionPtr;

        hipMemcpyFromSymbol(&hFunctionPtr,
                             HIP_SYMBOL(dMultiplyByTwoPtr),
                             sizeof(deviceFunc));
        transformVector<<<dimGrid, dimBlock>>>
        (dVector, hFunctionPtr, kVectorSize);
        checkCudaErrors(hipGetLastError());

        hipMemcpyFromSymbol(&hFunctionPtr,
                             HIP_SYMBOL(dDivideByTwoPtr),
                             sizeof(deviceFunc));
        transformVector<<<dimGrid, dimBlock>>>
        (dVector, hFunctionPtr, kVectorSize);
        checkCudaErrors(hipGetLastError());

        // Download results.
        vector<float> hResultVector(kVectorSize);

        checkCudaErrors(hipMemcpy(&hResultVector[0],
                                   dVector,
                                   kVectorSize *sizeof(float),
                                   hipMemcpyDeviceToHost));

        // Check results.
        for (int i = 0; i < kVectorSize; ++i)
        {
            if (fabs(hVector[i] - hResultVector[i]) > EPS)
            {
                cout << "Computations were incorrect..." << endl;
                testResult = false;
                break;
            }
        }

        // Free resources.
        if (dVector) checkCudaErrors(hipFree(dVector));
    }
    catch (...)
    {
        cout << "Error occured, exiting..." << endl;

        exit(EXIT_FAILURE);
    }
}

