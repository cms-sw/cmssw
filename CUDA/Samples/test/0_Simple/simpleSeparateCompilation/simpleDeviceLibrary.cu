
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

__device__ float multiplyByTwo(float number)
{
    return number * 2.0f;
}

__device__ float divideByTwo(float number)
{
    return number * 0.5f;
}
