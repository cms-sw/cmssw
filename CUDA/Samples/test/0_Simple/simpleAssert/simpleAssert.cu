#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#else
#  include <sys/utsname.h>
#endif

// Includes, system
#include <stdio.h>
#include <cassert>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

const char *sampleName = "simpleAssert";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Tests assert function.
//! Thread whose id > N will print assertion failed error message.
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(int N)
{
    int gtid = blockIdx.x*blockDim.x + threadIdx.x ;
    assert(gtid < N) ;
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n", sampleName);

    runTest(argc, argv);

    printf("%s completed, returned %s\n",
           sampleName,
           testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void runTest(int argc, char **argv)
{
    int devID;
    int Nblocks = 2;
    int Nthreads = 32;
    hipError_t error ;

#ifndef _WIN32
    utsname OS_System_Type;
    uname(&OS_System_Type);

    printf("OS_System_Type.release = %s\n", OS_System_Type.release);

    if (!strcasecmp(OS_System_Type.sysname, "Darwin"))
    {
        printf("simpleAssert is not current supported on Mac OSX\n\n");
        exit(EXIT_SUCCESS);
    }
    else
    {
        printf("OS Info: <%s>\n\n", OS_System_Type.version);
    }

#endif

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    dim3 dimGrid(Nblocks);
    dim3 dimBlock(Nthreads);

    printf("Launch kernel to generate assertion failures\n");
    testKernel<<<dimGrid, dimBlock>>>(60);

    //Synchronize (flushes assert output).
    printf("\n-- Begin assert output\n\n");
    error = hipDeviceSynchronize();
    printf("\n-- End assert output\n\n");

    //Check for errors and failed asserts in asynchronous kernel launch.
    if (error == hipErrorAssert)
    {
        printf("Device assert failed as expected, "
               "CUDA error message is: %s\n\n",
               hipGetErrorString(error));
    }


    testResult = error == hipErrorAssert;
}
