#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This example shows how to use the clock function to measure the performance of
// block of threads of a kernel accurately.
//
// Blocks are executed in parallel and out of order. Since there's no synchronization
// mechanism between blocks, we measure the clock once for each block. The clock
// samples are written to device memory.

// System includes
#include <stdio.h>
#include <stdint.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// This kernel computes a standard parallel reduction and evaluates the
// time it takes to do that for each block. The timing results are stored
// in device memory.
__global__ static void timedReduction(const float *input, float *output, clock_t *timer)
{
    // __shared__ float shared[2 * blockDim.x];
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    if (tid == 0) timer[bid] = clock();

    // Copy input.
    shared[tid] = input[tid];
    shared[tid + blockDim.x] = input[tid + blockDim.x];

    // Perform reduction to find minimum.
    for (int d = blockDim.x; d > 0; d /= 2)
    {
        __syncthreads();

        if (tid < d)
        {
            float f0 = shared[tid];
            float f1 = shared[tid + d];

            if (f1 < f0)
            {
                shared[tid] = f1;
            }
        }
    }

    // Write result.
    if (tid == 0) output[bid] = shared[0];

    __syncthreads();

    if (tid == 0) timer[bid+gridDim.x] = clock();
}

#define NUM_BLOCKS    64
#define NUM_THREADS   256

// It's interesting to change the number of blocks and the number of threads to
// understand how to keep the hardware busy.
//
// Here are some numbers I get on my G80:
//    blocks - clocks
//    1 - 3096
//    8 - 3232
//    16 - 3364
//    32 - 4615
//    64 - 9981
//
// With less than 16 blocks some of the multiprocessors of the device are idle. With
// more than 16 you are using all the multiprocessors, but there's only one block per
// multiprocessor and that doesn't allow you to hide the latency of the memory. With
// more than 32 the speed scales linearly.

// Start the main CUDA Sample here
int main(int argc, char **argv)
{
    printf("CUDA Clock sample\n");

    // This will pick the best possible CUDA capable device
    int dev = findCudaDevice(argc, (const char **)argv);

    float *dinput = NULL;
    float *doutput = NULL;
    clock_t *dtimer = NULL;

    clock_t timer[NUM_BLOCKS * 2];
    float input[NUM_THREADS * 2];

    for (int i = 0; i < NUM_THREADS * 2; i++)
    {
        input[i] = (float)i;
    }

    checkCudaErrors(hipMalloc((void **)&dinput, sizeof(float) * NUM_THREADS * 2));
    checkCudaErrors(hipMalloc((void **)&doutput, sizeof(float) * NUM_BLOCKS));
    checkCudaErrors(hipMalloc((void **)&dtimer, sizeof(clock_t) * NUM_BLOCKS * 2));

    checkCudaErrors(hipMemcpy(dinput, input, sizeof(float) * NUM_THREADS * 2, hipMemcpyHostToDevice));

    timedReduction<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 *NUM_THREADS>>>(dinput, doutput, dtimer);

    checkCudaErrors(hipMemcpy(timer, dtimer, sizeof(clock_t) * NUM_BLOCKS * 2, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(dinput));
    checkCudaErrors(hipFree(doutput));
    checkCudaErrors(hipFree(dtimer));

    long double avgElapsedClocks = 0;

    for (int i = 0; i < NUM_BLOCKS; i++)
    {
        avgElapsedClocks += (long double) (timer[i + NUM_BLOCKS] - timer[i]);
    }

    avgElapsedClocks = avgElapsedClocks/NUM_BLOCKS;
    printf("Average clocks/block = %Lf\n", avgElapsedClocks);

    return EXIT_SUCCESS;
}
