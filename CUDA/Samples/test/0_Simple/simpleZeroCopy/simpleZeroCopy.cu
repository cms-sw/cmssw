#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif


/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

// Allocate generic memory with malloc() and pin it laster instead of using hipHostAlloc()
bool bPinGenericMemory = false;

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

int main(int argc, char **argv)
{
    int n, nelem, deviceCount;
    int idev = 0; // use default device 0
    char *device = NULL;
    unsigned int flags;
    size_t bytes;
    float *a, *b, *c;                      // Pinned memory allocated on the CPU
    float *a_UA, *b_UA, *c_UA;             // Non-4K Aligned Pinned memory on the CPU
    float *d_a, *d_b, *d_c;                // Device pointers for mapped memory
    float errorNorm, refNorm, ref, diff;
    hipDeviceProp_t deviceProp;

    if (checkCmdLineFlag(argc, (const char **)argv, "help"))
    {
        printf("Usage:  simpleZeroCopy [OPTION]\n\n");
        printf("Options:\n");
        printf("  --device=[device #]  Specify the device to be used\n");
        printf("  --use_generic_memory (optional) use generic page-aligned for system memory\n");
        return EXIT_SUCCESS;
    }

    /* Get the device selected by the user or default to 0, and then set it. */
    if (getCmdLineArgumentString(argc, (const char **)argv, "device", &device))
    {
        hipGetDeviceCount(&deviceCount);
        idev = atoi(device);

        if (idev >= deviceCount || idev < 0)
        {
            fprintf(stderr, "Device number %d is invalid, will use default CUDA device 0.\n", idev);
            idev = 0;
        }
    }

    // if GPU found supports SM 1.2, then continue, otherwise we exit 
    if (!checkCudaCapabilities(1, 2))
    {
        exit(EXIT_SUCCESS);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "use_generic_memory"))
    {
#if defined(__APPLE__) || defined(MACOSX)
        bPinGenericMemory = false;  // Generic Pinning of System Paged memory is not currently supported on Mac OSX
#else
        bPinGenericMemory = true;
#endif
    }

    if (bPinGenericMemory)
    {
        printf("> Using Generic System Paged Memory (malloc)\n");
    }
    else
    {
        printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
    }

    checkCudaErrors(hipSetDevice(idev));

    /* Verify the selected device supports mapped memory and set the device
       flags for mapping host memory. */

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020

    if (!deviceProp.canMapHostMemory)
    {
        fprintf(stderr, "Device %d does not support mapping CPU host memory!\n", idev);

        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#else
    fprintf(stderr, "CUDART version %d.%d does not support <hipDeviceProp_t.canMapHostMemory> field\n", , CUDART_VERSION/1000, (CUDART_VERSION%100)/10);

    exit(EXIT_SUCCESS);
#endif

#if CUDART_VERSION < 4000

    if (bPinGenericMemory)
    {
        fprintf(stderr, "CUDART version %d.%d does not support <hipHostRegister> function\n", CUDART_VERSION/1000, (CUDART_VERSION%100)/10);

        exit(EXIT_SUCCESS);
    }

#endif

    /* Allocate mapped CPU memory. */

    nelem = 1048576;
    bytes = nelem*sizeof(float);

    if (bPinGenericMemory)
    {
#if CUDART_VERSION >= 4000
        a_UA = (float *) malloc(bytes + MEMORY_ALIGNMENT);
        b_UA = (float *) malloc(bytes + MEMORY_ALIGNMENT);
        c_UA = (float *) malloc(bytes + MEMORY_ALIGNMENT);

        // We need to ensure memory is aligned to 4K (so we will need to padd memory accordingly)
        a = (float *) ALIGN_UP(a_UA, MEMORY_ALIGNMENT);
        b = (float *) ALIGN_UP(b_UA, MEMORY_ALIGNMENT);
        c = (float *) ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

        checkCudaErrors(hipHostRegister(a, bytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(b, bytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(c, bytes, hipHostMallocMapped));
#endif
    }
    else
    {
#if CUDART_VERSION >= 2020
        flags = hipHostMallocMapped;
        checkCudaErrors(hipHostAlloc((void **)&a, bytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&b, bytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&c, bytes, flags));
#endif
    }

    /* Initialize the vectors. */

    for (n = 0; n < nelem; n++)
    {
        a[n] = rand() / (float)RAND_MAX;
        b[n] = rand() / (float)RAND_MAX;
    }

    /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */

#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_a, (void *)a, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_b, (void *)b, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
#endif

    /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */
    printf("> vectorAddGPU kernel will add vectors using mapped CPU memory...\n");
    dim3 block(256);
    dim3 grid((unsigned int)ceil(nelem/(float)block.x));
    vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);
    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("vectorAddGPU() execution failed");

    /* Compare the results */

    printf("> Checking the results from vectorAddGPU() ...\n");
    errorNorm = 0.f;
    refNorm = 0.f;

    for (n = 0; n < nelem; n++)
    {
        ref = a[n] + b[n];
        diff = c[n] - ref;
        errorNorm += diff*diff;
        refNorm += ref*ref;
    }

    errorNorm = (float)sqrt((double)errorNorm);
    refNorm = (float)sqrt((double)refNorm);

    /* Memory clean up */

    printf("> Releasing CPU memory...\n");

    if (bPinGenericMemory)
    {
#if CUDART_VERSION >= 4000
        checkCudaErrors(hipHostUnregister(a));
        checkCudaErrors(hipHostUnregister(b));
        checkCudaErrors(hipHostUnregister(c));
        free(a_UA);
        free(b_UA);
        free(c_UA);
#endif
    }
    else
    {
#if CUDART_VERSION >= 2020
        checkCudaErrors(hipHostFree(a));
        checkCudaErrors(hipHostFree(b));
        checkCudaErrors(hipHostFree(c));
#endif
    }

    exit(errorNorm/refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE);
}
