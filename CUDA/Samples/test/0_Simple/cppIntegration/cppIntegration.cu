#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void computeGold(char *reference, char *idata, const unsigned int len);
extern "C" void computeGold2(int2 *reference, int2 *idata, const unsigned int len);

///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel(int *g_data)
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid] = ((((data <<  0) >> 24) - 10) << 24)
                  | ((((data <<  8) >> 24) - 10) << 16)
                  | ((((data << 16) >> 24) - 10) <<  8)
                  | ((((data << 24) >> 24) - 10) <<  0);
}

///////////////////////////////////////////////////////////////////////////////
//! Demonstration that int2 data can be used in the cpp code
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernel2(int2 *g_data)
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int2 data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid].x = data.x - data.y;
}

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" bool
runTest(const int argc, const char **argv, char *data, int2 *data_int2, unsigned int len)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    const unsigned int num_threads = len / 4;
    assert(0 == (len % 4));
    const unsigned int mem_size = sizeof(char) * len;
    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    char *d_data;
    checkCudaErrors(hipMalloc((void **) &d_data, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data, data, mem_size,
                               hipMemcpyHostToDevice));
    // allocate device memory for int2 version
    int2 *d_data_int2;
    checkCudaErrors(hipMalloc((void **) &d_data_int2, mem_size_int2));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
                               hipMemcpyHostToDevice));

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
    // execute the kernel
    kernel<<< grid, threads >>>((int *) d_data);
    kernel2<<< grid, threads2 >>>(d_data_int2);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // compute reference solutions
    char *reference = (char *) malloc(mem_size);
    computeGold(reference, data, len);
    int2 *reference2 = (int2 *) malloc(mem_size_int2);
    computeGold2(reference2, data_int2, len);

    // copy results from device to host
    checkCudaErrors(hipMemcpy(data, d_data, mem_size,
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
                               hipMemcpyDeviceToHost));

    // check result
    bool success = true;

    for (unsigned int i = 0; i < len; i++)
    {
        if (reference[i] != data[i] ||
            reference2[i].x != data_int2[i].x ||
            reference2[i].y != data_int2[i].y)
        {
            success = false;
        }
    }

    // cleanup memory
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_data_int2));
    free(reference);
    free(reference2);

    return success;
}
