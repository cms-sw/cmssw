#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


/* Simple example demonstrating how to use MPI with CUDA
*
*  Generate some random numbers on one node.
*  Dispatch them to all nodes.
*  Compute their square root on each node's GPU.
*  Compute the average of the results using MPI.
*
*  simpleMPI.cu: GPU part, compiled with nvcc
*/

#include <iostream>
using std::cerr;
using std::endl;

#include "simpleMPI.h"

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
        my_abort(err); }


// Device code
// Very simple GPU Kernel that computes square roots of input numbers
__global__ void simpleMPIKernel(float *input, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = sqrt(input[tid]);
}


// Initialize an array with random data (between 0 and 1)
void initData(float *data, int dataSize)
{
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = (float)rand() / RAND_MAX;
    }
}

// CUDA computation on each node
// No MPI here, only CUDA
void computeGPU(float *hostData, int blockSize, int gridSize)
{
    int dataSize = blockSize * gridSize;

    // Allocate data on GPU memory
    float *deviceInputData = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceInputData, dataSize * sizeof(float)));

    float *deviceOutputData = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float)));

    // Copy to GPU memory
    CUDA_CHECK(hipMemcpy(deviceInputData, hostData, dataSize * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

    // Copy data back to CPU memory
    CUDA_CHECK(hipMemcpy(hostData, deviceOutputData, dataSize *sizeof(float), hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CHECK(hipFree(deviceInputData));
    CUDA_CHECK(hipFree(deviceOutputData));
}

float sum(float *data, int size)
{
    float accum = 0.f;

    for (int i = 0; i < size; i++)
    {
        accum += data[i];
    }

    return accum;
}
