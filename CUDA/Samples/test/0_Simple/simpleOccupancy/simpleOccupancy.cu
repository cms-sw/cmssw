#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <iostream>
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

const int manualBlockSize = 32;

////////////////////////////////////////////////////////////////////////////////
// Test kernel
//
// This kernel squares each array element. Each thread addresses
// himself with threadIdx and blockIdx, so that it can handle any
// execution configuration, including anything the launch configurator
// API suggests.
////////////////////////////////////////////////////////////////////////////////
__global__ void square(int *array, int arrayCount)
{
    extern __shared__ int dynamicSmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < arrayCount) {
        array[idx] *= array[idx];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Potential occupancy calculator
//
// The potential occupancy is calculated according to the kernel and
// execution configuration the user desires. Occupancy is defined in
// terms of active blocks per multiprocessor, and the user can convert
// it to other metrics.
//
// This wrapper routine computes the occupancy of kernel, and reports
// it in terms of active warps / maximum warps per SM.
////////////////////////////////////////////////////////////////////////////////
static double reportPotentialOccupancy(void *kernel, int blockSize, size_t dynamicSMem)
{
    int device;
    hipDeviceProp_t prop;

    int numBlocks;
    int activeWarps;
    int maxWarps;

    double occupancy;

    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipGetDeviceProperties(&prop, device));

    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
                        &numBlocks,
                        kernel,
                        blockSize,
                        dynamicSMem));

    activeWarps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    occupancy = (double)activeWarps / maxWarps;

    return occupancy;
}

////////////////////////////////////////////////////////////////////////////////
// Occupancy-based launch configurator
//
// The launch configurator, hipOccupancyMaxPotentialBlockSize and
// hipOccupancyMaxPotentialBlockSizeVariableSMem, suggests a block
// size that achieves the best theoretical occupancy. It also returns
// the minimum number of blocks needed to achieve the occupancy on the
// whole device.
//
// This launch configurator is purely occupancy-based. It doesn't
// translate directly to performance, but the suggestion should
// nevertheless be a good starting point for further optimizations.
//
// This function configures the launch based on the "automatic"
// argument, records the runtime, and reports occupancy and runtime.
////////////////////////////////////////////////////////////////////////////////
static int launchConfig(int *array, int arrayCount, bool automatic)
{
    int blockSize;
    int minGridSize;
    int gridSize;
    size_t dynamicSMemUsage = 0;

    hipEvent_t start;
    hipEvent_t end;

    float elapsedTime;
    
    double potentialOccupancy;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    if (automatic) {
        checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                            &minGridSize,
                            &blockSize,
                            (void*)square,
                            dynamicSMemUsage,
                            arrayCount));

        std::cout << "Suggested block size: " << blockSize << std::endl
                  << "Minimum grid size for maximum occupancy: " << minGridSize << std::endl;
    } else {
        // This block size is too small. Given limited number of
        // active blocks per multiprocessor, the number of active
        // threads will be limited, and thus unable to achieve maximum
        // occupancy.
        //
        blockSize = manualBlockSize;
    }

    // Round up
    //
    gridSize = (arrayCount + blockSize - 1) / blockSize;

    // Launch and profile
    //
    checkCudaErrors(hipEventRecord(start));
    square<<<gridSize, blockSize, dynamicSMemUsage>>>(array, arrayCount);
    checkCudaErrors(hipEventRecord(end));

    checkCudaErrors(hipDeviceSynchronize());

    // Calculate occupancy
    //
    potentialOccupancy = reportPotentialOccupancy((void*)square, blockSize, dynamicSMemUsage);

    std::cout << "Potential occupancy: " << potentialOccupancy * 100 << "%" << std::endl;

    // Report elapsed time
    //
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
    std::cout << "Elapsed time: " << elapsedTime << "ms" << std::endl;
    
    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// The test
//
// The test generates an array and squares it with a CUDA kernel, then
// verifies the result.
////////////////////////////////////////////////////////////////////////////////
static int test(bool automaticLaunchConfig, const int count = 1000000)
{
    int *array;
    int *dArray;
    int size = count * sizeof(int);

    array = new int[count];

    for (int i = 0; i < count; i += 1) {
        array[i] = i;
    }

    checkCudaErrors(hipMalloc(&dArray, size));
    checkCudaErrors(hipMemcpy(dArray, array, size, hipMemcpyHostToDevice));

    for (int i = 0; i < count; i += 1) {
        array[i] = 0;
    }

    launchConfig(dArray, count, automaticLaunchConfig);

    checkCudaErrors(hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(dArray));

    // Verify the return data
    //
    for (int i = 0; i < count; i += 1) {
        if (array[i] != i * i) {
            std::cout << "element " << i << " expected " << i * i << " actual " << array[i] << std::endl;
            return 1;
        }
    }
    delete[] array;

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Sample Main
//
// The sample runs the test with manually configured launch and
// automatically configured launch, and reports the occupancy and
// performance.
////////////////////////////////////////////////////////////////////////////////
int main()
{
    int status;

    std::cout << "starting Simple Occupancy" << std::endl << std::endl;

    std::cout << "[ Manual configuration with " << manualBlockSize
              << " threads per block ]" << std::endl;

    status = test(false);
    if (status) {
        std::cerr << "Test failed\n" << std::endl;
        return -1;
    }

    std::cout << std::endl;

    std::cout << "[ Automatic, occupancy-based configuration ]" << std::endl;
    status = test(true);
    if (status) {
        std::cerr << "Test failed\n" << std::endl;
        return -1;
    }        

    std::cout << std::endl;
    std::cout << "Test PASSED\n" << std::endl;
    
    return 0;
}
