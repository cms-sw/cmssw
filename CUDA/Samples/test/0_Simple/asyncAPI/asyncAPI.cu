#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

//
// This sample illustrates the usage of CUDA events for both GPU timing and
// overlapping CPU and GPU execution.  Events are inserted into a stream
// of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
// perform computations while GPU is executing (including DMA memcopies
// between the host and device).  CPU can query CUDA events to determine
// whether GPU has completed tasks.
//

// includes, system
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions 

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + inc_value;
}

bool correct_output(int *data, const int n, const int x)
{
    for (int i = 0; i < n; i++)
        if (data[i] != x)
        {
            printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
            return false;
        }

    return true;
}

int main(int argc, char *argv[])
{
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

    // allocate host memory
    int *a = 0;
    checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
    checkCudaErrors(hipMemset(d_a, 255, nbytes));

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
    increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;

    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
    printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    bool bFinalResults = correct_output(a, n, value);

    // release resources
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(a));
    checkCudaErrors(hipFree(d_a));

    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}
