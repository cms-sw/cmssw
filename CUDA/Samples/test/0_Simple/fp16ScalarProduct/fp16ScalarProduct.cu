#include "hip/hip_runtime.h"
/**
 * Copyright 2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "hip/hip_fp16.h"
#include "hip/hip_runtime_api.h"

#include <cstdio>
#include <cstdlib>
#include <ctime>

__forceinline__ __device__ void reduceInShared(half2 * const v)
{
    if(threadIdx.x<64)
        v[threadIdx.x] = __hadd2( v[threadIdx.x], v[threadIdx.x+64]);
    __syncthreads();
    if(threadIdx.x<32)
        v[threadIdx.x] = __hadd2( v[threadIdx.x], v[threadIdx.x+32]);
    __syncthreads();
    if(threadIdx.x<32)
        v[threadIdx.x] = __hadd2( v[threadIdx.x], v[threadIdx.x+16]);
    __syncthreads();
    if(threadIdx.x<32)
        v[threadIdx.x] = __hadd2( v[threadIdx.x], v[threadIdx.x+8]);
    __syncthreads();
    if(threadIdx.x<32)
        v[threadIdx.x] = __hadd2( v[threadIdx.x], v[threadIdx.x+4]);
    __syncthreads();
    if(threadIdx.x<32)
        v[threadIdx.x] = __hadd2( v[threadIdx.x], v[threadIdx.x+2]);
    __syncthreads();
    if(threadIdx.x<32)
        v[threadIdx.x] = __hadd2( v[threadIdx.x], v[threadIdx.x+1]);
    __syncthreads();
}

__global__ void scalarProductKernel(
        half2 const * const a,
        half2 const * const b,
        float * const results,
        size_t const size
        )
{
    const int stride = gridDim.x*blockDim.x;
    __shared__ half2 shArray[128];

    shArray[threadIdx.x] = __float2half2_rn(0.f);
    half2 value = __float2half2_rn(0.f);

    for (int i = threadIdx.x + blockDim.x + blockIdx.x; i < size; i+=stride)
    {
        value = __hfma2(a[i], b[i], value);
    }

    shArray[threadIdx.x] = value;
    __syncthreads();
    reduceInShared(shArray);

    if (threadIdx.x == 0)
    {
        half2 result = shArray[0];
        float f_result = __low2float(result) + __high2float(result);
        results[blockIdx.x] = f_result;
    }
}

void generateInput(half2 * a, size_t size)
{
    for (size_t i = 0; i < size; ++i)
    {
        unsigned temp = rand();
        temp &= 0x83FF83FF;
        temp |= 0x3C003C00;
        a[i] = *(half2*)&temp;
    }
}

int main(int argc, char *argv[])
{
    srand(time(NULL));
    const int blocks = 128;
    const int threads = 128;
    size_t size = blocks*threads*16;

    half2 * vec[2];
    half2 * devVec[2];

    float * results;
    float * devResults;

    int devID = findCudaDevice(argc, (const char **)argv);

    hipDeviceProp_t devProp;
    checkCudaErrors(hipGetDeviceProperties(&devProp, devID));

    if (devProp.major < 5 || (devProp.major == 5 && devProp.minor < 3))
    {
        printf("ERROR: fp16ScalarProduct requires GPU devices with compute SM 5.3 or higher.\n");
        return EXIT_WAIVED;
    }

    for (int i = 0; i < 2; ++i)
    {
        checkCudaErrors(hipHostMalloc((void**)&vec[i], size*sizeof*vec[i]));
        checkCudaErrors(hipMalloc((void**)&devVec[i], size*sizeof*devVec[i]));
    }

    checkCudaErrors(hipHostMalloc((void**)&results, blocks*sizeof*results));
    checkCudaErrors(hipMalloc((void**)&devResults, blocks*sizeof*devResults));

    for (int i = 0; i < 2; ++i)
    {
        generateInput(vec[i], size);
        checkCudaErrors(hipMemcpy(devVec[i], vec[i], size*sizeof*vec[i], hipMemcpyHostToDevice));
    }

    scalarProductKernel<<<blocks, threads>>>(devVec[0], devVec[1], devResults, size);

    checkCudaErrors(hipMemcpy(results, devResults, blocks*sizeof*results, hipMemcpyDeviceToHost));

    float result = 0;
    for (int i = 0; i < blocks; ++i)
    {
        result += results[i];
    }
    printf("Result: %f \n", result);

    for (int i = 0; i < 2; ++i)
    {
        checkCudaErrors(hipFree(devVec[i]));
        checkCudaErrors(hipHostFree(vec[i]));
    }

    checkCudaErrors(hipFree(devResults));
    checkCudaErrors(hipHostFree(results));

    return EXIT_SUCCESS;
}
