#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Multi-GPU sample using OpenMP for threading on the CPU side
 * needs a compiler that supports OpenMP 2.0
 */

#include <omp.h>
#include <stdio.h>  // stdio functions are used since C++ streams aren't necessarily thread safe
#include <hip/hip_runtime_api.h>

using namespace std;

// a simple kernel that simply increments each array element by b
__global__ void kernelAddConstant(int *g_a, const int b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_a[idx] += b;
}

// a predicate that checks whether each array element is set to its index plus b
int correctResult(int *data, const int n, const int b)
{
    for (int i = 0; i < n; i++)
        if (data[i] != i + b)
            return 0;

    return 1;
}

int main(int argc, char *argv[])
{
    int num_gpus = 0;   // number of CUDA GPUs

    printf("%s Starting...\n\n", argv[0]);

    /////////////////////////////////////////////////////////////////
    // determine the number of CUDA capable GPUs
    //
    hipGetDeviceCount(&num_gpus);

    if (num_gpus < 1)
    {
        printf("no CUDA capable devices were detected\n");
        return 1;
    }

    /////////////////////////////////////////////////////////////////
    // display CPU and GPU configuration
    //
    printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("number of CUDA devices:\t%d\n", num_gpus);

    for (int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }

    printf("---------------------------\n");


    /////////////////////////////////////////////////////////////////
    // initialize data
    //
    unsigned int n = num_gpus * 8192;
    unsigned int nbytes = n * sizeof(int);
    int *a = 0;     // pointer to data on the CPU
    int b = 3;      // value by which the array is incremented
    a = (int *)malloc(nbytes);

    if (0 == a)
    {
        printf("couldn't allocate CPU memory\n");
        return 1;
    }

    for (unsigned int i = 0; i < n; i++)
        a[i] = i;


    ////////////////////////////////////////////////////////////////
    // run as many CPU threads as there are CUDA devices
    //   each CPU thread controls a different device, processing its
    //   portion of the data.  It's possible to use more CPU threads
    //   than there are CUDA devices, in which case several CPU
    //   threads will be allocating resources and launching kernels
    //   on the same device.  For example, try omp_set_num_threads(2*num_gpus);
    //   Recall that all variables declared inside an "omp parallel" scope are
    //   local to each CPU thread
    //
    omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
    //omp_set_num_threads(2*num_gpus);// create twice as many CPU threads as there are CUDA devices
    #pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();

        // set and check the CUDA device for this CPU thread
        int gpu_id = -1;
        checkCudaErrors(hipSetDevice(cpu_thread_id % num_gpus));   // "% num_gpus" allows more CPU threads than GPU devices
        checkCudaErrors(hipGetDevice(&gpu_id));
        printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);

        int *d_a = 0;   // pointer to memory on the device associated with this CPU thread
        int *sub_a = a + cpu_thread_id * n / num_cpu_threads;   // pointer to this CPU thread's portion of data
        unsigned int nbytes_per_kernel = nbytes / num_cpu_threads;
        dim3 gpu_threads(128);  // 128 threads per block
        dim3 gpu_blocks(n / (gpu_threads.x * num_cpu_threads));

        checkCudaErrors(hipMalloc((void **)&d_a, nbytes_per_kernel));
        checkCudaErrors(hipMemset(d_a, 0, nbytes_per_kernel));
        checkCudaErrors(hipMemcpy(d_a, sub_a, nbytes_per_kernel, hipMemcpyHostToDevice));
        kernelAddConstant<<<gpu_blocks, gpu_threads>>>(d_a, b);

        checkCudaErrors(hipMemcpy(sub_a, d_a, nbytes_per_kernel, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_a));

    }
    printf("---------------------------\n");

    if (hipSuccess != hipGetLastError())
        printf("%s\n", hipGetErrorString(hipGetLastError()));


    ////////////////////////////////////////////////////////////////
    // check the result
    //
    bool bResult = correctResult(a, n, b);

    if (a)
        free(a); // free CPU memory

    exit(bResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
