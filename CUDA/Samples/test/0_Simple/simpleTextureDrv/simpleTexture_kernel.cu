#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _SIMPLETEXTURE_KERNEL_H_
#define _SIMPLETEXTURE_KERNEL_H_

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void
transformKernel(float *g_odata, int width, int height, float theta)
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = (float)x - (float)width/2; 
    float v = (float)y - (float)height/2; 
    float tu = u*cosf(theta) - v*sinf(theta); 
    float tv = v*cosf(theta) + u*sinf(theta); 

    tu /= (float)width; 
    tv /= (float)height; 

    // read from texture and write to global memory
    g_odata[y*width + x] = tex2D(tex, tu+0.5f, tv+0.5f);
}

#endif // #ifndef _SIMPLETEXTURE_KERNEL_H_
