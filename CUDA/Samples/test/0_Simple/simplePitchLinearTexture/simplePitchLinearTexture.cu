#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* pitchLinearTexture
*
* This example demonstrates how to use textures bound to pitch linear memory.
* It performs a shift of matrix elements using wrap addressing mode (aka
* periodic boundary conditions) on two arrays, a pitch linear and a CUDA array,
* in order to highlight the differences in using each.
*
* Textures binding to pitch linear memory is a new feature in CUDA 2.2,
* and allows use of texture features such as wrap addressing mode and
* filtering which are not possible with textures bound to regular linear memory
*/

// includes, system
#include <stdio.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define NUM_REPS 100  // number of repetitions performed
#define TILE_DIM 16   // tile/block size

const char *sSDKsample = "simplePitchLinearTexture";

////////////////////////////////////////////////////////////////////////////////
// Texture references
texture<float, 2, hipReadModeElementType> texRefPL;
texture<float, 2, hipReadModeElementType> texRefArray;

// Auto-Verification Code
bool bTestResult = true;

////////////////////////////////////////////////////////////////////////////////
// NB: (1) The second argument "pitch" is in elements, not bytes
//     (2) normalized coordinates are used (required for wrap address mode)
////////////////////////////////////////////////////////////////////////////////
//! Shifts matrix elements using pitch linear array
//! @param odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void shiftPitchLinear(float *odata,
                                 int pitch,
                                 int width,
                                 int height,
                                 int shiftX,
                                 int shiftY)
{
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    odata[yid * pitch + xid] = tex2D(texRefPL,
                                     (xid + shiftX) / (float) width,
                                     (yid + shiftY) / (float) height);
}

////////////////////////////////////////////////////////////////////////////////
//! Shifts matrix elements using regular array
//! @param odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void shiftArray(float *odata,
                           int pitch,
                           int width,
                           int height,
                           int shiftX,
                           int shiftY)
{
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    odata[yid * pitch + xid] = tex2D(texRefArray,
                                     (xid + shiftX) / (float) width,
                                     (yid + shiftY) / (float) height);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n\n", sSDKsample);

    runTest(argc, argv);

    printf("%s completed, returned %s\n",
           sSDKsample,
           bTestResult ? "OK" : "ERROR!");
    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
    // Set array size
    const int nx = 2048;
    const int ny = 2048;

    // Setup shifts applied to x and y data
    const int x_shift = 5;
    const int y_shift = 7;

    if ((nx % TILE_DIM != 0)  || (ny % TILE_DIM != 0))
    {
        printf("nx and ny must be multiples of TILE_DIM\n");
        exit(EXIT_FAILURE);
    }

    // Setup execution configuration parameters
    dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM), dimBlock(TILE_DIM, TILE_DIM);

    // This will pick the best possible CUDA capable device
    int devID = findCudaDevice(argc, (const char **)argv);

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host allocation and initialization
    float *h_idata = (float *) malloc(sizeof(float) * nx * ny);
    float *h_odata = (float *) malloc(sizeof(float) * nx * ny);
    float *gold = (float *) malloc(sizeof(float) * nx * ny);

    for (int i = 0; i < nx * ny; ++i)
    {
        h_idata[i] = (float) i;
    }

    // Device memory allocation
    // Pitch linear input data
    float *d_idataPL;
    size_t d_pitchBytes;

    checkCudaErrors(hipMallocPitch((void **) &d_idataPL,
                                    &d_pitchBytes,
                                    nx * sizeof(float),
                                    ny));

    // Array input data
    hipArray *d_idataArray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    checkCudaErrors(hipMallocArray(&d_idataArray, &channelDesc, nx, ny));

    // Pitch linear output data
    float *d_odata;
    checkCudaErrors(hipMallocPitch((void **) &d_odata,
                                    &d_pitchBytes,
                                    nx * sizeof(float),
                                    ny));

    // Copy host data to device
    // Pitch linear
    size_t h_pitchBytes = nx * sizeof(float);

    checkCudaErrors(hipMemcpy2D(d_idataPL,
                                 d_pitchBytes,
                                 h_idata,
                                 h_pitchBytes,
                                 nx * sizeof(float),
                                 ny,
                                 hipMemcpyHostToDevice));

    // Array
    checkCudaErrors(hipMemcpyToArray(d_idataArray,
                                      0,
                                      0,
                                      h_idata,
                                      nx * ny * sizeof(float),
                                      hipMemcpyHostToDevice));

    // Bind texture to memory
    // Pitch linear
    texRefPL.normalized = 1;
    texRefPL.filterMode = hipFilterModePoint;
    texRefPL.addressMode[0] = hipAddressModeWrap;
    texRefPL.addressMode[1] = hipAddressModeWrap;

    checkCudaErrors(hipBindTexture2D(0,
                                      &texRefPL,
                                      d_idataPL,
                                      &channelDesc,
                                      nx,
                                      ny,
                                      d_pitchBytes));

    // Array
    texRefArray.normalized = 1;
    texRefArray.filterMode = hipFilterModePoint;
    texRefArray.addressMode[0] = hipAddressModeWrap;
    texRefArray.addressMode[1] = hipAddressModeWrap;

    checkCudaErrors(hipBindTextureToArray(texRefArray,
                                           d_idataArray,
                                           channelDesc));

    // Reference calculation
    for (int j = 0; j < ny; ++j)
    {
        int jshift = (j + y_shift) % ny;

        for (int i = 0; i < nx; ++i)
        {
            int ishift = (i + x_shift) % nx;
            gold[j * nx + i] = h_idata[jshift * nx + ishift];
        }
    }

    // Run ShiftPitchLinear kernel
    checkCudaErrors(hipMemset2D(d_odata,
                                 d_pitchBytes,
                                 0,
                                 nx * sizeof(float),
                                 ny));

    checkCudaErrors(hipEventRecord(start, 0));

    for (int i = 0; i < NUM_REPS; ++i)
    {
        shiftPitchLinear<<<dimGrid, dimBlock>>>
        (d_odata,
         (int)(d_pitchBytes / sizeof(float)),
         nx,
         ny,
         x_shift,
         y_shift);
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float timePL;
    checkCudaErrors(hipEventElapsedTime(&timePL, start, stop));

    // Check results
    checkCudaErrors(hipMemcpy2D(h_odata,
                                 h_pitchBytes,
                                 d_odata,
                                 d_pitchBytes,
                                 nx * sizeof(float),
                                 ny,
                                 hipMemcpyDeviceToHost));

    bool res = compareData(gold, h_odata, nx*ny, 0.0f, 0.15f);

    bTestResult = true;

    if (res == false)
    {
        printf("*** shiftPitchLinear failed ***\n");
        bTestResult = false;
    }

    // Run ShiftArray kernel
    checkCudaErrors(hipMemset2D(d_odata,
                                 d_pitchBytes,
                                 0,
                                 nx * sizeof(float),
                                 ny));
    checkCudaErrors(hipEventRecord(start, 0));

    for (int i = 0; i < NUM_REPS; ++i)
    {
        shiftArray<<<dimGrid, dimBlock>>>
        (d_odata,
         (int)(d_pitchBytes / sizeof(float)),
         nx,
         ny,
         x_shift,
         y_shift);
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float timeArray;
    checkCudaErrors(hipEventElapsedTime(&timeArray, start, stop));

    // Check results
    checkCudaErrors(hipMemcpy2D(h_odata,
                                 h_pitchBytes,
                                 d_odata,
                                 d_pitchBytes,
                                 nx * sizeof(float),
                                 ny,
                                 hipMemcpyDeviceToHost));
    res = compareData(gold, h_odata, nx*ny, 0.0f, 0.15f);

    if (res == false)
    {
        printf("*** shiftArray failed ***\n");
        bTestResult = false;
    }

    float bandwidthPL =
        2.f * 1000.f * nx * ny * sizeof(float) /
        (1.e+9f) / (timePL / NUM_REPS);
    float bandwidthArray =
        2.f * 1000.f * nx * ny * sizeof(float) /
        (1.e+9f) / (timeArray / NUM_REPS);

    printf("\nBandwidth (GB/s) for pitch linear: %.2e; for array: %.2e\n",
           bandwidthPL, bandwidthArray);

    float fetchRatePL =
        nx * ny / 1.e+6f / (timePL / (1000.0f * NUM_REPS));
    float fetchRateArray =
        nx * ny / 1.e+6f / (timeArray / (1000.0f * NUM_REPS));

    printf("\nTexture fetch rate (Mpix/s) for pitch linear: "
           "%.2e; for array: %.2e\n\n",
           fetchRatePL, fetchRateArray);

    // Cleanup
    free(h_idata);
    free(h_odata);
    free(gold);

    checkCudaErrors(hipUnbindTexture(texRefPL));
    checkCudaErrors(hipUnbindTexture(texRefArray));
    checkCudaErrors(hipFree(d_idataPL));
    checkCudaErrors(hipFreeArray(d_idataArray));
    checkCudaErrors(hipFree(d_odata));

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
}
