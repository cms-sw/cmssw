/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* A simple program demonstrating trivial use of global memory atomic
 * device functions (atomic*() functions).
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

// Includes, kernels
#include "simpleAtomicIntrinsics_kernel.cuh"

const char *sampleName = "simpleAtomicIntrinsics";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

extern "C" bool computeGold(int *gpuData, const int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n", sampleName);

    runTest(argc, argv);

    printf("%s completed, returned %s\n",
           sampleName,
           testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;
    int dev = 0;

    // This will pick the best possible CUDA capable device
    dev = findCudaDevice(argc, (const char **)argv);

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, "
           "SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    unsigned int numThreads = 256;
    unsigned int numBlocks = 64;
    unsigned int numData = 11;
    unsigned int memSize = sizeof(int) * numData;

    //allocate mem for the result on host side
    int *hOData = (int *) malloc(memSize);

    //initialize the memory
    for (unsigned int i = 0; i < numData; i++)
        hOData[i] = 0;

    //To make the AND and XOR tests generate something other than 0...
    hOData[8] = hOData[10] = 0xff;

    // allocate device memory for result
    int *dOData;
    checkCudaErrors(hipMalloc((void **) &dOData, memSize));
    // copy host memory to device to initialize to zero
    checkCudaErrors(hipMemcpy(dOData,
                               hOData,
                               memSize,
                               hipMemcpyHostToDevice));

    // execute the kernel
    testKernel<<<numBlocks, numThreads>>>(dOData);
    getLastCudaError("Kernel execution failed");

    //Copy result from device to host
    checkCudaErrors(hipMemcpy(hOData,
                               dOData,
                               memSize,
                               hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // Compute reference solution
    testResult = computeGold(hOData, numThreads * numBlocks);

    // Cleanup memory
    free(hOData);
    checkCudaErrors(hipFree(dOData));
}
