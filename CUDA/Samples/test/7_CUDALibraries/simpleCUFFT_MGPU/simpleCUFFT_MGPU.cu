#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// System includes
#include <stdlib.h>
#include <stdio.h>

#include <string.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

//CUFFT Header file
#include <hipfft/hipfftXt.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Complex data type
typedef float2 Complex;

static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMulAndScale( hipfftComplex *, hipfftComplex *, int, float);

//Kernel for GPU
void multiplyCoefficient ( hipLibXtDesc *, hipLibXtDesc *, int, float, int);

// Filtering functions
void Convolve(const Complex *, int, const Complex *, int, Complex *);

// Padding functions
int PadData(const Complex *, Complex **, int,
            const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
// Data configuration
// The filter size is assumed to be a number smaller than the signal size
///////////////////////////////////////////////////////////////////////////////
const int SIGNAL_SIZE        = 1018;
const int FILTER_KERNEL_SIZE = 11;
const int GPU_COUNT          =  2;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("\n[simpleCUFFT_MGPU] is starting...\n\n");

    int GPU_N;
    checkCudaErrors(hipGetDeviceCount(&GPU_N));

    if (GPU_N < GPU_COUNT)
    {
        printf("No. of GPU on node %d\n", GPU_N);
        printf("Two GPUs are required to run simpleCUFFT_MGPU sample code\n");
        exit(EXIT_WAIVED);
    }

    // Allocate host memory for the signal
    Complex *h_signal = (Complex *)malloc(sizeof(Complex) * SIGNAL_SIZE);

    // Initialize the memory for the signal
    for (int i = 0; i < SIGNAL_SIZE; ++i)
    {
        h_signal[i].x = rand() / (float)RAND_MAX;
        h_signal[i].y = 0;
    }

    // Allocate host memory for the filter
    Complex *h_filter_kernel = (Complex *)malloc(sizeof(Complex) * FILTER_KERNEL_SIZE);

    // Initialize the memory for the filter
    for (int i = 0; i < FILTER_KERNEL_SIZE; ++i)
    {
        h_filter_kernel[i].x = rand() / (float)RAND_MAX;
        h_filter_kernel[i].y = 0;
    }

    // Pad signal and filter kernel
    Complex *h_padded_signal;
    Complex *h_padded_filter_kernel;
    int new_size = PadData(h_signal, &h_padded_signal, SIGNAL_SIZE,
                           h_filter_kernel, &h_padded_filter_kernel,
                           FILTER_KERNEL_SIZE);

    // hipfftCreate() - Create an empty plan
    hipfftResult result;
    hipfftHandle plan_input;
    checkCudaErrors (hipfftCreate (&plan_input));

    // hipfftXtSetGPUs() - Define which GPUs to use
    int nGPUs = 2;
    int *whichGPUs ;
    whichGPUs = (int*) malloc(sizeof(int) * nGPUs);

    // Iterate all device combinations to see if a supported combo exists
    for (int i = 0; i < GPU_N; i++)
    {
        for (int j = i+1; j < GPU_N; j++)
        {
            whichGPUs[0] = i;
            whichGPUs[1] = j;
            result = hipfftXtSetGPUs (plan_input, nGPUs, whichGPUs);

            if (result == HIPFFT_INVALID_DEVICE) { continue; }
            else if (result == HIPFFT_SUCCESS) { break; }
            else { printf ("hipfftXtSetGPUs failed\n"); exit(EXIT_FAILURE); }
        }

        if (result == HIPFFT_SUCCESS) { break; }
    }

    if (result == HIPFFT_INVALID_DEVICE)
    {
        printf ("This sample requires two GPUs on the same board.\n");
        printf ("No such board was found. Waiving sample.\n");
        exit (EXIT_WAIVED);
    }

    //Print the device information to run the code
    for (int i = 0 ; i < nGPUs ; i++)
    {
        hipDeviceProp_t deviceProp;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, whichGPUs[i]));
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", whichGPUs[i], deviceProp.name, deviceProp.major, deviceProp.minor);

    }

    size_t* worksize;
    worksize =(size_t*)malloc(sizeof(size_t) * nGPUs);

    // hipfftMakePlan1d() - Create the plan
    result = hipfftMakePlan1d(plan_input, new_size, HIPFFT_C2C, 1, worksize);
    if (result != HIPFFT_SUCCESS) { printf ("*MakePlan* failed\n"); exit (EXIT_FAILURE); }

    // hipfftXtMalloc() - Malloc data on multiple GPUs
    hipLibXtDesc *d_signal ;
    result = hipfftXtMalloc (plan_input, (hipLibXtDesc **)&d_signal, HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); exit (EXIT_FAILURE); }
    hipLibXtDesc *d_out_signal ;
    result = hipfftXtMalloc (plan_input, (hipLibXtDesc **)&d_out_signal, HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); exit (EXIT_FAILURE); }
    hipLibXtDesc *d_filter_kernel;
    result = hipfftXtMalloc (plan_input, (hipLibXtDesc **)&d_filter_kernel, HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); exit (EXIT_FAILURE); }
    hipLibXtDesc *d_out_filter_kernel;
    result = hipfftXtMalloc (plan_input, (hipLibXtDesc **)&d_out_filter_kernel, HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); exit (EXIT_FAILURE); }

    // hipfftXtMemcpy() - Copy data from host to multiple GPUs
    result = hipfftXtMemcpy (plan_input,d_signal, h_padded_signal, HIPFFT_COPY_HOST_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); exit (EXIT_FAILURE); }
    result = hipfftXtMemcpy (plan_input, d_filter_kernel, h_padded_filter_kernel, HIPFFT_COPY_HOST_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); exit (EXIT_FAILURE) ; }

    // hipfftXtExecDescriptorC2C() - Execute FFT on data on multiple GPUs
    result = hipfftXtExecDescriptorC2C(plan_input, d_signal,  d_signal, HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExecC2C  failed\n"); exit (EXIT_FAILURE) ; }
    result = hipfftXtExecDescriptorC2C(plan_input, d_filter_kernel, d_filter_kernel, HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExecC2C  failed\n"); exit (EXIT_FAILURE) ; }

    // hipfftXtMemcpy() - Copy the data to natural order on GPUs
    result = hipfftXtMemcpy (plan_input, d_out_signal, d_signal, HIPFFT_COPY_DEVICE_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); exit (EXIT_FAILURE) ; }
    result = hipfftXtMemcpy (plan_input, d_out_filter_kernel, d_filter_kernel, HIPFFT_COPY_DEVICE_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); exit (EXIT_FAILURE) ; }

    printf("\n\nValue of Library Descriptor\n");
    printf("Number of GPUs %d\n", d_out_signal->descriptor->nGPUs );
    printf("Device id  %d %d\n", d_out_signal->descriptor->GPUs[0], d_out_signal->descriptor->GPUs[1]);
    printf("Data size on GPU %ld %ld\n", (long)(d_out_signal->descriptor->size[0]/sizeof(hipfftComplex)), (long)(d_out_signal->descriptor->size[1]/sizeof(hipfftComplex))) ;

    //Multiply the coefficients together and normalize the result
    printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
    multiplyCoefficient(d_out_signal,d_out_filter_kernel, new_size, 1.0f / new_size, nGPUs );

    // hipfftXtExecDescriptorC2C() - Execute inverse  FFT on data on multiple GPUs
    printf("Transforming signal back hipfftExecC2C\n");
    result = hipfftXtExecDescriptorC2C(plan_input, d_out_signal,  d_out_signal, HIPFFT_BACKWARD);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExecC2C  failed\n"); exit (EXIT_FAILURE) ; }

    // Create host pointer pointing to padded signal
    Complex *h_convolved_signal = h_padded_signal;

    // Allocate host memory for the convolution result
    Complex *h_convolved_signal_ref = (Complex *)malloc(sizeof(Complex) * SIGNAL_SIZE);

    // hipfftXtMemcpy() - Copy data from multiple GPUs to host
    result = hipfftXtMemcpy (plan_input,h_convolved_signal, d_out_signal, HIPFFT_COPY_DEVICE_TO_HOST);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); exit (EXIT_FAILURE); }

    // Convolve on the host
    Convolve(h_signal, SIGNAL_SIZE, h_filter_kernel,
             FILTER_KERNEL_SIZE, h_convolved_signal_ref);

    // Compare CPU and GPU result
    bool bTestResult = sdkCompareL2fe((float *)h_convolved_signal_ref,
                                      (float *)h_convolved_signal, 2 * SIGNAL_SIZE,
                                      1e-5f);
    printf("\nvalue of TestResult %d\n", bTestResult);

    // Cleanup memory
    free(whichGPUs);
    free(worksize);
    free(h_signal);
    free(h_filter_kernel);
    free(h_padded_signal);
    free(h_padded_filter_kernel);
    free(h_convolved_signal_ref);

    // cudaXtFree() - Free GPU memory
    result = hipfftXtFree(d_signal);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); exit (EXIT_FAILURE); }
    result = hipfftXtFree(d_filter_kernel);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); exit (EXIT_FAILURE) ; }
    result = hipfftXtFree(d_out_signal);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); exit (EXIT_FAILURE) ; }
    result = hipfftXtFree(d_out_filter_kernel);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); exit (EXIT_FAILURE) ; }

    // hipfftDestroy() - Destroy FFT plan
    result = hipfftDestroy(plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("hipfftDestroy failed: code %d\n",(int)result); exit (EXIT_FAILURE); }

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////////
// Function for padding original data
//////////////////////////////////////////////////////////////////////////////////
int PadData(const Complex *signal, Complex **padded_signal, int signal_size,
            const Complex *filter_kernel, Complex **padded_filter_kernel, int filter_kernel_size)
{
    int minRadius = filter_kernel_size / 2;
    int maxRadius = filter_kernel_size - minRadius;
    int new_size = signal_size + maxRadius;

    // Pad signal
    Complex *new_data = (Complex *)malloc(sizeof(Complex) * new_size);
    memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
    memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
    *padded_signal = new_data;

    // Pad filter
    new_data = (Complex *)malloc(sizeof(Complex) * new_size);
    memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
    memset(new_data + maxRadius, 0, (new_size - filter_kernel_size) * sizeof(Complex));
    memcpy(new_data + new_size - minRadius, filter_kernel, minRadius * sizeof(Complex));
    *padded_filter_kernel = new_data;

    return new_size;
}

////////////////////////////////////////////////////////////////////////////////
// Filtering operations - Computing Convolution on the host
////////////////////////////////////////////////////////////////////////////////
void Convolve(const Complex *signal, int signal_size,
              const Complex *filter_kernel, int filter_kernel_size,
              Complex *filtered_signal)
{
    int minRadius = filter_kernel_size / 2;
    int maxRadius = filter_kernel_size - minRadius;

    // Loop over output element indices
    for (int i = 0; i < signal_size; ++i)
    {
        filtered_signal[i].x = filtered_signal[i].y = 0;

        // Loop over convolution indices
        for (int j = - maxRadius + 1; j <= minRadius; ++j)
        {
            int k = i + j;

            if (k >= 0 && k < signal_size)
            {
                filtered_signal[i] = ComplexAdd(filtered_signal[i], ComplexMul(signal[k], filter_kernel[minRadius - j]));
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//  Launch Kernel on multiple GPU
////////////////////////////////////////////////////////////////////////////////
void  multiplyCoefficient( hipLibXtDesc *d_signal,hipLibXtDesc *d_filter_kernel,
                           int new_size, float val , int nGPUs)
{
    int device ;
    //Launch the ComplexPointwiseMulAndScale<<< >>> kernel on multiple GPU
    for(int i=0; i < nGPUs ;i++)
    {
        device = d_signal->descriptor->GPUs[i];

        //Set device
        checkCudaErrors(hipSetDevice(device));

        //Perform GPU computations
        ComplexPointwiseMulAndScale<<<32, 256>>>((hipfftComplex*) d_signal->descriptor->data[i],
                                                 (hipfftComplex*) d_filter_kernel->descriptor->data[i],
                                                  int(d_signal->descriptor->size[i]/sizeof(hipfftComplex)), val);
    }

    // Wait for device to finish all operation
    for( int i=0; i< nGPUs ; i++ )
    {
        device = d_signal->descriptor->GPUs[i];
        checkCudaErrors(hipSetDevice(device));
        hipDeviceSynchronize();
        // Check if kernel execution generated and error
        getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");
    }
}


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
{
    Complex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}
// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, hipfftComplex *b, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
    }
}
