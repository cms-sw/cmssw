#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Routines for testing the device API of CUBLAS.
 */

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

/* Includes, cuda helper functions */
#include <hip/hip_runtime_api.h>

__global__ void invokeDeviceCublasSgemm(hipblasStatus_t *returnValue,
                                        int n,
                                        const float *d_alpha,
                                        const float *d_A,
                                        const float *d_B,
                                        const float *d_beta,
                                        float *d_C)
{
    hipblasHandle_t cnpHandle;
    hipblasStatus_t status = hipblasCreate(&cnpHandle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        *returnValue = status;
        return;
    }

    /* Perform operation using cublas */
    status =
        hipblasSgemm(cnpHandle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, n, n,
                    d_alpha,
                    d_A, n,
                    d_B, n,
                    d_beta,
                    d_C, n);

    hipblasDestroy(cnpHandle);

    *returnValue = status;
}

struct SGEMMScalarParams
{
    float alpha, beta;
};

extern "C" void device_cublas_sgemm(int n,
                                    float alpha,
                                    const float *d_A, const float *d_B,
                                    float beta,
                                    float *d_C)
{
    hipblasStatus_t *d_status;
    hipblasStatus_t status;

    if (hipMalloc((void **) &d_status, sizeof(hipblasStatus_t)) != hipSuccess)
    {
        fprintf(stderr,
                "!!!! device memory allocation error (allocate d_status)\n");
        exit(EXIT_FAILURE);
    }

    // Device API requires scalar arguments (alpha and beta)
    // to be allocated in the device memory.
    SGEMMScalarParams h_params = {alpha, beta};
    SGEMMScalarParams *d_params;

    if (hipMalloc((void **) &d_params, sizeof(SGEMMScalarParams)) !=
        hipSuccess)
    {
        fprintf(stderr,
                "!!!! device memory allocation error (allocate d_params)\n");
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(d_params,
                   &h_params,
                   sizeof(SGEMMScalarParams),
                   hipMemcpyHostToDevice) != hipSuccess)
    {
        fprintf(stderr,
                "!!!! host to device memory copy error\n");
        exit(EXIT_FAILURE);
    }

    // Launch hipblasSgemm wrapper kernel.
    invokeDeviceCublasSgemm<<<1, 1>>>
    (d_status, n, &d_params->alpha, d_A, d_B, &d_params->beta, d_C);

    hipError_t error;

    if ((error = hipGetLastError()) != hipSuccess)
    {
        fprintf(stderr,
                "!!!! kernel execution error: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(&status,
                   d_status,
                   sizeof(hipblasStatus_t),
                   hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr,
                "!!!! device to host memory copy error\n");
        exit(EXIT_FAILURE);
    }

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr,
                "!!!! CUBLAS Device API call failed with code %d\n",
                status);
        exit(EXIT_FAILURE);
    }

    // Free allocated device memory.
    if (hipFree(d_status) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (d_status)\n");
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_params) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (d_params)\n");
        exit(EXIT_FAILURE);
    }
}
