#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <assert.h>
#include "DataFormats/DetId/interface/DetId.h"
#include "DataFormats/HcalDetId/interface/HcalDetId.h"

__global__ void test_gen_detid(DetId* id, uint32_t const rawid) {
  DetId did{rawid};
  *id = did;
}

void test_detid() {
  // test det ids
  DetId h_id, h_id_test{100};
  DetId h_test0{1};
  DetId* d_id;

  hipMalloc((void**)&d_id, sizeof(DetId));
  hipMemcpy(d_id, &h_id, sizeof(DetId), hipMemcpyHostToDevice);
  test_gen_detid<<<1, 1>>>(d_id, 100);
  hipMemcpy(&h_id, d_id, sizeof(DetId), hipMemcpyDeviceToHost);

  assert(h_id_test == h_id);
  assert(h_id != h_test0);
}

int main(int argc, char** argv) {
  int nDevices;
  hipGetDeviceCount(&nDevices);
  std::cout << "nDevices = " << nDevices << std::endl;

  // test det id functionality
  if (nDevices > 0)
    test_detid();
}
