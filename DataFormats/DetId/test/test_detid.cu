#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "DataFormats/DetId/interface/DetId.h"
#include "DataFormats/HcalDetId/interface/HcalDetId.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

__global__ void test_gen_detid(DetId* id, uint32_t const rawid) {
  DetId did{rawid};
  *id = did;
}

void test_detid() {
  // test det ids
  DetId h_id, h_id_test{100};
  DetId h_test0{1};
  DetId* d_id;

  hipMalloc((void**)&d_id, sizeof(DetId));
  hipMemcpy(d_id, &h_id, sizeof(DetId), hipMemcpyHostToDevice);
  test_gen_detid<<<1, 1>>>(d_id, 100);
  hipMemcpy(&h_id, d_id, sizeof(DetId), hipMemcpyDeviceToHost);

  assert(h_id_test == h_id);
  assert(h_id != h_test0);
}

int main(int argc, char** argv) {
  cms::cudatest::requireDevices();

  // test det id functionality
  test_detid();
}
