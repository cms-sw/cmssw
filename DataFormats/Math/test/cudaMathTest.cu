#include "hip/hip_runtime.h"
/**
 * Derived from the nVIDIA CUDA 8.0 samples by
 *
 *   Eyal Rozenberg <E.Rozenberg@cwi.nl>
 *
 * The derivation is specifically permitted in the nVIDIA CUDA Samples EULA
 * and the deriver is the owner of this code according to the EULA.
 *
 * Use this reasonably. If you want to discuss licensing formalities, please
 * contact the author.
 *
 *  Modified by VinInn for testing math funcs
 */

/* to run test
foreach f ( $CMSSW_BASE/test/$SCRAM_ARCH/DFM_Vector* )
echo $f; $f
end
*/

#include <algorithm>
#include <cassert>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <memory>
#include <random>
#include <stdexcept>

#ifdef __HIPCC__
#define inline __host__ __device__ inline
#include <vdt/sin.h>
#undef inline
#else
#include <vdt/sin.h>
#endif

#include "DataFormats/Math/interface/approx_log.h"
#include "DataFormats/Math/interface/approx_exp.h"
#include "DataFormats/Math/interface/approx_atan2.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"
#include "HeterogeneousCore/CUDAUtilities/interface/launch.h"

std::mt19937 eng;
std::mt19937 eng2;
std::uniform_real_distribution<float> rgen(0., 1.);

constexpr float myExp(float x) { return unsafe_expf<6>(x); }

constexpr float myLog(float x) { return unsafe_logf<6>(x); }

__host__ __device__ inline float mySin(float x) { return vdt::fast_sinf(x); }

constexpr int USEEXP = 0, USESIN = 1, USELOG = 2;

template <int USE, bool ADDY = false>
// __host__ __device__
constexpr float testFunc(float x, float y) {
  float ret = 0;
  if (USE == USEEXP)
    ret = myExp(x);
  else if (USE == USESIN)
    ret = mySin(x);
  else
    ret = myLog(x);
  return ADDY ? ret + y : ret;
}

template <int USE, bool ADDY>
__global__ void vectorOp(const float *A, const float *B, float *C, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    C[i] = testFunc<USE, ADDY>(A[i], B[i]);
  }
}

template <int USE, bool ADDY>
void vectorOpH(const float *A, const float *B, float *C, int numElements) {
  for (int i = 0; i < numElements; ++i) {
    C[i] = testFunc<USE, ADDY>(A[i], B[i]);
  }
}

template <int USE, bool ADDY = false>
void go() {
  auto start = std::chrono::high_resolution_clock::now();
  auto delta = start - start;

  int numElements = 200000;
  size_t size = numElements * sizeof(float);
  std::cout << "[Vector of " << numElements << " elements]\n";

  auto h_A = std::make_unique<float[]>(numElements);
  auto h_B = std::make_unique<float[]>(numElements);
  auto h_C = std::make_unique<float[]>(numElements);
  auto h_C2 = std::make_unique<float[]>(numElements);

  std::generate(h_A.get(), h_A.get() + numElements, [&]() { return rgen(eng); });
  std::generate(h_B.get(), h_B.get() + numElements, [&]() { return rgen(eng); });

  delta -= (std::chrono::high_resolution_clock::now() - start);
  auto d_A = cms::cuda::make_device_unique<float[]>(numElements, nullptr);
  auto d_B = cms::cuda::make_device_unique<float[]>(numElements, nullptr);
  auto d_C = cms::cuda::make_device_unique<float[]>(numElements, nullptr);

  cudaCheck(hipMemcpy(d_A.get(), h_A.get(), size, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_B.get(), h_B.get(), size, hipMemcpyHostToDevice));
  delta += (std::chrono::high_resolution_clock::now() - start);
  std::cout << "cuda alloc+copy took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() << " ms"
            << std::endl;

  // Launch the Vector OP CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads\n";

  delta -= (std::chrono::high_resolution_clock::now() - start);
  cms::cuda::launch(
      vectorOp<USE, ADDY>, {blocksPerGrid, threadsPerBlock}, d_A.get(), d_B.get(), d_C.get(), numElements);
  delta += (std::chrono::high_resolution_clock::now() - start);
  std::cout << "cuda computation took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() << " ms"
            << std::endl;

  delta -= (std::chrono::high_resolution_clock::now() - start);
  cms::cuda::launch(
      vectorOp<USE, ADDY>, {blocksPerGrid, threadsPerBlock}, d_A.get(), d_B.get(), d_C.get(), numElements);
  delta += (std::chrono::high_resolution_clock::now() - start);
  std::cout << "cuda computation took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() << " ms"
            << std::endl;

  delta -= (std::chrono::high_resolution_clock::now() - start);
  cudaCheck(hipMemcpy(h_C.get(), d_C.get(), size, hipMemcpyDeviceToHost));
  delta += (std::chrono::high_resolution_clock::now() - start);
  std::cout << "cuda copy back took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() << " ms"
            << std::endl;

  // on host now...
  delta -= (std::chrono::high_resolution_clock::now() - start);
  vectorOpH<USE, ADDY>(h_A.get(), h_B.get(), h_C2.get(), numElements);
  delta += (std::chrono::high_resolution_clock::now() - start);
  std::cout << "host computation took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() << " ms"
            << std::endl;

  delta -= (std::chrono::high_resolution_clock::now() - start);
  vectorOpH<USE, ADDY>(h_A.get(), h_B.get(), h_C2.get(), numElements);
  delta += (std::chrono::high_resolution_clock::now() - start);
  std::cout << "host computation took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() << " ms"
            << std::endl;

  // Verify that the result vector is correct
  double ave = 0;
  int maxDiff = 0;
  long long ndiff = 0;
  double fave = 0;
  float fmaxDiff = 0;
  for (int i = 0; i < numElements; ++i) {
    approx_math::binary32 g, c;
    g.f = testFunc<USE, ADDY>(h_A[i], h_B[i]);
    c.f = h_C[i];
    auto diff = std::abs(g.i32 - c.i32);
    maxDiff = std::max(diff, maxDiff);
    ave += diff;
    if (diff != 0)
      ++ndiff;
    auto fdiff = std::abs(g.f - c.f);
    fave += fdiff;
    fmaxDiff = std::max(fdiff, fmaxDiff);
    //           if (diff>7)
    //           std::cerr << "Large diff at element " << i << ' ' << diff << ' ' << std::hexfloat
    //                                  << g.f << "!=" << c.f << "\n";
  }
  std::cout << "ndiff ave, max " << ndiff << ' ' << ave / numElements << ' ' << maxDiff << std::endl;
  std::cout << "float ave, max " << fave / numElements << ' ' << fmaxDiff << std::endl;
  if (!ndiff) {
    std::cout << "Test PASSED\n";
    std::cout << "SUCCESS" << std::endl;
  }
  hipDeviceSynchronize();
}

int main() {
  cms::cudatest::requireDevices();

  try {
    go<USEEXP>();
    go<USESIN>();
    go<USELOG>();
    go<USELOG, true>();
  } catch (std::runtime_error &ex) {
    std::cerr << "CUDA or std runtime error: " << ex.what() << std::endl;
    exit(EXIT_FAILURE);
  } catch (...) {
    std::cerr << "A non-CUDA error occurred" << std::endl;
    exit(EXIT_FAILURE);
  }

  return EXIT_SUCCESS;
}
