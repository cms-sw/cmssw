#include "hip/hip_runtime.h"
/**
 * Derived from the nVIDIA CUDA 8.0 samples by
 *
 *   Eyal Rozenberg <E.Rozenberg@cwi.nl>
 *
 * The derivation is specifically permitted in the nVIDIA CUDA Samples EULA
 * and the deriver is the owner of this code according to the EULA.
 *
 * Use this reasonably. If you want to discuss licensing formalities, please
 * contact the author.
 *
 *  Modified by VinInn for testing math funcs
 */

/* to run test
foreach f ( $CMSSW_BASE/test/$SCRAM_ARCH/DFM_Vector* )
echo $f; $f
end
*/

#include <algorithm>
#include <cassert>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <memory>
#include <random>
#include <stdexcept>

#include "DataFormats/Math/interface/approx_atan2.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"
#include "HeterogeneousCore/CUDAUtilities/interface/launch.h"

constexpr float xmin = -100.001;  // avoid 0
constexpr float incr = 0.04;
constexpr int Nsteps = 2. * std::abs(xmin) / incr;

template <int DEGREE>
__global__ void diffAtan(int *diffs) {
  auto mdiff = &diffs[0];
  auto idiff = &diffs[1];
  auto sdiff = &diffs[2];

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  auto x = xmin + incr * i;
  auto y = xmin + incr * j;

  auto approx = unsafe_atan2f<DEGREE>(y, x);
  auto iapprox = unsafe_atan2i<DEGREE>(y, x);
  auto sapprox = unsafe_atan2s<DEGREE>(y, x);
  auto std = std::atan2(y, x);
  auto fd = std::abs(std - approx);
  atomicMax(mdiff, int(fd * 1.e7));
  atomicMax(idiff, std::abs(phi2int(std) - iapprox));
  short dd = std::abs(phi2short(std) - sapprox);
  atomicMax(sdiff, int(dd));
}

template <int DEGREE>
void go() {
  auto start = std::chrono::high_resolution_clock::now();
  auto delta = start - start;

  // atan2
  delta -= (std::chrono::high_resolution_clock::now() - start);

  auto diff_d = cms::cuda::make_device_unique<int[]>(3, nullptr);

  int diffs[3];
  cudaCheck(hipMemset(diff_d.get(), 0, 3 * 4));

  // Launch the diff CUDA Kernel
  dim3 threadsPerBlock(32, 32, 1);
  dim3 blocksPerGrid(
      (Nsteps + threadsPerBlock.x - 1) / threadsPerBlock.x, (Nsteps + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
  std::cout << "CUDA kernel 'diff' launch with " << blocksPerGrid.x << " blocks of " << threadsPerBlock.y
            << " threads\n";

  cms::cuda::launch(diffAtan<DEGREE>, {blocksPerGrid, threadsPerBlock}, diff_d.get());

  cudaCheck(hipMemcpy(diffs, diff_d.get(), 3 * 4, hipMemcpyDeviceToHost));
  delta += (std::chrono::high_resolution_clock::now() - start);

  float mdiff = diffs[0] * 1.e-7;
  int idiff = diffs[1];
  int sdiff = diffs[2];

  std::cout << "for degree " << DEGREE << " max diff is " << mdiff << ' ' << idiff << ' ' << int2phi(idiff) << ' '
            << sdiff << ' ' << short2phi(sdiff) << std::endl;
  std::cout << "cuda computation took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() << " ms"
            << std::endl;
}

int main() {
  cms::cudatest::requireDevices();

  try {
    go<3>();
    go<5>();
    go<7>();
    go<9>();
  } catch (std::runtime_error &ex) {
    std::cerr << "CUDA or std runtime error: " << ex.what() << std::endl;
    exit(EXIT_FAILURE);
  } catch (...) {
    std::cerr << "A non-CUDA error occurred" << std::endl;
    exit(EXIT_FAILURE);
  }

  return EXIT_SUCCESS;
}
