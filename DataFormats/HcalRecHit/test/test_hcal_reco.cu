#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <type_traits>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "DataFormats/HcalRecHit/interface/HBHERecHit.h"
#include "DataFormats/HcalRecHit/interface/HFRecHit.h"
#include "DataFormats/HcalRecHit/interface/HORecHit.h"
#include "DataFormats/HcalRecHit/interface/HFQIE10Info.h"
#include "DataFormats/HcalRecHit/interface/HBHEChannelInfo.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

template <typename T>
__global__ void kernel_test_hcal_rechits(T *other) {
  T rh(HcalDetId(0), 10.0f, 10.0f);
  other->setEnergy(rh.energy());
  other->setTime(rh.time());
}

__global__ void kernel_test_hcal_hfqie10info() { HFQIE10Info info; }

__global__ void kernel_test_hcal_hbhechinfo(HBHEChannelInfo *other) {
  HBHEChannelInfo info{true, true};
  info.setChannelInfo(HcalDetId{0}, 10, 10, 10, 1, 2.0, 2.0, 2.0, 0.0, false, false, false);
  other->setChannelInfo(info.id(),
                        info.recoShape(),
                        info.nSamples(),
                        info.soi(),
                        info.capid(),
                        info.darkCurrent(),
                        info.fcByPE(),
                        info.lambda(),
                        info.noisecorr(),
                        info.hasLinkError(),
                        info.hasCapidError(),
                        info.isDropped());
}

void test_hcal_hfqie10info() {
  auto check_error = [](auto code) {
    if (code != hipSuccess) {
      std::cout << hipGetErrorString(code) << std::endl;
      assert(false);
    }
  };

  kernel_test_hcal_hfqie10info<<<1, 1>>>();
  check_error(hipGetLastError());
}

template <typename T>
void test_hcal_rechits() {
  auto check_error = [](auto code) {
    if (code != hipSuccess) {
      std::cout << hipGetErrorString(code) << std::endl;
      assert(false);
    }
  };

  T h_rh, h_rh_test{HcalDetId(0), 10.0f, 10.0f};
  T *d_rh;

  hipMalloc((void **)&d_rh, sizeof(T));
  hipMemcpy(d_rh, &h_rh, sizeof(T), hipMemcpyHostToDevice);
  kernel_test_hcal_rechits<T><<<1, 1>>>(d_rh);
  hipDeviceSynchronize();
  check_error(hipGetLastError());
  hipMemcpy(&h_rh, d_rh, sizeof(T), hipMemcpyDeviceToHost);

  std::cout << h_rh << std::endl;
  std::cout << h_rh_test << std::endl;
  assert(h_rh.energy() == h_rh_test.energy());
  assert(h_rh.time() == h_rh_test.time());

  std::cout << "all good in " << __FUNCTION__ << std::endl;
}

void test_hcal_hbhechinfo() {
  auto check_error = [](auto code) {
    if (code != hipSuccess) {
      std::cout << hipGetErrorString(code) << std::endl;
      assert(false);
    }
  };

  HBHEChannelInfo h_info, h_info_test{true, true};
  h_info_test.setChannelInfo(HcalDetId{0}, 10, 10, 10, 1, 2.0, 2.0, 2.0, 0.0, false, false, false);
  HBHEChannelInfo *d_info;

  hipMalloc((void **)&d_info, sizeof(HBHEChannelInfo));
  hipMemcpy(d_info, &h_info, sizeof(HBHEChannelInfo), hipMemcpyHostToDevice);
  kernel_test_hcal_hbhechinfo<<<1, 1>>>(d_info);
  hipDeviceSynchronize();
  check_error(hipGetLastError());
  hipMemcpy(&h_info, d_info, sizeof(HBHEChannelInfo), hipMemcpyDeviceToHost);

  assert(h_info.id() == h_info_test.id());
  assert(h_info.recoShape() == h_info_test.recoShape());
  assert(h_info.nSamples() == h_info_test.nSamples());
  assert(h_info.soi() == h_info_test.soi());
  assert(h_info.capid() == h_info_test.capid());
  assert(h_info.darkCurrent() == h_info_test.darkCurrent());
  assert(h_info.fcByPE() == h_info_test.fcByPE());
  assert(h_info.lambda() == h_info_test.lambda());
  assert(h_info.noisecorr() == h_info_test.noisecorr());
  assert(h_info.hasLinkError() == h_info_test.hasLinkError());
  assert(h_info.hasCapidError() == h_info_test.hasCapidError());

  std::cout << "all good in " << __FUNCTION__ << std::endl;
}

int main(int argc, char **argv) {
  cms::cudatest::requireDevices();

  test_hcal_rechits<HBHERecHit>();
  test_hcal_rechits<HFRecHit>();
  test_hcal_rechits<HORecHit>();
  test_hcal_hbhechinfo();

  std::cout << "all good" << std::endl;
  return 0;
}
