#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "DataFormats/CaloRecHit/interface/CaloRecHit.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

__global__ void kernel_test_calo_rechit(CaloRecHit* other) {
  CaloRecHit rh{DetId(0), 10, 1, 0, 0};
  other->setEnergy(rh.energy());
  other->setTime(rh.time());
  other->setFlagField(10, 31, 1);
}

void test_calo_rechit() {
  auto check_error = [](auto code) {
    if (code != hipSuccess) {
      std::cout << hipGetErrorString(code) << std::endl;
      assert(false);
    }
  };

  CaloRecHit h_rh, h_rh_test{DetId(0), 10, 1, 0, 0};
  h_rh_test.setFlagField(10, 31, 1);
  CaloRecHit* d_rh;

  hipMalloc((void**)&d_rh, sizeof(CaloRecHit));
  hipMemcpy(d_rh, &h_rh, sizeof(CaloRecHit), hipMemcpyHostToDevice);
  kernel_test_calo_rechit<<<1, 1>>>(d_rh);
  hipDeviceSynchronize();
  check_error(hipGetLastError());
  hipMemcpy(&h_rh, d_rh, sizeof(CaloRecHit), hipMemcpyDeviceToHost);

  std::cout << h_rh << std::endl;
  std::cout << h_rh_test << std::endl;
  assert(h_rh.energy() == h_rh_test.energy());
  assert(h_rh.time() == h_rh_test.time());
  assert(h_rh.flags() == h_rh_test.flags());
  assert(h_rh.aux() == h_rh_test.aux());
  assert(h_rh.detid() == h_rh_test.detid());
}

int main(int argc, char** argv) {
  cms::cudatest::requireDevices();

  test_calo_rechit();

  std::cout << "all good!" << std::endl;
  return 0;
}
