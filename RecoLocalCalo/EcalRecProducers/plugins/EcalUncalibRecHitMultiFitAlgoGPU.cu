#include <iostream>
#include <limits>

#include <hip/hip_runtime.h>

#include "CondFormats/EcalObjects/interface/EcalMGPAGainRatio.h"
#include "CondFormats/EcalObjects/interface/EcalPedestals.h"
#include "CondFormats/EcalObjects/interface/EcalPulseCovariances.h"
#include "CondFormats/EcalObjects/interface/EcalPulseShapes.h"
#include "CondFormats/EcalObjects/interface/EcalSampleMask.h"
#include "CondFormats/EcalObjects/interface/EcalSamplesCorrelation.h"
#include "CondFormats/EcalObjects/interface/EcalXtalGroupId.h"
#include "DataFormats/EcalDigi/interface/EcalDataFrame.h"
#include "DataFormats/EcalDigi/interface/EcalDigiCollections.h"

#include "AmplitudeComputationCommonKernels.h"
#include "AmplitudeComputationKernels.h"
#include "EcalUncalibRecHitMultiFitAlgoGPU.h"
#include "TimeComputationKernels.h"

//#define DEBUG

//#define ECAL_RECO_CUDA_DEBUG

namespace ecal {
  namespace multifit {

    void entryPoint(EventInputDataGPU const& eventInputGPU,
                    EventOutputDataGPU& eventOutputGPU,
                    EventDataForScratchGPU& scratch,
                    ConditionsProducts const& conditions,
                    ConfigurationParameters const& configParameters,
                    hipStream_t cudaStream) {
      using digis_type = std::vector<uint16_t>;
      using dids_type = std::vector<uint32_t>;
      // accodring to the cpu setup  //----> hardcoded
      bool const gainSwitchUseMaxSampleEB = true;
      // accodring to the cpu setup  //----> hardcoded
      bool const gainSwitchUseMaxSampleEE = false;

      uint32_t const offsetForHashes = conditions.offsetForHashes;
      uint32_t const offsetForInputs = eventInputGPU.ebDigis.size;
      unsigned int totalChannels = eventInputGPU.ebDigis.size + eventInputGPU.eeDigis.size;

      //
      // 1d preparation kernel
      //
      unsigned int nchannels_per_block = 32;
      unsigned int threads_1d = 10 * nchannels_per_block;
      unsigned int blocks_1d = threads_1d > 10 * totalChannels ? 1 : (totalChannels * 10 + threads_1d - 1) / threads_1d;
      int shared_bytes = nchannels_per_block * EcalDataFrame::MAXSAMPLES *
                         (sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(char) + sizeof(bool));
      kernel_prep_1d_and_initialize<<<blocks_1d, threads_1d, shared_bytes, cudaStream>>>(
          conditions.pulseShapes.values,
          eventInputGPU.ebDigis.data.get(),
          eventInputGPU.ebDigis.ids.get(),
          eventInputGPU.eeDigis.data.get(),
          eventInputGPU.eeDigis.ids.get(),
          (SampleVector*)scratch.samples.get(),
          (SampleVector*)eventOutputGPU.recHitsEB.amplitudesAll.get(),
          (SampleVector*)eventOutputGPU.recHitsEE.amplitudesAll.get(),
          (SampleGainVector*)scratch.gainsNoise.get(),
          conditions.pedestals.mean_x1,
          conditions.pedestals.mean_x12,
          conditions.pedestals.rms_x12,
          conditions.pedestals.mean_x6,
          conditions.gainRatios.gain6Over1,
          conditions.gainRatios.gain12Over6,
          scratch.hasSwitchToGain6.get(),
          scratch.hasSwitchToGain1.get(),
          scratch.isSaturated.get(),
          eventOutputGPU.recHitsEB.amplitude.get(),
          eventOutputGPU.recHitsEE.amplitude.get(),
          eventOutputGPU.recHitsEB.chi2.get(),
          eventOutputGPU.recHitsEE.chi2.get(),
          eventOutputGPU.recHitsEB.pedestal.get(),
          eventOutputGPU.recHitsEE.pedestal.get(),
          eventOutputGPU.recHitsEB.did.get(),
          eventOutputGPU.recHitsEE.did.get(),
          eventOutputGPU.recHitsEB.flags.get(),
          eventOutputGPU.recHitsEE.flags.get(),
          scratch.acState.get(),
          (BXVectorType*)scratch.activeBXs.get(),
          offsetForHashes,
          offsetForInputs,
          gainSwitchUseMaxSampleEB,
          gainSwitchUseMaxSampleEE,
          totalChannels);
      cudaCheck(hipGetLastError());

      //
      // 2d preparation kernel
      //
      int blocks_2d = totalChannels;
      dim3 threads_2d{10, 10};
      kernel_prep_2d<<<blocks_2d, threads_2d, 0, cudaStream>>>((SampleGainVector*)scratch.gainsNoise.get(),
                                                               eventInputGPU.ebDigis.ids.get(),
                                                               eventInputGPU.eeDigis.ids.get(),
                                                               conditions.pedestals.rms_x12,
                                                               conditions.pedestals.rms_x6,
                                                               conditions.pedestals.rms_x1,
                                                               conditions.gainRatios.gain12Over6,
                                                               conditions.gainRatios.gain6Over1,
                                                               conditions.samplesCorrelation.EBG12SamplesCorrelation,
                                                               conditions.samplesCorrelation.EBG6SamplesCorrelation,
                                                               conditions.samplesCorrelation.EBG1SamplesCorrelation,
                                                               conditions.samplesCorrelation.EEG12SamplesCorrelation,
                                                               conditions.samplesCorrelation.EEG6SamplesCorrelation,
                                                               conditions.samplesCorrelation.EEG1SamplesCorrelation,
                                                               (SampleMatrix*)scratch.noisecov.get(),
                                                               (PulseMatrixType*)scratch.pulse_matrix.get(),
                                                               conditions.pulseShapes.values,
                                                               scratch.hasSwitchToGain6.get(),
                                                               scratch.hasSwitchToGain1.get(),
                                                               scratch.isSaturated.get(),
                                                               offsetForHashes,
                                                               offsetForInputs);
      cudaCheck(hipGetLastError());

      // run minimization kernels
      v1::minimization_procedure(eventInputGPU, eventOutputGPU, scratch, conditions, configParameters, cudaStream);

      if (configParameters.shouldRunTimingComputation) {
        //
        // TODO: this guy can run concurrently with other kernels,
        // there is no dependence on the order of execution
        //
        unsigned int threads_time_init = threads_1d;
        unsigned int blocks_time_init = blocks_1d;
        int sharedBytesInit = 2 * threads_time_init * sizeof(SampleVector::Scalar);
        kernel_time_computation_init<<<blocks_time_init, threads_time_init, sharedBytesInit, cudaStream>>>(
            eventInputGPU.ebDigis.data.get(),
            eventInputGPU.ebDigis.ids.get(),
            eventInputGPU.eeDigis.data.get(),
            eventInputGPU.eeDigis.ids.get(),
            conditions.pedestals.rms_x12,
            conditions.pedestals.rms_x6,
            conditions.pedestals.rms_x1,
            conditions.pedestals.mean_x12,
            conditions.pedestals.mean_x6,
            conditions.pedestals.mean_x1,
            conditions.gainRatios.gain12Over6,
            conditions.gainRatios.gain6Over1,
            scratch.sample_values.get(),
            scratch.sample_value_errors.get(),
            scratch.ampMaxError.get(),
            scratch.useless_sample_values.get(),
            scratch.pedestal_nums.get(),
            offsetForHashes,
            offsetForInputs,
            conditions.sampleMask.getEcalSampleMaskRecordEB(),
            conditions.sampleMask.getEcalSampleMaskRecordEE(),
            totalChannels);
        cudaCheck(hipGetLastError());

        //
        // TODO: small kernel only for EB. It needs to be checked if
        /// fusing such small kernels is beneficial in here
        //
        // we are running only over EB digis
        // therefore we need to create threads/blocks only for that
        unsigned int const threadsFixMGPA = threads_1d;
        unsigned int const blocksFixMGPA =
            threadsFixMGPA > 10 * eventInputGPU.ebDigis.size
                ? 1
                : (10 * eventInputGPU.ebDigis.size + threadsFixMGPA - 1) / threadsFixMGPA;
        kernel_time_compute_fixMGPAslew<<<blocksFixMGPA, threadsFixMGPA, 0, cudaStream>>>(
            eventInputGPU.ebDigis.data.get(),
            eventInputGPU.eeDigis.data.get(),
            scratch.sample_values.get(),
            scratch.sample_value_errors.get(),
            scratch.useless_sample_values.get(),
            conditions.sampleMask.getEcalSampleMaskRecordEB(),
            totalChannels,
            offsetForInputs);
        cudaCheck(hipGetLastError());

        int sharedBytes = EcalDataFrame::MAXSAMPLES * nchannels_per_block * 4 * sizeof(SampleVector::Scalar);
        auto const threads_nullhypot = threads_1d;
        auto const blocks_nullhypot = blocks_1d;
        kernel_time_compute_nullhypot<<<blocks_nullhypot, threads_nullhypot, sharedBytes, cudaStream>>>(
            scratch.sample_values.get(),
            scratch.sample_value_errors.get(),
            scratch.useless_sample_values.get(),
            scratch.chi2sNullHypot.get(),
            scratch.sum0sNullHypot.get(),
            scratch.sumAAsNullHypot.get(),
            totalChannels);
        cudaCheck(hipGetLastError());

        unsigned int nchannels_per_block_makeratio = 10;
        unsigned int threads_makeratio = 45 * nchannels_per_block_makeratio;
        unsigned int blocks_makeratio = threads_makeratio > 45 * totalChannels
                                            ? 1
                                            : (totalChannels * 45 + threads_makeratio - 1) / threads_makeratio;
        int sharedBytesMakeRatio = 5 * threads_makeratio * sizeof(SampleVector::Scalar);
        kernel_time_compute_makeratio<<<blocks_makeratio, threads_makeratio, sharedBytesMakeRatio, cudaStream>>>(
            scratch.sample_values.get(),
            scratch.sample_value_errors.get(),
            eventInputGPU.ebDigis.ids.get(),
            eventInputGPU.eeDigis.ids.get(),
            scratch.useless_sample_values.get(),
            scratch.pedestal_nums.get(),
            configParameters.amplitudeFitParametersEB,
            configParameters.amplitudeFitParametersEE,
            configParameters.timeFitParametersEB,
            configParameters.timeFitParametersEE,
            scratch.sumAAsNullHypot.get(),
            scratch.sum0sNullHypot.get(),
            scratch.tMaxAlphaBetas.get(),
            scratch.tMaxErrorAlphaBetas.get(),
            scratch.accTimeMax.get(),
            scratch.accTimeWgt.get(),
            scratch.tcState.get(),
            configParameters.timeFitParametersSizeEB,
            configParameters.timeFitParametersSizeEE,
            configParameters.timeFitLimitsFirstEB,
            configParameters.timeFitLimitsFirstEE,
            configParameters.timeFitLimitsSecondEB,
            configParameters.timeFitLimitsSecondEE,
            totalChannels,
            offsetForInputs);
        cudaCheck(hipGetLastError());

        auto const threads_findamplchi2 = threads_1d;
        auto const blocks_findamplchi2 = blocks_1d;
        int const sharedBytesFindAmplChi2 = 2 * threads_findamplchi2 * sizeof(SampleVector::Scalar);
        kernel_time_compute_findamplchi2_and_finish<<<blocks_findamplchi2,
                                                      threads_findamplchi2,
                                                      sharedBytesFindAmplChi2,
                                                      cudaStream>>>(scratch.sample_values.get(),
                                                                    scratch.sample_value_errors.get(),
                                                                    eventInputGPU.ebDigis.ids.get(),
                                                                    eventInputGPU.eeDigis.ids.get(),
                                                                    scratch.useless_sample_values.get(),
                                                                    scratch.tMaxAlphaBetas.get(),
                                                                    scratch.tMaxErrorAlphaBetas.get(),
                                                                    scratch.accTimeMax.get(),
                                                                    scratch.accTimeWgt.get(),
                                                                    configParameters.amplitudeFitParametersEB,
                                                                    configParameters.amplitudeFitParametersEE,
                                                                    scratch.sumAAsNullHypot.get(),
                                                                    scratch.sum0sNullHypot.get(),
                                                                    scratch.chi2sNullHypot.get(),
                                                                    scratch.tcState.get(),
                                                                    scratch.ampMaxAlphaBeta.get(),
                                                                    scratch.ampMaxError.get(),
                                                                    scratch.timeMax.get(),
                                                                    scratch.timeError.get(),
                                                                    totalChannels,
                                                                    offsetForInputs);
        cudaCheck(hipGetLastError());

        auto const threads_timecorr = 32;
        auto const blocks_timecorr =
            threads_timecorr > totalChannels ? 1 : (totalChannels + threads_timecorr - 1) / threads_timecorr;
        kernel_time_correction_and_finalize<<<blocks_timecorr, threads_timecorr, 0, cudaStream>>>(
            eventOutputGPU.recHitsEB.amplitude.get(),
            eventOutputGPU.recHitsEE.amplitude.get(),
            eventInputGPU.ebDigis.data.get(),
            eventInputGPU.ebDigis.ids.get(),
            eventInputGPU.eeDigis.data.get(),
            eventInputGPU.eeDigis.ids.get(),
            conditions.timeBiasCorrections.ebTimeCorrAmplitudeBins,
            conditions.timeBiasCorrections.eeTimeCorrAmplitudeBins,
            conditions.timeBiasCorrections.ebTimeCorrShiftBins,
            conditions.timeBiasCorrections.eeTimeCorrShiftBins,
            scratch.timeMax.get(),
            scratch.timeError.get(),
            conditions.pedestals.rms_x12,
            conditions.timeCalibConstants.values,
            eventOutputGPU.recHitsEB.jitter.get(),
            eventOutputGPU.recHitsEE.jitter.get(),
            eventOutputGPU.recHitsEB.jitterError.get(),
            eventOutputGPU.recHitsEE.jitterError.get(),
            eventOutputGPU.recHitsEB.flags.get(),
            eventOutputGPU.recHitsEE.flags.get(),
            conditions.timeBiasCorrections.ebTimeCorrAmplitudeBinsSize,
            conditions.timeBiasCorrections.eeTimeCorrAmplitudeBinsSize,
            configParameters.timeConstantTermEB,
            configParameters.timeConstantTermEE,
            conditions.timeOffsetConstant.getEBValue(),
            conditions.timeOffsetConstant.getEEValue(),
            configParameters.timeNconstEB,
            configParameters.timeNconstEE,
            configParameters.amplitudeThreshEB,
            configParameters.amplitudeThreshEE,
            configParameters.outOfTimeThreshG12pEB,
            configParameters.outOfTimeThreshG12pEE,
            configParameters.outOfTimeThreshG12mEB,
            configParameters.outOfTimeThreshG12mEE,
            configParameters.outOfTimeThreshG61pEB,
            configParameters.outOfTimeThreshG61pEE,
            configParameters.outOfTimeThreshG61mEB,
            configParameters.outOfTimeThreshG61mEE,
            offsetForHashes,
            offsetForInputs,
            totalChannels);
        cudaCheck(hipGetLastError());
      }
    }

  }  // namespace multifit
}  // namespace ecal
