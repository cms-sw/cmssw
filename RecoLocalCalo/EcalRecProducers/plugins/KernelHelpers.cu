#include "DataFormats/EcalDetId/interface/EBDetId.h"
#include "DataFormats/EcalDetId/interface/EEDetId.h"

#include "KernelHelpers.h"

namespace ecal {
  namespace reconstruction {

    namespace internal {

      namespace barrel {

        __device__ __forceinline__ bool positiveZ(uint32_t id) { return id & 0x10000; }

        __device__ __forceinline__ uint32_t ietaAbs(uint32_t id) { return (id >> 9) & 0x7F; }

        __device__ __forceinline__ uint32_t iphi(uint32_t id) { return id & 0x1FF; }

        __device__ int dccFromSm(int ism) {
          int iz = 1;
          if (ism > 18)
            iz = -1;
          if (iz == -1)
            ism -= 18;
          int idcc = 9 + ism;
          if (iz == +1)
            idcc += 18;
          return idcc;
        }

        __device__ int sm(int ieta, int iphi) {
          int iz = 1;
          if (ieta < 0)
            iz = -1;
          ieta *= iz;
          int iphi_ = iphi;
          if (iphi_ > 360)
            iphi_ -= 360;
          int ism = (iphi_ - 1) / 20 + 1;
          if (iz == -1)
            ism += 18;
          return ism;
        }

        __device__ int dcc(int ieta, int iphi) {
          int ism = sm(ieta, iphi);
          return dccFromSm(ism);
        }

        __device__ int lm_channel(int iX, int iY) {
          static const int idx_[] = {
              // clang-format off
         // 0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16
            1, 2, 2, 2, 2, 4, 4, 4, 4, 6, 6, 6, 6, 8, 8, 8, 8,  // 3
            1, 2, 2, 2, 2, 4, 4, 4, 4, 6, 6, 6, 6, 8, 8, 8, 8,  // 2
            1, 3, 3, 3, 3, 5, 5, 5, 5, 7, 7, 7, 7, 9, 9, 9, 9,  // 1
            1, 3, 3, 3, 3, 5, 5, 5, 5, 7, 7, 7, 7, 9, 9, 9, 9  // 0
         // 0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16
              // clang-format on
          };

          int il, ic, ii;
          const int iym = 4;
          const int ixm = 17;
          int iX_ = iX + 1;
          int iY_ = iY + 1;
          il = iym - iY_;
          ic = iX_ - 1;
          ii = il * ixm + ic;
          if (ii < 0 || ii > (int)(sizeof(idx_) / sizeof(int))) {
            return -1;
          };
          return idx_[ii];
        }

        __device__ int localCoord_x(int ieta, int iphi) {
          int iz = 1;
          if (ieta < 0) {
            iz = -1;
          }
          ieta *= iz;
          int ix = ieta - 1;

          return ix;
        }

        __device__ int localCoord_y(int ieta, int iphi) {
          int iz = 1;
          if (ieta < 0) {
            iz = -1;
          }
          int iphi_ = iphi;
          if (iphi_ > 360) {
            iphi_ -= 360;
          }
          int iy = (iphi_ - 1) % 20;
          if (iz == -1) {
            iy = 19 - iy;
          }

          return iy;
        }

        __device__ int lmmod(int ieta, int iphi) {
          int ix = localCoord_x(ieta, iphi);
          int iy = localCoord_y(ieta, iphi);

          return lm_channel(ix / 5, iy / 5);
        }

        __device__ int side(int ieta, int iphi) {
          int ilmmod = lmmod(ieta, iphi);
          return (ilmmod % 2 == 0) ? 1 : 0;
        }

      }  // namespace barrel

    }  // namespace internal

    __device__ uint32_t hashedIndexEB(uint32_t id) {
      using namespace internal::barrel;
      return (EBDetId::MAX_IETA + (positiveZ(id) ? ietaAbs(id) - 1 : -ietaAbs(id))) * EBDetId::MAX_IPHI + iphi(id) - 1;
    }

    //
    // https://cmssdt.cern.ch/lxr/source/CalibCalorimetry/EcalLaserAnalyzer/src/MEEBGeom.cc
    //  function: "lmr"

    __device__ int laser_monitoring_region_EB(uint32_t id) {
      using namespace internal::barrel;

      int ieta;
      if (positiveZ(id)) {
        ieta = ietaAbs(id);
      } else {
        ieta = -ietaAbs(id);
      }

      int idcc = dcc(ieta, (int)(iphi(id)));
      int ism = idcc - 9;

      int iside = side(ieta, (int)(iphi(id)));

      return (1 + 2 * (ism - 1) + iside);
    }

    namespace internal {

      namespace endcap {

        __device__ __forceinline__ uint32_t ix(uint32_t id) { return (id >> 7) & 0x7F; }

        __device__ __forceinline__ uint32_t iy(uint32_t id) { return id & 0x7F; }

        __device__ __forceinline__ bool positiveZ(uint32_t id) { return id & 0x4000; }

        // these constants come from EE Det Id
        __constant__ const unsigned short kxf[] = {
            41, 51, 41, 51, 41, 51, 36, 51, 36, 51, 26, 51, 26, 51, 26, 51, 21, 51, 21, 51, 21, 51, 21, 51, 21,
            51, 16, 51, 16, 51, 14, 51, 14, 51, 14, 51, 14, 51, 14, 51, 9,  51, 9,  51, 9,  51, 9,  51, 9,  51,
            6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 4,  51, 4,  51, 4,
            51, 4,  51, 4,  56, 1,  58, 1,  59, 1,  60, 1,  61, 1,  61, 1,  62, 1,  62, 1,  62, 1,  62, 1,  62,
            1,  62, 1,  62, 1,  62, 1,  62, 1,  62, 1,  61, 1,  61, 1,  60, 1,  59, 1,  58, 4,  56, 4,  51, 4,
            51, 4,  51, 4,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51, 6,  51,
            9,  51, 9,  51, 9,  51, 9,  51, 9,  51, 14, 51, 14, 51, 14, 51, 14, 51, 14, 51, 16, 51, 16, 51, 21,
            51, 21, 51, 21, 51, 21, 51, 21, 51, 26, 51, 26, 51, 26, 51, 36, 51, 36, 51, 41, 51, 41, 51, 41, 51};

        __constant__ const unsigned short kdi[] = {
            0,    10,   20,   30,   40,   50,   60,   75,   90,   105,  120,  145,  170,  195,  220,  245,  270,
            300,  330,  360,  390,  420,  450,  480,  510,  540,  570,  605,  640,  675,  710,  747,  784,  821,
            858,  895,  932,  969,  1006, 1043, 1080, 1122, 1164, 1206, 1248, 1290, 1332, 1374, 1416, 1458, 1500,
            1545, 1590, 1635, 1680, 1725, 1770, 1815, 1860, 1905, 1950, 1995, 2040, 2085, 2130, 2175, 2220, 2265,
            2310, 2355, 2400, 2447, 2494, 2541, 2588, 2635, 2682, 2729, 2776, 2818, 2860, 2903, 2946, 2988, 3030,
            3071, 3112, 3152, 3192, 3232, 3272, 3311, 3350, 3389, 3428, 3467, 3506, 3545, 3584, 3623, 3662, 3701,
            3740, 3779, 3818, 3857, 3896, 3935, 3974, 4013, 4052, 4092, 4132, 4172, 4212, 4253, 4294, 4336, 4378,
            4421, 4464, 4506, 4548, 4595, 4642, 4689, 4736, 4783, 4830, 4877, 4924, 4969, 5014, 5059, 5104, 5149,
            5194, 5239, 5284, 5329, 5374, 5419, 5464, 5509, 5554, 5599, 5644, 5689, 5734, 5779, 5824, 5866, 5908,
            5950, 5992, 6034, 6076, 6118, 6160, 6202, 6244, 6281, 6318, 6355, 6392, 6429, 6466, 6503, 6540, 6577,
            6614, 6649, 6684, 6719, 6754, 6784, 6814, 6844, 6874, 6904, 6934, 6964, 6994, 7024, 7054, 7079, 7104,
            7129, 7154, 7179, 7204, 7219, 7234, 7249, 7264, 7274, 7284, 7294, 7304, 7314};

        __device__ int quadrant(int iX, int iY) {
          bool near = iX >= 11;
          bool far = !near;
          bool top = iY >= 11;
          bool bot = !top;

          int iquad = 0;
          if (near && top)
            iquad = 1;
          if (far && top)
            iquad = 2;
          if (far && bot)
            iquad = 3;
          if (near && bot)
            iquad = 4;

          return iquad;
        }

        __device__ int sector(int iX, int iY) {
          //  Y (towards the surface)
          //  T
          //  |
          //  |
          //  |
          //  o---------| X  (towards center of LHC)
          //
          static const int idx_[] = {
              // clang-format off
             // 1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20
                0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 9, 9, 9, 0, 0, 0, 0, 0, 0, 0,  // 20
                0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 9, 9, 9, 9, 9, 9, 0, 0, 0, 0,  // 19
                0, 0, 0, 2, 1, 1, 1, 1, 1, 1, 9, 9, 9, 9, 9, 9, 8, 0, 0, 0,  // 18
                0, 0, 2, 2, 2, 1, 1, 1, 1, 1, 9, 9, 9, 9, 9, 8, 8, 8, 0, 0,  // 17
                0, 2, 2, 2, 2, 1, 1, 1, 1, 1, 9, 9, 9, 9, 9, 8, 8, 8, 8, 0,  // 16
                0, 2, 2, 2, 2, 2, 1, 1, 1, 1, 9, 9, 9, 9, 8, 8, 8, 8, 8, 0,  // 15
                0, 2, 2, 2, 2, 2, 2, 1, 1, 1, 9, 9, 9, 8, 8, 8, 8, 8, 8, 0,  // 14
                2, 2, 2, 2, 2, 2, 2, 2, 1, 1, 9, 9, 8, 8, 8, 8, 8, 8, 8, 8,  // 13
                3, 3, 2, 2, 2, 2, 2, 2, 2, 0, 0, 8, 8, 8, 8, 8, 8, 8, 7, 7,  // 12
                3, 3, 3, 3, 3, 3, 3, 2, 0, 0, 0, 0, 8, 7, 7, 7, 7, 7, 7, 7,  // 11
                3, 3, 3, 3, 3, 3, 3, 3, 0, 0, 0, 0, 7, 7, 7, 7, 7, 7, 7, 7,  // 10
                3, 3, 3, 3, 3, 3, 3, 4, 4, 0, 0, 6, 6, 7, 7, 7, 7, 7, 7, 7,  // 9
                3, 3, 3, 3, 3, 3, 4, 4, 4, 5, 5, 6, 6, 6, 7, 7, 7, 7, 7, 7,  // 8
                0, 3, 3, 3, 4, 4, 4, 4, 4, 5, 5, 6, 6, 6, 6, 6, 7, 7, 7, 0,  // 7
                0, 3, 4, 4, 4, 4, 4, 4, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 7, 0,  // 6
                0, 4, 4, 4, 4, 4, 4, 4, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 0,  // 5
                0, 0, 4, 4, 4, 4, 4, 4, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 0, 0,  // 4
                0, 0, 0, 4, 4, 4, 4, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 0, 0, 0,  // 3
                0, 0, 0, 0, 4, 4, 4, 5, 5, 5, 5, 5, 5, 6, 6, 6, 0, 0, 0, 0,  // 2
                0, 0, 0, 0, 0, 0, 0, 5, 5, 5, 5, 5, 5, 0, 0, 0, 0, 0, 0, 0   // 1
             // 1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20
              // clang-format on
          };

          int iym, ixm, il, ic, ii;
          iym = 20;
          ixm = 20;
          int iX_ = iX;
          int iY_ = iY;
          il = iym - iY_;
          ic = iX_ - 1;
          ii = il * ixm + ic;

          if (ii < 0 || ii > (int)(sizeof(idx_) / sizeof(int)) || idx_[ii] == 0) {
            return -1;
          };
          return idx_[ii];
        }

      }  // namespace endcap

    }  // namespace internal

    __device__ uint32_t hashedIndexEE(uint32_t id) {
      using namespace internal::endcap;

      const uint32_t jx(ix(id));
      const uint32_t jd(2 * (iy(id) - 1) + (jx - 1) / 50);
      return ((positiveZ(id) ? EEDetId::kEEhalf : 0) + kdi[jd] + jx - kxf[jd]);
    }

    //
    // https://cmssdt.cern.ch/lxr/source/CalibCalorimetry/EcalLaserAnalyzer/src/MEEEGeom.cc
    // https://github.com/cms-sw/cmssw/blob/master/CalibCalorimetry/EcalLaserCorrection/src/EcalLaserDbService.cc
    //

    __device__ int laser_monitoring_region_EE(uint32_t id) {
      using namespace internal::endcap;

      // SuperCrysCoord
      uint32_t iX = (ix(id) - 1) / 5 + 1;
      uint32_t iY = (iy(id) - 1) / 5 + 1;

      // Correct convention
      //   * @param iz iz/zside index: -1 for EE-, +1 for EE+
      //   https://github.com/cms-sw/cmssw/blob/master/DataFormats/EcalDetId/interface/EEDetId.h#L68-L71
      //   zside in https://github.com/cms-sw/cmssw/blob/master/CalibCalorimetry/EcalLaserCorrection/src/EcalLaserDbService.cc#L63
      //
      int iz = positiveZ(id) ? 1 : -1;

      int iquad = quadrant(iX, iY);
      int isect = sector(iX, iY);
      if (isect < 0)
        return -1;

      int ilmr = 0;
      ilmr = isect - 6;
      if (ilmr <= 0)
        ilmr += 9;
      if (ilmr == 9)
        ilmr++;
      if (ilmr == 8 && iquad == 4)
        ilmr++;
      if (iz == +1)
        ilmr += 72;
      else
        ilmr += 82;

      return ilmr;
    }

  }  // namespace reconstruction
}  // namespace ecal
