#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "DataFormats/ForwardDetId/interface/HGCalDetId.h"
#include "RecoLocalCalo/HGCalRecProducers/plugins/HGCalCellPositionsKernelImpl.cuh"

__global__ void fill_positions_from_detids(
    const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds) {
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < conds->nelems_posmap; i += blockDim.x * gridDim.x) {
    HeterogeneousHGCSiliconDetId did(conds->posmap.detid[i]);
    const float cU = static_cast<float>(did.cellU());
    const float cV = static_cast<float>(did.cellV());
    const float wU = static_cast<float>(did.waferU());
    const float wV = static_cast<float>(did.waferV());
    const float ncells = static_cast<float>(did.nCellsSide());
    const int32_t layer = did.layer();

    //based on `std::pair<float, float> HGCalDDDConstants::locateCell(const HGCSiliconDetId&, bool)
    const float r_x2 = conds->posmap.waferSize + conds->posmap.sensorSeparation;
    const float r = 0.5f * r_x2;
    const float sqrt3 = __fsqrt_rn(3.f);
    const float rsqrt3 = __frsqrt_rn(3.f);  //rsqrt: 1 / sqrt
    const float R = r_x2 * rsqrt3;
    const float n2 = ncells / 2.f;
    const float yoff_abs = rsqrt3 * r_x2;
    const float yoff = (layer % 2 == 1) ? yoff_abs : -1.f * yoff_abs;  //CHANGE according to Sunanda's reply
    float xpos = (-2.f * wU + wV) * r;
    float ypos = yoff + (1.5f * wV * R);
    const float R1 = __fdividef(conds->posmap.waferSize, 3.f * ncells);
    const float r1_x2 = R1 * sqrt3;
    xpos += (1.5f * (cV - ncells) + 1.f) * R1;
    ypos += (cU - 0.5f * cV - n2) * r1_x2;
    conds->posmap.x[i] =
        xpos;  // times side; multiply by -1 if one wants to obtain the position from the opposite endcap. CAREFUL WITH LATER DETECTOR ALIGNMENT!!!
    conds->posmap.y[i] = ypos;
  }
}

__global__ void print_positions_from_detids(
    const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds) {
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < conds->nelems_posmap; i += blockDim.x * gridDim.x) {
    HeterogeneousHGCSiliconDetId did(conds->posmap.detid[i]);
    const int32_t layer = did.layer();
    float posz = conds->posmap.zLayer[layer - 1];
    printf("PosX: %lf\t PosY: %lf\t Posz: %lf\n", conds->posmap.x[i], conds->posmap.y[i], posz);
  }
}

//eventually this can also be written in parallel
__device__ unsigned map_cell_index(const float& cu, const float& cv, const unsigned& ncells_side) {
  unsigned counter = 0;
  //left side of wafer
  for (int cellUmax = ncells_side, icellV = 0; cellUmax < 2 * ncells_side && icellV < ncells_side;
       ++cellUmax, ++icellV) {
    for (int icellU = 0; icellU <= cellUmax; ++icellU) {
      if (cu == icellU and cv == icellV)
        return counter;
      else
        counter += 1;
    }
  }
  //right side of wafer
  for (int cellUmin = 1, icellV = ncells_side; cellUmin <= ncells_side && icellV < 2 * ncells_side;
       ++cellUmin, ++icellV) {
    for (int icellU = cellUmin; icellU < 2 * ncells_side; ++icellU) {
      if (cu == icellU and cv == icellV)
        return counter;
      else
        counter += 1;
    }
  }
  printf("ERROR: The cell was not found!");
  return 99;
}

//returns the index of the positions of a specific cell
//performs several geometry-related shifts, and adds them at the end:
//   1) number of cells up to the layer being inspected
//   2) number of cells up to the waferUchunk in question, only in the layer being inspected
//   3) number of cells up to the waferV in question, only in the layer and waferUchunk being inspected
//   4) cell index within this layer, waferUchunk and waferV
//Note: a 'waferUchunk' represents the first dimension of a 2D squared grid of wafers, and includes multiple waferV
__device__ unsigned hash_function(const int32_t& l,
                                  const int32_t& wU,
                                  const int32_t& wV,
                                  const int32_t& cu,
                                  const int32_t& cv,
                                  const int32_t& ncells_side,
                                  const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds) {
  const unsigned thislayer = l - conds->posmap.firstLayer;
  const unsigned thisUwafer = wU - conds->posmap.waferMin;
  const unsigned thisVwafer = wV - conds->posmap.waferMin;
  const unsigned nwafers1D = conds->posmap.waferMax - conds->posmap.waferMin;

  //layer shift in terms of cell number
  unsigned ncells_up_to_thislayer = 0;
  for (unsigned q = 0; q < thislayer; ++q)
    ncells_up_to_thislayer += conds->posmap.nCellsLayer[q];

  //waferU shift in terms of cell number
  unsigned ncells_up_to_thisUwafer = 0;
  unsigned nwaferUchunks_up_to_this_layer = thislayer * nwafers1D;
  for (unsigned q = 0; q < thisUwafer; ++q)
    ncells_up_to_thisUwafer += conds->posmap.nCellsWaferUChunk[nwaferUchunks_up_to_this_layer + q];

  //waferV shift in terms of cell number
  unsigned ncells_up_to_thisVwafer = 0;
  const unsigned nwafers_up_to_thisLayer = thislayer * nwafers1D * nwafers1D;
  const unsigned nwafers_up_to_thisUwafer = thisUwafer * nwafers1D;
  for (unsigned q = 0; q < thisVwafer; ++q)
    ncells_up_to_thisVwafer += conds->posmap.nCellsHexagon[nwafers_up_to_thisLayer + nwafers_up_to_thisUwafer + q];

  //cell shift in terms of cell number
  const unsigned cell_shift = map_cell_index(cu, cv, ncells_side);
  const unsigned shift_total = ncells_up_to_thislayer + ncells_up_to_thisUwafer + ncells_up_to_thisVwafer + cell_shift;
  return shift_total;
}

__global__ void test(uint32_t detid_test,
                     const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds) {
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid == 0) {
    //printf("Nelems: %u\n", static_cast<unsigned>(conds->nelems_posmap));
    for (unsigned i = 0; i < 1; ++i) {
      HeterogeneousHGCSiliconDetId did(detid_test);  // 2416969935, 2552165379, ...
      const int32_t cU = did.cellU();
      const int32_t cV = did.cellV();
      const int32_t wU = did.waferU();
      const int32_t wV = did.waferV();
      const int32_t ncs = did.nCellsSide();

      const int32_t layer = abs(did.layer());  //remove abs in case both endcaps are considered for x and y
      const unsigned shift = hash_function(layer, wU, wV, cU, cV, ncs, conds);
      //printf("id: cu: %d, cv: %d, wu: %d, wv: %d, ncells: %d, layer: %d\n", cU, cV, wU, wV, ncs, layer);
      printf("id: %u | shift: %u | x: %lf y: %lf\n",
             conds->posmap.detid[shift],
             shift,
             conds->posmap.x[shift],
             conds->posmap.y[shift]);
    }
  }
}
