#include "HeterogeneousCore/CUDAUtilities/interface/cudaMemoryPool.h"

#include "HeterogeneousCore/CUDAUtilities/interface/SimplePoolAllocator.h"


#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

struct CudaDeviceAlloc {

  using Pointer = void *;

  static Pointer alloc(size_t size) { Pointer p=nullptr; auto err = hipMalloc(&p,size); return err==hipSuccess ? p : nullptr;}
  static void free(Pointer ptr) { hipFree(ptr); }

};

struct CudaHostAlloc {

  using Pointer = void *;

  static Pointer alloc(size_t size) { Pointer p=nullptr; auto err = hipHostMalloc(&p,size); return err==hipSuccess ? p : nullptr;}
  static void free(Pointer ptr) { hipHostFree(ptr); }

};

namespace {

  SimplePoolAllocatorImpl<CudaHostAlloc>  hostPool(1024);

  struct DevicePools {
    using Pool = SimplePoolAllocatorImpl<CudaDeviceAlloc>;
    DevicePools(int size) {
      int devices = 0;
       auto status = hipGetDeviceCount(&devices);
       std::cout << "found " << devices << " cuda devices" << std::endl;
       if (status == hipSuccess && devices>0) {
          m_devicePools.reserve(devices);  
          for (int i=0; i<devices; ++i) m_devicePools.emplace_back(new Pool(size));
       }
    } 
    //return pool for current device
    Pool & operator()() {
       int dev=-1;
       hipGetDevice(&dev);
       return *m_devicePools[dev];
    }

    std::vector<std::unique_ptr<Pool>> m_devicePools;

  };

  DevicePools devicePool(128*1024);

}


namespace memoryPool {
  namespace cuda {

    void dumpStat() {
       std::cout << "device pool" << std::endl;
       devicePool().dumpStat();
       std::cout << "host pool" << std::endl;
       hostPool.dumpStat();

    }


   SimplePoolAllocator * getPool(Where where) {
      return onDevice==where ?  (SimplePoolAllocator *)(&devicePool()) : (SimplePoolAllocator *)(&hostPool);
   }

    struct Payload {
      SimplePoolAllocator * pool;
      std::vector<int> buckets;
    };

    // generic callback
    void CUDART_CB freeCallback(void * p){
      auto payload = (Payload*)(p);
      auto & pool = *(payload->pool);
      auto const & buckets = payload->buckets;
        std::cout << "do free " << buckets.size();
        if (!buckets.empty()) std::cout  << ' ' << buckets.front() << ' ' << buckets.back();
        std::cout << std::endl;
        for (auto i :  buckets) {
          pool.free(i);
        }
      delete payload;
    }

    // allocate either on current device or on host
    std::pair<void *,int> alloc(uint64_t size, SimplePoolAllocator & pool) {
       int i = pool.alloc(size);
       void * p = pool.pointer(i);
       return std::pair<void *,int>(p,i);
    }

    // schedule free
    void free(hipStream_t stream, std::vector<int> buckets, SimplePoolAllocator & pool) {
      // free
      std::cout << "schedule free " << buckets.size() << ' ';
      if (!buckets.empty()) std::cout << buckets[0]; 
      std::cout << std::endl;
      auto payload = new Payload{&pool, std::move(buckets)};
      hipLaunchHostFunc (stream, freeCallback, payload);
    }

  }
}
