#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <limits>
#include <memory>
#include <random>
#include <set>

#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"
#include "HeterogeneousCore/CUDAUtilities/interface/launch.h"
#include "HeterogeneousCore/CUDAUtilities/interface/radixSort.h"

using namespace cms::cuda;

template <typename T>
struct RS {
  using type = std::uniform_int_distribution<T>;
  static auto ud() { return type(std::numeric_limits<T>::min(), std::numeric_limits<T>::max()); }
  static constexpr T imax = std::numeric_limits<T>::max();
};

template <>
struct RS<float> {
  using T = float;
  using type = std::uniform_real_distribution<float>;
  static auto ud() { return type(-std::numeric_limits<T>::max() / 2, std::numeric_limits<T>::max() / 2); }
  //  static auto ud() { return type(0,std::numeric_limits<T>::max()/2);}
  static constexpr int imax = std::numeric_limits<int>::max();
};

template <typename T, int NS = sizeof(T), typename U = T, typename LL = long long>
void go(bool useShared) {
  std::mt19937 eng;
  //std::mt19937 eng2;
  auto rgen = RS<T>::ud();

  auto start = std::chrono::high_resolution_clock::now();
  auto delta = start - start;

  constexpr int blocks = 10;
  constexpr int blockSize = 256 * 32;
  constexpr int N = blockSize * blocks;
  T v[N];
  uint16_t ind[N];

  constexpr bool sgn = T(-1) < T(0);
  std::cout << "Will sort " << N << (sgn ? " signed" : " unsigned")
            << (std::numeric_limits<T>::is_integer ? " 'ints'" : " 'float'") << " of size " << sizeof(T) << " using "
            << NS << " significant bytes" << std::endl;

  for (int i = 0; i < 50; ++i) {
    if (i == 49) {
      for (long long j = 0; j < N; j++)
        v[j] = 0;
    } else if (i > 30) {
      for (long long j = 0; j < N; j++)
        v[j] = rgen(eng);
    } else {
      uint64_t imax = (i < 15) ? uint64_t(RS<T>::imax) + 1LL : 255;
      for (uint64_t j = 0; j < N; j++) {
        v[j] = (j % imax);
        if (j % 2 && i % 2)
          v[j] = -v[j];
      }
    }

    uint32_t offsets[blocks + 1];
    offsets[0] = 0;
    for (int j = 1; j < blocks + 1; ++j) {
      offsets[j] = offsets[j - 1] + blockSize - 3 * j;
      assert(offsets[j] <= N);
    }

    if (i == 1) {  // special cases...
      offsets[0] = 0;
      offsets[1] = 0;
      offsets[2] = 19;
      offsets[3] = 32 + offsets[2];
      offsets[4] = 123 + offsets[3];
      offsets[5] = 256 + offsets[4];
      offsets[6] = 311 + offsets[5];
      offsets[7] = 2111 + offsets[6];
      offsets[8] = 256 * 11 + offsets[7];
      offsets[9] = 44 + offsets[8];
      offsets[10] = 3297 + offsets[9];
    }

    std::random_shuffle(v, v + N);

    auto v_d = cms::cuda::make_device_unique<U[]>(N, nullptr);
    auto ind_d = cms::cuda::make_device_unique<uint16_t[]>(N, nullptr);
    auto ws_d = cms::cuda::make_device_unique<uint16_t[]>(N, nullptr);
    auto off_d = cms::cuda::make_device_unique<uint32_t[]>(blocks + 1, nullptr);

    cudaCheck(hipMemcpy(v_d.get(), v, N * sizeof(T), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(off_d.get(), offsets, 4 * (blocks + 1), hipMemcpyHostToDevice));

    if (i < 2)
      std::cout << "lauch for " << offsets[blocks] << std::endl;

    auto ntXBl __attribute__((unused)) = 1 == i % 4 ? 256 : 256;

    delta -= (std::chrono::high_resolution_clock::now() - start);
    constexpr int MaxSize = 256 * 32;
    if (useShared)
      cms::cuda::launch(
          radixSortMultiWrapper<U, NS>, {blocks, ntXBl, MaxSize * 2}, v_d.get(), ind_d.get(), off_d.get(), nullptr);
    else
      cms::cuda::launch(
          radixSortMultiWrapper2<U, NS>, {blocks, ntXBl}, v_d.get(), ind_d.get(), off_d.get(), ws_d.get());

    if (i == 0)
      std::cout << "done for " << offsets[blocks] << std::endl;

    cudaCheck(hipMemcpy(ind, ind_d.get(), 2 * N, hipMemcpyDeviceToHost));

    delta += (std::chrono::high_resolution_clock::now() - start);

    if (i == 0)
      std::cout << "done for " << offsets[blocks] << std::endl;

    if (32 == i) {
      std::cout << LL(v[ind[0]]) << ' ' << LL(v[ind[1]]) << ' ' << LL(v[ind[2]]) << std::endl;
      std::cout << LL(v[ind[3]]) << ' ' << LL(v[ind[10]]) << ' ' << LL(v[ind[blockSize - 1000]]) << std::endl;
      std::cout << LL(v[ind[blockSize / 2 - 1]]) << ' ' << LL(v[ind[blockSize / 2]]) << ' '
                << LL(v[ind[blockSize / 2 + 1]]) << std::endl;
    }
    for (int ib = 0; ib < blocks; ++ib) {
      std::set<uint16_t> inds;
      if (offsets[ib + 1] > offsets[ib])
        inds.insert(ind[offsets[ib]]);
      for (auto j = offsets[ib] + 1; j < offsets[ib + 1]; j++) {
        inds.insert(ind[j]);
        auto a = v + offsets[ib];
        auto k1 = a[ind[j]];
        auto k2 = a[ind[j - 1]];
        auto sh = sizeof(uint64_t) - NS;
        sh *= 8;
        auto shorten = [sh](T& t) {
          auto k = (uint64_t*)(&t);
          *k = (*k >> sh) << sh;
        };
        shorten(k1);
        shorten(k2);
        if (k1 < k2)
          std::cout << ib << " not ordered at " << ind[j] << " : " << a[ind[j]] << ' ' << a[ind[j - 1]] << std::endl;
      }
      if (!inds.empty()) {
        assert(0 == *inds.begin());
        assert(inds.size() - 1 == *inds.rbegin());
      }
      if (inds.size() != (offsets[ib + 1] - offsets[ib]))
        std::cout << "error " << i << ' ' << ib << ' ' << inds.size() << "!=" << (offsets[ib + 1] - offsets[ib])
                  << std::endl;
      assert(inds.size() == (offsets[ib + 1] - offsets[ib]));
    }
  }  // 50 times
  std::cout << "cuda computation took " << std::chrono::duration_cast<std::chrono::milliseconds>(delta).count() / 50.
            << " ms" << std::endl;
}

int main() {
  cms::cudatest::requireDevices();

  bool useShared = false;

  std::cout << "using Global memory" << std::endl;

  go<int8_t>(useShared);
  go<int16_t>(useShared);
  go<int32_t>(useShared);
  go<int32_t, 3>(useShared);
  go<int64_t>(useShared);
  go<float, 4, float, double>(useShared);
  go<float, 2, float, double>(useShared);

  go<uint8_t>(useShared);
  go<uint16_t>(useShared);
  go<uint32_t>(useShared);
  // go<uint64_t>(v);

  useShared = true;

  std::cout << "using Shared memory" << std::endl;

  go<int8_t>(useShared);
  go<int16_t>(useShared);
  go<int32_t>(useShared);
  go<int32_t, 3>(useShared);
  go<int64_t>(useShared);
  go<float, 4, float, double>(useShared);
  go<float, 2, float, double>(useShared);

  go<uint8_t>(useShared);
  go<uint16_t>(useShared);
  go<uint32_t>(useShared);
  // go<uint64_t>(v);

  return 0;
}
