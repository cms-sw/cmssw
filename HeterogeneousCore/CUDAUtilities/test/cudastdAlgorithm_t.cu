#include "hip/hip_runtime.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudastdAlgorithm.h"


#include "cuda/api_wrappers.h"
#include<cassert>

__global__
void testBinaryFind()
{
    int data[] = { 1, 1, 2, 3, 3, 3, 3, 4, 4, 4, 5, 5, 6 };
 
    auto lower = cuda_std::lower_bound(data, data+13, 4);
    auto upper = cuda_std::upper_bound(data, data+12, 4);
 
    assert(3 == upper-lower);
 
    // classic binary search, returning a value only if it is present
 
    constexpr int data2[] = { 1, 2, 4, 6, 9, 10 }; 
 
    assert(data2+2 == cuda_std::binary_find(data2, data2+6, 4)); 
    assert(data2+6 == cuda_std::binary_find(data2, data2+6, 5));
}

#include<iostream>
void wrapper()
{

  if (cuda::device::count() == 0) {
        std::cerr << "No CUDA devices on this system" << "\n";
        exit(EXIT_FAILURE);
  }

  auto current_device = cuda::device::current::get();

  cuda::launch(
                testBinaryFind,
                { 32, 64 }
              );


}

int main() {

  wrapper();


}
