#include "hip/hip_runtime.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cuda_assert.h"
#include "HeterogeneousCore/CUDAUtilities/interface/exitSansCUDADevices.h"

__global__
void testIt(int one){
  assert(one == 1);
}

int main(int argc, char* argv[]) {
  exitSansCUDADevices();

  testIt<<<1,1>>>(argc);
  hipDeviceSynchronize();

  return (argc == 1) ? EXIT_SUCCESS : EXIT_FAILURE;
}
