#include "hip/hip_runtime.h"
#include <iostream>

#include <hipcub/hipcub.hpp>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/prefixScan.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

using namespace cms::cuda;

template <typename T>
struct format_traits {
public:
  static const constexpr char *failed_msg = "failed %d %d %d: %d %d\n";
};

template <>
struct format_traits<float> {
public:
  static const constexpr char *failed_msg = "failed %d %d %d: %f %f\n";
};

template <typename T>
__global__ void testPrefixScan(uint32_t size) {
  __shared__ T ws[32];
  __shared__ T c[1024];
  __shared__ T co[1024];

  auto first = threadIdx.x;
  for (auto i = first; i < size; i += blockDim.x)
    c[i] = 1;
  __syncthreads();

  blockPrefixScan(c, co, size, ws);
  blockPrefixScan(c, size, ws);

  assert(1 == c[0]);
  assert(1 == co[0]);
  for (auto i = first + 1; i < size; i += blockDim.x) {
    if (c[i] != c[i - 1] + 1)
      printf(format_traits<T>::failed_msg, size, i, blockDim.x, c[i], c[i - 1]);
    assert(c[i] == c[i - 1] + 1);
    assert(c[i] == i + 1);
    assert(c[i] = co[i]);
  }
}

template <typename T>
__global__ void testWarpPrefixScan(uint32_t size) {
  assert(size <= 32);
  __shared__ T c[1024];
  __shared__ T co[1024];
  auto i = threadIdx.x;
  c[i] = 1;
  __syncthreads();

  warpPrefixScan(c, co, i, 0xffffffff);
  warpPrefixScan(c, i, 0xffffffff);
  __syncthreads();

  assert(1 == c[0]);
  assert(1 == co[0]);
  if (i != 0) {
    if (c[i] != c[i - 1] + 1)
      printf(format_traits<T>::failed_msg, size, i, blockDim.x, c[i], c[i - 1]);
    assert(c[i] == c[i - 1] + 1);
    assert(c[i] == i + 1);
    assert(c[i] = co[i]);
  }
}

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    v[i] = val;
  if (i == 0)
    printf("init\n");
}

__global__ void verify(uint32_t const *v, uint32_t n) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    assert(v[i] == i + 1);
  if (i == 0)
    printf("verify\n");
}

int main() {
  cms::cudatest::requireDevices();

  std::cout << "warp level" << std::endl;
  // std::cout << "warp 32" << std::endl;
  testWarpPrefixScan<int><<<1, 32>>>(32);
  hipDeviceSynchronize();
  // std::cout << "warp 16" << std::endl;
  testWarpPrefixScan<int><<<1, 32>>>(16);
  hipDeviceSynchronize();
  // std::cout << "warp 5" << std::endl;
  testWarpPrefixScan<int><<<1, 32>>>(5);
  hipDeviceSynchronize();

  std::cout << "block level" << std::endl;
  for (int bs = 32; bs <= 1024; bs += 32) {
    // std::cout << "bs " << bs << std::endl;
    for (int j = 1; j <= 1024; ++j) {
      // std::cout << j << std::endl;
      testPrefixScan<uint16_t><<<1, bs>>>(j);
      hipDeviceSynchronize();
      testPrefixScan<float><<<1, bs>>>(j);
      hipDeviceSynchronize();
    }
  }
  hipDeviceSynchronize();

  int num_items = 200;
  for (int ksize = 1; ksize < 4; ++ksize) {
    // test multiblock
    std::cout << "multiblok" << std::endl;
    // Declare, allocate, and initialize device-accessible pointers for input and output
    num_items *= 10;
    uint32_t *d_in;
    uint32_t *d_out1;
    uint32_t *d_out2;

    cudaCheck(hipMalloc(&d_in, num_items * sizeof(uint32_t)));
    cudaCheck(hipMalloc(&d_out1, num_items * sizeof(uint32_t)));
    cudaCheck(hipMalloc(&d_out2, num_items * sizeof(uint32_t)));

    auto nthreads = 256;
    auto nblocks = (num_items + nthreads - 1) / nthreads;

    init<<<nblocks, nthreads, 0>>>(d_in, 1, num_items);

    // the block counter
    int32_t *d_pc;
    cudaCheck(hipMalloc(&d_pc, sizeof(int32_t)));
    cudaCheck(hipMemset(d_pc, 0, 4));

    nthreads = 1024;
    nblocks = (num_items + nthreads - 1) / nthreads;
    multiBlockPrefixScan<<<nblocks, nthreads, 0>>>(d_in, d_out1, num_items, d_pc);
    verify<<<nblocks, nthreads, 0>>>(d_out1, num_items);
    hipDeviceSynchronize();

    // test cub
    std::cout << "cub" << std::endl;
    // Determine temporary device storage requirements for inclusive prefix sum
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out2, num_items);

    std::cout << "temp storage " << temp_storage_bytes << std::endl;

    // Allocate temporary storage for inclusive prefix sum
    // fake larger ws already available
    temp_storage_bytes *= 8;
    cudaCheck(hipMalloc(&d_temp_storage, temp_storage_bytes));
    std::cout << "temp storage " << temp_storage_bytes << std::endl;
    // Run inclusive prefix sum
    CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out2, num_items));
    std::cout << "temp storage " << temp_storage_bytes << std::endl;

    verify<<<nblocks, nthreads, 0>>>(d_out2, num_items);
    hipDeviceSynchronize();
  }  // ksize
  return 0;
}
