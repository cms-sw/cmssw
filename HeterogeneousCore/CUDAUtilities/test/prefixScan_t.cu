#include "hip/hip_runtime.h"
#include "HeterogeneousCore/CUDAUtilities/interface/prefixScan.h"


template<typename T>
__global__
void testPrefixScan(uint32_t size) {

  __shared__ T ws[32];
  __shared__ T c[1024];
  auto first = threadIdx.x;
  for (auto i=first; i<size; i+=blockDim.x) c[i]=1;
  __syncthreads();

  blockPrefixScan(c, size, ws);

  assert(1==c[0]);
  for (auto i=first+1; i<size; i+=blockDim.x) {
    if (c[i]!=c[i-1]+1) printf("failed %d %d %d: %d %d\n",size, i, blockDim.x, c[i],c[i-1]);
    assert(c[i]==c[i-1]+1); assert(c[i]==i+1);
  }
}


template<typename T>
__global__
void testWarpPrefixScan(uint32_t size) {
  assert(size<=32);
  __shared__ T c[1024];
  auto i = threadIdx.x;
  c[i]=1;
  __syncthreads();

  warpPrefixScan(c,i,0xffffffff);
 __syncthreads();

  assert(1==c[0]);
  if(i!=0) {
    if (c[i]!=c[i-1]+1) printf("failed %d %d %d: %d %d\n",size, i, blockDim.x, c[i],c[i-1]);
    assert(c[i]==c[i-1]+1); assert(c[i]==i+1);
  }
}

#include <hipcub/hipcub.hpp> 


__global__
void  init(uint32_t  * v, uint32_t  val, uint32_t n) {
     auto i  = blockIdx.x * blockDim.x + threadIdx.x;
     if(i<n) v[i]=val;
     if (i==0) printf("init\n");
}

__global__
void  verify(uint32_t  const * v, uint32_t n) {
     auto i  = blockIdx.x * blockDim.x + threadIdx.x;
     if(i<n) assert(v[i]==i+1);
     if (i==0) printf("verify\n");
}


#include<iostream>
int main() {

  std::cout << "warp level" << std::endl;
  // std::cout << "warp 32" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(32);
  hipDeviceSynchronize();
  // std::cout << "warp 16" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(16);
  hipDeviceSynchronize();
  // std::cout << "warp 5" << std::endl;
  testWarpPrefixScan<int><<<1,32>>>(5);
  hipDeviceSynchronize();

  std::cout << "block level" << std::endl;
  for(int bs=32; bs<=1024; bs+=32) {
//  std::cout << "bs " << bs << std::endl;
  for (int j=1;j<=1024; ++j) {
//   std::cout << j << std::endl;
   testPrefixScan<uint16_t><<<1,bs>>>(j);
   hipDeviceSynchronize();
   testPrefixScan<float><<<1,bs>>>(j);
  hipDeviceSynchronize();
  }}
  hipDeviceSynchronize();


  // test cub
  std::cout << "cub" << std::endl;
// Declare, allocate, and initialize device-accessible pointers for input and output
   int  num_items = 10000;
   uint32_t  *d_in;         
   uint32_t  *d_out;


   hipMalloc(&d_in,num_items*sizeof(uint32_t));
   // hipMalloc(&d_out,num_items*sizeof(uint32_t));

   d_out = d_in;
  
   auto nthreads = 256;
   auto nblocks = (num_items + nthreads - 1) / nthreads;

   init<<<nblocks, nthreads, 0>>>(d_in, 1, num_items);

   // Determine temporary device storage requirements for inclusive prefix sum
   void     *d_temp_storage = nullptr;
   size_t   temp_storage_bytes = 0;
   hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

   std::cout << "temp storage " << temp_storage_bytes << std::endl;

   // Allocate temporary storage for inclusive prefix sum
   // fake larger ws already available
   temp_storage_bytes *=8;
   hipMalloc(&d_temp_storage, temp_storage_bytes);
   std::cout << "temp storage " << temp_storage_bytes << std::endl;
   // Run inclusive prefix sum
   CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));
   std::cout << "temp storage " << temp_storage_bytes << std::endl;

   verify<<<nblocks, nthreads, 0>>>(d_out, num_items);
   hipDeviceSynchronize();

  return 0;
}
