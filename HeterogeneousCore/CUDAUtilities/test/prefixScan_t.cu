#include "hip/hip_runtime.h"
#include <iostream>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/prefixScan.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

using namespace cms::cuda;

template <typename T>
struct format_traits {
public:
  static const constexpr char *failed_msg = "failed %d %d %d: %d %d\n";
};

template <>
struct format_traits<float> {
public:
  static const constexpr char *failed_msg = "failed %d %d %d: %f %f\n";
};

template <typename T>
__global__ void testPrefixScan(uint32_t size) {
  __shared__ T ws[32];
  __shared__ T c[1024];
  __shared__ T co[1024];

  auto first = threadIdx.x;
  for (auto i = first; i < size; i += blockDim.x)
    c[i] = 1;
  __syncthreads();

  blockPrefixScan(c, co, size, ws);
  blockPrefixScan(c, size, ws);

  assert(1 == c[0]);
  assert(1 == co[0]);
  for (auto i = first + 1; i < size; i += blockDim.x) {
    if (c[i] != c[i - 1] + 1)
      printf(format_traits<T>::failed_msg, size, i, blockDim.x, c[i], c[i - 1]);
    assert(c[i] == c[i - 1] + 1);
    assert(c[i] == i + 1);
    assert(c[i] = co[i]);
  }
}

template <typename T>
__global__ void testWarpPrefixScan(uint32_t size) {
  assert(size <= 32);
  __shared__ T c[1024];
  __shared__ T co[1024];
  auto i = threadIdx.x;
  c[i] = 1;
  __syncthreads();

  warpPrefixScan(c, co, i, 0xffffffff);
  warpPrefixScan(c, i, 0xffffffff);
  __syncthreads();

  assert(1 == c[0]);
  assert(1 == co[0]);
  if (i != 0) {
    if (c[i] != c[i - 1] + 1)
      printf(format_traits<T>::failed_msg, size, i, blockDim.x, c[i], c[i - 1]);
    assert(c[i] == c[i - 1] + 1);
    assert(c[i] == i + 1);
    assert(c[i] = co[i]);
  }
}

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    v[i] = val;
  if (i == 0)
    printf("init\n");
}

__global__ void verify(uint32_t const *v, uint32_t n) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    assert(v[i] == i + 1);
  if (i == 0)
    printf("verify\n");
}

int main() {
  cms::cudatest::requireDevices();

  std::cout << "warp level" << std::endl;
  // std::cout << "warp 32" << std::endl;
  testWarpPrefixScan<int><<<1, 32>>>(32);
  hipDeviceSynchronize();
  // std::cout << "warp 16" << std::endl;
  testWarpPrefixScan<int><<<1, 32>>>(16);
  hipDeviceSynchronize();
  // std::cout << "warp 5" << std::endl;
  testWarpPrefixScan<int><<<1, 32>>>(5);
  hipDeviceSynchronize();

  std::cout << "block level" << std::endl;
  for (int bs = 32; bs <= 1024; bs += 32) {
    // std::cout << "bs " << bs << std::endl;
    for (int j = 1; j <= 1024; ++j) {
      // std::cout << j << std::endl;
      testPrefixScan<uint16_t><<<1, bs>>>(j);
      hipDeviceSynchronize();
      testPrefixScan<float><<<1, bs>>>(j);
      hipDeviceSynchronize();
    }
  }
  hipDeviceSynchronize();

  int num_items = 200;
  for (int ksize = 1; ksize < 4; ++ksize) {
    // test multiblock
    std::cout << "multiblok" << std::endl;
    // Declare, allocate, and initialize device-accessible pointers for input and output
    num_items *= 10;
    uint32_t *d_in;
    uint32_t *d_out1;
    uint32_t *d_out2;

    cudaCheck(hipMalloc(&d_in, num_items * sizeof(uint32_t)));
    cudaCheck(hipMalloc(&d_out1, num_items * sizeof(uint32_t)));
    cudaCheck(hipMalloc(&d_out2, num_items * sizeof(uint32_t)));

    auto nthreads = 256;
    auto nblocks = (num_items + nthreads - 1) / nthreads;

    init<<<nblocks, nthreads, 0>>>(d_in, 1, num_items);

    // the block counter
    int32_t *d_pc;
    cudaCheck(hipMalloc(&d_pc, sizeof(int32_t)));
    cudaCheck(hipMemset(d_pc, 0, sizeof(int32_t)));

    nthreads = 1024;
    nblocks = (num_items + nthreads - 1) / nthreads;
    std::cout << "launch multiBlockPrefixScan " << num_items << ' ' << nblocks << std::endl;
    multiBlockPrefixScan<<<nblocks, nthreads, 4 * nblocks>>>(d_in, d_out1, num_items, d_pc);
    cudaCheck(hipGetLastError());
    verify<<<nblocks, nthreads, 0>>>(d_out1, num_items);
    cudaCheck(hipGetLastError());
    hipDeviceSynchronize();

  }  // ksize
  return 0;
}
