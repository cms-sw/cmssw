#include "HeterogeneousCore/CUDAUtilities/interface/cudaMemoryPool.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include <iostream>

template <memoryPool::Where where>
struct dataProducer {
  auto operator()(hipStream_t stream) { return memoryPool::cuda::make_buffer<int>(20, stream, where); }
};

int main() {
  {
    int devices = 0;
    auto status = hipGetDeviceCount(&devices);
    if (status != hipSuccess || 0 == devices)
      return 0;
    std::cout << "found " << devices << " cuda devices" << std::endl;
  }
  const int NUMTHREADS = 1;

  printf("Using CUDA %d\n", CUDART_VERSION);
  int cuda_device = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cuda_device);
  printf("CUDA Capable: SM %d.%d hardware\n", deviceProp.major, deviceProp.minor);

  hipStream_t streams[NUMTHREADS];

  for (int i = 0; i < NUMTHREADS; i++) {
    hipStreamCreate(&(streams[i]));
  }

  memoryPool::cuda::dumpStat();

  auto& stream = streams[0];

  {
    auto pd = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onDevice);
    auto ph = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onHost);
    auto pc = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onCPU);

    auto dp = dataProducer<memoryPool::onDevice>()(stream);

    cudaCheck(memoryPool::cuda::copy(ph,pd,20,stream));
    cudaCheck(memoryPool::cuda::copy(pd,ph,20,stream));
    std::cout << "expect 2a 2u 1a 1u" << std::endl;
    memoryPool::cuda::dumpStat();

    {
      auto ph = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onHost);
      cudaCheck(memoryPool::cuda::copy(pd,ph,20,stream));
    }
    hipStreamSynchronize(stream);
    std::cout << "expect 2a 2u 2a 1u "    << std::endl;
    memoryPool::cuda::dumpStat();
   {
      auto ph = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onHost);
      cudaCheck(memoryPool::cuda::copy(pd,ph,20,stream));
    }
    std::cout << "expect 2a 2u 2a 1u "    << std::endl;
    hipStreamSynchronize(stream);
    memoryPool::cuda::dumpStat();
  }
  std::cout << "expect 2a 0u 2a 0u "    << std::endl;
   hipStreamSynchronize(stream);
   memoryPool::cuda::dumpStat();

  {
    memoryPool::Deleter devDeleter(std::make_shared<memoryPool::cuda::BundleDelete>(stream, memoryPool::onDevice));
    memoryPool::Deleter hosDeleter(std::make_shared<memoryPool::cuda::BundleDelete>(stream, memoryPool::onHost));

    auto p0 = memoryPool::cuda::make_buffer<int>(20, devDeleter);
    auto p1 = memoryPool::cuda::make_buffer<double>(20, devDeleter);
    auto p2 = memoryPool::cuda::make_buffer<bool>(20, devDeleter);
    auto p3 = memoryPool::cuda::make_buffer<int>(20, devDeleter);

    { 
      auto pd = memoryPool::cuda::make_buffer<int>(40, stream, memoryPool::onDevice);
      memoryPool::cuda::swapBuffer(p0,pd);
      memoryPool::cuda::dumpStat();
    }
    hipStreamSynchronize(stream);


    auto hp0 = memoryPool::cuda::make_buffer<int>(40, hosDeleter);
    auto hp1 = memoryPool::cuda::make_buffer<double>(20, hosDeleter);
    auto hp2 = memoryPool::cuda::make_buffer<bool>(20, hosDeleter);
    auto hp3 = memoryPool::cuda::make_buffer<int>(20, hosDeleter);

    cudaCheck(memoryPool::cuda::copy(hp3,p3,20,stream));
    cudaCheck(memoryPool::cuda::copy(p0,hp0,40,stream));;

    memoryPool::cuda::dumpStat();
  }

  hipStreamSynchronize(stream);
  memoryPool::cuda::dumpStat();

  return 0;
}
