#include "HeterogeneousCore/CUDAUtilities/interface/cudaMemoryPool.h"
#include <iostream>

template <memoryPool::Where where>
struct dataProducer {
  auto operator()(hipStream_t stream) { return memoryPool::cuda::make_buffer<int>(20, stream, where); }
};

int main() {
  {
    int devices = 0;
    auto status = hipGetDeviceCount(&devices);
    if (status != hipSuccess || 0 == devices)
      return 0;
    std::cout << "found " << devices << " cuda devices" << std::endl;
  }
  const int NUMTHREADS = 1;

  printf("Using CUDA %d\n", CUDART_VERSION);
  int cuda_device = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cuda_device);
  printf("CUDA Capable: SM %d.%d hardware\n", deviceProp.major, deviceProp.minor);

  hipStream_t streams[NUMTHREADS];

  for (int i = 0; i < NUMTHREADS; i++) {
    hipStreamCreate(&(streams[i]));
  }

  memoryPool::cuda::dumpStat();

  auto& stream = streams[0];

  {
    auto pd = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onDevice);
    auto ph = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onHost);
    auto pc = memoryPool::cuda::make_buffer<int>(20, stream, memoryPool::onCPU);

    auto dp = dataProducer<memoryPool::onDevice>()(stream);

    memoryPool::cuda::copy(ph,pd,20,stream);
    memoryPool::cuda::copy(pd,ph,20,stream);
    memoryPool::cuda::dumpStat();
  }

  {
    memoryPool::Deleter devDeleter(std::make_shared<memoryPool::cuda::BundleDelete>(stream, memoryPool::onDevice));
    memoryPool::Deleter hosDeleter(std::make_shared<memoryPool::cuda::BundleDelete>(stream, memoryPool::onHost));

    auto p0 = memoryPool::cuda::make_buffer<int>(20, devDeleter);
    auto p1 = memoryPool::cuda::make_buffer<double>(20, devDeleter);
    auto p2 = memoryPool::cuda::make_buffer<bool>(20, devDeleter);
    auto p3 = memoryPool::cuda::make_buffer<int>(20, devDeleter);

    auto hp0 = memoryPool::cuda::make_buffer<int>(20, hosDeleter);
    auto hp1 = memoryPool::cuda::make_buffer<double>(20, hosDeleter);
    auto hp2 = memoryPool::cuda::make_buffer<bool>(20, hosDeleter);
    auto hp3 = memoryPool::cuda::make_buffer<int>(20, hosDeleter);

    memoryPool::cuda::dumpStat();
  }

  hipStreamSynchronize(stream);
  memoryPool::cuda::dumpStat();

  return 0;
}
