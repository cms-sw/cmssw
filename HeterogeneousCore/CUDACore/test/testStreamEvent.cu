#include "hip/hip_runtime.h"
/**
 * The purpose of this test program is to ensure that the logic for
 * CUDA event use in cms::cuda::Product and cms::cuda::ScopedContext
 */

#include <iostream>
#include <memory>
#include <type_traits>
#include <chrono>
#include <thread>
#include <cassert>

#include <hip/hip_runtime.h>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

namespace {
  constexpr int ARRAY_SIZE = 20000000;
  constexpr int NLOOPS = 10;
}  // namespace

__global__ void kernel_looping(float *point, unsigned int num) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  for (int iloop = 0; iloop < NLOOPS; ++iloop) {
    for (size_t offset = idx; offset < num; offset += gridDim.x * blockDim.x) {
      point[offset] += 1;
    }
  }
}

int main() {
  cms::cudatest::requireDevices();

  constexpr bool debug = false;

  float *dev_points1;
  float *host_points1;
  hipStream_t stream1, stream2;
  hipEvent_t event1, event2;

  cudaCheck(hipMalloc(&dev_points1, ARRAY_SIZE * sizeof(float)));
  cudaCheck(hipHostMalloc(&host_points1, ARRAY_SIZE * sizeof(float)));
  hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
  hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
  hipEventCreate(&event1);
  hipEventCreate(&event2);

  for (size_t j = 0; j < ARRAY_SIZE; ++j) {
    host_points1[j] = static_cast<float>(j);
  }

  cudaCheck(hipMemcpyAsync(dev_points1, host_points1, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice, stream1));
  kernel_looping<<<1, 16, 0, stream1>>>(dev_points1, ARRAY_SIZE);
  if (debug)
    std::cout << "Kernel launched on stream1" << std::endl;

  auto status = hipStreamQuery(stream1);
  if (debug)
    std::cout << "Stream1 busy? " << (status == hipErrorNotReady) << " idle? " << (status == hipSuccess) << std::endl;
  hipEventRecord(event1, stream1);
  status = hipEventQuery(event1);
  if (debug)
    std::cout << "Event1 recorded? " << (status == hipErrorNotReady) << " occurred? " << (status == hipSuccess)
              << std::endl;
  assert(status == hipErrorNotReady);

  status = hipStreamQuery(stream2);
  if (debug)
    std::cout << "Stream2 busy? " << (status == hipErrorNotReady) << " idle? " << (status == hipSuccess) << std::endl;
  assert(status == hipSuccess);
  if (debug) {
    hipEventRecord(event2, stream2);
    status = hipEventQuery(event2);
    std::cout << "Event2 recorded? " << (status == hipErrorNotReady) << " occurred? " << (status == hipSuccess)
              << std::endl;
    std::this_thread::sleep_for(std::chrono::milliseconds(1));
    status = hipEventQuery(event2);
    std::cout << "Event2 recorded? " << (status == hipErrorNotReady) << " occurred? " << (status == hipSuccess)
              << std::endl;
  }

  hipStreamWaitEvent(stream2, event1, 0);
  if (debug)
    std::cout << "\nStream2 waiting for event1" << std::endl;
  status = hipStreamQuery(stream2);
  if (debug)
    std::cout << "Stream2 busy? " << (status == hipErrorNotReady) << " idle? " << (status == hipSuccess) << std::endl;
  assert(status == hipErrorNotReady);
  hipEventRecord(event2, stream2);
  status = hipEventQuery(event2);
  if (debug)
    std::cout << "Event2 recorded? " << (status == hipErrorNotReady) << " occurred? " << (status == hipSuccess)
              << std::endl;
  assert(status == hipErrorNotReady);
  if (debug) {
    std::this_thread::sleep_for(std::chrono::milliseconds(1));
    status = hipEventQuery(event2);
    std::cout << "Event2 recorded? " << (status == hipErrorNotReady) << " occurred? " << (status == hipSuccess)
              << std::endl;
  }

  status = hipStreamQuery(stream1);
  if (debug) {
    std::cout << "\nStream1 busy? " << (status == hipErrorNotReady) << " idle? " << (status == hipSuccess)
              << std::endl;
    std::cout << "Synchronizing stream1" << std::endl;
  }
  assert(status == hipErrorNotReady);
  hipStreamSynchronize(stream1);
  if (debug)
    std::cout << "Synchronized stream1" << std::endl;

  status = hipEventQuery(event1);
  if (debug)
    std::cout << "Event1 recorded? " << (status == hipErrorNotReady) << " occurred? " << (status == hipSuccess)
              << std::endl;
  assert(status == hipSuccess);
  status = hipEventQuery(event2);
  if (debug)
    std::cout << "Event2 recorded? " << (status == hipErrorNotReady) << " occurred? " << (status == hipSuccess)
              << std::endl;
  assert(status == hipSuccess);

  hipFree(dev_points1);
  hipHostFree(host_points1);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipEventDestroy(event1);
  hipEventDestroy(event2);

  return 0;
}
