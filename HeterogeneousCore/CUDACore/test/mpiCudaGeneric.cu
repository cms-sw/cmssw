#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <string>
#include <algorithm>
#include <vector>
#include <random>
#include <utility>
#include <mpi.h>
#include <unistd.h>
//////////////////////////////////////////// C U D A  /////////////////////////////////////////
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"

//called in the Host and excuted in the Device (GPU)
__global__ void addVectorsGpu(float *vect1, float *vect2, float *vect3, int size, int taskN) {
  //blockDim.x gives the number of threads in a block, in the x direction.
  //gridDim.x gives the number of blocks in a grid, in the x direction.
  //blockDim.x * gridDim.x gives the number of threads in a grid (in the x direction, in this case).
  int first = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = 0; i < taskN; ++i) {
    for (int j = first; j < size; j += stride) {
      vect3[j] = vect2[j] + vect1[j];
    }
  }
}  //add two vectors and save the result into the third vector.
//////////////////////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////  Global Varaibles  /////////////////////////////////////
unsigned int sizeVector = 2000;
int average = 5;
int task = 1;
int partsToRun = 1;
bool printStander = false;
bool saveFile = false;
bool help = false;
//////////////////////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////  Data Structure  /////////////////////////////////////
//Data For User's Choises Only
struct UserChoises {
  // unsigned int sizeVector;
  unsigned int sizeVectorBytes;  //Defualt vectors element float
  // unsigned int average;
  unsigned int extra;
  // unsigned int task;
  // unsigned int partsToRun;
  int root;
  // int numberProcess;
  int averageVectorSend;
  std::vector<int> partsToRunVector;  //vector for user's choice of part.
};

//Data For MPI Only
struct MPIData {
  int num_procs;
  int rank;

  std::pair<int, int> workSplit;
  float *mVect1;                  //declare vector 1.
  float *mVect2;                  //declare vector 2.
  float *mVect3;                  //declare vector fulled only by root to get result from workers.
  float *mVectChecking;           //declare vector to verify the results from each process.
  float *mVectWorker1;            //declare vector 1 for workers only.
  float *mVectWorker2;            //declare vector 2 for workers only.
  float *mVectWorker3;            //declare vector 2 for workers only.
  std::vector<int> displacement;  //declare vector for selecting location of each element to be sent.
  std::vector<int> numberToSend;
};

//Data For Cuda Only
struct Pointers {
  float *vect1;  //pointers only for Host
  float *vect2;
  float *vect3;

  float *dVect1;  //pointers only for device
  float *dVect2;
  float *dVect3;

  float *dVect1Extra;  //pointers only for device
  float *dVect2Extra;
  float *dVect3Extra;
};

//Data for Time Measurements Only
struct Timing {
  int partChosen;
  int unitChoice;
  double inputPreparationRoot[2];     // get time points from start and end on Root Side
  double inputPreparationHost[2];     // get time points from start and end on Host Side.
  double operationOnDeviceByHost[2];  //get time duration in Device with Host perspective.

  double outputPreparationRoot[2];
  double outputPreparationHost[2];

  std::vector<double> timeInputPreparationRoot;  //Save the Duration time.
  std::vector<double> timeInputPreparationHost;
  std::vector<double> timeOperationOnDeviceByRootHost;
  std::vector<double> timeOutputPreparationRoot;
  std::vector<double> timeOutputPreparationHost;

  hipEvent_t start, stop;                          //get time points in Device.
  float operationOnDeviceByDevice = 0;              //get time duration in Device with device perspective.
  std::vector<float> operationOnDeviceByDeviceAcc;  //get accumulating time duration in Device with device perspective.
  std::vector<float> averageResults;  ///declare vector for getting average calcualtion for Hosts and device to Root.
};

//////////////////////////////////////////////////////////////////////////////////////////////////
const std::vector<int> chooseFunction(int toInteger);
std::pair<int, int> splitProcess(int works, int numberOfProcess);
const std::vector<int> numberDataSend(int numberOfProcess, std::pair<int, int> splitWorks);
void setupMPIAndVectors(
    MPIData &mpiData,
    UserChoises &user);  //initialize communicator environment for MPI and Resize Vectors with Generating Random numbers.
void setupTime(Timing &timing, UserChoises &user);  //Resizing Vectors of Time.
void calculateTimeDuration(Timing &timing, int i, int &root);
void addVectorsHost(float *vect1, float *vect2, float *vect3);
void cleanBuffer(float *vect);
bool checkingResultsPrintout(float *vectCpu, float *vectGpu);
void calculateAverageDeviation(Timing &timing, int averg, int &root);
bool sendAverageToRoot(Timing &timing, UserChoises &user, int &rank);

Timing blockSendPart1(MPIData &mpidata, Timing &timing, Pointers &pointer, UserChoises &user);
Timing blockSendPart2(MPIData &mpiData, Timing &timing, Pointers &pointer, UserChoises &user);
Timing blockSendPart3(MPIData &mpiData, Timing &timing, Pointers &pointer, UserChoises &user);

void printTable(std::vector<Timing> &timing, bool standerDeviationPrint);
int getNumberofDigits(double number);
void newLineTitle(int line, const std::string &title);
void printResultEach(std::vector<Timing> &timing, int type, bool standerDeviationPrint);
bool saveToFile(const std::string &name, const Timing &timing);

void printHelp(void);
int main(int argc, char *argv[]) {
  cms::cudatest::requireDevices();
  int c;  //to get parameters from user.

  UserChoises user;  //Setup Uuser's input variables
  user.extra = 2;
  user.root = 0;
  user.averageVectorSend = 8;

  while ((c = getopt(argc, argv, "s:a:t:p:qfh")) != -1) {
    switch (c) {
      case 's':
        try {
          sizeVector = std::stoll(optarg, nullptr, 0);
        } catch (std::exception &err) {
          std::cout << "\n\tError Must be integer Argument!";
          std::cout << "\n\t" << err.what() << std::endl;
          return 0;
        }
        break;
      case 'a':
        try {
          average = std::stoll(optarg, nullptr, 0);

        } catch (std::exception &err) {
          std::cout << "\n\tError Must be integer Argument!";
          std::cout << "\n\t" << err.what() << std::endl;
          return 0;
        }
        break;
      case 't':
        try {
          task = std::stoll(optarg, nullptr, 0);
          //std::cout << "\nNumber of repeated Task is " << task << std::endl;
        } catch (std::exception &err) {
          std::cout << "\n\tError Must be integer Argument!";
          std::cout << "\n\t" << err.what() << std::endl;
          return 0;
        }
        break;
      case 'p':
        try {
          partsToRun = std::stoll(optarg, nullptr, 0);
          user.partsToRunVector = chooseFunction(partsToRun);
          //std::cout << "\nyou have chosen Part ";
          for (unsigned int j = 0; j < user.partsToRunVector.size(); ++j) {
            std::cout << user.partsToRunVector[j] << " ,";
          }
          std::cout << "\n";
        } catch (std::exception &err) {
          std::cout << "\n\tError Must be integer Argument!";
          std::cout << "\n\t" << err.what() << std::endl;
          return 0;
        }
        break;
      case 'q':
        try {
          printStander = true;
        } catch (std::exception &err) {
          std::cout << "\n\tError Must be integer Argument!";
          std::cout << "\n\t" << err.what() << std::endl;
          return 0;
        }
        break;
      case 'f':
        try {
          saveFile = true;
        } catch (std::exception &err) {
          std::cout << "\n\tError Must be integer Argument!";
          std::cout << "\n\t" << err.what() << std::endl;
          return 0;
        }
        break;
      case 'h':
        try {
          help = true;
        } catch (std::exception &err) {
          std::cout << "\n\tError Must be integer Argument!";
          std::cout << "\n\t" << err.what() << std::endl;
          return 0;
        }
        break;

      default:
        abort();
    }
  }

  MPIData mpiData;
  Timing timing;
  Timing resetTime;
  Pointers pointer;
  timing.unitChoice = 1000000;     //1M
  resetTime.unitChoice = 1000000;  //1M

  std::vector<Timing> allTiming;
  allTiming.resize(user.partsToRunVector.size());

  MPI_Init(&argc, &argv);  //initialize communicator environment.

  if (help) {
    printHelp();
    MPI::Finalize();
    exit(0);
  }
  setupMPIAndVectors(mpiData, user);

  setupTime(timing, user);
  setupTime(resetTime, user);

  for (long unsigned int i = 0; i < user.partsToRunVector.size(); ++i) {
    if (user.partsToRunVector[i] == 1) {
      //setupTime(allTiming[i], user);
      //blockSendPart1(mpiData, allTiming[i], pointer, user);
      allTiming[i] = blockSendPart1(mpiData, timing, pointer, user);
      timing = resetTime;

    } else if (user.partsToRunVector[i] == 2) {
      //setupTime(allTiming[i], user);
      //blockSendPart2(mpiData, allTiming[i], pointer, user);
      allTiming[i] = blockSendPart2(mpiData, timing, pointer, user);
      timing = resetTime;

    } else if (user.partsToRunVector[i] == 3) {
      allTiming[i] = blockSendPart3(mpiData, timing, pointer, user);
      timing = resetTime;
      // } else if (user.partsToRunVector[i] == 4) {
      //   allTiming[i] = cudaTimePart4(timing, vect, dvect, size);

      // } else if (user.partsToRunVector[i] == 5) {
      //   allTiming[i] = cudaTimePart5(timing, vect, dvect, size);

    } else {
      std::cout << "\n\n\tError the User has not chose any number of Function!\n";
      break;
    }
  }

  if (!mpiData.rank)
    printTable(allTiming, printStander);

  MPI::Finalize();
  return 0;
}
const std::vector<int> chooseFunction(int toInteger) {
  std::vector<int> digits(0, 0);
  std::vector<int> ERROR(0, 0);

  int digit{1};

  while (toInteger > 0) {
    digit = toInteger % 10;
    if (digit > 7) {
      std::cout << "\n\tError Must be integer Argument <= " << toInteger << std::endl;
      return ERROR;
    }
    digits.push_back(digit);
    toInteger /= 10;
  }
  std::reverse(digits.begin(), digits.end());
  return digits;
}

std::pair<int, int> splitProcess(int works, int numberOfProcess) {
  std::pair<int, int> Return{0, 0};
  if (numberOfProcess > 1 && numberOfProcess <= works) {
    Return.first = works / (numberOfProcess - 1);   //number of cycle for each process.
    Return.second = works % (numberOfProcess - 1);  //extra cycle for process.
  } else {
    std::cout << "\tError Either No worker are found OR Number Processes Larger than Length!!!\n";
  }

  return Return;
}
const std::vector<int> numberDataSend(int numberOfProcess, std::pair<int, int> splitWorks) {
  std::vector<int> dataSend(numberOfProcess, splitWorks.first);
  dataSend[0] = 0;
  for (int i = 1; i < splitWorks.second + 1; i++)  //neglect root
  {
    dataSend[i] += 1;  //extra work for each first processes.
  }
  return dataSend;
}
const std::vector<int> displacmentData(int numberOfProcess,
                                       std::pair<int, int> splitWorks,
                                       const std::vector<int> &numberDataSend) {
  std::vector<int> displacment(numberOfProcess, splitWorks.first);

  displacment[0] = 0;
  displacment[1] = 0;  //start Here.

  for (int i = 2; i < numberOfProcess; i++)  //neglect root
  {
    displacment[i] = numberDataSend[i - 1] + displacment[i - 1];  //extra work for each first processes.
  }
  return displacment;
}
void randomGenerator(float *vect) {
  std::random_device rand;
  std::default_random_engine gener(rand());
  std::uniform_real_distribution<> dis(0., 1.);
  for (unsigned int i = 0; i < sizeVector; ++i) {
    vect[i] = dis(gener);
  }
}
void setupMPIAndVectors(MPIData &mpiData, UserChoises &user) {
  mpiData.num_procs = MPI::COMM_WORLD.Get_size();  //get total size of processes.
  mpiData.rank = MPI::COMM_WORLD.Get_rank();       //get each process number.

  user.sizeVectorBytes = sizeVector * sizeof(float);  //get size in byte for vectors.

  mpiData.mVect1 = (float *)malloc(user.sizeVectorBytes);  //initialize size.
  mpiData.mVect2 = (float *)malloc(user.sizeVectorBytes);
  mpiData.mVect3 = (float *)malloc(user.sizeVectorBytes);
  mpiData.mVectChecking = (float *)malloc(user.sizeVectorBytes);

  //mpiData.mVectWorker1 = (float*) malloc(user.sizeVectorBytes);
  //mpiData.mVectWorker2 = (float*) malloc(user.sizeVectorBytes);
  mpiData.mVectWorker3 = (float *)malloc(user.sizeVectorBytes);

  mpiData.workSplit = splitProcess(sizeVector, mpiData.num_procs);

  if (!mpiData.workSplit.first) {
    MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    exit(-1);
  }

  mpiData.numberToSend = numberDataSend(mpiData.num_procs, mpiData.workSplit);
  mpiData.displacement = displacmentData(mpiData.num_procs, mpiData.workSplit, mpiData.numberToSend);

  // mpiData.mVectWorker1.resize(mpiData.numberToSend[mpiData.rank]);  //Resizing each process with appropriate Receiving Data.
  // mpiData.mVectWorker2.resize(mpiData.numberToSend[mpiData.rank]);
  mpiData.mVectWorker1 = (float *)malloc(mpiData.numberToSend[mpiData.rank] * sizeof(float));
  mpiData.mVectWorker2 = (float *)malloc(mpiData.numberToSend[mpiData.rank] * sizeof(float));

  if (!mpiData.rank)  //Only for root
  {
    randomGenerator(mpiData.mVect1);  //generate random floating numbers from(0,1) Only in the root.
    randomGenerator(mpiData.mVect2);
    std::cout << "\n\tNumber of Processes " << mpiData.num_procs << std::endl;
    std::cout << "\tNumber of workSplit First " << mpiData.workSplit.first << std::endl;
    std::cout << "\tNumber of workSplit Second " << mpiData.workSplit.second << std::endl;
    std::cout << "\tTotal size of a Buffer " << user.sizeVectorBytes << " B" << std::endl;
  }
}
void setupTime(Timing &timing, UserChoises &user) {
  //Setup Verctors for Taking Average and Standard deviation
  timing.timeInputPreparationRoot.resize(average + user.extra);  //extra for saving the average.
  timing.timeInputPreparationHost.resize(average + user.extra);
  timing.timeOperationOnDeviceByRootHost.resize(average + user.extra);
  timing.timeOutputPreparationRoot.resize(average + user.extra);
  timing.timeOutputPreparationHost.resize(average + user.extra);
  timing.operationOnDeviceByDeviceAcc.resize(average + user.extra);
  timing.averageResults.resize(user.averageVectorSend);
}

void calculateTimeDuration(Timing &timing, int i, int &root) {
  if (!root) {
    timing.timeInputPreparationRoot[i] =
        (timing.inputPreparationRoot[1] - timing.inputPreparationRoot[0]);  //getting the time in microseconds
    timing.timeOperationOnDeviceByRootHost[i] = (timing.outputPreparationRoot[0] - timing.inputPreparationRoot[1]);
    timing.timeOutputPreparationRoot[i] = (timing.outputPreparationRoot[1] - timing.outputPreparationRoot[0]);
  } else {
    timing.timeInputPreparationHost[i] = (timing.inputPreparationHost[1] - timing.inputPreparationHost[0]);
    timing.timeOperationOnDeviceByRootHost[i] =
        (timing.operationOnDeviceByHost[1] -
         timing.operationOnDeviceByHost[0]);  //time taking for Device operation with respect of Host.
    hipEventElapsedTime(&timing.operationOnDeviceByDevice,
                         timing.start,
                         timing.stop);  //get the time elapse in Device operation with device perspective.
    timing.operationOnDeviceByDeviceAcc[i] = (timing.operationOnDeviceByDevice * 1000);
    timing.timeOutputPreparationHost[i] = (timing.outputPreparationHost[1] - timing.outputPreparationHost[0]);
  }
}
void addVectorsHost(float *vect1, float *vect2, float *vect3) {
  for (unsigned int i = 0; i < sizeVector; ++i) {
    vect3[i] = vect2[i] + vect1[i];
  }
}
void cleanBuffer(float *vect) {
  for (unsigned int i = 0; i < sizeVector; ++i) {
    vect[i] = 0;
  }
}
bool checkingResultsPrintout(float *vectCpu, float *vectGpu) {
  float percent{0.0};
  float totalError{0.0};

  for (unsigned int j = 0; j < sizeVector; j++) {
    percent = ((vectCpu[j] - vectGpu[j]) / vectCpu[j]) * 100;
    totalError += percent;
  }
  if (totalError) {
    std::cout << "\n------------------------------------\n";
    std::cout << "| CpuSum | GpuSum | Error  | Error %| ";
    std::cout << "\n------------------------------------\n";
    //std::cout.precision(4);
    for (unsigned int j = 0; j < sizeVector; j++) {
      std::cout.flags(std::ios::fixed | std::ios::showpoint);
      std::cout.precision(4);
      std::cout << "| " << vectCpu[j] << " | " << vectGpu[j] << " | " << vectCpu[j] - vectGpu[j] << " | " << percent
                << " |\n";
    }
    std::cout << "-------------------------------------\n";
    std::cout << "-Total Error is " << totalError << std::endl;
    return false;
  }
  return true;
}
void calculateAverageDeviation(Timing &timing, int averg, int &root) {
  //Average
  for (int i = 0; i < averg; ++i) {
    if (!root) {
      timing.timeInputPreparationRoot[averg] += timing.timeInputPreparationRoot[i];
      timing.timeOperationOnDeviceByRootHost[averg] += timing.timeOperationOnDeviceByRootHost[i];
      timing.timeOutputPreparationRoot[averg] += timing.timeOutputPreparationRoot[i];
    } else {
      timing.timeInputPreparationHost[averg] += timing.timeInputPreparationHost[i];
      timing.timeOperationOnDeviceByRootHost[averg] += timing.timeOperationOnDeviceByRootHost[i];
      timing.timeOutputPreparationHost[averg] += timing.timeOutputPreparationHost[i];
      timing.operationOnDeviceByDeviceAcc[averg] += timing.operationOnDeviceByDeviceAcc[i];
    }
  }
  if (!root) {
    timing.timeInputPreparationRoot[averg] = timing.timeInputPreparationRoot[averg] / averg;
    timing.timeOperationOnDeviceByRootHost[averg] = timing.timeOperationOnDeviceByRootHost[averg] / averg;

    timing.timeOutputPreparationRoot[averg] = timing.timeOutputPreparationRoot[averg] / averg;

  } else {
    timing.timeInputPreparationHost[averg] = timing.timeInputPreparationHost[averg] / averg;

    timing.timeOperationOnDeviceByRootHost[averg] = timing.timeOperationOnDeviceByRootHost[averg] / averg;

    timing.timeOutputPreparationHost[averg] = timing.timeOutputPreparationHost[averg] / averg;

    timing.operationOnDeviceByDeviceAcc[averg] = (double)timing.operationOnDeviceByDeviceAcc[averg] / averg;
  }

  //Standard deviation
  for (int i = 0; i < averg; ++i) {
    if (!root) {
      timing.timeInputPreparationRoot[i] -= timing.timeInputPreparationRoot[averg];  //Take the different.
      timing.timeInputPreparationRoot[i] =
          timing.timeInputPreparationRoot[i] * timing.timeInputPreparationRoot[i];  // Square it.
      timing.timeInputPreparationRoot[averg + 1] +=
          timing.timeInputPreparationRoot[i];  //add them togather. averg+1 is location of the Deviation

      timing.timeOperationOnDeviceByRootHost[i] -= timing.timeOperationOnDeviceByRootHost[averg];
      timing.timeOperationOnDeviceByRootHost[i] *= timing.timeOperationOnDeviceByRootHost[i];
      timing.timeOperationOnDeviceByRootHost[averg + 1] += timing.timeOperationOnDeviceByRootHost[i];

      timing.timeOutputPreparationRoot[i] -= timing.timeOutputPreparationRoot[averg];
      timing.timeOutputPreparationRoot[i] *= timing.timeOutputPreparationRoot[i];
      timing.timeOutputPreparationRoot[averg + 1] += timing.timeOutputPreparationRoot[i];
    } else {
      timing.timeInputPreparationHost[i] -= timing.timeInputPreparationHost[averg];  //Take the different.
      timing.timeInputPreparationHost[i] =
          timing.timeInputPreparationHost[i] * timing.timeInputPreparationHost[i];  // Square it.
      timing.timeInputPreparationHost[averg + 1] +=
          timing.timeInputPreparationHost[i];  //add them togather. averg+1 is location of the Deviation

      timing.timeOperationOnDeviceByRootHost[i] -= timing.timeOperationOnDeviceByRootHost[averg];
      timing.timeOperationOnDeviceByRootHost[i] *= timing.timeOperationOnDeviceByRootHost[i];
      timing.timeOperationOnDeviceByRootHost[averg + 1] += timing.timeOperationOnDeviceByRootHost[i];

      timing.timeOutputPreparationHost[i] -= timing.timeOutputPreparationHost[averg];
      timing.timeOutputPreparationHost[i] *= timing.timeOutputPreparationHost[i];
      timing.timeOutputPreparationHost[averg + 1] += timing.timeOutputPreparationHost[i];

      timing.operationOnDeviceByDeviceAcc[i] -= timing.operationOnDeviceByDeviceAcc[averg];
      timing.operationOnDeviceByDeviceAcc[i] *= timing.operationOnDeviceByDeviceAcc[i];
      timing.operationOnDeviceByDeviceAcc[averg + 1] += timing.operationOnDeviceByDeviceAcc[i];
    }
  }

  if (!root) {
    timing.timeInputPreparationRoot[averg + 1] = timing.timeInputPreparationRoot[averg + 1] / averg;
    timing.timeInputPreparationRoot[averg + 1] = sqrt(timing.timeInputPreparationRoot[averg + 1]);

    timing.timeOperationOnDeviceByRootHost[averg + 1] = timing.timeOperationOnDeviceByRootHost[averg + 1] / averg;
    timing.timeOperationOnDeviceByRootHost[averg + 1] = sqrt(timing.timeOperationOnDeviceByRootHost[averg + 1]);

    timing.timeOutputPreparationRoot[averg + 1] = timing.timeOutputPreparationRoot[averg + 1] / averg;
    timing.timeOutputPreparationRoot[averg + 1] = sqrt(timing.timeOutputPreparationRoot[averg + 1]);

  } else {
    timing.timeInputPreparationHost[averg + 1] = timing.timeInputPreparationHost[averg + 1] / averg;  //*1000000
    timing.timeInputPreparationHost[averg + 1] = sqrt(timing.timeInputPreparationHost[averg + 1]);

    timing.timeOperationOnDeviceByRootHost[averg + 1] = timing.timeOperationOnDeviceByRootHost[averg + 1] / averg;
    timing.timeOperationOnDeviceByRootHost[averg + 1] = sqrt(timing.timeOperationOnDeviceByRootHost[averg + 1]);

    timing.timeOutputPreparationHost[averg + 1] = timing.timeOutputPreparationHost[averg + 1] / averg;
    timing.timeOutputPreparationHost[averg + 1] = sqrt(timing.timeOutputPreparationHost[averg + 1]);

    timing.operationOnDeviceByDeviceAcc[averg + 1] = (double)timing.operationOnDeviceByDeviceAcc[averg + 1] / averg;
    timing.operationOnDeviceByDeviceAcc[averg + 1] = sqrt(timing.operationOnDeviceByDeviceAcc[averg + 1]);
  }

  if (!root) {
    timing.timeInputPreparationRoot[averg] *= timing.unitChoice;
    timing.timeOperationOnDeviceByRootHost[averg] *= timing.unitChoice;
    timing.timeOutputPreparationRoot[averg] *= timing.unitChoice;

    timing.timeInputPreparationRoot[averg + 1] *= timing.unitChoice;
    timing.timeOperationOnDeviceByRootHost[averg + 1] *= timing.unitChoice;
    timing.timeOutputPreparationRoot[averg + 1] *= timing.unitChoice;
  } else {
    timing.timeInputPreparationHost[averg] *= timing.unitChoice;
    timing.timeOperationOnDeviceByRootHost[averg] *= timing.unitChoice;
    timing.timeOutputPreparationHost[averg] *= timing.unitChoice;

    timing.timeInputPreparationHost[averg + 1] *= timing.unitChoice;
    timing.timeOperationOnDeviceByRootHost[averg + 1] *= timing.unitChoice;
    timing.timeOutputPreparationHost[averg + 1] *= timing.unitChoice;
  }
}

bool sendAverageToRoot(Timing &timing, UserChoises &user, int &rank) {
  if (rank) {
    timing.averageResults[0] = timing.timeInputPreparationHost[average];
    timing.averageResults[1] = timing.timeInputPreparationHost[average + 1];  //Stander Deviation

    timing.averageResults[2] = timing.timeOperationOnDeviceByRootHost[average];
    timing.averageResults[3] = timing.timeOperationOnDeviceByRootHost[average + 1];

    timing.averageResults[4] = timing.timeOutputPreparationHost[average];
    timing.averageResults[5] = timing.timeOutputPreparationHost[average + 1];

    timing.averageResults[6] = timing.operationOnDeviceByDeviceAcc[average];
    timing.averageResults[7] = timing.operationOnDeviceByDeviceAcc[average + 1];

    MPI_Send(&timing.averageResults[0], user.averageVectorSend, MPI_FLOAT, user.root, 0, MPI_COMM_WORLD);

  } else if (!rank) {
    MPI_Recv(&timing.averageResults[0], user.averageVectorSend, MPI_FLOAT, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
  }
  return true;
}

Timing blockSendPart1(MPIData &mpiData, Timing &timing, Pointers &pointer, UserChoises &user) {
  cleanBuffer(mpiData.mVectWorker3);  //clear each value of vector's elements
  timing.partChosen = 1;

  if (mpiData.rank)  //Only for Workers
  {
    cudaCheck(hipMalloc((void **)&pointer.dVect1,
                         user.sizeVectorBytes));  //allocate memory space for vector in the global memory of the Device.
    cudaCheck(hipMalloc((void **)&pointer.dVect2, user.sizeVectorBytes));
    cudaCheck(hipMalloc((void **)&pointer.dVect3, user.sizeVectorBytes));
  }
  ///////////////////////////// Start of Average ////////////////////////
  for (int a = 0; a <= average; ++a) {
    if (!mpiData.rank)  //Only for root
    {
      ////////////////////////////////// Input Prepation for Root //////////////////////////////////
      timing.inputPreparationRoot[0] = MPI_Wtime();
      for (int i = 1; i < mpiData.num_procs; ++i) {
        MPI_Send(&mpiData.mVect1[mpiData.displacement[i]],
                 mpiData.numberToSend[i],
                 MPI_FLOAT,
                 i,
                 0,
                 MPI_COMM_WORLD);  //Tag is 0
        MPI_Send(&mpiData.mVect2[mpiData.displacement[i]], mpiData.numberToSend[i], MPI_FLOAT, i, 0, MPI_COMM_WORLD);
      }
      timing.inputPreparationRoot[1] = MPI_Wtime();
      /////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (mpiData.rank)  //Only for Workers
    {
      ////////////////////////////////// Input Prepation for Host //////////////////////////////////
      MPI_Probe(user.root, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      timing.inputPreparationHost[0] = MPI_Wtime();
      MPI_Recv(&mpiData.mVectWorker1[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD,
               MPI_STATUS_IGNORE);
      MPI_Recv(&mpiData.mVectWorker2[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD,
               MPI_STATUS_IGNORE);

      cudaCheck(hipMemcpy(pointer.dVect1,
                           mpiData.mVectWorker1,
                           user.sizeVectorBytes,
                           hipMemcpyHostToDevice));  //copy random vector from host to device.
      cudaCheck(hipMemcpy(pointer.dVect2, mpiData.mVectWorker2, user.sizeVectorBytes, hipMemcpyHostToDevice));

      timing.inputPreparationHost[1] = MPI_Wtime();
      ///////////////////////////////////////////////////////////////////////////////////////

      cudaCheck(hipEventCreate(&timing.start));  //inialize Event.
      cudaCheck(hipEventCreate(&timing.stop));

      ///////////////////////////// Operation on Device with respect of Host //////////////////

      int threads = 512;                                  //arbitrary number.
      int blocks = (sizeVector + threads - 1) / threads;  //get ceiling number of blocks.
      blocks = std::min(blocks, 8);  // Number 8 is least number can be got from lowest Nevedia GPUs.

      ////////////////////////// CAll Device Kernel //////////////////////////////////
      cudaCheck(hipEventRecord(timing.start));
      timing.operationOnDeviceByHost[0] = MPI_Wtime();

      addVectorsGpu<<<blocks, threads>>>(pointer.dVect1,
                                         pointer.dVect2,
                                         pointer.dVect3,
                                         sizeVector,
                                         task);  //call device function to add two vectors and save into vect3Gpu.

      cudaCheck(hipGetLastError());
      cudaCheck(hipDeviceSynchronize());
      cudaCheck(hipEventRecord(timing.stop));

      timing.operationOnDeviceByHost[1] = MPI_Wtime();
      /////////////////////////////////////////////////////////////////////////////////////////////

      /////////////////////////////////// Output Prepation for the Host //////////////////////////////////////
      timing.outputPreparationHost[0] = MPI_Wtime();
      cudaCheck(hipMemcpy(
          mpiData.mVectWorker3,
          pointer.dVect3,
          user.sizeVectorBytes,
          hipMemcpyDeviceToHost));  //copy summing result vector from Device to Host.// Try_Regist(3) delete this

      MPI_Send(&mpiData.mVectWorker3[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD);  //Tag is 0
      timing.outputPreparationHost[1] = MPI_Wtime();
      ////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (!mpiData.rank)  //Only for root
    {
      /////////////////////////////////// Output Prepation for the Root //////////////////////////////////////
      MPI_Probe(MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      timing.outputPreparationRoot[0] = MPI_Wtime();
      //MPI probe
      for (int i = 1; i < mpiData.num_procs; i++) {
        MPI_Recv(&mpiData.mVectWorker3[mpiData.displacement[i]],
                 mpiData.numberToSend[i],
                 MPI_FLOAT,
                 i,
                 0,
                 MPI_COMM_WORLD,
                 MPI_STATUS_IGNORE);
      }
      timing.outputPreparationRoot[1] = MPI_Wtime();
      ////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (a > 0)
      calculateTimeDuration(timing, a - 1, mpiData.rank);

    if (mpiData.rank) {
      cudaCheck(hipEventDestroy(timing.start));
      cudaCheck(hipEventDestroy(timing.stop));
    }
  }
  ///////////////////////////// End of Average ////////////////////////
  if (mpiData.rank) {
    cudaCheck(hipFree(pointer.dVect1));
    cudaCheck(hipFree(pointer.dVect2));
    cudaCheck(hipFree(pointer.dVect3));
  }
  ///
  bool test = 0;
  if (!mpiData.rank)  //Only for root
  {
    addVectorsHost(mpiData.mVect1, mpiData.mVect2, mpiData.mVectChecking);  //Host is adding vectors too.
    test = checkingResultsPrintout(mpiData.mVectChecking,
                                   mpiData.mVectWorker3);  //Checking the results, if error then Print out to the user.
    if (!test)
      exit(-1);
  }

  calculateAverageDeviation(timing, average, mpiData.rank);
  test = sendAverageToRoot(timing, user, mpiData.rank);
  if (test && !mpiData.rank) {
    if (saveFile) {
      test = saveToFile("dataPart1", timing);

      if (test)
        std::cout << "Done Part " << timing.partChosen << " And File saved" << std::endl;
      else
        std::cout << "Error Saving File!!" << std::endl;
    }
    std::cout << "Done Part " << timing.partChosen << std::endl;
  }
  return timing;
}

Timing blockSendPart2(MPIData &mpiData, Timing &timing, Pointers &pointer, UserChoises &user) {
  cleanBuffer(mpiData.mVectWorker3);  //clear each value of vector's elements
  timing.partChosen = 2;

  if (mpiData.rank)  //Only for Workers
  {
    cudaCheck(hipHostMalloc((void **)&pointer.vect1, user.sizeVectorBytes));  //allocate Pinned memory on the Host.
    cudaCheck(hipHostMalloc((void **)&pointer.vect2, user.sizeVectorBytes));
    cudaCheck(hipHostMalloc((void **)&pointer.vect3, user.sizeVectorBytes));
    cudaCheck(hipMalloc((void **)&pointer.dVect1,
                         user.sizeVectorBytes));  //allocate memory space for vector in the global memory of the Device.
    cudaCheck(hipMalloc((void **)&pointer.dVect2, user.sizeVectorBytes));
    cudaCheck(hipMalloc((void **)&pointer.dVect3, user.sizeVectorBytes));
  }
  ///////////////////////////// Start of Average ////////////////////////
  for (int a = 0; a <= average; ++a) {
    if (!mpiData.rank)  //Only for root
    {
      ////////////////////////////////// Input Prepation for Root //////////////////////////////////
      timing.inputPreparationRoot[0] = MPI_Wtime();
      for (int i = 1; i < mpiData.num_procs; ++i) {
        MPI_Send(&mpiData.mVect1[mpiData.displacement[i]],
                 mpiData.numberToSend[i],
                 MPI_FLOAT,
                 i,
                 0,
                 MPI_COMM_WORLD);  //Tag is 0
        MPI_Send(&mpiData.mVect2[mpiData.displacement[i]], mpiData.numberToSend[i], MPI_FLOAT, i, 0, MPI_COMM_WORLD);
      }
      timing.inputPreparationRoot[1] = MPI_Wtime();
      /////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (mpiData.rank)  //Only for Workers
    {
      ////////////////////////////////// Input Prepation for Host //////////////////////////////////
      MPI_Probe(user.root, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      timing.inputPreparationHost[0] = MPI_Wtime();

      MPI_Recv(&pointer.vect1[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD,
               MPI_STATUS_IGNORE);
      MPI_Recv(&pointer.vect2[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD,
               MPI_STATUS_IGNORE);

      cudaCheck(hipMemcpy(pointer.dVect1,
                           pointer.vect1,
                           user.sizeVectorBytes,
                           hipMemcpyHostToDevice));  //copy random vector from host to device.
      cudaCheck(hipMemcpy(pointer.dVect2, pointer.vect2, user.sizeVectorBytes, hipMemcpyHostToDevice));

      timing.inputPreparationHost[1] = MPI_Wtime();
      ///////////////////////////////////////////////////////////////////////////////////////

      cudaCheck(hipEventCreate(&timing.start));  //inialize Event.
      cudaCheck(hipEventCreate(&timing.stop));

      ///////////////////////////// Operation on Device with respect of Host //////////////////

      int threads = 512;                                  //arbitrary number.
      int blocks = (sizeVector + threads - 1) / threads;  //get ceiling number of blocks.
      blocks = std::min(blocks, 8);  // Number 8 is least number can be got from lowest Nevedia GPUs.

      ////////////////////////// CAll Device Kernel //////////////////////////////////
      cudaCheck(hipEventRecord(timing.start));
      timing.operationOnDeviceByHost[0] = MPI_Wtime();

      addVectorsGpu<<<blocks, threads>>>(pointer.dVect1,
                                         pointer.dVect2,
                                         pointer.dVect3,
                                         sizeVector,
                                         task);  //call device function to add two vectors and save into vect3Gpu.

      cudaCheck(hipGetLastError());
      cudaCheck(hipDeviceSynchronize());
      cudaCheck(hipEventRecord(timing.stop));

      timing.operationOnDeviceByHost[1] = MPI_Wtime();
      /////////////////////////////////////////////////////////////////////////////////////////////

      /////////////////////////////////// Output Prepation for the Host //////////////////////////////////////
      timing.outputPreparationHost[0] = MPI_Wtime();

      cudaCheck(hipMemcpy(
          pointer.vect3,
          pointer.dVect3,
          user.sizeVectorBytes,
          hipMemcpyDeviceToHost));  //copy summing result vector from Device to Host.// Try_Regist(3) delete this

      MPI_Send(&pointer.vect3[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD);  //Tag is 0

      timing.outputPreparationHost[1] = MPI_Wtime();
      ////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (!mpiData.rank)  //Only for root
    {
      /////////////////////////////////// Output Prepation for the Root //////////////////////////////////////
      MPI_Probe(MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      timing.outputPreparationRoot[0] = MPI_Wtime();
      //MPI probe
      for (int i = 1; i < mpiData.num_procs; i++) {
        MPI_Recv(&mpiData.mVectWorker3[mpiData.displacement[i]],
                 mpiData.numberToSend[i],
                 MPI_FLOAT,
                 i,
                 0,
                 MPI_COMM_WORLD,
                 MPI_STATUS_IGNORE);
      }
      timing.outputPreparationRoot[1] = MPI_Wtime();
      ////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (a > 0)
      calculateTimeDuration(timing, a - 1, mpiData.rank);

    if (mpiData.rank) {
      cudaCheck(hipEventDestroy(timing.start));
      cudaCheck(hipEventDestroy(timing.stop));
    }
  }
  ///////////////////////////// End of Average ////////////////////////
  if (mpiData.rank) {
    cudaCheck(hipHostFree(pointer.vect1));
    cudaCheck(hipHostFree(pointer.vect2));
    cudaCheck(hipHostFree(pointer.vect3));
    cudaCheck(hipFree(pointer.dVect1));
    cudaCheck(hipFree(pointer.dVect2));
    cudaCheck(hipFree(pointer.dVect3));
  }

  bool test = 0;
  if (!mpiData.rank)  //Only for root
  {
    addVectorsHost(mpiData.mVect1, mpiData.mVect2, mpiData.mVectChecking);  //Host is adding vectors too.
    test = checkingResultsPrintout(mpiData.mVectChecking,
                                   mpiData.mVectWorker3);  //Checking the results, if error then Print out to the user.
    if (!test)
      exit(-1);
  }

  calculateAverageDeviation(timing, average, mpiData.rank);
  test = sendAverageToRoot(timing, user, mpiData.rank);
  if (test && !mpiData.rank) {
    if (saveFile) {
      test = saveToFile("dataPart2", timing);

      if (test)
        std::cout << "Done Part " << timing.partChosen << " And File saved" << std::endl;
      else
        std::cout << "Error Saving File!!" << std::endl;
    }
    std::cout << "Done Part " << timing.partChosen << std::endl;
  }
  return timing;
}

Timing blockSendPart3(MPIData &mpiData, Timing &timing, Pointers &pointer, UserChoises &user) {
  cleanBuffer(mpiData.mVectWorker3);  //clear each value of vector's elements
  timing.partChosen = 3;

  if (mpiData.rank)  //Only for Workers
  {
    cudaCheck(hipMalloc((void **)&pointer.dVect1,
                         user.sizeVectorBytes));  //allocate memory space for vector in the global memory of the Device.
    cudaCheck(hipMalloc((void **)&pointer.dVect2, user.sizeVectorBytes));
    cudaCheck(hipMalloc((void **)&pointer.dVect3, user.sizeVectorBytes));
  }
  ///////////////////////////// Start of Average ////////////////////////
  for (int a = 0; a <= average; ++a) {
    if (!mpiData.rank)  //Only for root
    {
      ////////////////////////////////// Input Prepation for Root //////////////////////////////////
      timing.inputPreparationRoot[0] = MPI_Wtime();
      for (int i = 1; i < mpiData.num_procs; ++i) {
        MPI_Send(&mpiData.mVect1[mpiData.displacement[i]],
                 mpiData.numberToSend[i],
                 MPI_FLOAT,
                 i,
                 0,
                 MPI_COMM_WORLD);  //Tag is 0
        MPI_Send(&mpiData.mVect2[mpiData.displacement[i]], mpiData.numberToSend[i], MPI_FLOAT, i, 0, MPI_COMM_WORLD);
      }
      timing.inputPreparationRoot[1] = MPI_Wtime();
      /////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (mpiData.rank)  //Only for Workers
    {
      ////////////////////////////////// Input Prepation for Host //////////////////////////////////
      MPI_Probe(user.root, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      timing.inputPreparationHost[0] = MPI_Wtime();
      MPI_Recv(&pointer.dVect1[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD,
               MPI_STATUS_IGNORE);
      MPI_Recv(&pointer.dVect2[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD,
               MPI_STATUS_IGNORE);

      // cudaCheck(hipMemcpy(pointer.dVect1, mpiData.mVectWorker1, user.sizeVectorBytes, hipMemcpyHostToDevice));  //copy random vector from host to device.
      // cudaCheck(hipMemcpy(pointer.dVect2, mpiData.mVectWorker2, user.sizeVectorBytes, hipMemcpyHostToDevice));

      timing.inputPreparationHost[1] = MPI_Wtime();
      ///////////////////////////////////////////////////////////////////////////////////////

      cudaCheck(hipEventCreate(&timing.start));  //inialize Event.
      cudaCheck(hipEventCreate(&timing.stop));

      ///////////////////////////// Operation on Device with respect of Host //////////////////

      int threads = 512;                                  //arbitrary number.
      int blocks = (sizeVector + threads - 1) / threads;  //get ceiling number of blocks.
      blocks = std::min(blocks, 8);  // Number 8 is least number can be got from lowest Nevedia GPUs.

      ////////////////////////// CAll Device Kernel //////////////////////////////////
      cudaCheck(hipEventRecord(timing.start));
      timing.operationOnDeviceByHost[0] = MPI_Wtime();

      addVectorsGpu<<<blocks, threads>>>(pointer.dVect1,
                                         pointer.dVect2,
                                         pointer.dVect3,
                                         sizeVector,
                                         task);  //call device function to add two vectors and save into vect3Gpu.

      cudaCheck(hipGetLastError());
      cudaCheck(hipDeviceSynchronize());
      cudaCheck(hipEventRecord(timing.stop));

      timing.operationOnDeviceByHost[1] = MPI_Wtime();
      /////////////////////////////////////////////////////////////////////////////////////////////

      /////////////////////////////////// Output Prepation for the Host //////////////////////////////////////
      timing.outputPreparationHost[0] = MPI_Wtime();
      //cudaCheck(hipMemcpy(mpiData.mVectWorker3,pointer.dVect3,user.sizeVectorBytes,hipMemcpyDeviceToHost));  //copy summing result vector from Device to Host.// Try_Regist(3) delete this

      MPI_Send(&pointer.dVect3[0],
               mpiData.numberToSend[mpiData.rank],
               MPI_FLOAT,
               user.root,
               0,
               MPI_COMM_WORLD);  //Tag is 0
      timing.outputPreparationHost[1] = MPI_Wtime();
      ////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (!mpiData.rank)  //Only for root
    {
      /////////////////////////////////// Output Prepation for the Root //////////////////////////////////////
      MPI_Probe(MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      timing.outputPreparationRoot[0] = MPI_Wtime();
      //MPI probe
      for (int i = 1; i < mpiData.num_procs; i++) {
        MPI_Recv(&mpiData.mVectWorker3[mpiData.displacement[i]],
                 mpiData.numberToSend[i],
                 MPI_FLOAT,
                 i,
                 0,
                 MPI_COMM_WORLD,
                 MPI_STATUS_IGNORE);
      }
      timing.outputPreparationRoot[1] = MPI_Wtime();
      ////////////////////////////////////////////////////////////////////////////////////////////////
    }

    if (a > 0)
      calculateTimeDuration(timing, a - 1, mpiData.rank);

    if (mpiData.rank) {
      cudaCheck(hipEventDestroy(timing.start));
      cudaCheck(hipEventDestroy(timing.stop));
    }
  }
  ///////////////////////////// End of Average ////////////////////////
  if (mpiData.rank) {
    cudaCheck(hipFree(pointer.dVect1));
    cudaCheck(hipFree(pointer.dVect2));
    cudaCheck(hipFree(pointer.dVect3));
  }
  ///
  bool test = 0;
  if (!mpiData.rank)  //Only for root
  {
    addVectorsHost(mpiData.mVect1, mpiData.mVect2, mpiData.mVectChecking);  //Host is adding vectors too.
    test = checkingResultsPrintout(mpiData.mVectChecking,
                                   mpiData.mVectWorker3);  //Checking the results, if error then Print out to the user.
    if (!test)
      exit(-1);
  }

  calculateAverageDeviation(timing, average, mpiData.rank);
  test = sendAverageToRoot(timing, user, mpiData.rank);
  if (test && !mpiData.rank) {
    if (saveFile) {
      test = saveToFile("dataPart3", timing);

      if (test)
        std::cout << "Done Part " << timing.partChosen << " And File saved" << std::endl;
      else
        std::cout << "Error Saving File!!" << std::endl;
    }
    std::cout << "Done Part " << timing.partChosen << std::endl;
  }
  return timing;
}

void printTable(std::vector<Timing> &timing, bool standerDeviationPrint) {
  const std::string inPrepatRoot = " Duration Time Read Input Prepations On Root ";
  const std::string inPrepatHost = " Duration Time Read Input Prepations On Host ";
  const std::string timeCpuR = " Duration Time operation on Root point View  ";
  const std::string timeCpu = " Duration Time operation on Host point View  ";
  const std::string timeGpu = " Duration Time operation on Device point View";
  const std::string outPrepatRoot = " Duration Time Read Output Prepations On Root";
  const std::string outPrepatHost = " Duration Time Read Output Prepations On Host";

  const std::string averageTime = " AverTime ";
  const std::string standerDeviation = " StDeviation ";
  const std::string nameTiming = " Name Timing ";
  const std::string partsNumberall = "Part ";

  int totalFix = 0;

  if (standerDeviationPrint) {
    totalFix = timeGpu.size() + timing.size() * (averageTime.size() + standerDeviation.size() + 3);
  } else {
    totalFix = timeGpu.size() + timing.size() * (averageTime.size() + 3);
  }

  std::cout.flags(std::ios::fixed | std::ios::showpoint);
  std::cout.precision(4);

  std::cout << '\n';
  std::cout.width(totalFix);
  std::cout.fill('-');
  std::cout << '-' << '\n';
  std::cout.fill(' ');

  std::cout << "|";
  std::cout.width((timeGpu.size() - nameTiming.size()) / 2);
  std::cout.fill(' ');
  std::cout << " ";
  std::cout << nameTiming;
  std::cout.width((timeGpu.size() - nameTiming.size()) / 2);
  std::cout.fill(' ');
  std::cout << " ";
  std::cout << "  |";

  for (unsigned int i = 0; i < timing.size(); ++i) {
    if (standerDeviationPrint) {
      std::cout.width(((averageTime.size() + standerDeviation.size()) - partsNumberall.size() + 1) / 2);
    }  //9
    else {
      std::cout.width(((averageTime.size()) - partsNumberall.size()) / 2);
    }  //2

    std::cout << " ";
    std::cout << partsNumberall << timing[i].partChosen;

    if (standerDeviationPrint) {
      std::cout.width(((averageTime.size() + standerDeviation.size()) - partsNumberall.size() + 1) / 2);
    }  //9
    else {
      std::cout.width(((averageTime.size()) - partsNumberall.size()) / 2);
    }
    //2
    std::cout << " ";
    std::cout << "|";
  }

  std::cout << '\n';
  std::cout << "|";
  std::cout.width(inPrepatHost.size() + 3);
  std::cout.fill(' ');
  std::cout << "|";

  for (unsigned int i = 0; i < timing.size(); ++i) {
    std::cout << averageTime;
    std::cout << "|";
    if (standerDeviationPrint) {
      std::cout << standerDeviation;
      std::cout << "|";
    }
  }

  newLineTitle(totalFix, inPrepatRoot);
  printResultEach(timing, 1, standerDeviationPrint);

  newLineTitle(totalFix, inPrepatHost);
  printResultEach(timing, 2, standerDeviationPrint);

  newLineTitle(totalFix, timeCpuR);
  printResultEach(timing, 3, standerDeviationPrint);

  newLineTitle(totalFix, timeCpu);
  printResultEach(timing, 4, standerDeviationPrint);

  newLineTitle(totalFix, timeGpu);
  printResultEach(timing, 5, standerDeviationPrint);

  newLineTitle(totalFix, outPrepatRoot);
  printResultEach(timing, 6, standerDeviationPrint);

  newLineTitle(totalFix, outPrepatHost);
  printResultEach(timing, 7, standerDeviationPrint);

  std::cout << '\n';
  std::cout.width(totalFix);
  std::cout.fill('-');
  std::cout << '-' << '\n';
  std::cout.fill(' ');
}
int getNumberofDigits(double number) { return ((int)log10(number) + 1) + 4; }
void newLineTitle(int line, const std::string &title) {
  std::cout << '\n';
  std::cout.width(line);
  std::cout.fill('-');
  std::cout << '-' << '\n';
  std::cout.fill(' ');

  std::cout << "| ";
  std::cout << title;
  std::cout << " |";
}
void printResultEach(std::vector<Timing> &timing, int type, bool standerDeviationPrint) {
  int averageTimeWidth = 10;
  int standerDeviationWidth = 13;

  for (unsigned int i = 0; i < timing.size(); ++i) {
    if (type == 1) {
      std::cout.width(averageTimeWidth);
      std::cout.fill(' ');
      std::cout << timing[i].timeInputPreparationRoot[average];
      std::cout << "|";
      if (standerDeviationPrint) {
        std::cout.width(standerDeviationWidth);
        std::cout.fill(' ');
        std::cout << timing[i].timeInputPreparationRoot[average + 1];
        std::cout << "|";
      }
    } else if (type == 2) {
      std::cout.width(averageTimeWidth);
      std::cout.fill(' ');
      std::cout << timing[i].averageResults[0];
      std::cout << "|";
      if (standerDeviationPrint) {
        std::cout.width(standerDeviationWidth);
        std::cout.fill(' ');
        std::cout << timing[i].averageResults[1];
        std::cout << "|";
      }
    } else if (type == 3) {
      std::cout.width(averageTimeWidth);
      std::cout.fill(' ');
      std::cout << timing[i].timeOperationOnDeviceByRootHost[average];
      std::cout << "|";
      if (standerDeviationPrint) {
        std::cout.width(standerDeviationWidth);
        std::cout.fill(' ');
        std::cout << timing[i].timeOperationOnDeviceByRootHost[average + 1];
        std::cout << "|";
      }
    } else if (type == 4) {
      std::cout.width(averageTimeWidth);
      std::cout.fill(' ');
      std::cout << timing[i].averageResults[2];
      std::cout << "|";
      if (standerDeviationPrint) {
        std::cout.width(standerDeviationWidth);
        std::cout.fill(' ');
        std::cout << timing[i].averageResults[3];
        std::cout << "|";
      }
    } else if (type == 5) {
      std::cout.width(averageTimeWidth);
      std::cout.fill(' ');
      std::cout << timing[i].averageResults[6];
      std::cout << "|";
      if (standerDeviationPrint) {
        std::cout.width(standerDeviationWidth);
        std::cout.fill(' ');
        std::cout << timing[i].averageResults[7];
        std::cout << "|";
      }
    } else if (type == 6) {
      std::cout.width(averageTimeWidth);
      std::cout.fill(' ');
      std::cout << timing[i].timeOutputPreparationRoot[average];
      std::cout << "|";
      if (standerDeviationPrint) {
        std::cout.width(standerDeviationWidth);
        std::cout.fill(' ');
        std::cout << timing[i].timeOutputPreparationRoot[average + 1];
        std::cout << "|";
      }
    } else if (type == 7) {
      std::cout.width(averageTimeWidth);
      std::cout.fill(' ');
      std::cout << timing[i].averageResults[4];
      std::cout << "|";
      if (standerDeviationPrint) {
        std::cout.width(standerDeviationWidth);
        std::cout.fill(' ');
        std::cout << timing[i].averageResults[5];
        std::cout << "|";
      }
    }
  }
}
bool saveToFile(const std::string &name, const Timing &timing) {
  std::ofstream file(name + ".txt", std::ios::out | std::ios::app);

  if (!file.is_open()) {
    std::cout << "\nCannot open File nor Create File!" << std::endl;
    return 0;
  }

  file << sizeVector << std::endl;
  file << average << std::endl;
  file << task << std::endl;
  file << timing.timeInputPreparationRoot[average] << " " << timing.timeInputPreparationRoot[average + 1] << std::endl;
  file << timing.averageResults[0] << " " << timing.averageResults[1] << std::endl;
  file << timing.timeOperationOnDeviceByRootHost[average] << " " << timing.timeOperationOnDeviceByRootHost[average + 1]
       << std::endl;
  file << timing.averageResults[2] << " " << timing.averageResults[3] << std::endl;
  file << timing.averageResults[6] << " " << timing.averageResults[7] << std::endl;
  file << timing.timeOutputPreparationRoot[average] << " " << timing.timeOutputPreparationRoot[average + 1]
       << std::endl;
  file << timing.averageResults[4] << " " << timing.averageResults[5] << std::endl;

  file.close();
  if (!file.good()) {
    std::cout << "\n*ERROR While Writing The " + name + " file!!" << std::endl;
    return 0;
  }
  return 1;
}
void printHelp(void) {
  int rank = MPI::COMM_WORLD.Get_rank();
  if (!rank) {
    std::cout << "\n\n\t**************************************\n";
    std::cout << "\t* This is a Help for Command Opitions*";
    std::cout << "\n\t**************************************\n";
    std::cout << "\n\tYou as a user, can choose two ways to run the program:\n";
    std::cout << "\n\t1) mpirun -np <number of Process/ors> -s <size of Vector> -t <number of task> -a <average size> "
                 "-p <part to run>\n";
    std::cout << "\n\t2) cmsenv_mpirun -np <number of Process/ors> -s <size of Vector> -t <number of task> -a <average "
                 "size> -p <part to run>\n";
    std::cout << "\n\t[-np] is for number of processes or processors that you would like to run.";
    std::cout
        << "\n\t[-s] is the size of vector that you would like to send, the type is float and there are two vectors.";
    std::cout << "\n\t[-t] is the number of repeating of task on the Device(GPU) side.";
    std::cout << "\n\t[-a] is the number of repeating the part that user has chosen.";
    std::cout << "\n\t[-p] is the choice of what part to run in the program.";
    std::cout << "\n\t[-q] is to print Stander Deviation.";
    std::cout << "\n\t[-f] is to save the results into a file for each part.";
    std::cout << "\n\n\tExample for only local Machine: ";
    std::cout << "\n\tcmsenv_mpirun -np 2 mpiCudaGeneric -p1 -s200 -t1 -a1\n";
    std::cout << "\n\tExample for two Machines connected: ";
    std::cout
        << "\n\tcmsenv_mpirun -H <machine Name as Root>,<machine Name as Host> -np 2 mpiCudaGeneric -p1 -s200 -t1 -a1";
    std::cout << "\n\tExample for two Machines connected Using ucx: ";
    std::cout << "\n\tcmsenv_mpirun -H <machine Name as Root>,<machine Name as Host> -np 2 -mca pml ucx -- "
                 "mpiCudaGeneric -p1 -s200 -t1 -a1";
    std::cout << "\n\n\tFor the Parts, we have in this program 4 Parts:";
    std::cout << "\n\t1)The Root, who does not have a GPU, using MPI Blocking send and receive to Host, The Host is "
                 "who have a GPU, then Host:";
    std::cout << "\n\t  uses hipMalloc and copies the receiving values to GPU side. Next, the GPU does the compuation";
    std::cout << "\n\t  Finaly, the Host copies the results from GPU, sends them back to The Root using MPI Blocking "
                 "Send.\n\n";
  }
}