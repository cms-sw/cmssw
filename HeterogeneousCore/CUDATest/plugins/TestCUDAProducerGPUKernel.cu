#include "hip/hip_runtime.h"
#include "TestCUDAProducerGPUKernel.h"

#include "FWCore/MessageLogger/interface/MessageLogger.h"
#include "FWCore/Utilities/interface/Exception.h"
#include "FWCore/ServiceRegistry/interface/Service.h"
#include "HeterogeneousCore/CUDAServices/interface/CUDAService.h"

namespace {
  template<typename T>
  __global__
  void vectorAdd(const T *a, const T *b, T *c, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < numElements) { c[i] = a[i] + b[i]; }
  }

  template <typename T>
  __global__
  void vectorProd(const T *a, const T *b, T *c, int numElements) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if(row < numElements && col < numElements) {
      c[row*numElements + col] = a[row]*b[col];
    }
  }

  template <typename T>
  __global__
  void matrixMul(const T *a, const T *b, T *c, int numElements) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if(row < numElements && col < numElements) {
      T tmp = 0;
      for(int i=0; i<numElements; ++i) {
        tmp += a[row*numElements + i] * b[i*numElements + col];
      }
      c[row*numElements + col] = tmp;
    }
  }

  template <typename T>
  __global__
  void matrixMulVector(const T *a, const T *b, T *c, int numElements) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    if(row < numElements) {
      T tmp = 0;
      for(int i=0; i<numElements; ++i) {
        tmp += a[row*numElements + i] * b[i];
      }
      c[row] = tmp;
    }
  }
}

cudautils::device::unique_ptr<float[]> TestCUDAProducerGPUKernel::runAlgo(const std::string& label, const float *d_input, cuda::stream_t<>& stream) const {
  // First make the sanity check
  if(d_input != nullptr) {
    auto h_check = std::make_unique<float[]>(NUM_VALUES);
    cuda::memory::copy(h_check.get(), d_input, NUM_VALUES*sizeof(float));
    for(int i=0; i<NUM_VALUES; ++i) {
      if(h_check[i] != i) {
        throw cms::Exception("Assert") << "Sanity check on element " << i << " failed, expected " << i << " got " << h_check[i];
      }
    }
  }

  edm::Service<CUDAService> cs;

  auto h_a = cs->make_host_unique<float[]>(NUM_VALUES, stream);
  auto h_b = cs->make_host_unique<float[]>(NUM_VALUES, stream);

  for (auto i=0; i<NUM_VALUES; i++) {
    h_a[i] = i;
    h_b[i] = i*i;
  }

  auto d_a = cs->make_device_unique<float[]>(NUM_VALUES, stream);
  auto d_b = cs->make_device_unique<float[]>(NUM_VALUES, stream);

  cuda::memory::async::copy(d_a.get(), h_a.get(), NUM_VALUES*sizeof(float), stream.id());
  cuda::memory::async::copy(d_b.get(), h_b.get(), NUM_VALUES*sizeof(float), stream.id());

  int threadsPerBlock {32};
  int blocksPerGrid = (NUM_VALUES + threadsPerBlock - 1) / threadsPerBlock;

  auto d_c = cs->make_device_unique<float[]>(NUM_VALUES, stream);
  auto current_device = cuda::device::current::get();
  edm::LogVerbatim("TestHeterogeneousEDProducerGPU") << "  " << label << " GPU launching kernels device " << current_device.id() << " CUDA stream " << stream.id();
  vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream.id()>>>(d_a.get(), d_b.get(), d_c.get(), NUM_VALUES);

  auto d_ma = cs->make_device_unique<float[]>(NUM_VALUES*NUM_VALUES, stream);
  auto d_mb = cs->make_device_unique<float[]>(NUM_VALUES*NUM_VALUES, stream);
  auto d_mc = cs->make_device_unique<float[]>(NUM_VALUES*NUM_VALUES, stream);
  dim3 threadsPerBlock3{NUM_VALUES, NUM_VALUES};
  dim3 blocksPerGrid3{1,1};
  if(NUM_VALUES*NUM_VALUES > 32) {
    threadsPerBlock3.x = 32;
    threadsPerBlock3.y = 32;
    blocksPerGrid3.x = ceil(double(NUM_VALUES)/double(threadsPerBlock3.x));
    blocksPerGrid3.y = ceil(double(NUM_VALUES)/double(threadsPerBlock3.y));
  }
  vectorProd<<<blocksPerGrid3, threadsPerBlock3, 0, stream.id()>>>(d_a.get(), d_b.get(), d_ma.get(), NUM_VALUES);
  vectorProd<<<blocksPerGrid3, threadsPerBlock3, 0, stream.id()>>>(d_a.get(), d_c.get(), d_mb.get(), NUM_VALUES);
  matrixMul<<<blocksPerGrid3, threadsPerBlock3, 0, stream.id()>>>(d_ma.get(), d_mb.get(), d_mc.get(), NUM_VALUES);

  matrixMulVector<<<blocksPerGrid, threadsPerBlock, 0, stream.id()>>>(d_mc.get(), d_b.get(), d_c.get(), NUM_VALUES);

  edm::LogVerbatim("TestHeterogeneousEDProducerGPU") << "  " << label << " GPU kernels launched, returning return pointer device " << current_device.id() << " CUDA stream " << stream.id();
  return d_a;
}
