#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <Eigen/Eigenvalues>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "PixelTrackReconstructionGPU.h"

using namespace Eigen;

__global__ void
KernelFastFitAllHits(float *hits_and_covariances,
    int hits_in_fit,
    int cumulative_size,
    float B,
    Rfit::helix_fit *results,
    Rfit::Matrix3xNd<4> *hits,
    Eigen::Matrix<float,6,4> *hits_ge,
    Rfit::circle_fit *circle_fit,
    Vector4d *fast_fit,
    Rfit::line_fit *line_fit)
{
  // Reshape Eigen components from hits_and_covariances, using proper thread and block indices
  // Perform the fit
  // Store the results in the proper vector, using again correct indices

  // Loop for hits_in_fit times:
  //   first 3 are the points
  //   the rest is the covariance matrix, 3x3
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG
  printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, start: %d, cumulative_size: %d\n",
      blockDim.x, blockIdx.x, threadIdx.x, start, cumulative_size);
#endif


  // Prepare data structure (stack)
  for (unsigned int i = 0; i < hits_in_fit; ++i) {
    hits[helix_start].col(i) << hits_and_covariances[start],
        hits_and_covariances[start + 1], hits_and_covariances[start + 2];
    start += 3;

    hits_ge[helix_start].col(i) << hits_and_covariances[start],
        hits_and_covariances[start + 1], hits_and_covariances[start + 2],
        hits_and_covariances[start + 3], hits_and_covariances[start + 4],
        hits_and_covariances[start + 5];
    start += 6;
  }

  Rfit::Fast_fit(hits[helix_start],fast_fit[helix_start]);
}

__global__ void
KernelCircleFitAllHits(float *hits_and_covariances, int hits_in_fit,
                       int cumulative_size, float B, Rfit::helix_fit *results,
                       Rfit::Matrix3xNd<4> *hits, Eigen::Matrix<float,6,4> *hits_ge,
                       Rfit::circle_fit *circle_fit, Vector4d *fast_fit,
                       Rfit::line_fit *line_fit)
{
  // Reshape Eigen components from hits_and_covariances, using proper thread and block indices
  // Perform the fit
  // Store the results in the proper vector, using again correct indices

  // Loop for hits_in_fit times:
  //   first 3 are the points
  //   the rest is the covariance matrix, 3x3
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG
    printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, start: %d, "
           "cumulative_size: %d\n",
           blockDim.x, blockIdx.x, threadIdx.x, start, cumulative_size);
#endif
  u_int n = hits[helix_start].cols();

  constexpr uint32_t N = 4;

  Rfit::VectorNd<N> rad = (hits[helix_start].block(0, 0, 2, n).colwise().norm());
  Rfit::Matrix2Nd<N> hits_cov =  MatrixXd::Zero(2 * n, 2 * n);
  Rfit::loadCovariance2D(hits_ge[helix_start],hits_cov);
  circle_fit[helix_start] =
      Rfit::Circle_fit(hits[helix_start].block(0, 0, 2, n),
                       hits_cov,
                       fast_fit[helix_start], rad, B, true);

#ifdef GPU_DEBUG
    printf("KernelCircleFitAllHits circle.par(0): %d %f\n", helix_start,
           circle_fit[helix_start].par(0));
    printf("KernelCircleFitAllHits circle.par(1): %d %f\n", helix_start,
           circle_fit[helix_start].par(1));
    printf("KernelCircleFitAllHits circle.par(2): %d %f\n", helix_start,
           circle_fit[helix_start].par(2));
#endif

}

__global__ void
KernelLineFitAllHits(float *hits_and_covariances, int hits_in_fit,
                     int cumulative_size, float B, Rfit::helix_fit *results,
                      Rfit::Matrix3xNd<4> *hits, Eigen::Matrix<float,6,4> *hits_ge,
                     Rfit::circle_fit *circle_fit, Vector4d *fast_fit,
                     Rfit::line_fit *line_fit)
{
  // Reshape Eigen components from hits_and_covariances, using proper thread and block indices
  // Perform the fit
  // Store the results in the proper vector, using again correct indices

  // Loop for hits_in_fit times:
  //   first 3 are the points
  //   the rest is the covariance matrix, 3x3
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * hits_in_fit * 12;
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= cumulative_size) {
    return;
  }

#ifdef GPU_DEBUG

    printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, start: %d, "
           "cumulative_size: %d\n",
           blockDim.x, blockIdx.x, threadIdx.x, start, cumulative_size);
#endif

  line_fit[helix_start] =
      Rfit::Line_fit(hits[helix_start], hits_ge[helix_start],
                     circle_fit[helix_start], fast_fit[helix_start], B, true);

  par_uvrtopak(circle_fit[helix_start], B, true);

  // Grab helix_fit from the proper location in the output vector
  Rfit::helix_fit &helix = results[helix_start];
  helix.par << circle_fit[helix_start].par, line_fit[helix_start].par;

  // TODO: pass properly error booleans

  helix.cov = MatrixXd::Zero(5, 5);
  helix.cov.block(0, 0, 3, 3) = circle_fit[helix_start].cov;
  helix.cov.block(3, 3, 2, 2) = line_fit[helix_start].cov;

  helix.q = circle_fit[helix_start].q;
  helix.chi2_circle = circle_fit[helix_start].chi2;
  helix.chi2_line = line_fit[helix_start].chi2;

#ifdef GPU_DEBUG

    printf("KernelLineFitAllHits line.par(0): %d %f\n", helix_start,
           circle_fit[helix_start].par(0));
    printf("KernelLineFitAllHits line.par(1): %d %f\n", helix_start,
           line_fit[helix_start].par(1));
#endif
}

void PixelTrackReconstructionGPU::launchKernelFit(
    float *hits_and_covariancesGPU, int cumulative_size, int hits_in_fit,
    float B, Rfit::helix_fit *results)
{
  const dim3 threads_per_block(32, 1);
  int num_blocks = cumulative_size / (hits_in_fit * 12) / threads_per_block.x + 1;
  auto numberOfSeeds = cumulative_size / (hits_in_fit * 12);

  Rfit::Matrix3xNd<4> *hitsGPU;
  cudaCheck(hipMalloc(&hitsGPU, 48 * numberOfSeeds * sizeof(Rfit::Matrix3xNd<4>)));
  cudaCheck(hipMemset(hitsGPU, 0x00, 48 * numberOfSeeds * sizeof(Rfit::Matrix3xNd<4>)));

  Eigen::Matrix<float,6,4> *hits_geGPU = nullptr;
  cudaCheck(hipMalloc(&hits_geGPU, 48 * numberOfSeeds * sizeof(Eigen::Matrix<float,6,4>)));
  cudaCheck(hipMemset(hits_geGPU, 0x00, 48 * numberOfSeeds * sizeof(Eigen::Matrix<float,6,4>)));

  Vector4d *fast_fit_resultsGPU = nullptr;
  cudaCheck(hipMalloc(&fast_fit_resultsGPU, 48 * numberOfSeeds * sizeof(Vector4d)));
  cudaCheck(hipMemset(fast_fit_resultsGPU, 0x00, 48 * numberOfSeeds * sizeof(Vector4d)));

  Rfit::circle_fit *circle_fit_resultsGPU = nullptr;
  cudaCheck(hipMalloc(&circle_fit_resultsGPU, 48 * numberOfSeeds * sizeof(Rfit::circle_fit)));
  cudaCheck(hipMemset(circle_fit_resultsGPU, 0x00, 48 * numberOfSeeds * sizeof(Rfit::circle_fit)));

  Rfit::line_fit *line_fit_resultsGPU = nullptr;
  cudaCheck(hipMalloc(&line_fit_resultsGPU, numberOfSeeds * sizeof(Rfit::line_fit)));
  cudaCheck(hipMemset(line_fit_resultsGPU, 0x00, numberOfSeeds * sizeof(Rfit::line_fit)));

  KernelFastFitAllHits<<<num_blocks, threads_per_block>>>(
      hits_and_covariancesGPU, hits_in_fit, cumulative_size, B, results,
      hitsGPU, hits_geGPU, circle_fit_resultsGPU, fast_fit_resultsGPU,
      line_fit_resultsGPU);
  cudaCheck(hipGetLastError());

  KernelCircleFitAllHits<<<num_blocks, threads_per_block>>>(
      hits_and_covariancesGPU, hits_in_fit, cumulative_size, B, results,
      hitsGPU, hits_geGPU, circle_fit_resultsGPU, fast_fit_resultsGPU,
      line_fit_resultsGPU);
  cudaCheck(hipGetLastError());

  KernelLineFitAllHits<<<num_blocks, threads_per_block>>>(
      hits_and_covariancesGPU, hits_in_fit, cumulative_size, B, results,
      hitsGPU, hits_geGPU, circle_fit_resultsGPU, fast_fit_resultsGPU,
      line_fit_resultsGPU);
  cudaCheck(hipGetLastError());

  hipFree(hitsGPU);
  hipFree(hits_geGPU);
  hipFree(fast_fit_resultsGPU);
  hipFree(circle_fit_resultsGPU);
  hipFree(line_fit_resultsGPU);
}
