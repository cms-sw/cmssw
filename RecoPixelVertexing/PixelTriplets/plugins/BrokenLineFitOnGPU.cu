#include "BrokenLineFitOnGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"

void HelixFitOnGPU::launchBrokenLineKernels(HitsView const *hv,
                                            uint32_t hitsInFit,
                                            uint32_t maxNumberOfTuples,
                                            hipStream_t stream) {
  assert(tuples_d);

  auto blockSize = 64;
  auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

  //  Fit internals
  auto hitsGPU_ = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix3xNd<4>) / sizeof(double), stream);
  auto hits_geGPU_ = cms::cuda::make_device_unique<float[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix6x4f) / sizeof(float), stream);
  auto fast_fit_resultsGPU_ = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Vector4d) / sizeof(double), stream);

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // fit triplets
    kernelBLFastFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(
        tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 3, offset);
    cudaCheck(hipGetLastError());

    kernelBLFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                             bField_,
                                                             outputSoa_d,
                                                             hitsGPU_.get(),
                                                             hits_geGPU_.get(),
                                                             fast_fit_resultsGPU_.get(),
                                                             3,
                                                             offset);
    cudaCheck(hipGetLastError());

    // fit quads
    kernelBLFastFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
        tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 4, offset);
    cudaCheck(hipGetLastError());

    kernelBLFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                 bField_,
                                                                 outputSoa_d,
                                                                 hitsGPU_.get(),
                                                                 hits_geGPU_.get(),
                                                                 fast_fit_resultsGPU_.get(),
                                                                 4,
                                                                 offset);
    cudaCheck(hipGetLastError());

    if (fit5as4_) {
      // fit penta (only first 4)
      kernelBLFastFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
          tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 5, offset);
      cudaCheck(hipGetLastError());

      kernelBLFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                   bField_,
                                                                   outputSoa_d,
                                                                   hitsGPU_.get(),
                                                                   hits_geGPU_.get(),
                                                                   fast_fit_resultsGPU_.get(),
                                                                   5,
                                                                   offset);
      cudaCheck(hipGetLastError());
    } else {
      // fit penta (all 5)
      kernelBLFastFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
          tuples_d, tupleMultiplicity_d, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), 5, offset);
      cudaCheck(hipGetLastError());

      kernelBLFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                   bField_,
                                                                   outputSoa_d,
                                                                   hitsGPU_.get(),
                                                                   hits_geGPU_.get(),
                                                                   fast_fit_resultsGPU_.get(),
                                                                   5,
                                                                   offset);
      cudaCheck(hipGetLastError());
    }

  }  // loop on concurrent fits
}
