#include "hip/hip_runtime.h"
#include "CAHitQuadrupletGeneratorKernels.h"

void CAHitQuadrupletGeneratorKernels::deallocateOnGPU() {
  if (doStats_) {
    // crash on multi-gpu processes
    printCounters();
  }
  hipFree(counters_);

  hipFree(device_nCells_);
  hipFree(device_theCellNeighbors_);
  hipFree(device_theCellTracks_);

  hipFree(device_hitToTuple_);
  hipFree(device_hitToTuple_apc_);
  hipFree(device_tupleMultiplicity_);
  hipFree(device_tmws_);
}

void CAHitQuadrupletGeneratorKernels::allocateOnGPU() {
  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  cudaCheck(hipMalloc(&counters_, sizeof(Counters)));
  cudaCheck(hipMemset(counters_, 0, sizeof(Counters)));

  cudaCheck(hipMalloc(&device_nCells_, sizeof(uint32_t)));
  cudaCheck(hipMemset(device_nCells_, 0, sizeof(uint32_t)));

  cudaCheck(hipMalloc(&device_theCellNeighbors_, sizeof(CAConstants::CellNeighborsVector)));
  cudaCheck(hipMemset(device_theCellNeighbors_, 0, sizeof(CAConstants::CellNeighborsVector)));
  cudaCheck(hipMalloc(&device_theCellTracks_, sizeof(CAConstants::CellTracksVector)));
  cudaCheck(hipMemset(device_theCellTracks_, 0, sizeof(CAConstants::CellTracksVector)));

  cudaCheck(hipMalloc(&device_hitToTuple_, sizeof(HitToTuple)));
  cudaCheck(hipMemset(device_hitToTuple_, 0, sizeof(HitToTuple)));  // overkill
  cudaCheck(hipMalloc(&device_hitToTuple_apc_, sizeof(AtomicPairCounter)));

  cudaCheck(hipMalloc(&device_tupleMultiplicity_, sizeof(TupleMultiplicity)));
  cudaCheck(hipMemset(device_tupleMultiplicity_, 0, sizeof(TupleMultiplicity)));  // overkill

  cudaCheck(hipMalloc(&device_tmws_, std::max(TupleMultiplicity::wsSize(), HitToTuple::wsSize())));
}

void CAHitQuadrupletGeneratorKernels::cleanup(hipStream_t cudaStream) {
#ifdef GPU_DEBUG
  std::cout << "CAHitQuadrupletGeneratorKernels::cleanup" << std::endl;
#endif

  // this lazily resets temporary memory for the next event, and is not needed for reading the output
  device_theCells_ = nullptr;
  device_isOuterHitOfCell_ = nullptr;
  device_theCellNeighborsContainer_ = nullptr;
  device_theCellTracksContainer_ = nullptr;

  cudaCheck(hipMemsetAsync(device_nCells_, 0, sizeof(uint32_t), cudaStream));

  cudautils::launchZero(device_tupleMultiplicity_, cudaStream);

  cudautils::launchZero(device_hitToTuple_, cudaStream);  // we may wish to keep it in the edm...
}
