#include "hip/hip_runtime.h"
#include "RecoPixelVertexing/PixelTriplets/plugins/CAHitNtupletGeneratorKernelsImpl.h"

template <>
void CAHitNtupletGeneratorKernelsGPU::fillHitDetIndices(HitsView const *hv, TkSoA *tracks_d, hipStream_t cudaStream) {
  auto blockSize = 128;
  auto numberOfBlocks = (HitContainer::ctCapacity() + blockSize - 1) / blockSize;

  kernel_fillHitDetIndices<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      &tracks_d->hitIndices, hv, &tracks_d->detIndices);
  cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif
}

template <>
void CAHitNtupletGeneratorKernelsGPU::launchKernels(HitsOnCPU const &hh, TkSoA *tracks_d, hipStream_t cudaStream) {
  // these are pointer on GPU!
  auto *tuples_d = &tracks_d->hitIndices;
  auto *quality_d = tracks_d->qualityData();

  // zero tuples
  cms::cuda::launchZero(tuples_d, cudaStream);

  auto nhits = hh.nHits();

#ifdef NTUPLE_DEBUG
  std::cout << "start tuple building. N hits " << nhits << std::endl;
  if (nhits < 2)
    std::cout << "too few hits " << nhits << std::endl;
#endif

  //
  // applying conbinatoric cleaning such as fishbone at this stage is too expensive
  //

  auto nthTot = 64;
  auto stride = 4;
  auto blockSize = nthTot / stride;
  auto numberOfBlocks = nDoubletBlocks(blockSize);
  auto rescale = numberOfBlocks / 65536;
  blockSize *= (rescale + 1);
  numberOfBlocks = nDoubletBlocks(blockSize);
  assert(numberOfBlocks < 65536);
  assert(blockSize > 0 && 0 == blockSize % 16);
  dim3 blks(1, numberOfBlocks, 1);
  dim3 thrs(stride, blockSize, 1);

  kernel_connect<<<blks, thrs, 0, cudaStream>>>(
      device_hitTuple_apc_,
      device_hitToTuple_apc_,  // needed only to be reset, ready for next kernel
      hh.view(),
      device_theCells_.get(),
      device_nCells_,
      device_theCellNeighbors_.get(),
      device_isOuterHitOfCell_.get(),
      params_.hardCurvCut_,
      params_.ptmin_,
      params_.CAThetaCutBarrel_,
      params_.CAThetaCutForward_,
      params_.dcaCutInnerTriplet_,
      params_.dcaCutOuterTriplet_);
  cudaCheck(hipGetLastError());

  if (nhits > 1 && params_.earlyFishbone_) {
    auto nthTot = 128;
    auto stride = 16;
    auto blockSize = nthTot / stride;
    auto numberOfBlocks = (nhits + blockSize - 1) / blockSize;
    dim3 blks(1, numberOfBlocks, 1);
    dim3 thrs(stride, blockSize, 1);
    gpuPixelDoublets::fishbone<<<blks, thrs, 0, cudaStream>>>(
        hh.view(), device_theCells_.get(), device_nCells_, device_isOuterHitOfCell_.get(), nhits, false);
    cudaCheck(hipGetLastError());
  }

  blockSize = 64;
  numberOfBlocks = (3 * params_.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
  kernel_find_ntuplets<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                                     device_theCells_.get(),
                                                                     device_nCells_,
                                                                     device_theCellTracks_.get(),
                                                                     tuples_d,
                                                                     device_hitTuple_apc_,
                                                                     quality_d,
                                                                     params_.minHitsPerNtuplet_);
  cudaCheck(hipGetLastError());

  if (params_.doStats_)
    kernel_mark_used<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(), device_theCells_.get(), device_nCells_);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  blockSize = 128;
  numberOfBlocks = (HitContainer::ctNOnes() + blockSize - 1) / blockSize;
  cms::cuda::finalizeBulk<<<numberOfBlocks, blockSize, 0, cudaStream>>>(device_hitTuple_apc_, tuples_d);

  // remove duplicates (tracks that share a doublet)
  numberOfBlocks = nDoubletBlocks(blockSize);
  kernel_earlyDuplicateRemover<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      device_theCells_.get(), device_nCells_, tuples_d, quality_d, params_.dupPassThrough_);
  cudaCheck(hipGetLastError());

  blockSize = 128;
  numberOfBlocks = (3 * caConstants::maxTuples / 4 + blockSize - 1) / blockSize;
  kernel_countMultiplicity<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      tuples_d, quality_d, device_tupleMultiplicity_.get());
  cms::cuda::launchFinalize(device_tupleMultiplicity_.get(), cudaStream);
  kernel_fillMultiplicity<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      tuples_d, quality_d, device_tupleMultiplicity_.get());
  cudaCheck(hipGetLastError());

  if (nhits > 1 && params_.lateFishbone_) {
    auto nthTot = 128;
    auto stride = 16;
    auto blockSize = nthTot / stride;
    auto numberOfBlocks = (nhits + blockSize - 1) / blockSize;
    dim3 blks(1, numberOfBlocks, 1);
    dim3 thrs(stride, blockSize, 1);
    gpuPixelDoublets::fishbone<<<blks, thrs, 0, cudaStream>>>(
        hh.view(), device_theCells_.get(), device_nCells_, device_isOuterHitOfCell_.get(), nhits, true);
    cudaCheck(hipGetLastError());
  }

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  // free space asap
  // device_isOuterHitOfCell_.reset();
}

template <>
void CAHitNtupletGeneratorKernelsGPU::buildDoublets(HitsOnCPU const &hh, hipStream_t stream) {
  int32_t nhits = hh.nHits();

#ifdef NTUPLE_DEBUG
  std::cout << "building Doublets out of " << nhits << " Hits" << std::endl;
#endif

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  // in principle we can use "nhits" to heuristically dimension the workspace...
  device_isOuterHitOfCell_ = cms::cuda::make_device_unique<GPUCACell::OuterHitOfCell[]>(std::max(1, nhits), stream);
  assert(device_isOuterHitOfCell_.get());

  cellStorage_ = cms::cuda::make_device_unique<unsigned char[]>(
      caConstants::maxNumOfActiveDoublets * sizeof(GPUCACell::CellNeighbors) +
          caConstants::maxNumOfActiveDoublets * sizeof(GPUCACell::CellTracks),
      stream);
  device_theCellNeighborsContainer_ = (GPUCACell::CellNeighbors *)cellStorage_.get();
  device_theCellTracksContainer_ = (GPUCACell::CellTracks *)(cellStorage_.get() + caConstants::maxNumOfActiveDoublets *
                                                                                      sizeof(GPUCACell::CellNeighbors));

  {
    int threadsPerBlock = 128;
    // at least one block!
    int blocks = (std::max(1, nhits) + threadsPerBlock - 1) / threadsPerBlock;
    gpuPixelDoublets::initDoublets<<<blocks, threadsPerBlock, 0, stream>>>(device_isOuterHitOfCell_.get(),
                                                                           nhits,
                                                                           device_theCellNeighbors_.get(),
                                                                           device_theCellNeighborsContainer_,
                                                                           device_theCellTracks_.get(),
                                                                           device_theCellTracksContainer_);
    cudaCheck(hipGetLastError());
  }

  device_theCells_ = cms::cuda::make_device_unique<GPUCACell[]>(params_.maxNumberOfDoublets_, stream);

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  if (0 == nhits)
    return;  // protect against empty events

  // take all layer pairs into account
  auto nActualPairs = gpuPixelDoublets::nPairs;
  if (not params_.includeJumpingForwardDoublets_) {
    // exclude forward "jumping" layer pairs
    nActualPairs = gpuPixelDoublets::nPairsForTriplets;
  }
  if (params_.minHitsPerNtuplet_ > 3) {
    // for quadruplets, exclude all "jumping" layer pairs
    nActualPairs = gpuPixelDoublets::nPairsForQuadruplets;
  }

  assert(nActualPairs <= gpuPixelDoublets::nPairs);
  int stride = 4;
  int threadsPerBlock = gpuPixelDoublets::getDoubletsFromHistoMaxBlockSize / stride;
  int blocks = (4 * nhits + threadsPerBlock - 1) / threadsPerBlock;
  dim3 blks(1, blocks, 1);
  dim3 thrs(stride, threadsPerBlock, 1);
  gpuPixelDoublets::getDoubletsFromHisto<<<blks, thrs, 0, stream>>>(device_theCells_.get(),
                                                                    device_nCells_,
                                                                    device_theCellNeighbors_.get(),
                                                                    device_theCellTracks_.get(),
                                                                    hh.view(),
                                                                    device_isOuterHitOfCell_.get(),
                                                                    nActualPairs,
                                                                    params_.idealConditions_,
                                                                    params_.doClusterCut_,
                                                                    params_.doZ0Cut_,
                                                                    params_.doPtCut_,
                                                                    params_.maxNumberOfDoublets_);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif
}

template <>
void CAHitNtupletGeneratorKernelsGPU::classifyTuples(HitsOnCPU const &hh, TkSoA *tracks_d, hipStream_t cudaStream) {
  // these are pointer on GPU!
  auto const *tuples_d = &tracks_d->hitIndices;
  auto *quality_d = tracks_d->qualityData();

  int32_t nhits = hh.nHits();

  auto blockSize = 64;

  // classify tracks based on kinematics
  auto numberOfBlocks = nQuadrupletBlocks(blockSize);
  kernel_classifyTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, tracks_d, params_.cuts_, quality_d);
  cudaCheck(hipGetLastError());

  if (params_.lateFishbone_) {
    // apply fishbone cleaning to good tracks
    numberOfBlocks = nDoubletBlocks(blockSize);
    kernel_fishboneCleaner<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
        device_theCells_.get(), device_nCells_, quality_d);
    cudaCheck(hipGetLastError());
  }

  // mark duplicates (tracks that share a doublet)
  numberOfBlocks = nDoubletBlocks(blockSize);
  kernel_fastDuplicateRemover<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      device_theCells_.get(), device_nCells_, tuples_d, tracks_d, params_.dupPassThrough_);
  cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
  cudaCheck(hipDeviceSynchronize());
#endif

  if (params_.doSharedHitCut_ || params_.doStats_) {
    // fill hit->track "map"
    assert(hitToTupleView_.offSize > nhits);
    numberOfBlocks = nQuadrupletBlocks(blockSize);
    kernel_countHitInTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
        tuples_d, quality_d, device_hitToTuple_.get());
    cudaCheck(hipGetLastError());
    assert((hitToTupleView_.assoc == device_hitToTuple_.get()) &&
           (hitToTupleView_.offStorage == device_hitToTupleStorage_.get()) && (hitToTupleView_.offSize > 0));
    cms::cuda::launchFinalize(hitToTupleView_, cudaStream);
    cudaCheck(hipGetLastError());
    kernel_fillHitInTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, device_hitToTuple_.get());
    cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
    cudaCheck(hipDeviceSynchronize());
#endif
  }

  if (params_.doSharedHitCut_) {
    // mark duplicates (tracks that share at least one hit)
    numberOfBlocks = (hitToTupleView_.offSize + blockSize - 1) / blockSize;

    kernel_rejectDuplicate<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                                         tuples_d,
                                                                         tracks_d,
                                                                         quality_d,
                                                                         params_.minHitsForSharingCut_,
                                                                         params_.dupPassThrough_,
                                                                         device_hitToTuple_.get());

    kernel_sharedHitCleaner<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                                          tuples_d,
                                                                          tracks_d,
                                                                          quality_d,
                                                                          params_.minHitsForSharingCut_,
                                                                          params_.dupPassThrough_,
                                                                          device_hitToTuple_.get());

    if (params_.useSimpleTripletCleaner_) {
      kernel_simpleTripletCleaner<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                                                tuples_d,
                                                                                tracks_d,
                                                                                quality_d,
                                                                                params_.minHitsForSharingCut_,
                                                                                params_.dupPassThrough_,
                                                                                device_hitToTuple_.get());
    } else {
      kernel_tripletCleaner<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                                          tuples_d,
                                                                          tracks_d,
                                                                          quality_d,
                                                                          params_.minHitsForSharingCut_,
                                                                          params_.dupPassThrough_,
                                                                          device_hitToTuple_.get());
    }
    cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
    cudaCheck(hipDeviceSynchronize());
#endif
  }

  if (params_.doStats_) {
    numberOfBlocks = (std::max(nhits, int(params_.maxNumberOfDoublets_)) + blockSize - 1) / blockSize;
    kernel_checkOverflows<<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d,
                                                                        device_tupleMultiplicity_.get(),
                                                                        device_hitToTuple_.get(),
                                                                        device_hitTuple_apc_,
                                                                        device_theCells_.get(),
                                                                        device_nCells_,
                                                                        device_theCellNeighbors_.get(),
                                                                        device_theCellTracks_.get(),
                                                                        device_isOuterHitOfCell_.get(),
                                                                        nhits,
                                                                        params_.maxNumberOfDoublets_,
                                                                        counters_);
    cudaCheck(hipGetLastError());
  }

  if (params_.doStats_) {
    // counters (add flag???)
    numberOfBlocks = (hitToTupleView_.offSize + blockSize - 1) / blockSize;
    kernel_doStatsForHitInTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(device_hitToTuple_.get(), counters_);
    cudaCheck(hipGetLastError());
    numberOfBlocks = (3 * caConstants::maxNumberOfQuadruplets / 4 + blockSize - 1) / blockSize;
    kernel_doStatsForTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, counters_);
    cudaCheck(hipGetLastError());
  }
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

#ifdef DUMP_GPU_TK_TUPLES
  static std::atomic<int> iev(0);
  ++iev;
  kernel_print_found_ntuplets<<<1, 32, 0, cudaStream>>>(
      hh.view(), tuples_d, tracks_d, quality_d, device_hitToTuple_.get(), 100, iev);
#endif
}

template <>
void CAHitNtupletGeneratorKernelsGPU::printCounters(Counters const *counters) {
  kernel_printCounters<<<1, 1>>>(counters);
}
