#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include "RiemannFitOnGPU.h"
#include "RecoPixelVertexing/PixelTrackFitting/interface/RiemannFit.h"

#include <cstdint>
#include <hip/hip_runtime.h>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cuda_assert.h"
#include "RecoLocalTracker/SiPixelRecHits/interface/pixelCPEforGPU.h"
#include "RecoLocalTracker/SiPixelRecHits/plugins/siPixelRecHitsHeterogeneousProduct.h"


using HitsOnCPU = siPixelRecHitsHeterogeneousProduct::HitsOnCPU;

using HitsOnGPU = siPixelRecHitsHeterogeneousProduct::HitsOnGPU;
using TuplesOnGPU = pixelTuplesHeterogeneousProduct::TuplesOnGPU;

using namespace Eigen;

__global__
void kernelFastFitAllHits(TuplesOnGPU::Container const * __restrict__ foundNtuplets,
    HitsOnGPU const * __restrict__ hhp,
    int hits_in_fit,
    double * __restrict__ phits,
    float * __restrict__ phits_ge,
    double * __restrict__ pfast_fit,
    uint32_t offset)
{

  assert(hits_in_fit==4); // FixMe later template

  assert(pfast_fit); assert(foundNtuplets);

  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);
  auto helix_start = local_start + offset;

  if (helix_start>=foundNtuplets->nbins()) return;
  if (foundNtuplets->size(helix_start)<hits_in_fit) {
    return;
  }

  Rfit::Map3x4d hits(phits+local_start);
  Rfit::Map4d   fast_fit(pfast_fit+local_start);
  Rfit::Map6x4f hits_ge(phits_ge+local_start);

  // Prepare data structure
  auto const * hitId = foundNtuplets->begin(helix_start);
  for (unsigned int i = 0; i < hits_in_fit; ++i) {
    auto hit = hitId[i];
    // printf("Hit global: %f,%f,%f\n", hhp->xg_d[hit],hhp->yg_d[hit],hhp->zg_d[hit]);
    float ge[6];
    hhp->cpeParams->detParams(hhp->detInd_d[hit]).frame.toGlobal(hhp->xerr_d[hit], 0, hhp->yerr_d[hit], ge);
    // printf("Error: %d: %f,%f,%f,%f,%f,%f\n",hhp->detInd_d[hit],ge[0],ge[1],ge[2],ge[3],ge[4],ge[5]);

    hits.col(i) << hhp->xg_d[hit], hhp->yg_d[hit], hhp->zg_d[hit];
    hits_ge.col(i) << ge[0],ge[1],ge[2],ge[3],ge[4],ge[5];
  }
  Rfit::Fast_fit(hits,fast_fit);

  // no NaN here....
  assert(fast_fit(0)==fast_fit(0));
  assert(fast_fit(1)==fast_fit(1));
  assert(fast_fit(2)==fast_fit(2));
  assert(fast_fit(3)==fast_fit(3));

}

__global__
void kernelCircleFitAllHits(TuplesOnGPU::Container const * __restrict__ foundNtuplets,
    int hits_in_fit,
    double B,
    double * __restrict__ phits,
    float * __restrict__ phits_ge,
    double * __restrict__ pfast_fit_input,
    Rfit::circle_fit *circle_fit,
    uint32_t offset)
{
  assert(circle_fit); 

  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);
  auto helix_start = local_start + offset;

  if (helix_start>=foundNtuplets->nbins()) return;
  if (foundNtuplets->size(helix_start)<hits_in_fit) {
    return;
  }

  Rfit::Map3x4d hits(phits+local_start);
  Rfit::Map4d   fast_fit(pfast_fit_input+local_start);
  Rfit::Map6x4f hits_ge(phits_ge+local_start);

  constexpr uint32_t N = Rfit::Map3x4d::ColsAtCompileTime;
  constexpr auto n = N;

  assert(4==n); // later will be templated...

  Rfit::VectorNd<N> rad = (hits.block(0, 0, 2, n).colwise().norm());

  Rfit::Matrix2Nd<N> hits_cov =  Rfit::Matrix2Nd<4>::Zero();
  Rfit::loadCovariance2D(hits_ge,hits_cov);

  circle_fit[local_start] =
      Rfit::Circle_fit(hits.block(0, 0, 2, n),
                       hits_cov,
                       fast_fit, rad, B, true);

#ifdef GPU_DEBUG
//  printf("kernelCircleFitAllHits circle.par(0,1,2): %d %f,%f,%f\n", helix_start, 
//         circle_fit[local_start].par(0), circle_fit[local_start].par(1), circle_fit[local_start].par(2));
#endif
}

__global__
void kernelLineFitAllHits(TuplesOnGPU::Container const * __restrict__ foundNtuplets,
    int hits_in_fit,
    double B,
    Rfit::helix_fit *results,
    double * __restrict__ phits,
    float * __restrict__ phits_ge,
    double * __restrict__ pfast_fit,
    Rfit::circle_fit * __restrict__ circle_fit,
    uint32_t offset)
{

  assert(results); assert(circle_fit);

  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);
  auto helix_start = local_start + offset;

  if (helix_start>=foundNtuplets->nbins()) return;
  if (foundNtuplets->size(helix_start)<hits_in_fit) {
    return;
  }

  Rfit::Map3x4d hits(phits+local_start);
  Rfit::Map4d   fast_fit(pfast_fit+local_start);
  Rfit::Map6x4f hits_ge(phits_ge+local_start);
  auto const & line_fit = Rfit::Line_fit(hits, hits_ge, circle_fit[local_start], fast_fit, B, true);

  par_uvrtopak(circle_fit[local_start], B, true);

  // Grab helix_fit from the proper location in the output vector
  auto & helix = results[helix_start];
  helix.par << circle_fit[local_start].par, line_fit.par;

  // TODO: pass properly error booleans

  helix.cov = Rfit::Matrix5d::Zero();
  helix.cov.block(0, 0, 3, 3) = circle_fit[local_start].cov;
  helix.cov.block(3, 3, 2, 2) = line_fit.cov;

  helix.q = circle_fit[local_start].q;
  helix.chi2_circle = circle_fit[local_start].chi2;
  helix.chi2_line = line_fit.chi2;

#ifdef GPU_DEBUG
  printf("kernelLineFitAllHits circle.par(0,1,2): %d %f,%f,%f\n", helix_start,
         circle_fit[local_start].par(0), circle_fit[local_start].par(1), circle_fit[local_start].par(2));
  printf("kernelLineFitAllHits line.par(0,1): %d %f,%f\n", helix_start, line_fit.par(0),line_fit.par(1));
  printf("kernelLineFitAllHits chi2 cov %f/%f %f,%f,%f,%f,%f\n",helix.chi2_circle,helix.chi2_line, 
         helix.cov(0,0),helix.cov(1,1),helix.cov(2,2),helix.cov(3,3),helix.cov(4,4));
#endif
}


void RiemannFitOnGPU::launchKernels(HitsOnCPU const & hh, uint32_t nhits, uint32_t maxNumberOfTuples, hipStream_t cudaStream)
{
    assert(tuples_d); assert(fast_fit_resultsGPU_);

    auto blockSize = 128;
    auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

    for (uint32_t offset=0; offset<maxNumberOfTuples; offset+=maxNumberOfConcurrentFits_) {
      kernelFastFitAllHits<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
          tuples_d, hh.gpu_d, 4,
          hitsGPU_, hits_geGPU_, fast_fit_resultsGPU_,offset);
      cudaCheck(hipGetLastError());

      kernelCircleFitAllHits<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
          tuples_d, 4, bField_,
          hitsGPU_, hits_geGPU_, fast_fit_resultsGPU_, circle_fit_resultsGPU_, offset);
      cudaCheck(hipGetLastError());


      kernelLineFitAllHits<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
             tuples_d, 4,  bField_, helix_fit_results_d,
             hitsGPU_, hits_geGPU_, fast_fit_resultsGPU_, circle_fit_resultsGPU_,
             offset);
      cudaCheck(hipGetLastError());
    }
}
