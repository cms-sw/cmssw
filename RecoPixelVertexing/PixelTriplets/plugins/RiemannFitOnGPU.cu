#include "RiemannFitOnGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"

void HelixFitOnGPU::launchRiemannKernels(HitsView const *hv,
                                         uint32_t nhits,
                                         uint32_t maxNumberOfTuples,
                                         hipStream_t stream) {
  assert(tuples_d);

  auto blockSize = 64;
  auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

  //  Fit internals
  auto hitsGPU_ = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix3xNd<4>) / sizeof(double), stream);
  auto hits_geGPU_ = cms::cuda::make_device_unique<float[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix6x4f) / sizeof(float), stream);
  auto fast_fit_resultsGPU_ = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Vector4d) / sizeof(double), stream);
  auto circle_fit_resultsGPU_holder =
      cms::cuda::make_device_unique<char[]>(maxNumberOfConcurrentFits_ * sizeof(Rfit::circle_fit), stream);
  Rfit::circle_fit *circle_fit_resultsGPU_ = (Rfit::circle_fit *)(circle_fit_resultsGPU_holder.get());

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // triplets
    kernelFastFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(
        tuples_d, tupleMultiplicity_d, 3, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), offset);
    cudaCheck(hipGetLastError());

    kernelCircleFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                 3,
                                                                 bField_,
                                                                 hitsGPU_.get(),
                                                                 hits_geGPU_.get(),
                                                                 fast_fit_resultsGPU_.get(),
                                                                 circle_fit_resultsGPU_,
                                                                 offset);
    cudaCheck(hipGetLastError());

    kernelLineFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                               3,
                                                               bField_,
                                                               outputSoa_d,
                                                               hitsGPU_.get(),
                                                               hits_geGPU_.get(),
                                                               fast_fit_resultsGPU_.get(),
                                                               circle_fit_resultsGPU_,
                                                               offset);
    cudaCheck(hipGetLastError());

    // quads
    kernelFastFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
        tuples_d, tupleMultiplicity_d, 4, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), offset);
    cudaCheck(hipGetLastError());

    kernelCircleFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                     4,
                                                                     bField_,
                                                                     hitsGPU_.get(),
                                                                     hits_geGPU_.get(),
                                                                     fast_fit_resultsGPU_.get(),
                                                                     circle_fit_resultsGPU_,
                                                                     offset);
    cudaCheck(hipGetLastError());

    kernelLineFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                   4,
                                                                   bField_,
                                                                   outputSoa_d,
                                                                   hitsGPU_.get(),
                                                                   hits_geGPU_.get(),
                                                                   fast_fit_resultsGPU_.get(),
                                                                   circle_fit_resultsGPU_,
                                                                   offset);
    cudaCheck(hipGetLastError());

    if (fit5as4_) {
      // penta
      kernelFastFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
          tuples_d, tupleMultiplicity_d, 5, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), offset);
      cudaCheck(hipGetLastError());

      kernelCircleFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                       5,
                                                                       bField_,
                                                                       hitsGPU_.get(),
                                                                       hits_geGPU_.get(),
                                                                       fast_fit_resultsGPU_.get(),
                                                                       circle_fit_resultsGPU_,
                                                                       offset);
      cudaCheck(hipGetLastError());

      kernelLineFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                     5,
                                                                     bField_,
                                                                     outputSoa_d,
                                                                     hitsGPU_.get(),
                                                                     hits_geGPU_.get(),
                                                                     fast_fit_resultsGPU_.get(),
                                                                     circle_fit_resultsGPU_,
                                                                     offset);
      cudaCheck(hipGetLastError());
    } else {
      // penta all 5
      kernelFastFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
          tuples_d, tupleMultiplicity_d, 5, hv, hitsGPU_.get(), hits_geGPU_.get(), fast_fit_resultsGPU_.get(), offset);
      cudaCheck(hipGetLastError());

      kernelCircleFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                       5,
                                                                       bField_,
                                                                       hitsGPU_.get(),
                                                                       hits_geGPU_.get(),
                                                                       fast_fit_resultsGPU_.get(),
                                                                       circle_fit_resultsGPU_,
                                                                       offset);
      cudaCheck(hipGetLastError());

      kernelLineFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_d,
                                                                     5,
                                                                     bField_,
                                                                     outputSoa_d,
                                                                     hitsGPU_.get(),
                                                                     hits_geGPU_.get(),
                                                                     fast_fit_resultsGPU_.get(),
                                                                     circle_fit_resultsGPU_,
                                                                     offset);
      cudaCheck(hipGetLastError());
    }
  }
}
