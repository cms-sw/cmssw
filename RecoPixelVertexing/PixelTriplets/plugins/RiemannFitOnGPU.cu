#include "RiemannFitOnGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"

void HelixFitOnGPU::launchRiemannKernels(HitsView const *hv,
                                         uint32_t nhits,
                                         uint32_t maxNumberOfTuples,
                                         hipStream_t stream) {
  assert(tuples_);

  auto blockSize = 64;
  auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

  //  Fit internals
  auto hitsGPU = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Matrix3xNd<4>) / sizeof(double), stream);
  auto hits_geGPU = cms::cuda::make_device_unique<float[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Matrix6x4f) / sizeof(float), stream);
  auto fast_fit_resultsGPU = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Vector4d) / sizeof(double), stream);
  auto circle_fit_resultsGPU_holder =
      cms::cuda::make_device_unique<char[]>(maxNumberOfConcurrentFits_ * sizeof(riemannFit::CircleFit), stream);
  riemannFit::CircleFit *circle_fit_resultsGPU_ = (riemannFit::CircleFit *)(circle_fit_resultsGPU_holder.get());

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // triplets
    kernel_FastFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(
        tuples_, tupleMultiplicity_, 3, hv, hitsGPU.get(), hits_geGPU.get(), fast_fit_resultsGPU.get(), offset);
    cudaCheck(hipGetLastError());

    kernel_CircleFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                  3,
                                                                  bField_,
                                                                  hitsGPU.get(),
                                                                  hits_geGPU.get(),
                                                                  fast_fit_resultsGPU.get(),
                                                                  circle_fit_resultsGPU_,
                                                                  offset);
    cudaCheck(hipGetLastError());

    kernel_LineFit<3><<<numberOfBlocks, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                3,
                                                                bField_,
                                                                outputSoa_,
                                                                hitsGPU.get(),
                                                                hits_geGPU.get(),
                                                                fast_fit_resultsGPU.get(),
                                                                circle_fit_resultsGPU_,
                                                                offset);
    cudaCheck(hipGetLastError());

    // quads
    kernel_FastFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
        tuples_, tupleMultiplicity_, 4, hv, hitsGPU.get(), hits_geGPU.get(), fast_fit_resultsGPU.get(), offset);
    cudaCheck(hipGetLastError());

    kernel_CircleFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                      4,
                                                                      bField_,
                                                                      hitsGPU.get(),
                                                                      hits_geGPU.get(),
                                                                      fast_fit_resultsGPU.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
    cudaCheck(hipGetLastError());

    kernel_LineFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                    4,
                                                                    bField_,
                                                                    outputSoa_,
                                                                    hitsGPU.get(),
                                                                    hits_geGPU.get(),
                                                                    fast_fit_resultsGPU.get(),
                                                                    circle_fit_resultsGPU_,
                                                                    offset);
    cudaCheck(hipGetLastError());

    if (fit5as4_) {
      // penta
      kernel_FastFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
          tuples_, tupleMultiplicity_, 5, hv, hitsGPU.get(), hits_geGPU.get(), fast_fit_resultsGPU.get(), offset);
      cudaCheck(hipGetLastError());

      kernel_CircleFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                        5,
                                                                        bField_,
                                                                        hitsGPU.get(),
                                                                        hits_geGPU.get(),
                                                                        fast_fit_resultsGPU.get(),
                                                                        circle_fit_resultsGPU_,
                                                                        offset);
      cudaCheck(hipGetLastError());

      kernel_LineFit<4><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                      5,
                                                                      bField_,
                                                                      outputSoa_,
                                                                      hitsGPU.get(),
                                                                      hits_geGPU.get(),
                                                                      fast_fit_resultsGPU.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
      cudaCheck(hipGetLastError());
    } else {
      // penta all 5
      kernel_FastFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(
          tuples_, tupleMultiplicity_, 5, hv, hitsGPU.get(), hits_geGPU.get(), fast_fit_resultsGPU.get(), offset);
      cudaCheck(hipGetLastError());

      kernel_CircleFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                        5,
                                                                        bField_,
                                                                        hitsGPU.get(),
                                                                        hits_geGPU.get(),
                                                                        fast_fit_resultsGPU.get(),
                                                                        circle_fit_resultsGPU_,
                                                                        offset);
      cudaCheck(hipGetLastError());

      kernel_LineFit<5><<<numberOfBlocks / 4, blockSize, 0, stream>>>(tupleMultiplicity_,
                                                                      5,
                                                                      bField_,
                                                                      outputSoa_,
                                                                      hitsGPU.get(),
                                                                      hits_geGPU.get(),
                                                                      fast_fit_resultsGPU.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
      cudaCheck(hipGetLastError());
    }
  }
}
