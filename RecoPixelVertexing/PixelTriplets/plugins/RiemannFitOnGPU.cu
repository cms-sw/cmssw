#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include <cstdint>

#include <hip/hip_runtime.h>

#include "CUDADataFormats/TrackingRecHit/interface/TrackingRecHit2DCUDA.h"
#include "FWCore/ServiceRegistry/interface/Service.h"
#include "HeterogeneousCore/CUDAServices/interface/CUDAService.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cuda_assert.h"
#include "RecoLocalTracker/SiPixelRecHits/interface/pixelCPEforGPU.h"
#include "RecoPixelVertexing/PixelTrackFitting/interface/RiemannFit.h"

#include "HelixFitOnGPU.h"

using HitsOnGPU = TrackingRecHit2DSOAView;
using TuplesOnGPU = pixelTuplesHeterogeneousProduct::TuplesOnGPU;

using namespace Eigen;

template <int N>
__global__ void kernelFastFit(TuplesOnGPU::Container const *__restrict__ foundNtuplets,
                              CAConstants::TupleMultiplicity const *__restrict__ tupleMultiplicity,
                              uint32_t nHits,
                              HitsOnGPU const *__restrict__ hhp,
                              double *__restrict__ phits,
                              float *__restrict__ phits_ge,
                              double *__restrict__ pfast_fit,
                              uint32_t offset) {
  constexpr uint32_t hitsInFit = N;

  assert(hitsInFit <= nHits);

  assert(pfast_fit);
  assert(foundNtuplets);
  assert(tupleMultiplicity);

  // look in bin for this hit multiplicity
  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);

#ifdef RIEMANN_DEBUG
  if (0 == local_start)
    printf("%d Ntuple of size %d for %d hits to fit\n", tupleMultiplicity->size(nHits), nHits, hitsInFit);
#endif

  auto tuple_start = local_start + offset;
  if (tuple_start >= tupleMultiplicity->size(nHits))
    return;

  // get it from the ntuple container (one to one to helix)
  auto helix_start = *(tupleMultiplicity->begin(nHits) + tuple_start);
  assert(helix_start < foundNtuplets->nbins());

  assert(foundNtuplets->size(helix_start) == nHits);

  Rfit::Map3xNd<N> hits(phits + local_start);
  Rfit::Map4d fast_fit(pfast_fit + local_start);
  Rfit::Map6xNf<N> hits_ge(phits_ge + local_start);

  // Prepare data structure
  auto const *hitId = foundNtuplets->begin(helix_start);
  for (unsigned int i = 0; i < hitsInFit; ++i) {
    auto hit = hitId[i];
    // printf("Hit global: %f,%f,%f\n", hhp->xg_d[hit],hhp->yg_d[hit],hhp->zg_d[hit]);
    float ge[6];
    hhp->cpeParams().detParams(hhp->detectorIndex(hit)).frame.toGlobal(hhp->xerrLocal(hit), 0, hhp->yerrLocal(hit), ge);
    // printf("Error: %d: %f,%f,%f,%f,%f,%f\n",hhp->detInd_d[hit],ge[0],ge[1],ge[2],ge[3],ge[4],ge[5]);

    hits.col(i) << hhp->xGlobal(hit), hhp->yGlobal(hit), hhp->zGlobal(hit);
    hits_ge.col(i) << ge[0], ge[1], ge[2], ge[3], ge[4], ge[5];
  }
  Rfit::Fast_fit(hits, fast_fit);

  // no NaN here....
  assert(fast_fit(0) == fast_fit(0));
  assert(fast_fit(1) == fast_fit(1));
  assert(fast_fit(2) == fast_fit(2));
  assert(fast_fit(3) == fast_fit(3));
}

template <int N>
__global__ void kernelCircleFit(CAConstants::TupleMultiplicity const *__restrict__ tupleMultiplicity,
                                uint32_t nHits,
                                double B,
                                double *__restrict__ phits,
                                float *__restrict__ phits_ge,
                                double *__restrict__ pfast_fit_input,
                                Rfit::circle_fit *circle_fit,
                                uint32_t offset) {
  assert(circle_fit);
  assert(N <= nHits);

  // same as above...

  // look in bin for this hit multiplicity
  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);
  auto tuple_start = local_start + offset;
  if (tuple_start >= tupleMultiplicity->size(nHits))
    return;

  // get it for the ntuple container (one to one to helix)
  auto helix_start = *(tupleMultiplicity->begin(nHits) + tuple_start);

  Rfit::Map3xNd<N> hits(phits + local_start);
  Rfit::Map4d fast_fit(pfast_fit_input + local_start);
  Rfit::Map6xNf<N> hits_ge(phits_ge + local_start);

  Rfit::VectorNd<N> rad = (hits.block(0, 0, 2, N).colwise().norm());

  Rfit::Matrix2Nd<N> hits_cov = Rfit::Matrix2Nd<N>::Zero();
  Rfit::loadCovariance2D(hits_ge, hits_cov);

  circle_fit[local_start] = Rfit::Circle_fit(hits.block(0, 0, 2, N), hits_cov, fast_fit, rad, B, true);

#ifdef RIEMANN_DEBUG
//  printf("kernelCircleFit circle.par(0,1,2): %d %f,%f,%f\n", helix_start,
//         circle_fit[local_start].par(0), circle_fit[local_start].par(1), circle_fit[local_start].par(2));
#endif
}

template <int N>
__global__ void kernelLineFit(CAConstants::TupleMultiplicity const *__restrict__ tupleMultiplicity,
                              uint32_t nHits,
                              double B,
                              Rfit::helix_fit *results,
                              double *__restrict__ phits,
                              float *__restrict__ phits_ge,
                              double *__restrict__ pfast_fit_input,
                              Rfit::circle_fit *__restrict__ circle_fit,
                              uint32_t offset) {
  assert(results);
  assert(circle_fit);
  assert(N <= nHits);

  // same as above...

  // look in bin for this hit multiplicity
  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);
  auto tuple_start = local_start + offset;
  if (tuple_start >= tupleMultiplicity->size(nHits))
    return;

  // get it for the ntuple container (one to one to helix)
  auto helix_start = *(tupleMultiplicity->begin(nHits) + tuple_start);

  Rfit::Map3xNd<N> hits(phits + local_start);
  Rfit::Map4d fast_fit(pfast_fit_input + local_start);
  Rfit::Map6xNf<N> hits_ge(phits_ge + local_start);

  auto const &line_fit = Rfit::Line_fit(hits, hits_ge, circle_fit[local_start], fast_fit, B, true);

  par_uvrtopak(circle_fit[local_start], B, true);

  // Grab helix_fit from the proper location in the output vector
  auto &helix = results[helix_start];
  helix.par << circle_fit[local_start].par, line_fit.par;

  // TODO: pass properly error booleans

  helix.cov = Rfit::Matrix5d::Zero();
  helix.cov.block(0, 0, 3, 3) = circle_fit[local_start].cov;
  helix.cov.block(3, 3, 2, 2) = line_fit.cov;

  helix.q = circle_fit[local_start].q;
  helix.chi2_circle = circle_fit[local_start].chi2;
  helix.chi2_line = line_fit.chi2;

#ifdef RIEMANN_DEBUG
  printf("kernelLineFit size %d for %d hits circle.par(0,1,2): %d %f,%f,%f\n",
         N,
         nHits,
         helix_start,
         circle_fit[local_start].par(0),
         circle_fit[local_start].par(1),
         circle_fit[local_start].par(2));
  printf("kernelLineFit line.par(0,1): %d %f,%f\n", helix_start, line_fit.par(0), line_fit.par(1));
  printf("kernelLineFit chi2 cov %f/%f %e,%e,%e,%e,%e\n",
         helix.chi2_circle,
         helix.chi2_line,
         helix.cov(0, 0),
         helix.cov(1, 1),
         helix.cov(2, 2),
         helix.cov(3, 3),
         helix.cov(4, 4));
#endif
}

void HelixFitOnGPU::launchRiemannKernels(HitsOnCPU const &hh,
                                         uint32_t nhits,
                                         uint32_t maxNumberOfTuples,
                                         cuda::stream_t<> &stream) {
  assert(tuples_d);

  auto blockSize = 64;
  auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

  //  Fit internals
  edm::Service<CUDAService> cs;
  auto hitsGPU_ = cs->make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix3xNd<4>) / sizeof(double), stream);
  auto hits_geGPU_ =
      cs->make_device_unique<float[]>(maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix6x4f) / sizeof(float), stream);
  auto fast_fit_resultsGPU_ =
      cs->make_device_unique<double[]>(maxNumberOfConcurrentFits_ * sizeof(Rfit::Vector4d) / sizeof(double), stream);
  auto circle_fit_resultsGPU_holder =
      cs->make_device_unique<char[]>(maxNumberOfConcurrentFits_ * sizeof(Rfit::circle_fit), stream);
  Rfit::circle_fit *circle_fit_resultsGPU_ = (Rfit::circle_fit *)(circle_fit_resultsGPU_holder.get());

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // triplets
    kernelFastFit<3><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                    tupleMultiplicity_d,
                                                                    3,
                                                                    hh.view(),
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    offset);
    cudaCheck(hipGetLastError());

    kernelCircleFit<3><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      3,
                                                                      bField_,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
    cudaCheck(hipGetLastError());

    kernelLineFit<3><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                    3,
                                                                    bField_,
                                                                    helix_fit_results_d,
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    circle_fit_resultsGPU_,
                                                                    offset);
    cudaCheck(hipGetLastError());

    // quads
    kernelFastFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                    tupleMultiplicity_d,
                                                                    4,
                                                                    hh.view(),
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    offset);
    cudaCheck(hipGetLastError());

    kernelCircleFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      4,
                                                                      bField_,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
    cudaCheck(hipGetLastError());

    kernelLineFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                    4,
                                                                    bField_,
                                                                    helix_fit_results_d,
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    circle_fit_resultsGPU_,
                                                                    offset);
    cudaCheck(hipGetLastError());

    if (fit5as4_) {
      // penta
      kernelFastFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                      tupleMultiplicity_d,
                                                                      5,
                                                                      hh.view(),
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      offset);
      cudaCheck(hipGetLastError());

      kernelCircleFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                        5,
                                                                        bField_,
                                                                        hitsGPU_.get(),
                                                                        hits_geGPU_.get(),
                                                                        fast_fit_resultsGPU_.get(),
                                                                        circle_fit_resultsGPU_,
                                                                        offset);
      cudaCheck(hipGetLastError());

      kernelLineFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      5,
                                                                      bField_,
                                                                      helix_fit_results_d,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
      cudaCheck(hipGetLastError());
    } else {
      // penta all 5
      kernelFastFit<5><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                      tupleMultiplicity_d,
                                                                      5,
                                                                      hh.view(),
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      offset);
      cudaCheck(hipGetLastError());

      kernelCircleFit<5><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                        5,
                                                                        bField_,
                                                                        hitsGPU_.get(),
                                                                        hits_geGPU_.get(),
                                                                        fast_fit_resultsGPU_.get(),
                                                                        circle_fit_resultsGPU_,
                                                                        offset);
      cudaCheck(hipGetLastError());

      kernelLineFit<5><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      5,
                                                                      bField_,
                                                                      helix_fit_results_d,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
      cudaCheck(hipGetLastError());
    }
  }
}
