#include "hip/hip_runtime.h"
#include "gpuClusterTracks.h"
#include "gpuFitVertices.h"
#include "gpuSortByPt2.h"
#include "gpuSplitVertices.h"

namespace gpuVertexFinder {


  void Producer::allocateOnGPU() {
    cudaCheck(hipMalloc(&onGPU.ntrks, sizeof(uint32_t)));
    cudaCheck(hipMemset(onGPU.ntrks, 0, sizeof(uint32_t)));
    cudaCheck(hipMalloc(&onGPU.itrk, OnGPU::MAXTRACKS*sizeof(uint16_t)));
    cudaCheck(hipMalloc(&onGPU.zt, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ezt2, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ptt2, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.iv, OnGPU::MAXTRACKS*sizeof(int32_t)));

    cudaCheck(hipMalloc(&onGPU.nvFinal, sizeof(uint32_t)));
    cudaCheck(hipMalloc(&onGPU.nvIntermediate, sizeof(uint32_t)));
    cudaCheck(hipMalloc(&onGPU.zv, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.wv, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.chi2, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ptv2, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.sortInd, OnGPU::MAXVTX*sizeof(uint16_t)));


    cudaCheck(hipMalloc(&onGPU.izt, OnGPU::MAXTRACKS*sizeof(uint8_t)));
    cudaCheck(hipMalloc(&onGPU.nn, OnGPU::MAXTRACKS*sizeof(int32_t)));

    cudaCheck(hipMalloc(&onGPU_d,sizeof(OnGPU)));
    cudaCheck(hipMemcpy(onGPU_d,&onGPU,sizeof(OnGPU),hipMemcpyHostToDevice));

  }
	      
  void Producer::deallocateOnGPU() {
    cudaCheck(hipFree(onGPU.ntrks));
    cudaCheck(hipFree(onGPU.itrk));
    cudaCheck(hipFree(onGPU.zt));
    cudaCheck(hipFree(onGPU.ezt2));
    cudaCheck(hipFree(onGPU.ptt2));
    cudaCheck(hipFree(onGPU.iv));

    cudaCheck(hipFree(onGPU.nvFinal));
    cudaCheck(hipFree(onGPU.nvIntermediate));
    cudaCheck(hipFree(onGPU.zv));
    cudaCheck(hipFree(onGPU.wv));
    cudaCheck(hipFree(onGPU.chi2));
    cudaCheck(hipFree(onGPU.ptv2));
    cudaCheck(hipFree(onGPU.sortInd));

    cudaCheck(hipFree(onGPU.izt));
    cudaCheck(hipFree(onGPU.nn));

    cudaCheck(hipFree(onGPU_d));

  }

  
  __global__
  void loadTracks(pixelTuplesHeterogeneousProduct::TuplesOnGPU const * tracks,
                  OnGPU * pdata,
                  float ptMin
                 ){

    auto const & tuples = *tracks->tuples_d;
    auto const * fit = tracks->helix_fit_results_d;
    auto const * quality = tracks->quality_d;

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx>= tuples.nbins()) return;
    if (tuples.size(idx)==0) {
      return;
    }

    if(quality[idx] != pixelTuplesHeterogeneousProduct::loose ) return;

    auto const & fittedTrack = fit[idx];

    if (fittedTrack.par(2)<ptMin) return; 

    auto & data = *pdata;   
    auto it = atomicAdd(data.ntrks,1);
    data.itrk[it] = idx;
    data.zt[it] = fittedTrack.par(4);
    data.ezt2[it] = fittedTrack.cov(4, 4);
    data.ptt2[it] = fittedTrack.par(2)*fittedTrack.par(2);
 
  }

  void Producer::produce(hipStream_t stream, TuplesOnCPU const & tracks, float ptMin) {
    
    assert(onGPU_d);assert(tracks.gpu_d);
    cudaCheck(hipMemsetAsync(onGPU.ntrks, 0, sizeof(uint32_t),stream));
    auto blockSize = 128;
    auto numberOfBlocks = (CAConstants::maxTuples() + blockSize - 1) / blockSize;
    loadTracks<<<numberOfBlocks,blockSize,0,stream>>>(tracks.gpu_d,onGPU_d, ptMin);
    cudaCheck(hipGetLastError());

    clusterTracks<<<1,1024-256,0,stream>>>(onGPU_d,minT,eps,errmax,chi2max);
    cudaCheck(hipGetLastError());
    fitVertices<<<1,1024-256,0,stream>>>(onGPU_d,50.);
    cudaCheck(hipGetLastError());

    splitVertices<<<1024,128,0,stream>>>(onGPU_d,9.f);
    cudaCheck(hipGetLastError());
    fitVertices<<<1,1024-256,0,stream>>>(onGPU_d,5000.);
    cudaCheck(hipGetLastError());

    sortByPt2<<<1,256,0,stream>>>(onGPU_d);
    cudaCheck(hipGetLastError());

    if(enableTransfer) {
      cudaCheck(hipMemcpyAsync(&gpuProduct.nVertices, onGPU.nvFinal, sizeof(uint32_t),
                                hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(&gpuProduct.nTracks, onGPU.ntrks, sizeof(uint32_t),
                              hipMemcpyDeviceToHost, stream));
    }
  }
  
  Producer::OnCPU const & Producer::fillResults(hipStream_t stream) {

    if(!enableTransfer) return gpuProduct;

    // finish copy
    gpuProduct.ivtx.resize(gpuProduct.nTracks);
    cudaCheck(hipMemcpyAsync(gpuProduct.ivtx.data(),onGPU.iv,sizeof(int32_t)*gpuProduct.nTracks,
                              hipMemcpyDeviceToHost, stream));
    gpuProduct.itrk.resize(gpuProduct.nTracks);
    cudaCheck(hipMemcpyAsync(gpuProduct.itrk.data(),onGPU.itrk,sizeof(int16_t)*gpuProduct.nTracks,
                              hipMemcpyDeviceToHost, stream));

    gpuProduct.z.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.z.data(),onGPU.zv,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
    gpuProduct.zerr.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.zerr.data(),onGPU.wv,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
    gpuProduct.chi2.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.chi2.data(),onGPU.chi2,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
        
    gpuProduct.sortInd.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.sortInd.data(),onGPU.sortInd,sizeof(uint16_t)*gpuProduct.nVertices,
                              hipMemcpyDeviceToHost, stream));

    hipStreamSynchronize(stream);
    
    return gpuProduct;
  }
	
} // end namespace
