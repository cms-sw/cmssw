#include "gpuClusterTracks.h"

namespace gpuVertexFinder {


  void Producer::allocateOnGPU() {
    cudaCheck(hipMalloc(&onGPU.zt, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ezt2, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ptt2, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.iv, OnGPU::MAXTRACKS*sizeof(int32_t)));

    cudaCheck(hipMalloc(&onGPU.nv, sizeof(uint32_t)));
    cudaCheck(hipMalloc(&onGPU.zv, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.wv, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.chi2, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ptv2, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.sortInd, OnGPU::MAXVTX*sizeof(uint16_t)));


    cudaCheck(hipMalloc(&onGPU.izt, OnGPU::MAXTRACKS*sizeof(uint8_t)));
    cudaCheck(hipMalloc(&onGPU.nn, OnGPU::MAXTRACKS*sizeof(int32_t)));

    cudaCheck(hipMalloc(&onGPU_d,sizeof(OnGPU)));
    cudaCheck(hipMemcpy(onGPU_d,&onGPU,sizeof(OnGPU),hipMemcpyHostToDevice));

  }
	      
  void Producer::deallocateOnGPU() {
    cudaCheck(hipFree(onGPU.zt));
    cudaCheck(hipFree(onGPU.ezt2));
    cudaCheck(hipFree(onGPU.ptt2));
    cudaCheck(hipFree(onGPU.iv));

    cudaCheck(hipFree(onGPU.nv));
    cudaCheck(hipFree(onGPU.zv));
    cudaCheck(hipFree(onGPU.wv));
    cudaCheck(hipFree(onGPU.chi2));
    cudaCheck(hipFree(onGPU.ptv2));
    cudaCheck(hipFree(onGPU.sortInd));

    cudaCheck(hipFree(onGPU.izt));
    cudaCheck(hipFree(onGPU.nn));

    cudaCheck(hipFree(onGPU_d));

  }


  void Producer::produce(hipStream_t stream,
			 float const * __restrict__ zt,
			 float const * __restrict__ ezt2,
                         float const * __restrict__ ptt2,
			 uint32_t ntrks
			 ) {
    
    cudaCheck(hipMemcpyAsync(onGPU.zt,zt,ntrks*sizeof(float),
			      hipMemcpyHostToDevice,stream));
    cudaCheck(hipMemcpyAsync(onGPU.ezt2,ezt2,ntrks*sizeof(float),
			      hipMemcpyHostToDevice,stream));
    cudaCheck(hipMemcpyAsync(onGPU.ptt2,ptt2,ntrks*sizeof(float),
                              hipMemcpyHostToDevice,stream));
    
    assert(onGPU_d);
    clusterTracks<<<1,1024-256,0,stream>>>(ntrks,onGPU_d,minT,eps,errmax,chi2max);
    cudaCheck(hipGetLastError());
    sortByPt2<<<1,256,0,stream>>>(ntrks,onGPU_d);
    cudaCheck(hipGetLastError());

    cudaCheck(hipMemcpyAsync(&gpuProduct.nVertices, onGPU.nv, sizeof(uint32_t),
			      hipMemcpyDeviceToHost, stream));
    
    gpuProduct.ivtx.resize(ntrks);
    cudaCheck(hipMemcpyAsync(gpuProduct.ivtx.data(),onGPU.iv,sizeof(int32_t)*ntrks,
			      hipMemcpyDeviceToHost, stream));
  }
  
  Producer::GPUProduct const & Producer::fillResults(hipStream_t stream) {

    // finish copy
    gpuProduct.z.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.z.data(),onGPU.zv,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
    gpuProduct.zerr.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.zerr.data(),onGPU.wv,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
    gpuProduct.chi2.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.chi2.data(),onGPU.chi2,sizeof(float)*gpuProduct.nVertices,
			      hipMemcpyDeviceToHost, stream));
        
    gpuProduct.sortInd.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.sortInd.data(),onGPU.sortInd,sizeof(uint16_t)*gpuProduct.nVertices,
                              hipMemcpyDeviceToHost, stream));

    hipStreamSynchronize(stream);
    
    return gpuProduct;
  }
	
} // end namespace
