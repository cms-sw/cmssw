#include "hip/hip_runtime.h"
/* Sushil Dubey, Shashi Dugad, TIFR, July 2017
 *
 * File Name: RawToClusterGPU.cu
 * Description: It converts Raw data into Digi Format on GPU
 * then it converts adc -> electron and
 * applies the adc threshold to needed for clustering
 * Finaly the Output of RawToDigi data is given to pixelClusterizer
 *
**/

// C++ includes
#include <cassert>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>

// cub includes
#include <hipcub/hipcub.hpp>

// CMSSW includes
#include "FWCore/ServiceRegistry/interface/Service.h"
#include "HeterogeneousCore/CUDAServices/interface/CUDAService.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuCalibPixel.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClustering.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClusterChargeCut.h"
#include "RecoLocalTracker/SiPixelClusterizer/interface/SiPixelFedCablingMapGPU.h"

// local includes
#include "SiPixelRawToClusterGPUKernel.h"

namespace pixelgpudetails {

  // data structures size
  constexpr uint32_t vsize = sizeof(GPU::SimpleVector<pixelgpudetails::error_obj>);
  constexpr uint32_t esize = sizeof(pixelgpudetails::error_obj);

  // number of words for all the FEDs
  constexpr uint32_t MAX_FED_WORDS   = pixelgpudetails::MAX_FED * pixelgpudetails::MAX_WORD;
  constexpr uint32_t MAX_ERROR_SIZE  = MAX_FED_WORDS * esize;

  SiPixelRawToClusterGPUKernel::WordFedAppender::WordFedAppender(cuda::stream_t<>& cudaStream) {
    edm::Service<CUDAService> cs;
    word_ = cs->make_host_unique<unsigned int[]>(MAX_FED_WORDS, cudaStream);
    fedId_ = cs->make_host_unique<unsigned char[]>(MAX_FED_WORDS, cudaStream);
  }

  void SiPixelRawToClusterGPUKernel::WordFedAppender::initializeWordFed(int fedId, unsigned int wordCounterGPU, const cms_uint32_t *src, unsigned int length) {
    std::memcpy(word_.get()+wordCounterGPU, src, sizeof(cms_uint32_t)*length);
    std::memset(fedId_.get()+wordCounterGPU/2, fedId - 1200, length/2);
  }

  ////////////////////

  __device__ uint32_t getLink(uint32_t ww)  {
    return ((ww >> pixelgpudetails::LINK_shift) & pixelgpudetails::LINK_mask);
  }


  __device__ uint32_t getRoc(uint32_t ww) {
    return ((ww >> pixelgpudetails::ROC_shift ) & pixelgpudetails::ROC_mask);
  }


  __device__ uint32_t getADC(uint32_t ww) {
    return ((ww >> pixelgpudetails::ADC_shift) & pixelgpudetails::ADC_mask);
  }


  __device__ bool isBarrel(uint32_t rawId) {
    return (1==((rawId>>25)&0x7));
  }

  __device__ pixelgpudetails::DetIdGPU getRawId(const SiPixelFedCablingMapGPU * cablingMap, uint32_t fed, uint32_t link, uint32_t roc) {
    uint32_t index = fed * MAX_LINK * MAX_ROC + (link-1) * MAX_ROC + roc;
    pixelgpudetails::DetIdGPU detId = { cablingMap->RawId[index], cablingMap->rocInDet[index], cablingMap->moduleId[index] };
    return detId;
  }

  //reference http://cmsdoxygen.web.cern.ch/cmsdoxygen/CMSSW_9_2_0/doc/html/dd/d31/FrameConversion_8cc_source.html
  //http://cmslxr.fnal.gov/source/CondFormats/SiPixelObjects/src/PixelROC.cc?v=CMSSW_9_2_0#0071
  // Convert local pixel to pixelgpudetails::global pixel
  __device__ pixelgpudetails::Pixel frameConversion(bool bpix, int side, uint32_t layer, uint32_t rocIdInDetUnit, pixelgpudetails::Pixel local) {

    int slopeRow  = 0, slopeCol = 0;
    int rowOffset = 0, colOffset = 0;

    if (bpix) {

      if (side == -1 && layer != 1) { // -Z side: 4 non-flipped modules oriented like 'dddd', except Layer 1
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
        else {
          slopeRow  = -1;
          slopeCol  = 1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = (rocIdInDetUnit-8)*pixelgpudetails::numColsInRoc;
        } // if roc
      }
      else { // +Z side: 4 non-flipped modules oriented like 'pppp', but all 8 in layer1
        if (rocIdInDetUnit < 8) {
          slopeRow  = -1;
          slopeCol  =  1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = rocIdInDetUnit * pixelgpudetails::numColsInRoc;
        }
        else {
          slopeRow  = 1;
          slopeCol  = -1;
          rowOffset = 0;
          colOffset = (16-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
      }

    }
    else { // fpix
      if (side==-1) { // pannel 1
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
        else {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = (rocIdInDetUnit-8)*pixelgpudetails::numColsInRoc;
        }
      }
      else { // pannel 2
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
        else {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = (rocIdInDetUnit-8)*pixelgpudetails::numColsInRoc;
        }

      } // side

    }

    uint32_t gRow = rowOffset+slopeRow*local.row;
    uint32_t gCol = colOffset+slopeCol*local.col;
    //printf("Inside frameConversion row: %u, column: %u\n", gRow, gCol);
    pixelgpudetails::Pixel global = {gRow, gCol};
    return global;
  }


  __device__ uint32_t conversionError(uint32_t fedId, uint32_t status, bool debug = false)
  {
    uint32_t errorType = 0;

    // debug = true;

    switch (status) {
      case(1) : {
        if (debug) printf("Error in Fed: %i, invalid channel Id (errorType = 35\n)", fedId );
        errorType = 35;
        break;
      }
      case(2) : {
        if (debug) printf("Error in Fed: %i, invalid ROC Id (errorType = 36)\n", fedId);
        errorType = 36;
        break;
      }
      case(3) : {
        if (debug) printf("Error in Fed: %i, invalid dcol/pixel value (errorType = 37)\n", fedId);
        errorType = 37;
        break;
      }
      case(4) : {
        if (debug) printf("Error in Fed: %i, dcol/pixel read out of order (errorType = 38)\n", fedId);
        errorType = 38;
        break;
      }
      default:
        if (debug) printf("Cabling check returned unexpected result, status = %i\n", status);
    };

    return errorType;
  }

  __device__ bool rocRowColIsValid(uint32_t rocRow, uint32_t rocCol)
  {
    uint32_t numRowsInRoc = 80;
    uint32_t numColsInRoc = 52;

    /// row and collumn in ROC representation
    return ((rocRow < numRowsInRoc) & (rocCol < numColsInRoc));
  }

  __device__ bool dcolIsValid(uint32_t dcol, uint32_t pxid)
  {
    return ((dcol < 26) &  (2 <= pxid) & (pxid < 162));
  }

  __device__ uint32_t checkROC(uint32_t errorWord, uint32_t fedId, uint32_t link, const SiPixelFedCablingMapGPU *cablingMap, bool debug = false)
  {
    int errorType = (errorWord >> pixelgpudetails::ROC_shift) & pixelgpudetails::ERROR_mask;
    if (errorType < 25) return false;
    bool errorFound = false;

    switch (errorType) {
      case(25) : {
        errorFound = true;
        uint32_t index = fedId * MAX_LINK * MAX_ROC + (link-1) * MAX_ROC + 1;
        if (index > 1 && index <= cablingMap->size) {
          if (!(link == cablingMap->link[index] && 1 == cablingMap->roc[index])) errorFound = false;
        }
        if (debug&errorFound) printf("Invalid ROC = 25 found (errorType = 25)\n");
        break;
      }
      case(26) : {
        if (debug) printf("Gap word found (errorType = 26)\n");
        errorFound = true;
        break;
      }
      case(27) : {
        if (debug) printf("Dummy word found (errorType = 27)\n");
        errorFound = true;
        break;
      }
      case(28) : {
        if (debug) printf("Error fifo nearly full (errorType = 28)\n");
        errorFound = true;
        break;
      }
      case(29) : {
        if (debug) printf("Timeout on a channel (errorType = 29)\n");
        if ((errorWord >> pixelgpudetails::OMIT_ERR_shift) & pixelgpudetails::OMIT_ERR_mask) {
          if (debug) printf("...first errorType=29 error, this gets masked out\n");
        }
        errorFound = true;
        break;
      }
      case(30) : {
        if (debug) printf("TBM error trailer (errorType = 30)\n");
        int StateMatch_bits = 4;
        int StateMatch_shift = 8;
        uint32_t StateMatch_mask = ~(~uint32_t(0) << StateMatch_bits);
        int StateMatch = (errorWord >> StateMatch_shift) & StateMatch_mask;
        if ( StateMatch != 1 && StateMatch != 8 ) {
          if (debug) printf("FED error 30 with unexpected State Bits (errorType = 30)\n");
        }
        if ( StateMatch == 1 ) errorType = 40; // 1=Overflow -> 40, 8=number of ROCs -> 30
        errorFound = true;
        break;
      }
      case(31) : {
        if (debug) printf("Event number error (errorType = 31)\n");
        errorFound = true;
        break;
      }
      default:
        errorFound = false;
    };

    return errorFound? errorType : 0;
  }

  __device__ uint32_t getErrRawID(uint32_t fedId, uint32_t errWord, uint32_t errorType, const SiPixelFedCablingMapGPU *cablingMap, bool debug = false)
  {
    uint32_t rID = 0xffffffff;

    switch (errorType) {
      case 25 : case 30 : case 31 : case 36 : case 40 : {
        //set dummy values for cabling just to get detId from link
        //cabling.dcol = 0;
        //cabling.pxid = 2;
        uint32_t roc  = 1;
        uint32_t link = (errWord >> pixelgpudetails::LINK_shift) & pixelgpudetails::LINK_mask;
        uint32_t rID_temp = getRawId(cablingMap, fedId, link, roc).RawId;
        if (rID_temp != 9999) rID = rID_temp;
        break;
      }
      case 29 : {
        int chanNmbr = 0;
        const int DB0_shift = 0;
        const int DB1_shift = DB0_shift + 1;
        const int DB2_shift = DB1_shift + 1;
        const int DB3_shift = DB2_shift + 1;
        const int DB4_shift = DB3_shift + 1;
        const uint32_t DataBit_mask = ~(~uint32_t(0) << 1);

        int CH1 = (errWord >> DB0_shift) & DataBit_mask;
        int CH2 = (errWord >> DB1_shift) & DataBit_mask;
        int CH3 = (errWord >> DB2_shift) & DataBit_mask;
        int CH4 = (errWord >> DB3_shift) & DataBit_mask;
        int CH5 = (errWord >> DB4_shift) & DataBit_mask;
        int BLOCK_bits      = 3;
        int BLOCK_shift     = 8;
        uint32_t BLOCK_mask = ~(~uint32_t(0) << BLOCK_bits);
        int BLOCK = (errWord >> BLOCK_shift) & BLOCK_mask;
        int localCH = 1*CH1+2*CH2+3*CH3+4*CH4+5*CH5;
        if (BLOCK%2==0) chanNmbr=(BLOCK/2)*9+localCH;
        else chanNmbr = ((BLOCK-1)/2)*9+4+localCH;
        if ((chanNmbr < 1)||(chanNmbr > 36)) break;  // signifies unexpected result

        // set dummy values for cabling just to get detId from link if in Barrel
        //cabling.dcol = 0;
        //cabling.pxid = 2;
        uint32_t roc  = 1;
        uint32_t link = chanNmbr;
        uint32_t rID_temp = getRawId(cablingMap, fedId, link, roc).RawId;
        if(rID_temp != 9999) rID = rID_temp;
        break;
      }
      case 37 : case 38: {
        //cabling.dcol = 0;
        //cabling.pxid = 2;
        uint32_t roc  = (errWord >> pixelgpudetails::ROC_shift) & pixelgpudetails::ROC_mask;
        uint32_t link = (errWord >> pixelgpudetails::LINK_shift) & pixelgpudetails::LINK_mask;
        uint32_t rID_temp = getRawId(cablingMap, fedId, link, roc).RawId;
        if(rID_temp != 9999) rID = rID_temp;
        break;
      }
      default:
        break;
    };

    return rID;
  }

  /*----------
   * Name: applyADCthreshold_kernel()
   * Desc: converts adc count to electrons and then applies the
   * threshold on each channel.
   * make pixel to 0 if it is below the threshold
   * Input: xx_d[], yy_d[], layer_d[], wordCounter, adc[], ADCThreshold
   *-----------
   * Output: xx_adc[], yy_adc[] with pixel threshold applied
   */
  // kernel to apply adc threshold on the channels


  // Felice: gains and pedestals are not the same for each pixel. This code should be rewritten to take
  // in account local gains/pedestals
  // __global__ void applyADCthreshold_kernel(const uint32_t *xx_d, const uint32_t *yy_d, const uint32_t *layer_d, uint32_t *adc, const uint32_t wordCounter,
  //  const ADCThreshold adcThreshold, uint32_t *xx_adc, uint32_t *yy_adc ) {
  //   int tid = threadIdx.x;
  //   int gIndex = blockDim.x*blockIdx.x+tid;
  //   if (gIndex<wordCounter) {
  //     uint32_t adcOld = adc[gIndex];
  //     const float gain = adcThreshold.theElectronPerADCGain_; // default: 1 adc = 135 electrons
  //     const float pedestal = 0; //
  //     int adcNew = int(adcOld*gain+pedestal);
  //     // rare chance of entering into the if ()
  //     if (layer_d[gIndex]>=adcThreshold.theFirstStack_) {
  //       if (adcThreshold.theStackADC_==1 && adcOld==1) {
  //         adcNew = int(255*135); // Arbitrarily use overflow value.
  //       }
  //       if (adcThreshold.theStackADC_ >1 && adcThreshold.theStackADC_!=255 && adcOld>=1){
  //         adcNew = int((adcOld-1) * gain * 255/float(adcThreshold.theStackADC_-1));
  //       }
  //     }
  //
  //     if (adcNew >adcThreshold.thePixelThreshold ) {
  //       xx_adc[gIndex]=xx_d[gIndex];
  //       yy_adc[gIndex]=yy_d[gIndex];
  //     }
  //     else {
  //       xx_adc[gIndex]=0; // 0: dead pixel
  //       yy_adc[gIndex]=0;
  //     }
  //     adc[gIndex] = adcNew;
  //   }
  // }


  // Kernel to perform Raw to Digi conversion
  __global__ void RawToDigi_kernel(const SiPixelFedCablingMapGPU *cablingMap, const unsigned char *modToUnp,
      const uint32_t wordCounter, const uint32_t *word, const uint8_t *fedIds,
      uint16_t *xx, uint16_t *yy, uint16_t *adc,
      uint32_t *pdigi, uint32_t *rawIdArr, uint16_t *moduleId,
      GPU::SimpleVector<pixelgpudetails::error_obj> *err,
      bool useQualityInfo, bool includeErrors, bool debug)
  {
    //if (threadIdx.x==0) printf("Event: %u blockIdx.x: %u start: %u end: %u\n", eventno, blockIdx.x, begin, end);

    auto gIndex  = threadIdx.x + blockIdx.x * blockDim.x;
    xx[gIndex]   = 0;
    yy[gIndex]   = 0;
    adc[gIndex]  = 0;
    bool skipROC = false;

    do {  // too many coninue below.... (to be fixed)
      if (gIndex < wordCounter) {
        uint32_t fedId = fedIds[gIndex/2]; // +1200;

        // initialize (too many coninue below)
        pdigi[gIndex]  = 0;
        rawIdArr[gIndex] = 0;
        moduleId[gIndex] = 9999;

        uint32_t ww = word[gIndex]; // Array containing 32 bit raw data
        if (ww == 0) {
          // 0 is an indicator of a noise/dead channel, skip these pixels during clusterization
          continue;
        }

        uint32_t link  = getLink(ww);            // Extract link
        uint32_t roc   = getRoc(ww);             // Extract Roc in link
        pixelgpudetails::DetIdGPU detId = getRawId(cablingMap, fedId, link, roc);

        uint32_t errorType = checkROC(ww, fedId, link, cablingMap, debug);
        skipROC = (roc < pixelgpudetails::maxROCIndex) ? false : (errorType != 0);
        if (includeErrors and skipROC)
        {
          uint32_t rID = getErrRawID(fedId, ww, errorType, cablingMap, debug);
          err->push_back(pixelgpudetails::error_obj{rID, ww, errorType, fedId});
          continue;
        }

        uint32_t rawId  = detId.RawId;
        uint32_t rocIdInDetUnit = detId.rocInDet;
        bool barrel = isBarrel(rawId);

        uint32_t index = fedId * MAX_LINK * MAX_ROC + (link-1) * MAX_ROC + roc;
        if (useQualityInfo) {
          skipROC = cablingMap->badRocs[index];
          if (skipROC) continue;
        }
        skipROC = modToUnp[index];
        if (skipROC) continue;

        uint32_t layer = 0;//, ladder =0;
        int side = 0, panel = 0, module = 0;//disk = 0, blade = 0

        if (barrel)
        {
          layer  = (rawId >> pixelgpudetails::layerStartBit) & pixelgpudetails::layerMask;
          module = (rawId >> pixelgpudetails::moduleStartBit) & pixelgpudetails::moduleMask;
          side   = (module < 5)? -1 : 1;
        }
        else {
          // endcap ids
          layer = 0;
          panel = (rawId >> pixelgpudetails::panelStartBit) & pixelgpudetails::panelMask;
          //disk  = (rawId >> diskStartBit_) & diskMask_;
          side  = (panel == 1)? -1 : 1;
          //blade = (rawId >> bladeStartBit_) & bladeMask_;
        }

        // ***special case of layer to 1 be handled here
        pixelgpudetails::Pixel localPix;
        if (layer == 1) {
          uint32_t col = (ww >> pixelgpudetails::COL_shift) & pixelgpudetails::COL_mask;
          uint32_t row = (ww >> pixelgpudetails::ROW_shift) & pixelgpudetails::ROW_mask;
          localPix.row = row;
          localPix.col = col;
          if (includeErrors) {
            if (not rocRowColIsValid(row, col)) {
              uint32_t error = conversionError(fedId, 3, debug); //use the device function and fill the arrays
              err->push_back(pixelgpudetails::error_obj{rawId, ww, error, fedId});
              if(debug) printf("BPIX1  Error status: %i\n", error);
              continue;
            }
          }
        } else {
          // ***conversion rules for dcol and pxid
          uint32_t dcol = (ww >> pixelgpudetails::DCOL_shift) & pixelgpudetails::DCOL_mask;
          uint32_t pxid = (ww >> pixelgpudetails::PXID_shift) & pixelgpudetails::PXID_mask;
          uint32_t row  = pixelgpudetails::numRowsInRoc - pxid/2;
          uint32_t col  = dcol*2 + pxid%2;
          localPix.row = row;
          localPix.col = col;
          if (includeErrors and not dcolIsValid(dcol, pxid)) {
            uint32_t error = conversionError(fedId, 3, debug);
            err->push_back(pixelgpudetails::error_obj{rawId, ww, error, fedId});
            if(debug) printf("Error status: %i %d %d %d %d\n", error, dcol, pxid, fedId, roc);
            continue;
          }
        }

        pixelgpudetails::Pixel globalPix = frameConversion(barrel, side, layer, rocIdInDetUnit, localPix);
        xx[gIndex]    = globalPix.row;  // origin shifting by 1 0-159
        yy[gIndex]    = globalPix.col;  // origin shifting by 1 0-415
        adc[gIndex]   = getADC(ww);
        pdigi[gIndex] = pixelgpudetails::pack(globalPix.row, globalPix.col, adc[gIndex]);
        moduleId[gIndex] = detId.moduleId;
        rawIdArr[gIndex] = rawId;
      } // end of if (gIndex < end)
    } while (false); // end fake loop
  } // end of Raw to Digi kernel

  // Interface to outside
  void SiPixelRawToClusterGPUKernel::makeClustersAsync(
      const SiPixelFedCablingMapGPU *cablingMap,
      const unsigned char *modToUnp,
      const SiPixelGainForHLTonGPU *gains,
      const WordFedAppender& wordFed,
      const uint32_t wordCounter, const uint32_t fedCounter,
      bool convertADCtoElectrons,
      bool useQualityInfo, bool includeErrors, bool transferToCPU, bool debug,
      cuda::stream_t<>& stream)
  {
    nDigis = wordCounter;

    constexpr uint32_t MAX_FED_WORDS   = pixelgpudetails::MAX_FED * pixelgpudetails::MAX_WORD;
    digis_d = SiPixelDigisCUDA(MAX_FED_WORDS, stream);
    clusters_d = SiPixelClustersCUDA(MAX_FED_WORDS, gpuClustering::MaxNumModules, stream);

    edm::Service<CUDAService> cs;
    digis_clusters_h.nModules_Clusters = cs->make_host_unique<uint32_t[]>(2, stream);

    {
      const int threadsPerBlock = 512;
      const int blocks = (wordCounter + threadsPerBlock-1) /threadsPerBlock; // fill it all

      assert(0 == wordCounter%2);
      // wordCounter is the total no of words in each event to be trasfered on device
      auto word_d = cs->make_device_unique<uint32_t[]>(wordCounter, stream);
      auto fedId_d = cs->make_device_unique<uint8_t[]>(wordCounter, stream);

      auto error_d = cs->make_device_unique<GPU::SimpleVector<pixelgpudetails::error_obj>>(stream);
      auto data_d = cs->make_device_unique<pixelgpudetails::error_obj[]>(MAX_FED_WORDS, stream);
      cudaCheck(hipMemsetAsync(data_d.get(), 0x00, MAX_ERROR_SIZE, stream.id()));
      auto error_h_tmp = cs->make_host_unique<GPU::SimpleVector<pixelgpudetails::error_obj>>(stream);
      GPU::make_SimpleVector(error_h_tmp.get(), MAX_FED_WORDS, data_d.get());
      assert(error_h_tmp->size() == 0);
      assert(error_h_tmp->capacity() == static_cast<int>(MAX_FED_WORDS));

      cudaCheck(hipMemcpyAsync(word_d.get(),  wordFed.word(), wordCounter*sizeof(uint32_t),    hipMemcpyDefault, stream.id()));
      cudaCheck(hipMemcpyAsync(fedId_d.get(), wordFed.fedId(), wordCounter*sizeof(uint8_t) / 2, hipMemcpyDefault, stream.id()));
      cudaCheck(hipMemcpyAsync(error_d.get(), error_h_tmp.get(), vsize, hipMemcpyDefault, stream.id()));

      auto pdigi_d = cs->make_device_unique<uint32_t[]>(wordCounter, stream);
      auto rawIdArr_d = cs->make_device_unique<uint32_t[]>(wordCounter, stream);

      // Launch rawToDigi kernel
      RawToDigi_kernel<<<blocks, threadsPerBlock, 0, stream.id()>>>(
          cablingMap,
          modToUnp,
          wordCounter,
          word_d.get(),
          fedId_d.get(),
          digis_d.xx(), digis_d.yy(), digis_d.adc(),
          pdigi_d.get(),
          rawIdArr_d.get(),
          digis_d.moduleInd(),
          error_d.get(),
          useQualityInfo,
          includeErrors,
          debug);
      cudaCheck(hipGetLastError());

      // copy data to host variable
      if(transferToCPU) {
        digis_clusters_h.pdigi = cs->make_host_unique<uint32_t[]>(MAX_FED_WORDS, stream);
        digis_clusters_h.rawIdArr = cs->make_host_unique<uint32_t[]>(MAX_FED_WORDS, stream);
        cudaCheck(hipMemcpyAsync(digis_clusters_h.pdigi.get(), pdigi_d.get(), wordCounter*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
        cudaCheck(hipMemcpyAsync(digis_clusters_h.rawIdArr.get(), rawIdArr_d.get(), wordCounter*sizeof(uint32_t), hipMemcpyDefault, stream.id()));

        if (includeErrors) {
          digis_clusters_h.data = cs->make_host_unique<pixelgpudetails::error_obj[]>(MAX_FED_WORDS, stream);
          digis_clusters_h.error = cs->make_host_unique<GPU::SimpleVector<pixelgpudetails::error_obj>>(stream);
          GPU::make_SimpleVector(digis_clusters_h.error.get(), MAX_FED_WORDS, digis_clusters_h.data.get());
          assert(digis_clusters_h.error->size() == 0);
          assert(digis_clusters_h.error->capacity() == static_cast<int>(MAX_FED_WORDS));

          cudaCheck(hipMemcpyAsync(digis_clusters_h.error.get(), error_d.get(), vsize, hipMemcpyDefault, stream.id()));
          cudaCheck(hipMemcpyAsync(digis_clusters_h.data.get(), data_d.get(), MAX_ERROR_SIZE, hipMemcpyDefault, stream.id()));
          // If we want to transfer only the minimal amount of data, we
          // need a synchronization point. A single ExternalWork (of
          // SiPixelRawToClusterHeterogeneous) does not help because it is
          // already used to synchronize the data movement. So we'd need
          // two ExternalWorks (or explicit use of TBB tasks). The
          // prototype of #100 would allow this easily (as there would be
          // two ExternalWorks).
          //
          //cudaCheck(hipStreamSynchronize(stream.id()));
          //int size = digis_clusters_h.error->size();
          //cudaCheck(hipMemcpyAsync(digis_clusters_h.data.get(), data_d.get(), size*esize, hipMemcpyDefault, stream.id()));
        }
      }
    }
    // End  of Raw2Digi and passing data for cluserisation

    {
      // clusterizer ...
      using namespace gpuClustering;
      int threadsPerBlock = 256;
      int blocks = (wordCounter + threadsPerBlock - 1) / threadsPerBlock;

      gpuCalibPixel::calibDigis<<<blocks, threadsPerBlock, 0, stream.id()>>>(
          digis_d.moduleInd(),
          digis_d.c_xx(), digis_d.c_yy(), digis_d.adc(),
          gains,
          wordCounter);
      cudaCheck(hipGetLastError());

      // calibrated adc
      if(transferToCPU) {
        digis_clusters_h.adc = cs->make_host_unique<uint16_t[]>(MAX_FED_WORDS, stream);
        cudaCheck(hipMemcpyAsync(digis_clusters_h.adc.get(), digis_d.adc(), wordCounter*sizeof(uint16_t), hipMemcpyDefault, stream.id()));
      }

#ifdef GPU_DEBUG
       std::cout
         << "CUDA countModules kernel launch with " << blocks
         << " blocks of " << threadsPerBlock << " threads\n";
#endif

      cudaCheck(hipMemsetAsync(clusters_d.moduleStart(), 0x00, sizeof(uint32_t), stream.id()));

      countModules<<<blocks, threadsPerBlock, 0, stream.id()>>>(digis_d.c_moduleInd(), clusters_d.moduleStart(), clusters_d.clus(), wordCounter);
      cudaCheck(hipGetLastError());

      // read the number of modules into a data member, used by getProduct())
      cudaCheck(hipMemcpyAsync(&(digis_clusters_h.nModules_Clusters[0]), clusters_d.moduleStart(), sizeof(uint32_t), hipMemcpyDefault, stream.id()));

      threadsPerBlock = 256;
      blocks = MaxNumModules;
#ifdef GPU_DEBUG
         std::cout << "CUDA findClus kernel launch with " << blocks
         << " blocks of " << threadsPerBlock << " threads\n";
#endif
      cudaCheck(hipMemsetAsync(clusters_d.clusInModule(), 0, (MaxNumModules)*sizeof(uint32_t), stream.id()));
      findClus<<<blocks, threadsPerBlock, 0, stream.id()>>>(
          digis_d.c_moduleInd(),
          digis_d.c_xx(), digis_d.c_yy(),
          clusters_d.c_moduleStart(),
          clusters_d.clusInModule(), clusters_d.moduleId(),
          clusters_d.clus(),
          wordCounter);
      cudaCheck(hipGetLastError());

      // apply charge cut
      clusterChargeCut<<<blocks, threadsPerBlock, 0, stream.id()>>>(
          digis_d.moduleInd(),
          digis_d.c_adc(),
          clusters_d.c_moduleStart(),
          clusters_d.clusInModule(), clusters_d.c_moduleId(),
          clusters_d.clus(),
          wordCounter);
      cudaCheck(hipGetLastError());



      // count the module start indices already here (instead of
      // rechits) so that the number of clusters/hits can be made
      // available in the rechit producer without additional points of
      // synchronization/ExternalWork
      //
      // Temporary storage
      size_t tempScanStorageSize = 0;
      {
        uint32_t *tmp = nullptr;
        cudaCheck(hipcub::DeviceScan::InclusiveSum(nullptr, tempScanStorageSize, tmp, tmp, MaxNumModules));
      }
      auto tempScanStorage_d = cs->make_device_unique<uint32_t[]>(tempScanStorageSize, stream);
      // Set first the first element to 0
      cudaCheck(hipMemsetAsync(clusters_d.clusModuleStart(), 0, sizeof(uint32_t), stream.id()));
      // Then use inclusive_scan to get the partial sum to the rest
      cudaCheck(hipcub::DeviceScan::InclusiveSum(tempScanStorage_d.get(), tempScanStorageSize,
                                              clusters_d.c_clusInModule(), &clusters_d.clusModuleStart()[1], gpuClustering::MaxNumModules,
                                              stream.id()));
      // last element holds the number of all clusters
      cudaCheck(hipMemcpyAsync(&(digis_clusters_h.nModules_Clusters[1]), clusters_d.clusModuleStart()+gpuClustering::MaxNumModules, sizeof(uint32_t), hipMemcpyDefault, stream.id()));


      // clusters
      if(transferToCPU) {
        digis_clusters_h.clus = cs->make_host_unique<int32_t[]>(MAX_FED_WORDS, stream);
        cudaCheck(hipMemcpyAsync(digis_clusters_h.clus.get(), clusters_d.clus(), wordCounter*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
      }
    } // end clusterizer scope
  }

}
