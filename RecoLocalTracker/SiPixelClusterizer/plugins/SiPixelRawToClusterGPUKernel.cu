#include "hip/hip_runtime.h"
/* Sushil Dubey, Shashi Dugad, TIFR, July 2017
 *
 * File Name: RawToClusterGPU.cu
 * Description: It converts Raw data into Digi Format on GPU
 * Finaly the Output of RawToDigi data is given to pixelClusterizer
**/

// C++ includes
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream>

// CUDA includes
#include <hip/hip_runtime.h>

// CMSSW includes
#include "CUDADataFormats/SiPixelCluster/interface/gpuClusteringConstants.h"
#include "CondFormats/SiPixelObjects/interface/SiPixelROCsStatusAndMapping.h"
#include "DataFormats/FEDRawData/interface/FEDNumbering.h"
#include "DataFormats/TrackerCommon/interface/TrackerTopology.h"
#include "DataFormats/SiPixelDigi/interface/SiPixelDigiConstants.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"
#include "HeterogeneousCore/CUDAUtilities/interface/host_unique_ptr.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuCalibPixel.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClusterChargeCut.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClustering.h"
// local includes
#include "SiPixelRawToClusterGPUKernel.h"

// #define GPU_DEBUG

namespace pixelgpudetails {

  __device__ bool isBarrel(uint32_t rawId) {
    return (PixelSubdetector::PixelBarrel == ((rawId >> DetId::kSubdetOffset) & DetId::kSubdetMask));
  }

  __device__ pixelgpudetails::DetIdGPU getRawId(const SiPixelROCsStatusAndMapping *cablingMap,
                                                uint8_t fed,
                                                uint32_t link,
                                                uint32_t roc) {
    uint32_t index = fed * MAX_LINK * MAX_ROC + (link - 1) * MAX_ROC + roc;
    pixelgpudetails::DetIdGPU detId = {
        cablingMap->rawId[index], cablingMap->rocInDet[index], cablingMap->moduleId[index]};
    return detId;
  }

  //reference http://cmsdoxygen.web.cern.ch/cmsdoxygen/CMSSW_9_2_0/doc/html/dd/d31/FrameConversion_8cc_source.html
  //http://cmslxr.fnal.gov/source/CondFormats/SiPixelObjects/src/PixelROC.cc?v=CMSSW_9_2_0#0071
  // Convert local pixel to pixelgpudetails::global pixel
  __device__ pixelgpudetails::Pixel frameConversion(
      bool bpix, int side, uint32_t layer, uint32_t rocIdInDetUnit, pixelgpudetails::Pixel local) {
    int slopeRow = 0, slopeCol = 0;
    int rowOffset = 0, colOffset = 0;

    if (bpix) {
      if (side == -1 && layer != 1) {  // -Z side: 4 non-flipped modules oriented like 'dddd', except Layer 1
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8 - rocIdInDetUnit) * pixelgpudetails::numColsInRoc - 1;
        } else {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2 * pixelgpudetails::numRowsInRoc - 1;
          colOffset = (rocIdInDetUnit - 8) * pixelgpudetails::numColsInRoc;
        }       // if roc
      } else {  // +Z side: 4 non-flipped modules oriented like 'pppp', but all 8 in layer1
        if (rocIdInDetUnit < 8) {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2 * pixelgpudetails::numRowsInRoc - 1;
          colOffset = rocIdInDetUnit * pixelgpudetails::numColsInRoc;
        } else {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (16 - rocIdInDetUnit) * pixelgpudetails::numColsInRoc - 1;
        }
      }

    } else {             // fpix
      if (side == -1) {  // pannel 1
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8 - rocIdInDetUnit) * pixelgpudetails::numColsInRoc - 1;
        } else {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2 * pixelgpudetails::numRowsInRoc - 1;
          colOffset = (rocIdInDetUnit - 8) * pixelgpudetails::numColsInRoc;
        }
      } else {  // pannel 2
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8 - rocIdInDetUnit) * pixelgpudetails::numColsInRoc - 1;
        } else {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2 * pixelgpudetails::numRowsInRoc - 1;
          colOffset = (rocIdInDetUnit - 8) * pixelgpudetails::numColsInRoc;
        }

      }  // side
    }

    uint32_t gRow = rowOffset + slopeRow * local.row;
    uint32_t gCol = colOffset + slopeCol * local.col;
    // inside frameConversion row: gRow, column: gCol
    pixelgpudetails::Pixel global = {gRow, gCol};
    return global;
  }

  // error decoding and handling copied from EventFilter/SiPixelRawToDigi/src/ErrorChecker.cc
  template <bool debug = false>
  __device__ uint8_t conversionError(uint8_t fedId, uint8_t status) {
    uint8_t errorType = 0;

    switch (status) {
      case (1): {
        if constexpr (debug)
          printf("Error in Fed: %i, invalid channel Id (errorType = 35\n)", fedId);
        errorType = 35;
        break;
      }
      case (2): {
        if constexpr (debug)
          printf("Error in Fed: %i, invalid ROC Id (errorType = 36)\n", fedId);
        errorType = 36;
        break;
      }
      case (3): {
        if constexpr (debug)
          printf("Error in Fed: %i, invalid dcol/pixel value (errorType = 37)\n", fedId);
        errorType = 37;
        break;
      }
      case (4): {
        if constexpr (debug)
          printf("Error in Fed: %i, dcol/pixel read out of order (errorType = 38)\n", fedId);
        errorType = 38;
        break;
      }
      default:
        if constexpr (debug)
          printf("Cabling check returned unexpected result, status = %i\n", status);
    };

    return errorType;
  }

  __device__ bool rocRowColIsValid(uint32_t rocRow, uint32_t rocCol) {
    /// row and column in ROC representation
    return ((rocRow < pixelgpudetails::numRowsInRoc) & (rocCol < pixelgpudetails::numColsInRoc));
  }

  __device__ bool dcolIsValid(uint32_t dcol, uint32_t pxid) { return ((dcol < 26) & (2 <= pxid) & (pxid < 162)); }

  // error decoding and handling copied from EventFilter/SiPixelRawToDigi/src/ErrorChecker.cc
  template <bool debug = false>
  __device__ uint8_t
  checkROC(uint32_t errorWord, uint8_t fedId, uint32_t link, const SiPixelROCsStatusAndMapping *cablingMap) {
    uint8_t errorType = (errorWord >> sipixelconstants::ROC_shift) & sipixelconstants::ERROR_mask;
    if (errorType < 25)
      return 0;
    bool errorFound = false;

    switch (errorType) {
      case (25): {
        errorFound = true;
        uint32_t index = fedId * MAX_LINK * MAX_ROC + (link - 1) * MAX_ROC + 1;
        if (index > 1 && index <= cablingMap->size) {
          if (!(link == cablingMap->link[index] && 1 == cablingMap->roc[index]))
            errorFound = false;
        }
        if constexpr (debug)
          if (errorFound)
            printf("Invalid ROC = 25 found (errorType = 25)\n");
        break;
      }
      case (26): {
        if constexpr (debug)
          printf("Gap word found (errorType = 26)\n");
        break;
      }
      case (27): {
        if constexpr (debug)
          printf("Dummy word found (errorType = 27)\n");
        break;
      }
      case (28): {
        if constexpr (debug)
          printf("Error fifo nearly full (errorType = 28)\n");
        errorFound = true;
        break;
      }
      case (29): {
        if constexpr (debug)
          printf("Timeout on a channel (errorType = 29)\n");
        if (!((errorWord >> sipixelconstants::OMIT_ERR_shift) & sipixelconstants::OMIT_ERR_mask)) {
          if constexpr (debug)
            printf("...2nd errorType=29 error, skip\n");
          break;
        }
        errorFound = true;
        break;
      }
      case (30): {
        if constexpr (debug)
          printf("TBM error trailer (errorType = 30)\n");
        int stateMatch_bits = 4;
        int stateMatch_shift = 8;
        uint32_t stateMatch_mask = ~(~uint32_t(0) << stateMatch_bits);
        int stateMatch = (errorWord >> stateMatch_shift) & stateMatch_mask;
        if (stateMatch != 1 && stateMatch != 8) {
          if constexpr (debug)
            printf("FED error 30 with unexpected State Bits (errorType = 30)\n");
          break;
        }
        if (stateMatch == 1)
          errorType = 40;  // 1=Overflow -> 40, 8=number of ROCs -> 30
        errorFound = true;
        break;
      }
      case (31): {
        if constexpr (debug)
          printf("Event number error (errorType = 31)\n");
        errorFound = true;
        break;
      }
      default:
        errorFound = false;
    };

    return errorFound ? errorType : 0;
  }

  // error decoding and handling copied from EventFilter/SiPixelRawToDigi/src/ErrorChecker.cc
  template <bool debug = false>
  __device__ uint32_t
  getErrRawID(uint8_t fedId, uint32_t errWord, uint32_t errorType, const SiPixelROCsStatusAndMapping *cablingMap) {
    uint32_t rID = 0xffffffff;

    switch (errorType) {
      case 25:
      case 29:
      case 30:
      case 31:
      case 36:
      case 40: {
        uint32_t roc = 1;
        uint32_t link = sipixelconstants::getLink(errWord);
        uint32_t rID_temp = getRawId(cablingMap, fedId, link, roc).rawId;
        if (rID_temp != gpuClustering::invalidModuleId)
          rID = rID_temp;
        break;
      }
      case 37:
      case 38: {
        uint32_t roc = sipixelconstants::getROC(errWord);
        uint32_t link = sipixelconstants::getLink(errWord);
        uint32_t rID_temp = getRawId(cablingMap, fedId, link, roc).rawId;
        if (rID_temp != gpuClustering::invalidModuleId)
          rID = rID_temp;
        break;
      }
      default:
        break;
    };

    return rID;
  }

  // Kernel to perform Raw to Digi conversion
  template <bool debug = false>
  __global__ void RawToDigi_kernel(const SiPixelROCsStatusAndMapping *cablingMap,
                                   const unsigned char *modToUnp,
                                   const uint32_t wordCounter,
                                   const uint32_t *word,
                                   const uint8_t *fedIds,
                                   SiPixelDigisCUDASOAView digisView,
                                   cms::cuda::SimpleVector<SiPixelErrorCompact> *err,
                                   bool useQualityInfo,
                                   bool includeErrors) {
    //if (threadIdx.x==0) printf("Event: %u blockIdx.x: %u start: %u end: %u\n", eventno, blockIdx.x, begin, end);

    int32_t first = threadIdx.x + blockIdx.x * blockDim.x;
    for (int32_t iloop = first, nend = wordCounter; iloop < nend; iloop += blockDim.x * gridDim.x) {
      auto gIndex = iloop;
      auto dvgi = digisView[gIndex];
      dvgi.xx() = 0;
      dvgi.yy() = 0;
      dvgi.adc() = 0;
      bool skipROC = false;

      uint8_t fedId = fedIds[gIndex / 2];  // +1200;

      // initialize (too many coninue below)
      dvgi.pdigi() = 0;
      dvgi.rawIdArr() = 0;
      dvgi.moduleId() = gpuClustering::invalidModuleId;

      uint32_t ww = word[gIndex];  // Array containing 32 bit raw data
      if (ww == 0) {
        // 0 is an indicator of a noise/dead channel, skip these pixels during clusterization
        continue;
      }

      uint32_t link = sipixelconstants::getLink(ww);  // Extract link
      uint32_t roc = sipixelconstants::getROC(ww);    // Extract ROC in link

      uint8_t errorType = checkROC<debug>(ww, fedId, link, cablingMap);
      skipROC = (roc < pixelgpudetails::maxROCIndex) ? false : (errorType != 0);
      if (includeErrors and skipROC) {
        uint32_t rID = getErrRawID<debug>(fedId, ww, errorType, cablingMap);
        if (rID != 0xffffffff)  // store errors only for valid DetIds
          err->push_back(SiPixelErrorCompact{rID, ww, errorType, fedId});
        continue;
      }

      // check for spurious channels
      if (roc > MAX_ROC or link > MAX_LINK) {
        uint32_t rawId = getRawId(cablingMap, fedId, link, 1).rawId;
        if constexpr (debug) {
          printf("spurious roc %d found on link %d, detector %d (index %d)\n", roc, link, rawId, gIndex);
        }
        if (roc > MAX_ROC and roc < 25) {
          uint8_t error = conversionError<debug>(fedId, 2);
          err->push_back(SiPixelErrorCompact{rawId, ww, error, fedId});
        }
        continue;
      }

      uint32_t index = fedId * MAX_LINK * MAX_ROC + (link - 1) * MAX_ROC + roc;
      if (useQualityInfo) {
        skipROC = cablingMap->badRocs[index];
        if (skipROC)
          continue;
      }
      skipROC = modToUnp[index];
      if (skipROC)
        continue;

      pixelgpudetails::DetIdGPU detId = getRawId(cablingMap, fedId, link, roc);
      uint32_t rawId = detId.rawId;
      uint32_t layer = 0;
      int side = 0, panel = 0, module = 0;
      bool barrel = isBarrel(rawId);
      if (barrel) {
        layer = (rawId >> pixelgpudetails::layerStartBit) & pixelgpudetails::layerMask;
        module = (rawId >> pixelgpudetails::moduleStartBit) & pixelgpudetails::moduleMask;
        side = (module < 5) ? -1 : 1;
      } else {
        // endcap ids
        layer = 0;
        panel = (rawId >> pixelgpudetails::panelStartBit) & pixelgpudetails::panelMask;
        side = (panel == 1) ? -1 : 1;
      }

      // ***special case of layer to 1 be handled here
      pixelgpudetails::Pixel localPix;
      if (layer == 1) {
        uint32_t col = sipixelconstants::getCol(ww);
        uint32_t row = sipixelconstants::getRow(ww);
        localPix.row = row;
        localPix.col = col;
        if (includeErrors) {
          if (not rocRowColIsValid(row, col)) {
            uint8_t error = conversionError<debug>(fedId, 3);  //use the device function and fill the arrays
            err->push_back(SiPixelErrorCompact{rawId, ww, error, fedId});
            if constexpr (debug)
              printf("BPIX1  Error status: %i\n", error);
            continue;
          }
        }
      } else {
        // ***conversion rules for dcol and pxid
        uint32_t dcol = sipixelconstants::getDCol(ww);
        uint32_t pxid = sipixelconstants::getPxId(ww);
        uint32_t row = pixelgpudetails::numRowsInRoc - pxid / 2;
        uint32_t col = dcol * 2 + pxid % 2;
        localPix.row = row;
        localPix.col = col;
        if (includeErrors and not dcolIsValid(dcol, pxid)) {
          uint8_t error = conversionError<debug>(fedId, 3);
          err->push_back(SiPixelErrorCompact{rawId, ww, error, fedId});
          if constexpr (debug)
            printf("Error status: %i %d %d %d %d\n", error, dcol, pxid, fedId, roc);
          continue;
        }
      }

      pixelgpudetails::Pixel globalPix = frameConversion(barrel, side, layer, detId.rocInDet, localPix);
      dvgi.xx() = globalPix.row;  // origin shifting by 1 0-159
      dvgi.yy() = globalPix.col;  // origin shifting by 1 0-415
      dvgi.adc() = sipixelconstants::getADC(ww);
      dvgi.pdigi() = pixelgpudetails::pack(globalPix.row, globalPix.col, dvgi.adc());
      dvgi.moduleId() = detId.moduleId;
      dvgi.rawIdArr() = rawId;
    }  // end of loop (gIndex < end)

  }  // end of Raw to Digi kernel

  template <typename TrackerTraits>
  __global__ void fillHitsModuleStart(uint32_t const *__restrict__ clusInModule,
                                      uint32_t *__restrict__ moduleStart,
                                      uint32_t const *__restrict__ nModules,
                                      uint32_t *__restrict__ nModules_Clusters) {
    constexpr int nMaxModules = TrackerTraits::numberOfModules;
    constexpr int startBPIX2 = TrackerTraits::layerStart[1];

    constexpr uint32_t maxHitsInModule = TrackerTraits::maxHitsInModule;

    assert(startBPIX2 < nMaxModules);
    assert(nMaxModules < 4096);  // easy to extend at least till 32*1024
    assert(nMaxModules > 1024);

    assert(1 == gridDim.x);
    assert(0 == blockIdx.x);

    int first = threadIdx.x;

    // limit to MaxHitsInModule;
    for (int i = first, iend = nMaxModules; i < iend; i += blockDim.x) {
      moduleStart[i + 1] = std::min(maxHitsInModule, clusInModule[i]);
    }

    constexpr bool isPhase2 = std::is_base_of<pixelTopology::Phase2, TrackerTraits>::value;
    __shared__ uint32_t ws[32];
    cms::cuda::blockPrefixScan(moduleStart + 1, moduleStart + 1, 1024, ws);
    constexpr int lastModules = isPhase2 ? 1024 : nMaxModules - 1024;
    cms::cuda::blockPrefixScan(moduleStart + 1024 + 1, moduleStart + 1024 + 1, lastModules, ws);

    if constexpr (isPhase2) {
      cms::cuda::blockPrefixScan(moduleStart + 2048 + 1, moduleStart + 2048 + 1, 1024, ws);
      cms::cuda::blockPrefixScan(moduleStart + 3072 + 1, moduleStart + 3072 + 1, nMaxModules - 3072, ws);
    }

    for (int i = first + 1025, iend = isPhase2 ? 2049 : nMaxModules + 1; i < iend; i += blockDim.x) {
      moduleStart[i] += moduleStart[1024];
    }
    __syncthreads();

    if constexpr (isPhase2) {
      for (int i = first + 2049, iend = 3073; i < iend; i += blockDim.x) {
        moduleStart[i] += moduleStart[2048];
      }
      __syncthreads();
      for (int i = first + 3073, iend = nMaxModules + 1; i < iend; i += blockDim.x) {
        moduleStart[i] += moduleStart[3072];
      }
      __syncthreads();
    }

    if (threadIdx.x == 0) {
      // copy the number of modules
      nModules_Clusters[0] = *nModules;
      // last element holds the number of all clusters
      nModules_Clusters[1] = moduleStart[nMaxModules];
      // element 96 is the start of BPIX2 (i.e. the number of clusters in BPIX1)
      nModules_Clusters[2] = moduleStart[startBPIX2];
    }

#ifdef GPU_DEBUG
    uint16_t maxH = isPhase2 ? 3027 : 1024;
    assert(0 == moduleStart[0]);
    auto c0 = std::min(maxHitsInModule, clusInModule[0]);
    assert(c0 == moduleStart[1]);
    assert(moduleStart[maxH] >= moduleStart[maxH - 1]);
    assert(moduleStart[maxH + 1] >= moduleStart[maxH]);
    assert(moduleStart[nMaxModules] >= moduleStart[maxH + 1]);

    constexpr int startFP1 = TrackerTraits::numberOfModulesInBarrel;
    constexpr int startLastFwd = TrackerTraits::layerStart[TrackerTraits::numberOfLayers];
    for (int i = first, iend = nMaxModules + 1; i < iend; i += blockDim.x) {
      if (0 != i)
        assert(moduleStart[i] >= moduleStart[i - i]);
      // [BPX1, BPX2, BPX3, BPX4,  FP1,  FP2,  FP3,  FN1,  FN2,  FN3, LAST_VALID]
      // [   0,   96,  320,  672, 1184, 1296, 1408, 1520, 1632, 1744,       1856]
      if (i == startBPIX2 || i == startFP1 || i == startLastFwd || i == nMaxModules)
        printf("moduleStart %d %d\n", i, moduleStart[i]);
    }

#endif
  }

  // Interface to outside
  template <typename TrackerTraits>
  void SiPixelRawToClusterGPUKernel<TrackerTraits>::makePhase1ClustersAsync(
      const SiPixelClusterThresholds clusterThresholds,
      const SiPixelROCsStatusAndMapping *cablingMap,
      const unsigned char *modToUnp,
      const SiPixelGainForHLTonGPU *gains,
      const WordFedAppender &wordFed,
      SiPixelFormatterErrors &&errors,
      const uint32_t wordCounter,
      const uint32_t fedCounter,
      bool useQualityInfo,
      bool includeErrors,
      bool debug,
      hipStream_t stream) {
    // we're not opting for calling this function in case of early events
    assert(wordCounter != 0);
    nDigis = wordCounter;

#ifdef GPU_DEBUG
    std::cout << "decoding " << wordCounter << " digis." << std::endl;
#endif

    // since wordCounter != 0 we're not allocating 0 bytes,
    // digis_d = SiPixelDigisCUDA(wordCounter, stream);
    digis_d = SiPixelDigisCUDA(size_t(wordCounter), stream);
    if (includeErrors) {
      digiErrors_d = SiPixelDigiErrorsCUDA(wordCounter, std::move(errors), stream);
    }
    clusters_d = SiPixelClustersCUDA(TrackerTraits::numberOfModules, stream);

    // Begin Raw2Digi block
    {
      const int threadsPerBlock = 512;
      const int blocks = (wordCounter + threadsPerBlock - 1) / threadsPerBlock;  // fill it all

      assert(0 == wordCounter % 2);
      // wordCounter is the total no of words in each event to be trasfered on device
      auto word_d = cms::cuda::make_device_unique<uint32_t[]>(wordCounter, stream);
      auto fedId_d = cms::cuda::make_device_unique<uint8_t[]>(wordCounter, stream);

      cudaCheck(
          hipMemcpyAsync(word_d.get(), wordFed.word(), wordCounter * sizeof(uint32_t), hipMemcpyDefault, stream));
      cudaCheck(hipMemcpyAsync(
          fedId_d.get(), wordFed.fedId(), wordCounter * sizeof(uint8_t) / 2, hipMemcpyDefault, stream));

      // Launch rawToDigi kernel
      if (debug)
        RawToDigi_kernel<true><<<blocks, threadsPerBlock, 0, stream>>>(  //
            cablingMap,
            modToUnp,
            wordCounter,
            word_d.get(),
            fedId_d.get(),
            digis_d.view(),
            digiErrors_d.error(),  // returns nullptr if default-constructed
            useQualityInfo,
            includeErrors);
      else
        RawToDigi_kernel<false><<<blocks, threadsPerBlock, 0, stream>>>(  //
            cablingMap,
            modToUnp,
            wordCounter,
            word_d.get(),
            fedId_d.get(),
            digis_d.view(),
            digiErrors_d.error(),  // returns nullptr if default-constructed
            useQualityInfo,
            includeErrors);
      cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
      cudaCheck(hipStreamSynchronize(stream));
#endif

      if (includeErrors) {
        digiErrors_d.copyErrorToHostAsync(stream);
      }
    }
    // End of Raw2Digi and passing data for clustering

    {
      // clusterizer ...
      using namespace gpuClustering;
      int threadsPerBlock = 256;
      int blocks =
          (std::max(int(wordCounter), int(TrackerTraits::numberOfModules)) + threadsPerBlock - 1) / threadsPerBlock;

      gpuCalibPixel::calibDigis<<<blocks, threadsPerBlock, 0, stream>>>(clusterThresholds,
                                                                        digis_d.view().moduleId(),
                                                                        digis_d.view().xx(),
                                                                        digis_d.view().yy(),
                                                                        digis_d.view().adc(),
                                                                        gains,
                                                                        wordCounter,
                                                                        clusters_d->moduleStart(),
                                                                        clusters_d->clusInModule(),
                                                                        clusters_d->clusModuleStart());

      cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
      cudaCheck(hipStreamSynchronize(stream));
#endif

#ifdef GPU_DEBUG
      std::cout << "CUDA countModules kernel launch with " << blocks << " blocks of " << threadsPerBlock
                << " threads\n";
#endif

      countModules<TrackerTraits><<<blocks, threadsPerBlock, 0, stream>>>(
          digis_d->moduleId(), clusters_d->moduleStart(), digis_d->clus(), wordCounter);
      cudaCheck(hipGetLastError());

      threadsPerBlock = ((TrackerTraits::maxPixInModule / 16 + 128 - 1) / 128) *
                        128;  /// should be larger than maxPixInModule/16 aka (maxPixInModule/maxiter in the kernel)
      blocks = TrackerTraits::numberOfModules;
#ifdef GPU_DEBUG
      std::cout << "CUDA findClus kernel launch with " << blocks << " blocks of " << threadsPerBlock << " threads\n";
#endif

      findClus<TrackerTraits><<<blocks, threadsPerBlock, 0, stream>>>(digis_d->rawIdArr(),
                                                                      digis_d->moduleId(),
                                                                      digis_d->xx(),
                                                                      digis_d->yy(),
                                                                      clusters_d->moduleStart(),
                                                                      clusters_d->clusInModule(),
                                                                      clusters_d->moduleId(),
                                                                      digis_d->clus(),
                                                                      wordCounter);

      cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
      cudaCheck(hipStreamSynchronize(stream));
#endif

      // apply charge cut
      clusterChargeCut<TrackerTraits><<<blocks, threadsPerBlock, 0, stream>>>(clusterThresholds,
                                                                              digis_d->moduleId(),
                                                                              digis_d->adc(),
                                                                              clusters_d->moduleStart(),
                                                                              clusters_d->clusInModule(),
                                                                              clusters_d->moduleId(),
                                                                              digis_d->clus(),
                                                                              wordCounter);

      cudaCheck(hipGetLastError());

      // count the module start indices already here (instead of
      // rechits) so that the number of clusters/hits can be made
      // available in the rechit producer without additional points of
      // synchronization/ExternalWork
      auto nModules_Clusters_d = cms::cuda::make_device_unique<uint32_t[]>(3, stream);
      // MUST be ONE block
      fillHitsModuleStart<TrackerTraits><<<1, 1024, 0, stream>>>(clusters_d->clusInModule(),
                                                                 clusters_d->clusModuleStart(),
                                                                 clusters_d->moduleStart(),
                                                                 nModules_Clusters_d.get());

      // copy to host
      nModules_Clusters_h = cms::cuda::make_host_unique<uint32_t[]>(3, stream);
      cudaCheck(hipMemcpyAsync(
          nModules_Clusters_h.get(), nModules_Clusters_d.get(), 3 * sizeof(uint32_t), hipMemcpyDefault, stream));

#ifdef GPU_DEBUG
      cudaCheck(hipStreamSynchronize(stream));
#endif

    }  // end clusterizer scope
  }

  template <typename TrackerTraits>
  void SiPixelRawToClusterGPUKernel<TrackerTraits>::makePhase2ClustersAsync(
      const SiPixelClusterThresholds clusterThresholds,
      const uint16_t *moduleIds,
      const uint16_t *xDigis,
      const uint16_t *yDigis,
      const uint16_t *adcDigis,
      const uint32_t *packedData,
      const uint32_t *rawIds,
      const uint32_t numDigis,
      hipStream_t stream) {
    using namespace gpuClustering;
    nDigis = numDigis;
    digis_d = SiPixelDigisCUDA(numDigis, stream);

    cudaCheck(hipMemcpyAsync(digis_d->moduleId(), moduleIds, sizeof(uint16_t) * numDigis, hipMemcpyDefault, stream));
    cudaCheck(hipMemcpyAsync(digis_d->xx(), xDigis, sizeof(uint16_t) * numDigis, hipMemcpyDefault, stream));
    cudaCheck(hipMemcpyAsync(digis_d->yy(), yDigis, sizeof(uint16_t) * numDigis, hipMemcpyDefault, stream));
    cudaCheck(hipMemcpyAsync(digis_d->adc(), adcDigis, sizeof(uint16_t) * numDigis, hipMemcpyDefault, stream));
    cudaCheck(hipMemcpyAsync(digis_d->pdigi(), packedData, sizeof(uint32_t) * numDigis, hipMemcpyDefault, stream));
    cudaCheck(hipMemcpyAsync(digis_d->rawIdArr(), rawIds, sizeof(uint32_t) * numDigis, hipMemcpyDefault, stream));

    clusters_d = SiPixelClustersCUDA(TrackerTraits::numberOfModules, stream);

    nModules_Clusters_h = cms::cuda::make_host_unique<uint32_t[]>(2, stream);

    int threadsPerBlock = 512;
    int blocks = (int(numDigis) + threadsPerBlock - 1) / threadsPerBlock;

    gpuCalibPixel::calibDigisPhase2<<<blocks, threadsPerBlock, 0, stream>>>(clusterThresholds,
                                                                            digis_d->moduleId(),
                                                                            digis_d->adc(),
                                                                            numDigis,
                                                                            clusters_d->moduleStart(),
                                                                            clusters_d->clusInModule(),
                                                                            clusters_d->clusModuleStart());

    cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
    cudaCheck(hipStreamSynchronize(stream));
    std::cout << "CUDA countModules kernel launch with " << blocks << " blocks of " << threadsPerBlock << " threads\n";
#endif

    countModules<TrackerTraits><<<blocks, threadsPerBlock, 0, stream>>>(
        digis_d->moduleId(), clusters_d->moduleStart(), digis_d->clus(), numDigis);
    cudaCheck(hipGetLastError());

    // read the number of modules into a data member, used by getProduct())
    cudaCheck(hipMemcpyAsync(
        &(nModules_Clusters_h[0]), clusters_d->moduleStart(), sizeof(uint32_t), hipMemcpyDefault, stream));

    threadsPerBlock = 256;
    blocks = TrackerTraits::numberOfModules;

#ifdef GPU_DEBUG
    cudaCheck(hipStreamSynchronize(stream));
    std::cout << "CUDA findClus kernel launch with " << blocks << " blocks of " << threadsPerBlock << " threads\n";
#endif
    findClus<TrackerTraits><<<blocks, threadsPerBlock, 0, stream>>>(digis_d->rawIdArr(),
                                                                    digis_d->moduleId(),
                                                                    digis_d->xx(),
                                                                    digis_d->yy(),
                                                                    clusters_d->moduleStart(),
                                                                    clusters_d->clusInModule(),
                                                                    clusters_d->moduleId(),
                                                                    digis_d->clus(),
                                                                    numDigis);

    cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
    cudaCheck(hipStreamSynchronize(stream));
    std::cout << "CUDA clusterChargeCut kernel launch with " << blocks << " blocks of " << threadsPerBlock
              << " threads\n";
#endif

    // apply charge cut
    clusterChargeCut<TrackerTraits><<<blocks, threadsPerBlock, 0, stream>>>(clusterThresholds,
                                                                            digis_d->moduleId(),
                                                                            digis_d->adc(),
                                                                            clusters_d->moduleStart(),
                                                                            clusters_d->clusInModule(),
                                                                            clusters_d->moduleId(),
                                                                            digis_d->clus(),
                                                                            numDigis);
    cudaCheck(hipGetLastError());

    auto nModules_Clusters_d = cms::cuda::make_device_unique<uint32_t[]>(3, stream);
    // MUST be ONE block

#ifdef GPU_DEBUG
    cudaCheck(hipStreamSynchronize(stream));
    std::cout << "CUDA fillHitsModuleStart kernel launch \n";
#endif

    fillHitsModuleStart<TrackerTraits><<<1, 1024, 0, stream>>>(clusters_d->clusInModule(),
                                                               clusters_d->clusModuleStart(),
                                                               clusters_d->moduleStart(),
                                                               nModules_Clusters_d.get());

    nModules_Clusters_h = cms::cuda::make_host_unique<uint32_t[]>(3, stream);
    cudaCheck(hipMemcpyAsync(
        nModules_Clusters_h.get(), nModules_Clusters_d.get(), 3 * sizeof(uint32_t), hipMemcpyDefault, stream));

#ifdef GPU_DEBUG
    cudaCheck(hipStreamSynchronize(stream));
#endif
  }  //

  template class SiPixelRawToClusterGPUKernel<pixelTopology::Phase1>;
  template class SiPixelRawToClusterGPUKernel<pixelTopology::Phase2>;
  template class SiPixelRawToClusterGPUKernel<pixelTopology::HIonPhase1>;
}  // namespace pixelgpudetails
