#include "hip/hip_runtime.h"
// C++ headers
#include <algorithm>
#include <numeric>

// CUDA runtime
#include <hip/hip_runtime.h>

// CMSSW headers
#include "RecoLocalTracker/SiPixelClusterizer/plugins/SiPixelRawToClusterGPUKernel.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClusteringConstants.h"
#include "PixelRecHits.h"
#include "gpuPixelRecHits.h"

namespace {
  __global__
  void setHitsLayerStart(const uint32_t* hitsModuleStart, const uint32_t* layerStart, uint32_t* hitsLayerStart) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < 10) {
      hitsLayerStart[i] = hitsModuleStart[layerStart[i]];
    }
    else if(i == 10) {
      hitsLayerStart[i] = hitsModuleStart[gpuClustering::MaxNumModules];
    }
  }

  template <typename T>
  T *slicePitch(void *ptr, size_t pitch, size_t row) {
    return reinterpret_cast<T *>( reinterpret_cast<char *>(ptr) + pitch*row);
  }
}

namespace pixelgpudetails {
  PixelRecHitGPUKernel::PixelRecHitGPUKernel(cuda::stream_t<>& cudaStream) {

    constexpr auto MAX_HITS = siPixelRecHitsHeterogeneousProduct::maxHits();

    cudaCheck(hipMalloc((void **) & gpu_.bs_d, 3 * sizeof(float)));
    cudaCheck(hipMalloc((void **) & gpu_.hitsLayerStart_d, 11 * sizeof(uint32_t)));

    // Coalesce all 32bit and 16bit arrays to two big blobs
    //
    // This is just a toy. Please don't copy-paste the logic but
    // create a proper abstraction (e.g. along FWCore/SOA, or
    // FWCore/Utilities/interface/SoATuple.h
    //
    // Order such that the first ones are the ones transferred to CPU
    static_assert(sizeof(uint32_t) == sizeof(float)); // just stating the obvious
    cudaCheck(hipMallocPitch(&gpu_.owner_32bit_, &gpu_.owner_32bit_pitch_, MAX_HITS*sizeof(uint32_t), 9));
    cudaCheck(hipMemsetAsync(gpu_.owner_32bit_, 0x0, gpu_.owner_32bit_pitch_*9, cudaStream.id()));
    //edm::LogPrint("Foo") << "Allocate 32bit with pitch " << gpu_.owner_32bit_pitch_;
    gpu_.charge_d = slicePitch<int32_t>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 0);
    gpu_.xl_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 1);
    gpu_.yl_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 2);
    gpu_.xerr_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 3);
    gpu_.yerr_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 4);
    gpu_.xg_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 5);
    gpu_.yg_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 6);
    gpu_.zg_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 7);
    gpu_.rg_d = slicePitch<float>(gpu_.owner_32bit_, gpu_.owner_32bit_pitch_, 8);

    // Order such that the first ones are the ones transferred to CPU
    cudaCheck(hipMallocPitch(&gpu_.owner_16bit_, &gpu_.owner_16bit_pitch_, MAX_HITS*sizeof(uint16_t), 5));
    cudaCheck(hipMemsetAsync(gpu_.owner_16bit_, 0x0, gpu_.owner_16bit_pitch_*5, cudaStream.id()));
    //edm::LogPrint("Foo") << "Allocate 16bit with pitch " << gpu_.owner_16bit_pitch_;
    gpu_.detInd_d = slicePitch<uint16_t>(gpu_.owner_16bit_, gpu_.owner_16bit_pitch_, 0);
    gpu_.mr_d = slicePitch<uint16_t>(gpu_.owner_16bit_, gpu_.owner_16bit_pitch_, 1);
    gpu_.mc_d = slicePitch<uint16_t>(gpu_.owner_16bit_, gpu_.owner_16bit_pitch_, 2);
    gpu_.iphi_d = slicePitch<int16_t>(gpu_.owner_16bit_, gpu_.owner_16bit_pitch_, 3);
    gpu_.sortIndex_d = slicePitch<uint16_t>(gpu_.owner_16bit_, gpu_.owner_16bit_pitch_, 4);

    cudaCheck(hipMalloc((void **) & gpu_.hist_d, sizeof(HitsOnGPU::Hist)));
    cudaCheck(hipMalloc((void **) & gpu_.hws_d, 4*HitsOnGPU::Hist::totbins()));
    cudaCheck(hipMalloc((void **) & gpu_d, sizeof(HitsOnGPU)));
    gpu_.me_d = gpu_d;
    cudaCheck(hipMemcpyAsync(gpu_d, &gpu_, sizeof(HitsOnGPU), hipMemcpyDefault, cudaStream.id()));

    // Feels a bit dumb but constexpr arrays are not supported for device code
    // TODO: should be moved to EventSetup (or better ideas?)
    // Would it be better to use "constant memory"?
    cudaCheck(hipMalloc((void **) & d_phase1TopologyLayerStart_, 11 * sizeof(uint32_t)));
    cudaCheck(hipMemcpyAsync(d_phase1TopologyLayerStart_, phase1PixelTopology::layerStart, 11 * sizeof(uint32_t), hipMemcpyDefault, cudaStream.id()));

    cudaCheck(hipHostMalloc(&h_hitsModuleStart_, (gpuClustering::MaxNumModules+1) * sizeof(uint32_t)));

    // On CPU we can safely use MAX_HITS*sizeof as the pitch. Thanks
    // to '*256' it is even aligned by cache line
    h_owner_32bit_pitch_ = MAX_HITS*sizeof(uint32_t); 
    cudaCheck(hipHostMalloc(&h_owner_32bit_, h_owner_32bit_pitch_ * 5));
    h_charge_ = slicePitch<int32_t>(h_owner_32bit_, h_owner_32bit_pitch_, 0);
    h_xl_ = slicePitch<float>(h_owner_32bit_, h_owner_32bit_pitch_, 1);
    h_yl_ = slicePitch<float>(h_owner_32bit_, h_owner_32bit_pitch_, 2);
    h_xe_ = slicePitch<float>(h_owner_32bit_, h_owner_32bit_pitch_, 3);
    h_ye_ = slicePitch<float>(h_owner_32bit_, h_owner_32bit_pitch_, 4);

    h_owner_16bit_pitch_ = MAX_HITS*sizeof(uint16_t);
    cudaCheck(hipHostMalloc(&h_owner_16bit_, h_owner_16bit_pitch_ * 3));
    h_detInd_ = slicePitch<uint16_t>(h_owner_16bit_, h_owner_16bit_pitch_, 0);
    h_mr_ = slicePitch<uint16_t>(h_owner_16bit_, h_owner_16bit_pitch_, 1);
    h_mc_ = slicePitch<uint16_t>(h_owner_16bit_, h_owner_16bit_pitch_, 2);

#ifdef GPU_DEBUG
    cudaCheck(hipHostMalloc(&h_hitsLayerStart_, 11 * sizeof(uint32_t)));
#endif
  }
  PixelRecHitGPUKernel::~PixelRecHitGPUKernel() {
    cudaCheck(hipFree(gpu_.bs_d));
    cudaCheck(hipFree(gpu_.hitsLayerStart_d));
    cudaCheck(hipFree(gpu_.owner_32bit_));
    cudaCheck(hipFree(gpu_.owner_16bit_));
    cudaCheck(hipFree(gpu_.hist_d));
    cudaCheck(hipFree(gpu_.hws_d));
    cudaCheck(hipFree(gpu_d));
    cudaCheck(hipFree(d_phase1TopologyLayerStart_));

    cudaCheck(hipHostFree(h_hitsModuleStart_));
    cudaCheck(hipHostFree(h_owner_32bit_));
    cudaCheck(hipHostFree(h_owner_16bit_));
#ifdef GPU_DEBUG
    cudaCheck(hipHostFree(h_hitsLayerStart_));
#endif
  }

  void PixelRecHitGPUKernel::makeHitsAsync(const siPixelRawToClusterHeterogeneousProduct::GPUProduct& input,
                                           float const * bs,
                                           pixelCPEforGPU::ParamsOnGPU const * cpeParams,
                                           bool transferToCPU,
                                           cuda::stream_t<>& stream) {
    cudaCheck(hipMemcpyAsync(gpu_.bs_d, bs, 3 * sizeof(float), hipMemcpyDefault, stream.id()));
    gpu_.hitsModuleStart_d = input.clusters_d.clusModuleStart();
    gpu_.cpeParams = cpeParams; // copy it for use in clients
    cudaCheck(hipMemcpyAsync(gpu_d, &gpu_, sizeof(HitsOnGPU), hipMemcpyDefault, stream.id()));

    int threadsPerBlock = 256;
    int blocks = input.nModules; // active modules (with digis)

#ifdef GPU_DEBUG
    std::cout << "launching getHits kernel for " << blocks << " blocks" << std::endl;
#endif
    gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, stream.id()>>>(
      cpeParams,
      gpu_.bs_d,
      input.digis_d.moduleInd(),
      input.digis_d.xx(), input.digis_d.yy(), input.digis_d.adc(),
      input.clusters_d.moduleStart(),
      input.clusters_d.clusInModule(), input.clusters_d.moduleId(),
      input.clusters_d.clus(),
      input.nDigis,
      gpu_.hitsModuleStart_d,
      gpu_.charge_d,
      gpu_.detInd_d,
      gpu_.xg_d, gpu_.yg_d, gpu_.zg_d, gpu_.rg_d,
      gpu_.iphi_d,
      gpu_.xl_d, gpu_.yl_d,
      gpu_.xerr_d, gpu_.yerr_d,
      gpu_.mr_d, gpu_.mc_d
    );
    cudaCheck(hipGetLastError());

    // assuming full warp of threads is better than a smaller number...
    setHitsLayerStart<<<1, 32, 0, stream.id()>>>(gpu_.hitsModuleStart_d, d_phase1TopologyLayerStart_, gpu_.hitsLayerStart_d);
    cudaCheck(hipGetLastError());

    // needed only if hits on CPU are required...
    nhits_ = input.nClusters;
    if(transferToCPU) {
      cudaCheck(hipMemcpyAsync(h_hitsModuleStart_, gpu_.hitsModuleStart_d, (gpuClustering::MaxNumModules+1) * sizeof(uint32_t), hipMemcpyDefault, stream.id()));
#ifdef GPU_DEBUG
      cudaCheck(hipMemcpyAsync(h_hitsLayerStart_, gpu_.hitsLayerStart_d, 11 * sizeof(uint32_t), hipMemcpyDefault, stream.id()));
#endif

      cudaCheck(hipMemcpy2DAsync(h_owner_16bit_, h_owner_16bit_pitch_,
                                  gpu_.owner_16bit_, gpu_.owner_16bit_pitch_,
                                  nhits_*sizeof(uint16_t), 3,
                                  hipMemcpyDefault, stream.id()));

      cudaCheck(hipMemcpy2DAsync(h_owner_32bit_, h_owner_32bit_pitch_,
                                  gpu_.owner_32bit_, gpu_.owner_32bit_pitch_,
                                  nhits_*sizeof(uint32_t), 5,
                                  hipMemcpyDefault, stream.id()));

#ifdef GPU_DEBUG
      hipStreamSynchronize(stream.id());

      std::cout << "hit layerStart ";
      for (int i=0;i<10;++i) std::cout << phase1PixelTopology::layerName[i] << ':' << h_hitsLayerStart_[i] << ' ';
      std::cout << "end:" << h_hitsLayerStart_[10] << std::endl;
#endif

      // for timing test
      // hipStreamSynchronize(stream.id());
      // auto nhits_ = h_hitsLayerStart_[10];
      // radixSortMultiWrapper<int16_t><<<10, 256, 0, c.stream>>>(gpu_.iphi_d, gpu_.sortIndex_d, gpu_.hitsLayerStart_d);
    }

    cudautils::fillManyFromVector(gpu_.hist_d, gpu_.hws_d, 10, gpu_.iphi_d, gpu_.hitsLayerStart_d, nhits_, 256, stream.id());
  }
}
