#include "PixelRecHits.h"
#include "gpuPixelRecHits.h"

#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClustering.h"

#include "EventFilter/SiPixelRawToDigi/plugins/RawToDigiGPU.h" // for context....
#include "EventFilter/SiPixelRawToDigi/plugins/cudaCheck.h"

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <algorithm>

HitsOnGPU allocHitsOnGPU() {
   HitsOnGPU hh;
   cudaCheck(hipMalloc((void**) & hh.hitsModuleStart_d,(gpuClustering::MaxNumModules+1)*sizeof(uint32_t)));
   cudaCheck(hipMalloc((void**) & hh.charge_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
   cudaCheck(hipMalloc((void**) & hh.xg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
   cudaCheck(hipMalloc((void**) & hh.yg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
   cudaCheck(hipMalloc((void**) & hh.zg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
   hipDeviceSynchronize();

   return hh;
}


void pixelRecHits_wrapper(
      context const & c,
      pixelCPEforGPU::ParamsOnGPU const * cpeParams,
      uint32_t ndigis,
      uint32_t nModules, // active modules (with digis)
      HitsOnGPU & hh
)
{

 
 uint32_t hitsModuleStart[gpuClustering::MaxNumModules+1];
 hitsModuleStart[0] =0;
 cudaCheck(hipMemcpyAsync(&hitsModuleStart[1], c.clusInModule_d, gpuClustering::MaxNumModules*sizeof(uint32_t), hipMemcpyDeviceToHost, c.stream)); 

 std::partial_sum(std::begin(hitsModuleStart),std::end(hitsModuleStart),std::begin(hitsModuleStart));

 auto nhits = hitsModuleStart[gpuClustering::MaxNumModules];
 std::cout << " total number of clusters " << nhits << std::endl;

 cudaCheck(hipMemcpyAsync(hh.hitsModuleStart_d, &hitsModuleStart, (gpuClustering::MaxNumModules+1)*sizeof(uint32_t), hipMemcpyHostToDevice, c.stream));

  
 int threadsPerBlock = 256;
 int blocks = nModules;
 gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, c.stream>>>(
               cpeParams,
               c.moduleInd_d,
               c.xx_d, c.yy_d, c.adc_d,
               c.moduleStart_d,
               c.clusInModule_d, c.moduleId_d,
               c.clus_d,
               ndigis,
               hh.hitsModuleStart_d,
               hh.charge_d,
               hh.xg_d,hh.yg_d,hh.zg_d,
               false
  );

  int32_t charge[nhits];
  cudaCheck(hipMemcpyAsync(charge, hh.charge_d, nhits*sizeof(uint32_t), hipMemcpyDeviceToHost, c.stream));
  int ngood=0;
  auto l1 = hitsModuleStart[96];
  for (auto i=0U; i<nhits; ++i) if( charge[i]>4000 || (i<l1 &&charge[i]>2000) ) ++ngood;
  std::cout << " total number of good clusters " << ngood << std::endl;

   
}
