#include "hip/hip_runtime.h"
// C++ headers
#include <algorithm>
#include <numeric>

// CUDA runtime
#include <hip/hip_runtime.h>

// CMSSW headers
#include "CUDADataFormats/SiPixelCluster/interface/gpuClusteringConstants.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"

#include "PixelRecHitGPUKernel.h"
#include "gpuPixelRecHits.h"

namespace {
  __global__ void setHitsLayerStart(uint32_t const* __restrict__ hitsModuleStart,
                                    pixelCPEforGPU::ParamsOnGPU const* cpeParams,
                                    uint32_t* hitsLayerStart) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto m =
        cpeParams->commonParams().isPhase2 ? phase2PixelTopology::numberOfLayers : phase1PixelTopology::numberOfLayers;

    assert(0 == hitsModuleStart[0]);

    if (i <= m) {
      hitsLayerStart[i] = hitsModuleStart[cpeParams->layerGeometry().layerStart[i]];
#ifdef GPU_DEBUG
      printf("LayerStart %d/%d at module %d: %d\n", i, m, cpeParams->layerGeometry().layerStart[i], hitsLayerStart[i]);
#endif
    }
  }
}  // namespace

namespace pixelgpudetails {

  TrackingRecHit2DGPU PixelRecHitGPUKernel::makeHitsAsync(SiPixelDigisCUDA const& digis_d,
                                                          SiPixelClustersCUDA const& clusters_d,
                                                          BeamSpotCUDA const& bs_d,
                                                          pixelCPEforGPU::ParamsOnGPU const* cpeParams,
                                                          bool isPhase2,
                                                          hipStream_t stream) const {
    auto nHits = clusters_d.nClusters();

    TrackingRecHit2DGPU hits_d(
        nHits, isPhase2, clusters_d.offsetBPIX2(), cpeParams, clusters_d.clusModuleStart(), stream);
    assert(hits_d.nMaxModules() == isPhase2 ? phase2PixelTopology::numberOfModules
                                            : phase1PixelTopology::numberOfModules);

    int activeModulesWithDigis = digis_d.nModules();
    // protect from empty events
    if (activeModulesWithDigis) {
      int threadsPerBlock = 128;
      int blocks = activeModulesWithDigis;

#ifdef GPU_DEBUG
      std::cout << "launching getHits kernel for " << blocks << " blocks" << std::endl;
#endif
      gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, stream>>>(
          cpeParams, bs_d.data(), digis_d.view(), digis_d.nDigis(), clusters_d.view(), hits_d.view());
      cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
      cudaCheck(hipDeviceSynchronize());
#endif

      // assuming full warp of threads is better than a smaller number...
      if (nHits) {
        setHitsLayerStart<<<1, 32, 0, stream>>>(clusters_d.clusModuleStart(), cpeParams, hits_d.hitsLayerStart());
        cudaCheck(hipGetLastError());
        auto nLayers = isPhase2 ? phase2PixelTopology::numberOfLayers : phase1PixelTopology::numberOfLayers;
        cms::cuda::fillManyFromVector(hits_d.phiBinner(),
                                      nLayers,
                                      hits_d.iphi(),
                                      hits_d.hitsLayerStart(),
                                      nHits,
                                      256,
                                      hits_d.phiBinnerStorage(),
                                      stream);
        cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
        cudaCheck(hipDeviceSynchronize());
#endif
      }
    }

    return hits_d;
  }

}  // namespace pixelgpudetails
