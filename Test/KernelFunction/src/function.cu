#include <cstdio>
#include <hip/hip_runtime.h>

#include "Test/KernelFunction/interface/function.h"

namespace KernelFunction
{

  __device__
  void function()
  {
    printf("block %d,%d,%d, thread %d,%d,%d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
  }

}
