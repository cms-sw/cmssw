#include <hip/hip_runtime.h>

#include "Test/Kernel_uses_Function/interface/kernel.h"
#include "wrapper.h"

namespace BinaryWrapper_uses_Kernel_uses_Function {

  __host__
  void wrapper()
  {
    Kernel_uses_Function::kernel<<<1, 1>>>();
    hipDeviceSynchronize();
  }

}
