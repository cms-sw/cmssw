#include <cstdio>
#include <hip/hip_runtime.h>

#include "Test/WrapperKernelFunction/interface/function.h"

namespace WrapperKernelFunction
{

  __device__
  void function()
  {
    printf("block %d,%d,%d, thread %d,%d,%d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
  }

}
